#include "def.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include "hipsparse.h"
#include "hipsolver.h"
#include "hipsolver.h"
#include "hipsolver.h"
#include "hipsolver.h"
#include "hiprand/hiprand_kernel.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include "hip/hip_runtime_api.h"
#include "interop.cuh"

int Conti_Flag;
GFC *Fluid_sp;
GFG *Fluid_Den, *Fluid_Src;
int Conti_xnum, Conti_ynum;
Con_RegionX *Conti_x;
Con_RegionY *Conti_y;
int *dev_StructureIndex;
int *ReArgFlag;

int h_nvel;
float *vsave;
float time_sum;
float gputime;
hipEvent_t start, stop;
float 	totaltime,gputime_field,gputime_efield;
float 	gputime_move,gputime_mcc,gputime_deposit;
float 	gputime_diag,gputime_sort,gputime_Tec;
float 	gputime_continue,gputime_dump;
int		TotalT_D;
int		TotalT_H;
int		TotalT_M;
int		TotalT_S;

int PRINT_Flag;
int device_num;
float xlength,ylength,zlength;
int ngx,ngy,Gsize;
int ncx,ncy,Csize;
float dx,dy;
float idxy,idx,idy,dx2,dy2,dxdy2,hdx,hdy,r_eps0;
float fncx,fncy,fngx,fngy;
float *x_Garray,*y_Garray;
float *x_Carray,*y_Carray;
int BoundaryNUM;
int *BoundaryX0,*BoundaryY0,*BoundaryX1,*BoundaryY1,*BoundaryBC;
float *BoundaryTEMP;
int CondNUM,CondNUMR;  // CondNUMR is Real Conductor number
int*CondM_ID, *CondX0,*CondX1,*CondY0,*CondY1;
float *CondTEMP,*CondR,*CondL,*CondC;
int SrcNUM;
int *SrcM_ID;
float *SrcDC, *SrcPOWER, *SrcAC, *SrcFREQ, *Src2piFREQ, *SrcPHASE, *SrcRPHASE;
int External_Flag; // 0 : Voltage driven, 1: Power driven
float Min_FREQ, Max_FREQ;
int DielNUM,DielNUMR;
int *DielM_ID, *DielX0, *DielX1, *DielY0, *DielY1;
float *DielEPS;
GGA *vec_G;
GCA *vec_C;
int **StructureIndex;
int *vec_StructureIndex;
int MainGas; // Gas type 0:argon, 1:oxygen, 2:argon/oxygen
int nsp, nfsp, nBG;
Species *SP;// particle species
Fluid *FG;	// fluid species
BackG *BG;	// background species
HCP *PtD;
GPG *Host_G_sp;
float Total_Pressure;
int DT_PIC;  // Number of 1 cycle step
int DT_CONTI; // How many times PIC dt?
int Lap_Field_Solver_Test,Lap_Field_Solver_Flag,Lap_Field_Solver_Save;
float PCGtol;
float PCGtol2;
int HISTMAX;
int dHIST;
int NP_LIMIT; //Each of particle limit
int N_ave;
int N_smt;  // Number of smoothing every timestep
int ConstB_Flag; // Magnetic field 
int PD_intv;
float PD_Ratio;
int CSS_Flag;
float EndTime;
float Margin_ave_np;
int Flag_ave_np, Same_ave_np;
int Basic_Flag; // 0 : Basic, 1: OTHERS
int nRct_cx,nRct_rc; // Number of reaction _ cross section or Reaction rate
int TnRct; // Total Number of reaction 
int mMnum;
CollF *Coll_Flag;
int Msize;
float *MCC_rate,*ave_MCC_rate;
int FieldIter;
int A_size;
float *A_val,*TA_val;
int *Ai,*Aj;
int **A_idx;
float *MatM,**cond_b,*temp_b;
float *phi_cond;
float **AM,*V_t,*b_t,*extq,*extq_1,*extq_2,*extq_3;
float *CondCharge;
float *Surf_charge,*Old_Surf_charge,*Old2_Surf_charge;
int init_dump_num;
int OVER_dump_order;
int dump_order;
int dump_num;
float *dump_cycle;
int TecplotS_CX_Flag;
int TecplotS_Gsize_Flag;
int TecplotS_Particle_Flag;
int TecplotS_Particle_Num;
int TecplotS_2D_Flag;
int TecplotS_2D_Ncycle;
int TecplotS_Movie_Flag;
int TecplotS_Movie_Ncycle;
int TecplotS_Movie_Frame;
int TecplotS_Movie_SCYCLE;
int TecplotS_Movie_Count;
int TecplotS_PT_Movie_Flag;
int TecplotS_PT_Movie_Ncycle;
int TecplotS_PT_Movie_Frame;
int TecplotS_PT_Movie_SCYCLE;
int TecplotS_PT_Movie_Count;

int nz;
int N;
float *dev_A,*dev_TA;	
int *dev_Aj,*dev_Ai;
float *dev_b,*dev_Tb;
int   *vec_A_idx;
int   *dev_A_idx;
float *dev_M;
float *dev_AP,*dev_X,*dev_R,*dev_Z,*dev_P;
double *dot_result;
float *dot_result2;
int *FIter;
GGA *dev_GvecSet;
GCA *dev_CvecSet;
float *dev_CondCharge;
float *dev_phi;     // PCG Solution
float *dev_phi_buf; // Sigma or buf
size_t pitch;
float *Lap_TEMP_Sol; // Temperature Solution of Laplace Equation
float *Lap_PHI_Sol; // Each of conductor Phi Solution of Laplace Equation, This is Device value
float *TotPotential;
float *LapPotential;
float *dev_Sigma, *dev_Source;
											   
											   
											   


float LOGX_MIN,LOGX_MAX;
float dLOGX,idLOGX;
int N_LOGX;
ArCollD *Ar_Data;
O2CollD *O2_Data;
ArO2CollD *ArO2_Data;
float *Host_G_buf, *Host_C_buf;
float *dev_sum_Potential, *dev_ave_Potential;
float *dev_sum_Source, *dev_ave_Source;
float *dev_sum_Sigma, *dev_ave_Sigma;
float *dev_sum_Ex, *dev_ave_Ex;
float *dev_sum_Ey, *dev_ave_Ey;





hiprandState *devStates;
float *dev_vsave;
int sMemSize;
int sMemSize_MCC;
dim3 FIELD_GRID,FIELD_BLOCK;
dim3 FIELD_GRID2,FIELD_BLOCK2;
dim3 DEPOSIT_GRID,DEPOSIT_BLOCK;
dim3 EFIELD_GRID,EFIELD_BLOCK;
dim3 MOVE_GRID, MOVE_BLOCK;
dim3 SORT_GRID, SORT_BLOCK;
dim3 MCC_GRID, MCC_BLOCK;
dim3 DIAG_G_GRID, DIAG_G_BLOCK;
dim3 DIAG_NSPG_GRID, DIAG_NSPG_BLOCK;

float *dev_MCC_rate,*dev_ave_MCC_rate;
float *dev_RCstack,*dev_stack;
CollF *dev_Coll_Flag;
ArCollD *dev_ArCX;
O2CollD *dev_O2CX;
ArO2CollD *dev_ArO2CX;
MCC_sigmav *Host_SigmaV;
MCC_sigmav *dev_SigmaV;

long int seed; // related to Random number
double t;  // real time
int tstep; // number of time step
int cstep; // Number of Cycle step
int DumpFlag; // Dump File ON,OFF
char InputFile[80]; // INPUT FILE NAME
char DumpFile[80];  // DUMP FILE NAME
char *ConstBFile;  // DUMP FILE NAME
Fluid *dev_FG;
GFG *dev_FG_Den, *dev_FG_Src;

int Total_maxnp;
GCP *Host_sp, *dev_sp;
GPG *dev_G_sp;
Species *dev_info_sp;
hipDeviceProp_t prop;
float dt;   // timestsep for PIC
float dtc; // time step for continuity equation
float dt_dx, dt_dy;
int DT_MCCn; // mcc count for each step
float dt_mcc; // timestsep for MCC Module
int CYCLE_NUM; // Minimum frequency number of cycle
int *Efield_Flag, *Cond_Source_num, *Cond_count,**Cond_Power_ID;
float *Cond_Power;
float **CC_a; //Circuit_Const a
int Hcount;
int hist_count, hist_ave_count;
int nave_count;
float *t_array;
float *iter_array;    
Hist *HistPt,*HistFG;
float *t_ave_array;
Hist *Hist_ave_Pt,*Hist_ave_Pt_stack;
float **Current_hist,**Surf_charge_hist,**Volt_hist,**Volt_cond_hist;
float ***SP_current_hist;
float *Current_Now;
float *vec_Potential, *sum_Potential, *ave_Potential; // [Gsize] potential
float *vec_Source, *sum_Source, *ave_Source; // [Gsize] Charge density 
float *vec_Sigma, *sum_Sigma, *ave_Sigma; // [Gsize] [Dielectric] surface charge density, [Conductor] Surface current
float *sum_Ex, *ave_Ex, *sum_Ey, *ave_Ey;
int *Stack_ave_np, *ave_np;
float *new_ave_np, *old_ave_np;

int PT_Movie_S_count;