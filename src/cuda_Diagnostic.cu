#include "hip/hip_runtime.h"
#include "cuda_Diagnostic.cuh"

void Diagnostic_Basic(){
    int i, j, k, isp, index;
    static int power_init = 0;
    float cond_current, dis_current, phi_now, power_total;
    int buf1=0, now_np;
    float buf;
    if(nave_count==N_ave){  // average calculate
        Conti_Flag = 1;  
        Average_Particle_Density<<<DIAG_NSPG_GRID, DIAG_NSPG_BLOCK>>>(nsp, Gsize, N_ave, dev_info_sp, dev_G_sp);
        Average_Field_Data<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, N_ave, TotPotential,dev_Source,dev_Sigma,dev_GvecSet
                        ,dev_sum_Potential,dev_sum_Source,dev_sum_Sigma,dev_sum_Ex,dev_sum_Ey
                        ,dev_ave_Potential,dev_ave_Source,dev_ave_Sigma,dev_ave_Ex,dev_ave_Ey);
        switch(MainGas){ 
		case ARGON:
            Average_Argon_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
            break;
		case OXYGEN: 
            Average_Oxygen_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
			break;
		case ARO2: 
            Average_ArO2_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
			break;
        }
        nave_count = 0;
    }else{ // accomulation data
        Accomulate_Particle_Density<<<DIAG_NSPG_GRID, DIAG_NSPG_BLOCK>>>(nsp, Gsize, dev_G_sp);
        Accomulate_Field_Data<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize,TotPotential,dev_Source,dev_Sigma,dev_GvecSet
                        ,dev_sum_Potential,dev_sum_Source,dev_sum_Sigma,dev_sum_Ex,dev_sum_Ey);
        nave_count++;
    }  
    // Steady-state check
    if(Flag_ave_np){
        for(isp = 0;isp<nsp;isp++){
            hipMemcpy(&now_np, &dev_info_sp[isp].np, sizeof(int),hipMemcpyDeviceToHost);
            ave_np[isp] += (int)((float)now_np/(float)DT_PIC);
        }
        if(tstep%DT_PIC == 0){
            buf1 = 0;
            for(isp = 0;isp<nsp;isp++){
                new_ave_np[isp] = (float)ave_np[isp];
                ave_np[isp] = 0;
                buf = 100.0f * fabs((new_ave_np[isp]-old_ave_np[isp])/new_ave_np[isp]);
                old_ave_np[isp] = new_ave_np[isp];
                if(buf < Margin_ave_np){
                    Stack_ave_np[isp]++;
                }else{
                    Stack_ave_np[isp] = 0;
                }
                if(Stack_ave_np[isp] >= Same_ave_np){
                    buf1++;
                }
                t_ave_array[hist_ave_count] = (float) t;
                Hist_ave_Pt[isp].np[hist_ave_count] = new_ave_np[isp];
                Hist_ave_Pt_stack[isp].np[hist_ave_count] = (float)Stack_ave_np[isp];
            }
            hist_ave_count++;
            if(buf1>=nsp){
                Flag_ave_np = 0;
                Basic_Flag = 0;
            }
        }
    }
    if(Basic_Flag<-1) if(cstep > abs(Basic_Flag)) Basic_Flag = 0;
    // Calculate Current for Power driven or External circuit 
    hipMemcpy(Host_G_buf, dev_Sigma, Gsize * sizeof(float),	hipMemcpyDeviceToHost);
	for (i = 0; i < CondNUMR; i++) {
		Old2_Surf_charge[i] = Old_Surf_charge[i];
		Old_Surf_charge[i] = Surf_charge[i];
		Surf_charge[i] = 0.0f;
	}
	for (i = 0; i < Gsize; i++) {
		if (vec_G[i].CondID) {
			index = vec_G[i].CondID - 1;
			Surf_charge[index] += Host_G_buf[i] * vec_G[i].Area;
		}
	}
    // Power driven and Dual frequency;
    hipMemcpy(CondCharge, dev_CondCharge, nsp * CondNUMR * sizeof(float),hipMemcpyDeviceToHost);
    power_total = 0;
    for (i = 0; i < CondNUMR; i++) {
        // Current calculator
        Current_Now[i] = 0.0f;
        cond_current = 0.0f;
        for (isp = 0; isp < nsp; isp++) {
			cond_current += SP[isp].q_density * CondCharge[isp*CondNUMR + i] / dt;
		}
        dis_current = 0.5 * (3 * Surf_charge[i] - 4 * Old_Surf_charge[i] + Old2_Surf_charge[i]) / dt;
		Current_Now[i] = dis_current - cond_current;
        // Power calculator
        if(Cond_Source_num[i] == 1){ // Single Power Calculator
            //Accumulation
            phi_now = V_t[i];
			Cond_Power[i] += phi_now * Current_Now[i];
            Cond_count[i]++;
            //Averaged power
            if(Cond_count[i] == PD_intv){
                Cond_Power[i] /= (float)Cond_count[i];
                power_total+=Cond_Power[i];
                Cond_count[i] = 0;
                index = Cond_Power_ID[i][0];
                if(Cond_Power[i]>0) phi_now = SrcAC[index]*pow(fabs(SrcPOWER[index]/Cond_Power[i]),0.2);
                if(power_init) {
					if(phi_now>(1.0f + PD_Ratio)*SrcAC[index]){
						SrcAC[index]*=1.05;
					}
					else{
						SrcAC[index]=phi_now;
					}
				}
				printf("\nConductor %d : Fix Power %g, Current Power %g, Voltage %g\n",i,SrcPOWER[index],Cond_Power[i],SrcAC[index]);
				Cond_Power[i] = 0.0f;
                Cond_count[i]=0;
				power_init++;
            }
        }else if(Cond_Source_num[i] == 2){ // Dual Power Calculator

        }
    }
    if (hist_count >= HISTMAX) {
        for (k = 1, i = 4; k < HISTMAX / 4; k++, i += 4) {
			t_array[k] = t_array[i];
			for (isp = 0; isp < nsp; isp++) {
                HistPt[isp].np[k] =  HistPt[isp].np[i];
			}
			iter_array[k] = iter_array[i];
            for (j = 0; j < CondNUMR; j++) {
				for (isp = 0; isp < nsp; isp++) {
					SP_current_hist[isp][j][k] = SP_current_hist[isp][j][i];
				}
				Current_hist[j][k] = Current_hist[j][i];
				Surf_charge_hist[j][k] = Surf_charge_hist[j][i];
				Volt_hist[j][k] = Volt_hist[j][i];
				Volt_cond_hist[j][k] = Volt_cond_hist[j][i];
			}
		}
		hist_count = k;
		dHIST *= 4;
    }
    if((--Hcount)==0){
        t_array[hist_count] = (float) t;
        iter_array[hist_count] = (float) *FIter;
        for (isp = 0; isp < nsp; isp++) {
            hipMemcpy(&now_np, &dev_info_sp[isp].np, sizeof(int),hipMemcpyDeviceToHost);
            HistPt[isp].np[hist_count] = now_np;
        }
        for (i = 0; i < CondNUMR; i++) {
            Current_hist[i][hist_count] = Current_Now[i];
            Surf_charge_hist[i][hist_count] = Surf_charge[i];
            Volt_hist[i][hist_count] = V_t[i];
		    Volt_cond_hist[i][hist_count] = phi_cond[i];
            for (isp = 0; isp < nsp; isp++) {
			    SP_current_hist[isp][i][hist_count] = SP[isp].q_density * CondCharge[isp*CondNUMR + i] / dt;
		    }
        }
        hist_count++;
        Hcount = dHIST;
    }
}
__global__ void Accomulate_Field_Data(int Gsize, float *TotPot, float *Source, float *Sigma, GGA *vecG
                        ,float *sum_Potential, float *sum_Source, float *sum_Sigma, float *sum_Ex, float *sum_Ey){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    sum_Potential[TID] += TotPot[TID];
    //if(TotPot[TID] !=0) printf("TotPot[%d]= %g\n",TID,TotPot[TID]);
    sum_Source[TID] += Source[TID];
    sum_Sigma[TID] += Sigma[TID];
    sum_Ex[TID] += vecG[TID].Ex;
    sum_Ey[TID] += vecG[TID].Ey;           
}
__global__ void Average_Field_Data(int Gsize, int N_ave, float *TotPot, float *Source, float *Sigma, GGA *vecG
                        ,float *sum_Potential, float *sum_Source, float *sum_Sigma, float *sum_Ex, float *sum_Ey
                        ,float *ave_Potential, float *ave_Source, float *ave_Sigma, float *ave_Ex, float *ave_Ey){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    float oneofN = 1/(float)N_ave;
    ave_Potential[TID] = oneofN * sum_Potential[TID];// average
    //printf("ave_Potential[%d]= %g\n",TID,oneofN);
    sum_Potential[TID] = TotPot[TID];// Init && start
    ave_Source[TID] = oneofN * sum_Source[TID];
    sum_Source[TID] = Source[TID];
    ave_Sigma[TID] = oneofN * sum_Sigma[TID];
    sum_Sigma[TID] = Sigma[TID];
    ave_Ex[TID] = oneofN * sum_Ex[TID];
    sum_Ex[TID] = vecG[TID].Ex;
    ave_Ey[TID] = oneofN * sum_Ey[TID];    
    sum_Ey[TID] = vecG[TID].Ey;   
}
__global__ void Accomulate_Particle_Density(int nsp, int Gsize, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    data[TID].sum_den += data[TID].den;
}
__global__ void Average_Particle_Density(int nsp, int Gsize, int N_ave, Species *info, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    int isp = TID/Gsize;
    float temp = info[isp].Denscale/(float)N_ave;
    data[TID].sum_den += data[TID].den;
    data[TID].ave_den = temp * data[TID].sum_den;
    data[TID].sum_den = 0.0f;
}
__global__ void Average_Argon_MCC_rate(int Gsize, int TnRct, int N_ave, float dt, float *MCCR, float *ave_MCCR, Species *info){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int i;
    float temp = info[0].Denscale/(float)N_ave/dt;

    for(i=0;i<7;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    ave_MCCR[TID*TnRct+8] = temp * MCCR[TID*TnRct+8];
    MCCR[TID*TnRct+8] = 0.0f;
}
__global__ void Average_Oxygen_MCC_rate(int Gsize, int TnRct, int N_ave, float dt, float *MCCR, float *ave_MCCR, Species *info){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int i;
    float temp;

    temp = info[0].Denscale/(float)N_ave/dt;
    for(i=0;i<41;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[3].Denscale/(float)N_ave/dt;
    for(i=41;i<47;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[1].Denscale/(float)N_ave/dt;
    for(i=47;i<53;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[2].Denscale/(float)N_ave/dt;
    for(i=53;i<58;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = 1/(float)N_ave/dt;
    ave_MCCR[TID*TnRct+58] = info[3].Denscale * temp * MCCR[TID*TnRct+58];
    MCCR[TID*TnRct+58] = 0.0f;
}
__global__ void Average_ArO2_MCC_rate(int Gsize, int TnRct, int N_ave, float dt, float *MCCR, float *ave_MCCR, Species *info){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int i;
    float temp;

    temp = info[0].Denscale/(float)N_ave/dt;
    for(i=0;i<46;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[4].Denscale/(float)N_ave/dt;
    for(i=46;i<52;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[2].Denscale/(float)N_ave/dt;
    for(i=52;i<60;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[3].Denscale/(float)N_ave/dt;
    for(i=60;i<65;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[1].Denscale/(float)N_ave/dt;
    for(i=65;i<68;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = 1/(float)N_ave/dt;
    ave_MCCR[TID*TnRct+69] = info[3].Denscale * temp * MCCR[TID*TnRct+69];
    MCCR[TID*TnRct+69] = 0.0f;
    ave_MCCR[TID*TnRct+70] = info[3].Denscale * temp * MCCR[TID*TnRct+70];
    MCCR[TID*TnRct+70] = 0.0f;
    ave_MCCR[TID*TnRct+79] = info[1].Denscale * temp * MCCR[TID*TnRct+79];
    MCCR[TID*TnRct+79] = 0.0f;
    ave_MCCR[TID*TnRct+80] = info[1].Denscale * temp * MCCR[TID*TnRct+80];
    MCCR[TID*TnRct+80] = 0.0f;
    ave_MCCR[TID*TnRct+81] = info[0].Denscale * temp * MCCR[TID*TnRct+81];
    MCCR[TID*TnRct+81] = 0.0f;
}
void Set_Diagnostic_cuda(){
    // Host BUF VECTOR
    Host_G_buf = VFMalloc(Gsize);
    Host_C_buf = VFMalloc(Csize);
    VFInit(Host_G_buf,0.0,Gsize);
    VFInit(Host_C_buf,0.0,Csize);

    checkCudaErrors(hipMalloc((void**)&dev_sum_Potential, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Potential, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Potential, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Potential, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Potential, ave_Potential, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Source, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Source, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Source, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Source, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Source, ave_Source, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Sigma, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Sigma, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Sigma, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Sigma, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Sigma, ave_Sigma, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Ex, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Ex, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Ex, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Ex, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Ex, ave_Ex, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Ey, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Ey, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Ey, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Ey, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Ey, ave_Ey, Gsize * sizeof(float), hipMemcpyHostToDevice));
}
	