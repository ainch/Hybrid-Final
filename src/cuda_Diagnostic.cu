#include "hip/hip_runtime.h"
#include "cuda_Diagnostic.cuh"

void Diagnostic(){
    int i, k, isp;
    // Data Copy
    hipMemcpy(vec_Potential, TotPotential, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vec_Source, dev_Source, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vec_Sigma, dev_Sigma, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(vec_G, dev_GvecSet, Gsize * sizeof(GGA), hipMemcpyDeviceToHost);
    if(nave_count==N_ave){  // average calculate
        //Conti_Flag = 1;  
        Average_Particle_Density<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(nsp, Gsize, N_ave, dev_info_sp, dev_G_sp);
        // field data
        for(i=0;i<Gsize;i++){
            ave_Potential[i] = sum_Potential[i]/nave_count;
            ave_Source[i] += sum_Source[i]/nave_count;
            ave_Sigma[i] += sum_Sigma[i]/nave_count;
            ave_Ex[i] += sum_Ex[i]/nave_count;
            ave_Ey[i] += sum_Ey[i]/nave_count;
            sum_Potential[i] = 0.0f;
            sum_Source[i] = 0.0f;
            sum_Sigma[i] = 0.0f;
            sum_Ex[i] = 0.0f;
            sum_Ey[i] = 0.0f;
            sum_Potential[i] += vec_Potential[i];
            sum_Source[i] += vec_Source[i];
            sum_Sigma[i] += vec_Sigma[i];
            sum_Ex[i] += vec_G[i].Ex;
            sum_Ey[i] += vec_G[i].Ey;
        }
        nave_count = 0;
    }else{ // accomulation data
        Accomulate_Particle_Density<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(nsp, Gsize, dev_G_sp);

        // field data
        for(i=0;i<Gsize;i++){
            sum_Potential[i] += vec_Potential[i];
            sum_Source[i] += vec_Source[i];
            sum_Sigma[i] += vec_Sigma[i];
            sum_Ex[i] += vec_G[i].Ex;
            sum_Ey[i] += vec_G[i].Ey;
        }
        nave_count++;
    }
    if (hist_count >= HISTMAX) {
        for (k = 1, i = 4; k < HISTMAX / 4; k++, i += 4) {
			t_array[k] = t_array[i];
			for (isp = 0; isp < nsp; isp++) {
                HistPt[isp].np[k] =  HistPt[isp].np[i];
			}
			iter_array[k] = iter_array[i];
		}
		hist_count = k;
		dHIST *= 4;
    }
    
    
    if((--Hcount)==0){
        t_array[hist_count] = (float) t;
        iter_array[hist_count] = (float) *FIter;
        hipMemcpy(SP, dev_info_sp, nsp * sizeof(Species), hipMemcpyDeviceToHost);
        for (isp = 0; isp < nsp; isp++) {
            HistPt[isp].np[hist_count] = SP[isp].np;
        }
        hist_count++;
        Hcount = dHIST;
    }
}
__global__ void Accomulate_Particle_Density(int nsp, int Gsize, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    data[TID].sum_den += data[TID].den;
}
__global__ void Average_Particle_Density(int nsp, int Gsize, int N_ave, Species *info, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    int isp = TID/Gsize;
    float temp = info[isp].Denscale/N_ave;
    data[TID].sum_den += data[TID].den;
    data[TID].ave_den = temp * data[TID].sum_den;
    data[TID].sum_den = 0.0f;
}
void Set_Diagnostic_cuda(){
    // Host BUF VECTOR
    Host_G_buf = VFMalloc(Gsize);
    Host_C_buf = VFMalloc(Csize);
    VFInit(Host_G_buf,0.0,Gsize);
    VFInit(Host_C_buf,0.0,Csize);

}