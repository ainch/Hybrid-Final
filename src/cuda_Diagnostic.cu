#include "hip/hip_runtime.h"
#include "cuda_Diagnostic.cuh"

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

static float * Host_G_buf_tmp;
static float * Surf_charge_tmp;

static void __global__ update
(
	float * Surf_charge_tmp,
	GGA * const Field, 
	float * const dev_phi_buf,
	int Gsize,
	int CondNUMR
)
{
	cg::thread_block group_block  = cg::this_thread_block();
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float a_shd[128];
	
	int id_cond = 0;
	float area;
	float potential;
	if(tid < Gsize)
	{
		id_cond = Field[tid].CondID;
		area = Field[tid].Area;
		potential = dev_phi_buf[tid];
        if(id_cond != 0)
           atomicAdd(Surf_charge_tmp + id_cond - 1, area * potential);
	}
}

void Diagnostic(){
    int i, j, k, isp, index;
    static int power_init = 0;
    float cond_current, dis_current, phi_now, power_total;
    int buf1=0, now_np[nsp];
    float oldDen[nfsp],newDen[nfsp];
    float buf;
    // Cal NP
    for (isp = 0; isp < nsp; isp++) {
		hipMemset((void *) &dev_info_sp[isp].np, 0, sizeof(int));
		SumReductionINT1024All<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, isp, dev_G_sp, &dev_info_sp[isp].np);
		hipMemcpy(&SP[isp].np, &dev_info_sp[isp].np, sizeof(int),hipMemcpyDeviceToHost);
        now_np[isp] = SP[isp].np;
	}
    if(nave_count==N_ave){  // average calculate
        Conti_Flag = 1;  
        Average_Particle_Density<<<DIAG_NSPG_GRID, DIAG_NSPG_BLOCK>>>(nsp, Gsize, N_ave, dev_info_sp, dev_G_sp);
        Average_Field_Data<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, N_ave, TotPotential,dev_Source,dev_Sigma,dev_GvecSet
                        ,dev_sum_Potential,dev_sum_Source,dev_sum_Sigma,dev_sum_Ex,dev_sum_Ey
                        ,dev_ave_Potential,dev_ave_Source,dev_ave_Sigma,dev_ave_Ex,dev_ave_Ey);
        switch(MainGas){ 
		case ARGON:
            Average_Argon_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
            Argon_Update_Source<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, ngy, TnRct, dev_ave_MCC_rate, dev_Coll_Flag, dev_G_sp, dev_FG_Den, dev_FG_Src, dev_GvecSet);
            break;
		case OXYGEN: 
            Average_Oxygen_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
			Oxygen_Update_Source<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, dev_ave_MCC_rate, dev_Coll_Flag, dev_G_sp, dev_FG_Den, dev_FG_Src, dev_GvecSet);
            break;
		case ARO2: 
            Average_ArO2_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
			ArO2_Update_Source<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, ngy, TnRct, dev_ave_MCC_rate, dev_Coll_Flag, dev_G_sp, dev_FG_Den, dev_FG_Src, dev_GvecSet);
            break;
        }
        hipMemcpy(Fluid_Src, dev_FG_Src, nfsp * Gsize * sizeof(GFG), hipMemcpyDeviceToHost);
        if(CSS_Flag){
            for(isp = 0;isp<nfsp;isp++){ // Continuity on
                FG[isp].CSS_Flag = 1;
            }
        }
        Sync_Fluid_GFGtoGFC_forSource(Fluid_Src, Fluid_sp);
        nave_count = 0;
    }else{ // accomulation data
        Accomulate_Particle_Density<<<DIAG_NSPG_GRID, DIAG_NSPG_BLOCK>>>(nsp, Gsize, dev_G_sp);
        Accomulate_Field_Data<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize,TotPotential,dev_Source,dev_Sigma,dev_GvecSet
                        ,dev_sum_Potential,dev_sum_Source,dev_sum_Sigma,dev_sum_Ex,dev_sum_Ey);
        nave_count++;
    }  
    if(Conti_Flag){//Fluid density copy
        if(CSS_Flag){
            for(isp = 0;isp<nfsp;isp++){
                oldDen[isp] = FG[isp].ave_Den;
            }
        }
        Sync_Fluid_GFCtoGFG_forDen(Fluid_sp, Fluid_Den); 
        hipMemcpy(dev_FG_Den, Fluid_Den, nfsp * Gsize * sizeof(GFG), hipMemcpyHostToDevice);
        if(CSS_Flag){
            for(isp = 0;isp<nfsp;isp++){
                newDen[isp] = FG[isp].ave_Den;
                if(nave_count > FG[isp].CSS_Check){
                    if(abs(oldDen[isp]-newDen[isp])/newDen[isp] <= FG[isp].CSS_Conver)
                        FG[isp].CSS_Flag = 1;
                }
            }
        }   
    }
    // Steady-state check
    if(Flag_ave_np){
        for(isp = 0;isp<nsp;isp++){
            ave_np[isp] += (int)((float)now_np[isp]/(float)DT_PIC);
        }
        if(tstep%DT_PIC == 0){
            buf1 = 0;
            for(isp = 0;isp<nsp;isp++){
                new_ave_np[isp] = (float)ave_np[isp];
                ave_np[isp] = 0;
                buf = 100.0f * fabs((new_ave_np[isp]-old_ave_np[isp])/new_ave_np[isp]);
                old_ave_np[isp] = new_ave_np[isp];
                if(buf < Margin_ave_np){
                    Stack_ave_np[isp]++;
                }else{
                    Stack_ave_np[isp] = 0;
                }
                if(Stack_ave_np[isp] >= Same_ave_np){
                    buf1++;
                }
                t_ave_array[hist_ave_count] = (float) t;
                Hist_ave_Pt[isp].np[hist_ave_count] = new_ave_np[isp];
                Hist_ave_Pt_stack[isp].np[hist_ave_count] = (float)Stack_ave_np[isp];
            }
            hist_ave_count++;
            if(buf1>=nsp){
                Flag_ave_np = 0;
                Basic_Flag = 0;
            }
        }
    }
    if(Basic_Flag<-1) if(cstep > abs(Basic_Flag)) Basic_Flag = 0;
    // Calculate Current for Power driven or External circuit 
    
    hipMemcpy(Host_G_buf, dev_Sigma, Gsize * sizeof(float),	hipMemcpyDeviceToHost);
	for (i = 0; i < CondNUMR; i++) {
		Old2_Surf_charge[i] = Old_Surf_charge[i];
		Old_Surf_charge[i] = Surf_charge[i];
		Surf_charge[i] = 0.0f;
	}
    
    if(true)
    {
        checkCudaErrors(hipMemset((void *) Surf_charge_tmp, 0.0, CondNUMR * sizeof(float)));
            
        dim3 dim_num_block = dim3(Gsize / 128 + 1);
        dim3 dim_size_block = dim3(128);
    
        hipMemcpy
        (
            Host_G_buf_tmp, Host_G_buf, Gsize * sizeof(float), hipMemcpyHostToDevice
        );
    
        update<<<dim_num_block, dim_size_block>>>
        (
            Surf_charge_tmp, dev_GvecSet,
            Host_G_buf_tmp, Gsize, CondNUMR
        );
    
        checkCudaErrors
        (
            hipMemcpy
            (
                Surf_charge, Surf_charge_tmp, CondNUMR * sizeof(float), hipMemcpyDeviceToHost
            )
        );
    }
    else
    {
        for (i = 0; i < Gsize; i++) {
            if (vec_G[i].CondID) {
                index = vec_G[i].CondID - 1;
                Surf_charge[index] += Host_G_buf[i] * vec_G[i].Area;
            }
        }
    }
	
    // Power driven and Dual frequency;
    hipMemcpy(CondCharge, dev_CondCharge, nsp * CondNUMR * sizeof(float),hipMemcpyDeviceToHost);
    power_total = 0;
    for (i = 0; i < CondNUMR; i++) {
        // Current calculator
        Current_Now[i] = 0.0f;
        cond_current = 0.0f;
        for (isp = 0; isp < nsp; isp++) {
			cond_current += SP[isp].q_density * CondCharge[isp*CondNUMR + i] / dt;
		}
        dis_current = 0.5 * (3 * Surf_charge[i] - 4 * Old_Surf_charge[i] + Old2_Surf_charge[i]) / dt;
		Current_Now[i] = dis_current - cond_current;
        // Power calculator
        if(Cond_Source_num[i] == 1){ // Single Power Calculator
            //Accumulation
            phi_now = V_t[i];
			Cond_Power[i] += phi_now * Current_Now[i];
            Cond_count[i]++;
            //Averaged power
            if(Cond_count[i] == PD_intv){
                Cond_Power[i] /= (float)Cond_count[i];
                power_total+=Cond_Power[i];
                Cond_count[i] = 0;
                index = Cond_Power_ID[i][0];
                if(Cond_Power[i]>0) phi_now = SrcAC[index]*pow(fabs(SrcPOWER[index]/Cond_Power[i]),0.2);
                if(power_init) {
					if(phi_now>(1.0f + PD_Ratio)*SrcAC[index]){
						SrcAC[index]*=1.05;
					}
					else{
						SrcAC[index]=phi_now;
					}
				}
				printf("\nConductor %d : Fix Power %g, Current Power %g, Voltage %g\n",i,SrcPOWER[index],Cond_Power[i],SrcAC[index]);
				Cond_Power[i] = 0.0f;
                Cond_count[i]=0;
				power_init++;
            }
        }else if(Cond_Source_num[i] == 2){ // Dual Power Calculator

        }
    }
    if (hist_count >= HISTMAX) {
        for (k = 1, i = 4; k < HISTMAX / 4; k++, i += 4) {
			t_array[k] = t_array[i];
			for (isp = 0; isp < nsp; isp++) {
                HistPt[isp].np[k] =  HistPt[isp].np[i];
			}
            for (isp = 0; isp < nfsp; isp++) {
                HistFG[isp].np[k] = HistFG[isp].np[i];
            }
			iter_array[k] = iter_array[i];
            for (j = 0; j < CondNUMR; j++) {
				for (isp = 0; isp < nsp; isp++) {
					SP_current_hist[isp][j][k] = SP_current_hist[isp][j][i];
				}
				Current_hist[j][k] = Current_hist[j][i];
				Surf_charge_hist[j][k] = Surf_charge_hist[j][i];
				Volt_hist[j][k] = Volt_hist[j][i];
				Volt_cond_hist[j][k] = Volt_cond_hist[j][i];
			}
		}
		hist_count = k;
		dHIST *= 4;
    }
    if((--Hcount)==0){
        t_array[hist_count] = (float) t;
        iter_array[hist_count] = (float) *FIter;
        for (isp = 0; isp < nsp; isp++) {
            hipMemcpy(&now_np, &dev_info_sp[isp].np, sizeof(int),hipMemcpyDeviceToHost);
            HistPt[isp].np[hist_count] = now_np[isp];
        }
        for (isp = 0; isp < nfsp; isp++) {
            HistFG[isp].np[hist_count] = FG[isp].ave_Den;
        }
        for (i = 0; i < CondNUMR; i++) {
            Current_hist[i][hist_count] = Current_Now[i];
            Surf_charge_hist[i][hist_count] = Surf_charge[i];
            Volt_hist[i][hist_count] = V_t[i];
		    Volt_cond_hist[i][hist_count] = phi_cond[i];
            for (isp = 0; isp < nsp; isp++) {
			    SP_current_hist[isp][i][hist_count] = SP[isp].q_density * CondCharge[isp*CondNUMR + i] / dt;
		    }
        }
        hist_count++;
        Hcount = dHIST;
    }
}
void Diagnostic_Basic(){
    int i, j, k, isp, index;
    static int power_init = 0;
    float cond_current, dis_current, phi_now, power_total;
    int buf1=0, now_np[nsp];
    float oldDen[nfsp],newDen[nfsp];
    float buf;
    // Cal NP
    for (isp = 0; isp < nsp; isp++) {
		hipMemset((void *) &dev_info_sp[isp].np, 0, sizeof(int));
		SumReductionINT1024All<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, isp, dev_G_sp, &dev_info_sp[isp].np);
		hipMemcpy(&SP[isp].np, &dev_info_sp[isp].np, sizeof(int),hipMemcpyDeviceToHost);
        now_np[isp] = SP[isp].np;
	}
    if(nave_count==N_ave){  // average calculate
        Conti_Flag = 1;  
        Average_Particle_Density<<<DIAG_NSPG_GRID, DIAG_NSPG_BLOCK>>>(nsp, Gsize, N_ave, dev_info_sp, dev_G_sp);
        Average_Field_Data<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, N_ave, TotPotential,dev_Source,dev_Sigma,dev_GvecSet
                        ,dev_sum_Potential,dev_sum_Source,dev_sum_Sigma,dev_sum_Ex,dev_sum_Ey
                        ,dev_ave_Potential,dev_ave_Source,dev_ave_Sigma,dev_ave_Ex,dev_ave_Ey);
        switch(MainGas){ 
		case ARGON:
            Average_Argon_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
            Argon_Update_Source<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, ngy, TnRct, dev_ave_MCC_rate, dev_Coll_Flag, dev_G_sp, dev_FG_Den, dev_FG_Src, dev_GvecSet);
            break;
		case OXYGEN: 
            Average_Oxygen_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
			Oxygen_Update_Source<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, dev_ave_MCC_rate, dev_Coll_Flag, dev_G_sp, dev_FG_Den, dev_FG_Src, dev_GvecSet);
            break;
		case ARO2: 
            Average_ArO2_MCC_rate<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, TnRct, N_ave, dt, dev_MCC_rate, dev_ave_MCC_rate, dev_info_sp);
			ArO2_Update_Source<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize, ngy, TnRct, dev_ave_MCC_rate, dev_Coll_Flag, dev_G_sp, dev_FG_Den, dev_FG_Src, dev_GvecSet);
            break;
        }
        hipMemcpy(Fluid_Src, dev_FG_Src, nfsp * Gsize * sizeof(GFG), hipMemcpyDeviceToHost);
        Sync_Fluid_GFGtoGFC_forSource(Fluid_Src, Fluid_sp);
        if(CSS_Flag){
            for(isp = 0;isp<nfsp;isp++){ // Continuity on
                FG[isp].CSS_Flag = 1;
            }
        }
        nave_count = 0;
    }else{ // accomulation data
        Accomulate_Particle_Density<<<DIAG_NSPG_GRID, DIAG_NSPG_BLOCK>>>(nsp, Gsize, dev_G_sp);
        Accomulate_Field_Data<<<DIAG_G_GRID, DIAG_G_BLOCK>>>(Gsize,TotPotential,dev_Source,dev_Sigma,dev_GvecSet
                        ,dev_sum_Potential,dev_sum_Source,dev_sum_Sigma,dev_sum_Ex,dev_sum_Ey);
        nave_count++;
    }  
    if(Conti_Flag){//Fluid density copy
        if(CSS_Flag){
            for(isp = 0;isp<nfsp;isp++){
                oldDen[isp] = FG[isp].ave_Den;
            }
        }
        Sync_Fluid_GFCtoGFG_forDen(Fluid_sp, Fluid_Den); 
        hipMemcpy(dev_FG_Den, Fluid_Den, nfsp * Gsize * sizeof(GFG), hipMemcpyHostToDevice);
        if(CSS_Flag){
            for(isp = 0;isp<nfsp;isp++){
                newDen[isp] = FG[isp].ave_Den;
                if(nave_count > FG[isp].CSS_Check){
                    if(abs(oldDen[isp]-newDen[isp])/newDen[isp] <= FG[isp].CSS_Conver)
                        FG[isp].CSS_Flag = 1;
                }
            }
        }   
    }
    // Steady-state check
    if(Flag_ave_np){
        for(isp = 0;isp<nsp;isp++){
            ave_np[isp] += (int)((float)now_np[isp]/(float)DT_PIC);
        }
        if(tstep%DT_PIC == 0){
            buf1 = 0;
            for(isp = 0;isp<nsp;isp++){
                new_ave_np[isp] = (float)ave_np[isp];
                ave_np[isp] = 0;
                buf = 100.0f * fabs((new_ave_np[isp]-old_ave_np[isp])/new_ave_np[isp]);
                old_ave_np[isp] = new_ave_np[isp];
                if(buf < Margin_ave_np){
                    Stack_ave_np[isp]++;
                }else{
                    Stack_ave_np[isp] = 0;
                }
                if(Stack_ave_np[isp] >= Same_ave_np){
                    buf1++;
                }
                t_ave_array[hist_ave_count] = (float) t;
                Hist_ave_Pt[isp].np[hist_ave_count] = new_ave_np[isp];
                Hist_ave_Pt_stack[isp].np[hist_ave_count] = (float)Stack_ave_np[isp];
            }
            hist_ave_count++;
            if(buf1>=nsp){
                Flag_ave_np = 0;
                Basic_Flag = 0;
            }
        }
    }
    if(Basic_Flag<-1) if(cstep > abs(Basic_Flag)) Basic_Flag = 0;
    // Calculate Current for Power driven or External circuit 
    hipMemcpy(Host_G_buf, dev_Sigma, Gsize * sizeof(float),	hipMemcpyDeviceToHost);
	for (i = 0; i < CondNUMR; i++) {
		Old2_Surf_charge[i] = Old_Surf_charge[i];
		Old_Surf_charge[i] = Surf_charge[i];
		Surf_charge[i] = 0.0f;
	}
	for (i = 0; i < Gsize; i++) {
		if (vec_G[i].CondID) {
			index = vec_G[i].CondID - 1;
			Surf_charge[index] += Host_G_buf[i] * vec_G[i].Area;
		}
	}
    // Power driven and Dual frequency;
    hipMemcpy(CondCharge, dev_CondCharge, nsp * CondNUMR * sizeof(float),hipMemcpyDeviceToHost);
    power_total = 0;
    for (i = 0; i < CondNUMR; i++) {
        // Current calculator
        Current_Now[i] = 0.0f;
        cond_current = 0.0f;
        for (isp = 0; isp < nsp; isp++) {
			cond_current += SP[isp].q_density * CondCharge[isp*CondNUMR + i] / dt;
		}
        dis_current = 0.5 * (3 * Surf_charge[i] - 4 * Old_Surf_charge[i] + Old2_Surf_charge[i]) / dt;
		Current_Now[i] = dis_current - cond_current;
        // Power calculator
        if(Cond_Source_num[i] == 1){ // Single Power Calculator
            //Accumulation
            phi_now = V_t[i];
			Cond_Power[i] += phi_now * Current_Now[i];
            Cond_count[i]++;
            //Averaged power
            if(Cond_count[i] == PD_intv){
                Cond_Power[i] /= (float)Cond_count[i];
                power_total+=Cond_Power[i];
                Cond_count[i] = 0;
                index = Cond_Power_ID[i][0];
                if(Cond_Power[i]>0) phi_now = SrcAC[index]*pow(fabs(SrcPOWER[index]/Cond_Power[i]),0.2);
                if(power_init) {
					if(phi_now>(1.0f + PD_Ratio)*SrcAC[index]){
						SrcAC[index]*=1.05;
					}
					else{
						SrcAC[index]=phi_now;
					}
				}
				printf("\nConductor %d : Fix Power %g, Current Power %g, Voltage %g\n",i,SrcPOWER[index],Cond_Power[i],SrcAC[index]);
				Cond_Power[i] = 0.0f;
                Cond_count[i]=0;
				power_init++;
            }
        }else if(Cond_Source_num[i] == 2){ // Dual Power Calculator

        }
    }
    if (hist_count >= HISTMAX) {
        for (k = 1, i = 4; k < HISTMAX / 4; k++, i += 4) {
			t_array[k] = t_array[i];
			for (isp = 0; isp < nsp; isp++) {
                HistPt[isp].np[k] =  HistPt[isp].np[i];
			}
            for (isp = 0; isp < nfsp; isp++) {
                HistFG[isp].np[k] = HistFG[isp].np[i]; 
            }
			iter_array[k] = iter_array[i];
            for (j = 0; j < CondNUMR; j++) {
				for (isp = 0; isp < nsp; isp++) {
					SP_current_hist[isp][j][k] = SP_current_hist[isp][j][i];
				}
				Current_hist[j][k] = Current_hist[j][i];
				Surf_charge_hist[j][k] = Surf_charge_hist[j][i];
				Volt_hist[j][k] = Volt_hist[j][i];
				Volt_cond_hist[j][k] = Volt_cond_hist[j][i];
			}
		}
		hist_count = k;
		dHIST *= 4;
    }
    if((--Hcount)==0){
        t_array[hist_count] = (float) t;
        iter_array[hist_count] = (float) *FIter;
        for (isp = 0; isp < nsp; isp++) {
            hipMemcpy(&now_np, &dev_info_sp[isp].np, sizeof(int),hipMemcpyDeviceToHost);
            HistPt[isp].np[hist_count] = now_np[isp];
        }
        for (isp = 0; isp < nfsp; isp++) {
            HistFG[isp].np[hist_count] = FG[isp].ave_Den;
        }
        for (i = 0; i < CondNUMR; i++) {
            Current_hist[i][hist_count] = Current_Now[i];
            Surf_charge_hist[i][hist_count] = Surf_charge[i];
            Volt_hist[i][hist_count] = V_t[i];
		    Volt_cond_hist[i][hist_count] = phi_cond[i];
            for (isp = 0; isp < nsp; isp++) {
			    SP_current_hist[isp][i][hist_count] = SP[isp].q_density * CondCharge[isp*CondNUMR + i] / dt;
		    }
        }
        hist_count++;
        Hcount = dHIST;
    }
}
__global__ void SumReductionINT1024All(int n, int isp, GPG *g_data, int *g_max){
	__shared__ int sdata[1024];
	unsigned int TID=threadIdx.x;
	unsigned int i=blockDim.x*blockIdx.x+TID ; // global thread index
    
	//if(i>=n) g_data[isp*n + i].PtNumInCell = 0;
    if(i>=n) return;
    
	sdata[TID] = g_data[isp*n + i].PtNumInCell;
	__syncthreads();

	if(TID<512) sdata[TID]=sdata[TID]+sdata[TID+512];
		__syncthreads();

	if(TID<256) sdata[TID]=sdata[TID]+sdata[TID+256];
		__syncthreads();

	if(TID<128) sdata[TID]=sdata[TID]+sdata[TID+128];
		__syncthreads();

	if(TID<64) sdata[TID]=sdata[TID]+sdata[TID+64];
		__syncthreads();

	if(TID<32) warpSumReduceINT(sdata,TID);

	if(TID==0) {
		atomicAdd(g_max,sdata[0]);
	}
}
__device__ void warpSumReduceINT(volatile int* sdata,int TID)
{
	sdata[TID]+=sdata[TID+32];
	sdata[TID]+=sdata[TID+16];
	sdata[TID]+=sdata[TID+8];
	sdata[TID]+=sdata[TID+4];
	sdata[TID]+=sdata[TID+2];
	sdata[TID]+=sdata[TID+1];
}
__global__ void Accomulate_Field_Data(int Gsize, float *TotPot, float *Source, float *Sigma, GGA *vecG
                        ,float *sum_Potential, float *sum_Source, float *sum_Sigma, float *sum_Ex, float *sum_Ey){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    sum_Potential[TID] += TotPot[TID];
    //if(TotPot[TID] !=0) printf("TotPot[%d]= %g\n",TID,TotPot[TID]);
    sum_Source[TID] += Source[TID];
    sum_Sigma[TID] += Sigma[TID];
    sum_Ex[TID] += vecG[TID].Ex;
    sum_Ey[TID] += vecG[TID].Ey;           
}
__global__ void Average_Field_Data(int Gsize, int N_ave, float *TotPot, float *Source, float *Sigma, GGA *vecG
                        ,float *sum_Potential, float *sum_Source, float *sum_Sigma, float *sum_Ex, float *sum_Ey
                        ,float *ave_Potential, float *ave_Source, float *ave_Sigma, float *ave_Ex, float *ave_Ey){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    float oneofN = 1/(float)N_ave;
    ave_Potential[TID] = oneofN * sum_Potential[TID];// average
    //printf("ave_Potential[%d]= %g\n",TID,oneofN);
    sum_Potential[TID] = TotPot[TID];// Init && start
    ave_Source[TID] = oneofN * sum_Source[TID];
    sum_Source[TID] = Source[TID];
    ave_Sigma[TID] = oneofN * sum_Sigma[TID];
    sum_Sigma[TID] = Sigma[TID];
    ave_Ex[TID] = oneofN * sum_Ex[TID];
    sum_Ex[TID] = vecG[TID].Ex;
    ave_Ey[TID] = oneofN * sum_Ey[TID];    
    sum_Ey[TID] = vecG[TID].Ey;   
}
__global__ void Accomulate_Particle_Density(int nsp, int Gsize, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    data[TID].sum_den += data[TID].den;
}
__global__ void Average_Particle_Density(int nsp, int Gsize, int N_ave, Species *info, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    int isp = TID/Gsize;
    float temp = info[isp].Denscale/(float)N_ave;
    data[TID].sum_den += data[TID].den;
    data[TID].ave_den = temp * data[TID].sum_den;
    data[TID].sum_den = 0.0f;
}
__global__ void Average_Argon_MCC_rate(int Gsize, int TnRct, int N_ave, float dt, float *MCCR, float *ave_MCCR, Species *info){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int i;
    float temp = info[0].Denscale/(float)N_ave/dt;

    for(i=0;i<7;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    ave_MCCR[TID*TnRct+8] = temp * MCCR[TID*TnRct+8];
    MCCR[TID*TnRct+8] = 0.0f;
}
__global__ void Average_Oxygen_MCC_rate(int Gsize, int TnRct, int N_ave, float dt, float *MCCR, float *ave_MCCR, Species *info){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int i;
    float temp;

    temp = info[0].Denscale/(float)N_ave/dt;
    for(i=0;i<41;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[3].Denscale/(float)N_ave/dt;
    for(i=41;i<47;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[1].Denscale/(float)N_ave/dt;
    for(i=47;i<53;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[2].Denscale/(float)N_ave/dt;
    for(i=53;i<58;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = 1/(float)N_ave/dt;
    ave_MCCR[TID*TnRct+58] = info[3].Denscale * temp * MCCR[TID*TnRct+58];
    MCCR[TID*TnRct+58] = 0.0f;
}
__global__ void Average_ArO2_MCC_rate(int Gsize, int TnRct, int N_ave, float dt, float *MCCR, float *ave_MCCR, Species *info){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int i;
    float temp;

    temp = info[0].Denscale/(float)N_ave/dt;
    for(i=0;i<46;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[4].Denscale/(float)N_ave/dt;
    for(i=46;i<52;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[2].Denscale/(float)N_ave/dt;
    for(i=52;i<60;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[3].Denscale/(float)N_ave/dt;
    for(i=60;i<65;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = info[1].Denscale/(float)N_ave/dt;
    for(i=65;i<68;i++){
        ave_MCCR[TID*TnRct+i] = temp * MCCR[TID*TnRct+i];
        MCCR[TID*TnRct+i] = 0.0f;
    }
    temp = 1/(float)N_ave/dt;
    ave_MCCR[TID*TnRct+69] = info[3].Denscale * temp * MCCR[TID*TnRct+69];
    MCCR[TID*TnRct+69] = 0.0f;
    ave_MCCR[TID*TnRct+70] = info[3].Denscale * temp * MCCR[TID*TnRct+70];
    MCCR[TID*TnRct+70] = 0.0f;
    ave_MCCR[TID*TnRct+79] = info[1].Denscale * temp * MCCR[TID*TnRct+79];
    MCCR[TID*TnRct+79] = 0.0f;
    ave_MCCR[TID*TnRct+80] = info[1].Denscale * temp * MCCR[TID*TnRct+80];
    MCCR[TID*TnRct+80] = 0.0f;
    ave_MCCR[TID*TnRct+81] = info[0].Denscale * temp * MCCR[TID*TnRct+81];
    MCCR[TID*TnRct+81] = 0.0f;
}
__global__ void Argon_Update_Source(int Gsize, int ngy, int TnRct, float*MCCR, CollF *CollP, GPG *SP, GFG *FG, GFG *FG_S, GGA *BG){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    //float Eden = 0.25*(SP[TID].ave_den+SP[TID+ngy].ave_den+SP[TID+1].ave_den+SP[TID+ngy+1].ave_den);
    float Eden = 0.25*SP[TID].ave_den;
    MCCR[TID*TnRct + 7] = CollP[7].RR*Eden*FG[TID].n;
    MCCR[TID*TnRct + 9] = CollP[9].RR*BG[TID].BackDen1*FG[TID].n;      
    FG_S[TID].n = MCCR[TID*TnRct + 2] - MCCR[TID*TnRct + 4] - MCCR[TID*TnRct + 7] - 2 * MCCR[TID*TnRct + 8] - MCCR[TID*TnRct + 9];
} 
__global__ void Oxygen_Update_Source(int Gsize, int TnRct, float*MCCR, CollF *CollP, GPG *SP, GFG *FG, GFG *FG_S, GGA *BG){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int RID;
    float Sum_O2A, Diff_O2A, Sum_O2B, Diff_O2B,Sum_OP, Diff_OP,Sum_OD, Diff_OD;
    RID = TnRct * TID;

    MCCR[RID + 59] = CollP[59].RR*FG[2*Gsize + TID].n*FG[3*Gsize + TID].n;
    MCCR[RID + 60] = CollP[60].RR*FG[3*Gsize + TID].n*BG[TID].BackDen1;
    MCCR[RID + 61] = CollP[61].RR*FG[3*Gsize + TID].n*BG[TID].BackDen1;
    MCCR[RID + 62] = CollP[62].RR*FG[3*Gsize + TID].n*BG[TID].BackDen1;
    MCCR[RID + 63] = CollP[63].RR*FG[TID].n*FG[2*Gsize + TID].n;
    MCCR[RID + 64] = CollP[64].RR*FG[TID].n*BG[TID].BackDen1;   
    MCCR[RID + 65] = CollP[65].RR*FG[TID].n*FG[TID].n;
    MCCR[RID + 66] = CollP[66].RR*FG[Gsize + TID].n*BG[TID].BackDen1;  
    //
    Sum_O2A  = MCCR[RID + 3] + MCCR[RID + 61];
    Diff_O2A = MCCR[RID + 14] + MCCR[RID + 15] + MCCR[RID + 16] + MCCR[RID + 17] + MCCR[RID + 18]
                + MCCR[RID + 19] + MCCR[RID + 20] + MCCR[RID + 21] + MCCR[RID + 46] + MCCR[RID + 51]
                 + MCCR[RID + 56] + MCCR[RID + 63] + MCCR[RID + 64] + 2*MCCR[RID + 65];
    Sum_O2B = MCCR[RID + 4] + MCCR[RID + 62];
    Diff_O2B = MCCR[RID + 22] + MCCR[RID + 23] + MCCR[RID + 24] + MCCR[RID + 25] + MCCR[RID + 26]
                 + MCCR[RID + 27] + MCCR[RID + 28] + MCCR[RID + 29] + MCCR[RID + 52] + MCCR[RID + 57]
                 + MCCR[RID + 58] + MCCR[RID + 66];
    Sum_OP = MCCR[RID + 6] + 2 * MCCR[RID + 7] + MCCR[RID + 8] + MCCR[RID + 11] + MCCR[RID + 13]
                + MCCR[RID + 15] + 2*MCCR[RID + 18] + MCCR[RID + 19] + MCCR[RID + 21] + MCCR[RID + 23]
                 + 2*MCCR[RID + 26] + MCCR[RID + 27] + MCCR[RID + 29] + MCCR[RID + 30] + MCCR[RID + 31]
                  + MCCR[RID + 40] + MCCR[RID + 42] + MCCR[RID + 44] + 2*MCCR[RID + 45] + MCCR[RID + 46]
                   + MCCR[RID + 50] + MCCR[RID + 53] + MCCR[RID + 56] + MCCR[RID + 57] + MCCR[RID + 58]
                    + MCCR[RID + 59]  + MCCR[RID + 60]  + MCCR[RID + 61] + MCCR[RID + 62];
    Diff_OP =  MCCR[RID + 33] + MCCR[RID + 34] + MCCR[RID + 35] + MCCR[RID + 36] + MCCR[RID + 37]
                 + MCCR[RID + 38] + MCCR[RID + 43] + MCCR[RID + 47];
    Sum_OD = MCCR[RID + 8] + 2*MCCR[RID + 9] + MCCR[RID + 19] + 2*MCCR[RID + 20] + MCCR[RID + 27]
                 + 2*MCCR[RID + 28] + MCCR[RID + 31] + MCCR[RID + 33];
    Diff_OD = MCCR[RID + 39] + MCCR[RID + 40] + MCCR[RID + 59] + MCCR[RID + 60] + MCCR[RID + 61] + MCCR[RID + 62];

    FG_S[TID].n = Sum_O2A - Diff_O2A;
    FG_S[Gsize + TID].n = Sum_O2B - Diff_O2B;
    FG_S[2*Gsize + TID].n = Sum_OP - Diff_OP;
    FG_S[3*Gsize + TID].n = Sum_OD - Diff_OD;   
}
__global__ void ArO2_Update_Source(int Gsize, int ngy, int TnRct, float*MCCR, CollF *CollP, GPG *SP, GFG *FG, GFG *FG_S, GGA *BG){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize) return;
    int RID;
    float Eden;
    float Sum_ARM, Diff_ARM,Sum_O2A, Diff_O2A, Sum_O2B, Diff_O2B,Sum_OP, Diff_OP,Sum_OD, Diff_OD;
    float N1,N2,N4,N5;
    //Eden = 0.25*(SP[TID].ave_den+SP[TID+ngy].ave_den+SP[TID+1].ave_den+SP[TID+ngy+1].ave_den);
    Eden = 0.25*SP[TID].ave_den;
    RID = TnRct * TID;
    N1 = FG[TID].n;
    N2 = FG[Gsize + TID].n;
    N4 = FG[3*Gsize + TID].n;
    N5 = FG[4*Gsize + TID].n;
    MCCR[RID + 68] = CollP[68].RR*Eden*N1;
    MCCR[RID + 71] = CollP[71].RR*N4*N5;
    MCCR[RID + 72] = CollP[72].RR*N5*BG[TID].BackDen2;
    MCCR[RID + 73] = CollP[73].RR*N5*BG[TID].BackDen2;
    MCCR[RID + 74] = CollP[74].RR*N5*BG[TID].BackDen2;
    MCCR[RID + 75] = CollP[75].RR*N2*N4;
    MCCR[RID + 76] = CollP[76].RR*N2*BG[TID].BackDen2;
    MCCR[RID + 77] = CollP[77].RR*N2*N2;
    MCCR[RID + 78] = CollP[78].RR*FG[2*Gsize + TID].n*BG[TID].BackDen2;
    MCCR[RID + 82] = CollP[82].RR*N1*BG[TID].BackDen1;
    MCCR[RID + 83] = CollP[83].RR*N1*N4;
    MCCR[RID + 84] = CollP[84].RR*N1*N4;
    MCCR[RID + 85] = CollP[85].RR*N1*BG[TID].BackDen2;
    MCCR[RID + 86] = CollP[86].RR*N1*BG[TID].BackDen2;
    MCCR[RID + 87] = CollP[87].RR*N1*BG[TID].BackDen2;

    Sum_ARM = MCCR[RID + 2];
    Diff_ARM = MCCR[RID + 4] + MCCR[RID + 68] + 2*MCCR[RID + 81] + MCCR[RID + 82]
                + MCCR[RID + 83]+ MCCR[RID + 84]+ MCCR[RID + 85]
                + MCCR[RID + 86] + MCCR[RID + 87];
    Sum_O2A  = MCCR[RID + 8] + MCCR[RID + 73];
    Diff_O2A = MCCR[RID + 19] + MCCR[RID + 20] + MCCR[RID + 21] + MCCR[RID + 22] + MCCR[RID + 23]
                + MCCR[RID + 24] + MCCR[RID + 25] + MCCR[RID + 26] + MCCR[RID + 51] + MCCR[RID + 56]
                 + MCCR[RID + 30] + MCCR[RID + 75] + MCCR[RID + 76] + 2*MCCR[RID + 77];
    Sum_O2B = MCCR[RID + 9] + MCCR[RID + 74];
    Diff_O2B = MCCR[RID + 27] + MCCR[RID + 28] + MCCR[RID + 29] + MCCR[RID + 30] + MCCR[RID + 31]
                 + MCCR[RID + 32] + MCCR[RID + 33] + MCCR[RID + 34] + MCCR[RID + 57] + MCCR[RID + 64]
                 + MCCR[RID + 70] + MCCR[RID + 78];
    Sum_OP = MCCR[RID + 11] + 2 * MCCR[RID + 12] + MCCR[RID + 13] + MCCR[RID + 16] + MCCR[RID + 18]
                + MCCR[RID + 20] + 2*MCCR[RID + 23] + MCCR[RID + 24] + MCCR[RID + 26] + MCCR[RID + 28]
                 + 2*MCCR[RID + 31] + MCCR[RID + 32] + MCCR[RID + 34] + MCCR[RID + 35] + MCCR[RID + 36]
                  + MCCR[RID + 45] + MCCR[RID + 47] + MCCR[RID + 49] + 2*MCCR[RID + 50] + MCCR[RID + 51]
                   + MCCR[RID + 55] + MCCR[RID + 60] + MCCR[RID + 63] + MCCR[RID + 64] + MCCR[RID + 70]
                   + MCCR[RID + 71] + MCCR[RID + 72] + MCCR[RID + 73] + MCCR[RID + 74] + MCCR[RID + 69]
                   +2*MCCR[RID + 85] + MCCR[RID + 86];
    Diff_OP =  MCCR[RID + 38] + MCCR[RID + 39] + MCCR[RID + 40] + MCCR[RID + 41] + MCCR[RID + 42]
                 + MCCR[RID + 43] + MCCR[RID + 48] + MCCR[RID + 52] + MCCR[RID + 79] + MCCR[RID + 83];
    Sum_OD = MCCR[RID + 13] + 2*MCCR[RID + 14] + MCCR[RID + 24] + 2*MCCR[RID + 25] + MCCR[RID + 32]
                 + 2*MCCR[RID + 33] + MCCR[RID + 36] + MCCR[RID + 38] + MCCR[RID + 83] + MCCR[RID + 86];
    Diff_OD = MCCR[RID + 44] + MCCR[RID + 45] + MCCR[RID + 71]+ MCCR[RID + 72]+ MCCR[RID + 73]
                    + MCCR[RID + 74];
    FG_S[TID].n = Sum_ARM - Diff_ARM;
    FG_S[Gsize + TID].n = Sum_O2A - Diff_O2A;
    FG_S[2*Gsize + TID].n = Sum_O2B - Diff_O2B;
    FG_S[3*Gsize + TID].n = Sum_OP - Diff_OP;
    FG_S[4*Gsize + TID].n = Sum_OD - Diff_OD;
}
void Set_Diagnostic_cuda(){
    // Host BUF VECTOR
    Host_G_buf = VFMalloc(Gsize);
    Host_C_buf = VFMalloc(Csize);
    VFInit(Host_G_buf,0.0,Gsize);
    VFInit(Host_C_buf,0.0,Csize);

    checkCudaErrors(hipMalloc((void**)&Host_G_buf_tmp, sizeof(float) * Gsize));
    checkCudaErrors(hipMalloc((void**) &Surf_charge_tmp, CondNUMR * sizeof(float)));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Potential, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Potential, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Potential, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Potential, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Potential, ave_Potential, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Source, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Source, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Source, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Source, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Source, ave_Source, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Sigma, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Sigma, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Sigma, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Sigma, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Sigma, ave_Sigma, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Ex, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Ex, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Ex, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Ex, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Ex, ave_Ex, Gsize * sizeof(float), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_sum_Ey, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_sum_Ey, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&dev_ave_Ey, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_ave_Ey, 0.0f, Gsize * sizeof(float)));
	checkCudaErrors(hipMemcpy(dev_ave_Ey, ave_Ey, Gsize * sizeof(float), hipMemcpyHostToDevice));
}
	
