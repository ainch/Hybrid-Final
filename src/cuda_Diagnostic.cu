#include "hip/hip_runtime.h"
#include "cuda_Diagnostic.cuh"

void Diagnostic(){
    int i, k, isp;
    nave_count++;
    if(nave_count==N_ave){  // average calculate
        //Conti_Flag = 1;  
        Average_Particle_Density<<<MCC_GRID, MCC_BLOCK>>>(nsp, Gsize, N_ave, dev_info_sp, dev_G_sp);
        nave_count = 0;
    }else{ // accomulation data
        Accomulate_Particle_Density<<<MCC_GRID, MCC_BLOCK>>>(nsp, Gsize, dev_G_sp);
    }
    if (hist_count >= HISTMAX) {
        for (k = 1, i = 4; k < HISTMAX / 4; k++, i += 4) {
			t_array[k] = t_array[i];
			for (isp = 0; isp < nsp; isp++) {
                HistPt[isp].np[k] =  HistPt[isp].np[i];
			}
			iter_array[k] = iter_array[i];
		}
		hist_count = k;
		dHIST *= 4;
    }
    if((--Hcount)==0){
        t_array[hist_count] = (float) t;
        iter_array[hist_count] = (float) *FIter;
        hipMemcpy(SP, dev_info_sp, nsp * sizeof(Species), hipMemcpyDeviceToHost);
        for (isp = 0; isp < nsp; isp++) {
            HistPt[isp].np[hist_count] = SP[isp].np;
        }
        hist_count++;
        Hcount = dHIST;
    }
}
__global__ void Accomulate_Particle_Density(int nsp, int Gsize, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    data[TID].sum_den += data[TID].den;
}
__global__ void Average_Particle_Density(int nsp, int Gsize, int N_ave, Species *info, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nsp*Gsize) return;
    int isp = TID/Gsize;
    float temp = info[isp].Denscale/N_ave;
    data[TID].sum_den += data[TID].den;
    data[TID].ave_den = temp * data[TID].sum_den;
    data[TID].sum_den = 0.0f;
}
void Set_Diagnostic_cuda(){
    // Host BUF VECTOR
    Host_G_buf = VFMalloc(Gsize);
    Host_C_buf = VFMalloc(Csize);
    VFInit(Host_G_buf,0.0,Gsize);
    VFInit(Host_C_buf,0.0,Csize);

}