#include "hip/hip_runtime.h"
#include "cuda_Deposit.cuh"

void Deposit_cuda(){
    int i,j;
    // Function list
    // 1. DepositAtom - Particle > density
    // 2. boundary density up
    // 3. density smoothing
    // 4. charge density cal --> dev_Source
    // 5. dev_Source --> dev_b  (PCG setting)
    // <<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>
    printf("Deposit start.\n");
    DepositAtom<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(Gsize,ngy,dev_info_sp,dev_sp,dev_G_sp);
    DepositBoundary<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(Gsize,ngy,nsp,dev_GvecSet,dev_G_sp);
    // Smoothing

    // 4. charge density cal --> dev_Source

    // 5. dev_Source --> dev_b  (PCG setting)

    // TEST
    hipMemcpy(Host_G_sp, dev_G_sp, Gsize * nsp * sizeof(GPG), hipMemcpyDeviceToHost);
    for(j=ngy-1;j>=0;j--){
        for(i=0;i<ngx;i++){
            printf("%8.2g ",Host_G_sp[i*ngy+j].den);
        } printf("\n");
    }

}
__global__ void DepositBoundary(int Gsize, int ngy, int nsp, GGA *vecSet, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>Gsize*nsp) return;
    int ID;
    ID = (int)TID%Gsize;

	if(vecSet[ID].Face==LEFT || vecSet[ID].Face==RIGHT || vecSet[ID].Face==UP || vecSet[ID].Face==DOWN) data[TID].den *= 2;
	else if(vecSet[ID].Face==UL_CORN || vecSet[ID].Face==LL_CORN || vecSet[ID].Face==UR_CORN || vecSet[ID].Face==LR_CORN) data[TID].den *= 4;
	if(vecSet[ID].Boundary==NEUMANN) data[TID].den *= 2;
	else if(vecSet[ID].Boundary==DIRICHLET && vecSet[ID].Face!=NO_FACE && vecSet[ID].CondID!=0) data[TID].den *= 2;
}
__global__ void DepositAtom(int Gsize, int ngy, Species *info, GCP *sp, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
	int i,k,PNC,isp,ID;
    float lx,ly;
	float WS,WN,ES,EN;

    if(TID>Gsize*info[0].spnum) return;

    data[TID].den = 0.0;

    PNC = data[TID].PtNumInCell;
    if(PNC==0) return;

    isp = (int)TID/Gsize;
    ID = (int)TID%Gsize;

    //printf("TID[%d] = [%d]\n",TID,isp);
    WS=0.0; WN=0.0; ES=0.0; EN=0.0;
	i=info[isp].St_num + ID;

	for(k=0;k<PNC;k++){
		lx=sp[i].x; ly=sp[i].y;
		WS+=(1-lx)*(1-ly);
		WN+=(1-lx)*ly;
		ES+=lx*(1-ly);
		EN+=lx*ly;
		i+=Gsize;
	}

	atomicAdd(&data[TID].den,WS);
	atomicAdd(&data[TID+1].den,WN);
	atomicAdd(&data[TID+ngy].den,ES);
	atomicAdd(&data[TID+ngy+1].den,EN); 
}