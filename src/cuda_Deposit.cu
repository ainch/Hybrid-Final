#include "hip/hip_runtime.h"
#include "cuda_Deposit.cuh"

void Deposit_cuda(){
    int i;
    // DepositAtom - Particle > density
	DepositInitDensity<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(Gsize,dev_info_sp,dev_G_sp);
	//start plot Gsize values
	hipDeviceSynchronize();
    DepositAtom<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(Gsize,ngy,dev_info_sp,dev_sp,dev_G_sp);
	hipDeviceSynchronize();
    DepositBoundary<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(Gsize,ngy,nsp,dev_GvecSet,dev_G_sp);
    // Smoothing
    for(i=0;i<N_smt;i++){
        Smooth_121_A<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(ngx, ngy, nsp, dev_GvecSet, dev_G_sp);
        Smooth_121_B<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(ngx, ngy, nsp, dev_GvecSet, dev_G_sp);
    }
    // charge density cal --> dev_Source
    SumSource<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(ngx, ngy, dev_info_sp, dev_GvecSet, dev_G_sp, dev_Sigma, dev_Source);
    // dev_Source --> dev_b  (PCG setting)
    PCG_Set<<<DEPOSIT_GRID,DEPOSIT_BLOCK>>>(Gsize,dev_A_idx, dev_GvecSet, dev_Source, dev_R);
	hipDeviceSynchronize();
}
__global__ void PCG_Set(int Gsize, int *IDX, GGA *vecSet, float *Source, float *B){
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	if(TID>=Gsize) return;
	if(IDX[TID]) B[IDX[TID]-1]=Source[TID];
}
__global__ void SumSource(int ngx, int ngy, Species *info, GGA *vecSet, GPG *data, float *Sigma, float *Source){
     int TID = threadIdx.x + blockIdx.x * blockDim.x;
     if(TID>=ngx*ngy) return;
     int i;
     int x,y;
     x=TID/ngy; y=TID%ngy;
	float sum, SD;
    float SumSig = 0.0;
     
	if(x==0) {
		if(vecSet[TID+ngy].Boundary==DIELECTRIC) {
            for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].sigma * info[i].q_density;
			Sigma[TID] = 2 * SumSig/vecSet[TID].Area;
		    SD = SumSig;
		}
		else if(vecSet[TID+ngy].Boundary==CONDUCTOR) {
               for(i=0;i<info[0].spnum;i++) data[TID + i*ngx*ngy].sigma = 0;
			SD=0;
		}
	}
	else if(x==ngx-1) {
		if(vecSet[TID-ngy].Boundary==DIELECTRIC) {
			for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].sigma * info[i].q_density;
			Sigma[TID] = 2 * SumSig/vecSet[TID].Area;
		    SD = SumSig;
		}
		else if(vecSet[TID-ngy].Boundary==CONDUCTOR) {
			for(i=0;i<info[0].spnum;i++) data[TID + i*ngx*ngy].sigma = 0;
			SD=0;
		}
	}
	else if(y==0) {
		if(vecSet[TID+1].Boundary==DIELECTRIC) {
			for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].sigma * info[i].q_density;
			Sigma[TID] = 2 * SumSig/vecSet[TID].Area;
		    SD = SumSig;
		}
		else if(vecSet[TID+1].Boundary==CONDUCTOR) {
			for(i=0;i<info[0].spnum;i++) data[TID + i*ngx*ngy].sigma = 0;
			SD=0;
		}
	}
	else if(y==ngy-1) {
		if(vecSet[TID-1].Boundary==DIELECTRIC) {
			for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].sigma * info[i].q_density;
			Sigma[TID] = 2 * SumSig/vecSet[TID].Area;
		    SD = SumSig;
		}
		else if(vecSet[TID-1].Boundary==CONDUCTOR) {
			for(i=0;i<info[0].spnum;i++) data[TID + i*ngx*ngy].sigma = 0;
			SD=0;
		}
	}
	else if(vecSet[TID].Boundary==DIELECTRIC) {
		for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].sigma * info[i].q_density;
		Sigma[TID] = SumSig/vecSet[TID].Area;
		SD = SumSig;
	}
	else if(vecSet[TID].Boundary==CONDUCTOR) {
		for(i=0;i<info[0].spnum;i++) data[TID + i*ngx*ngy].sigma = 0;
		SD=0;
	}
	else {
		SD=0;
	}
    SumSig = 0.0;
    for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].den * info[i].q_density;
	sum=(SumSig + SD)/EPS0;
	Source[TID]=sum;
}
__global__ void Smooth_121_A(int ngx, int ngy, int nsp, GGA *vecSet, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=ngx*ngy*nsp) return;
    if(data[TID].den == 0) return;
    int ID;
    int x,y;
    ID = (int)TID%(ngx*ngy);
    x=ID/ngy; y=ID%ngy;
    // den >> smt_den at x direction
     if(vecSet[ID].Boundary==0){
        data[TID].smt_den = 0.25 * (data[TID-ngy].den + 2  *data[TID].den + data[TID+ngy].den);
	}else if(vecSet[ID].Boundary==CONDUCTOR || vecSet[ID].Boundary==DIELECTRIC) {
		if(vecSet[ID].Face==LEFT || vecSet[ID].Face==UL_CORN || vecSet[ID].Face==LL_CORN) 
               data[TID].smt_den = 0.5 * (data[TID].den + data[TID-ngy].den);
		else if(vecSet[ID].Face==RIGHT || vecSet[ID].Face==UR_CORN || vecSet[ID].Face==LR_CORN) 
               data[TID].smt_den = 0.5 * (data[TID].den + data[TID+ngy].den);
		else if(vecSet[ID].Face==UP || vecSet[ID].Face==DOWN) 
               data[TID].smt_den = 0.25 * (data[TID-ngy].den + 2*data[TID].den + data[TID+ngy].den);
	}else if(vecSet[ID].Boundary==DIRICHLET) {
		if(vecSet[ID].Face==LEFT || vecSet[ID].Face==UL_CORN || vecSet[ID].Face==LL_CORN) 
               data[TID].smt_den = 0.5 * (data[TID].den + data[TID-ngy].den);
		else if(vecSet[ID].Face==RIGHT || vecSet[ID].Face==UR_CORN || vecSet[ID].Face==LR_CORN) 
               data[TID].smt_den = 0.5 * (data[TID].den + data[TID+ngy].den);
		else if(vecSet[ID].Face==UP || vecSet[ID].Face==DOWN) {
			if(x==0) 
                    data[TID].smt_den = 0.5 * (data[TID].den+data[TID+ngy].den);
			else if(x==ngx-1) 
                    data[TID].smt_den = 0.5 * (data[TID].den+data[TID-ngy].den);
			else if(y==0 || y==ngy-1) 
                    data[TID].smt_den = 0.25 * (data[TID-ngy].den+2*data[TID].den+data[TID+ngy].den);
		}
	}else if(vecSet[ID].Boundary==NEUMANN) {
		if(vecSet[ID].Face==NO_FACE && vecSet[ID].CondID==0) {
			if(x==0) 
                    data[TID].smt_den=0.5*(data[TID].den+data[TID+ngy].den);
			else if(x==ngx-1) 
                    data[TID].smt_den=0.5*(data[TID].den+data[TID-ngy].den);
			else if(y==0 || y==ngy-1) 
                    data[TID].smt_den=0.25*(data[TID-ngy].den+2*data[TID].den+data[TID+ngy].den);
		}else if(vecSet[ID].Face==LEFT || vecSet[ID].Face==UL_CORN || vecSet[ID].Face==LL_CORN) 
               data[TID].smt_den=0.5*(data[TID].den+data[TID-ngy].den);
		else if(vecSet[ID].Face==RIGHT || vecSet[ID].Face==UR_CORN || vecSet[ID].Face==LR_CORN) 
               data[TID].smt_den=0.5*(data[TID].den+data[TID+ngy].den);
		else if(vecSet[ID].Face==UP || vecSet[ID].Face==DOWN) {
			if(x==0) 
                    data[TID].smt_den=0.5*(data[TID].den+data[TID+ngy].den);
			else if(x==ngx-1) 
                    data[TID].smt_den=0.5*(data[TID].den+data[TID-ngy].den);
			else if(y==0 || y==ngy-1) 
                    data[TID].smt_den=0.25*(data[TID-ngy].den+2*data[TID].den+data[TID+ngy].den);
		}
	}
}
__global__ void Smooth_121_B(int ngx, int ngy, int nsp, GGA *vecSet, GPG *data){
     int TID = threadIdx.x + blockIdx.x * blockDim.x;
     if(TID>=ngx*ngy*nsp) return;
     if(data[TID].smt_den == 0) return;
     int ID;
     int x,y;
     ID = (int)TID%(ngx*ngy);
     x=ID/ngy; y=ID%ngy;
     // smt_den >> den at y direction
     if(vecSet[ID].Boundary==0) {
          data[TID].den=0.25*(data[TID-1].smt_den+2*data[TID].smt_den+data[TID+1].smt_den);
	}else if(vecSet[ID].Boundary==CONDUCTOR || vecSet[ID].Boundary==DIELECTRIC) {
		if(vecSet[ID].Face==UP || vecSet[ID].Face==UL_CORN || vecSet[ID].Face==UR_CORN) 
               data[TID].den=0.5*(data[TID].smt_den+data[TID+1].smt_den);
		else if(vecSet[ID].Face==DOWN || vecSet[ID].Face==LR_CORN || vecSet[ID].Face==LL_CORN) 
               data[TID].den=0.5*(data[TID].smt_den+data[TID-1].smt_den);
		else if(vecSet[ID].Face==LEFT || vecSet[ID].Face==RIGHT) 
               data[TID].den=0.25*(data[TID-1].smt_den+2*data[TID].smt_den+data[TID+1].smt_den);
	}else if(vecSet[ID].Boundary==DIRICHLET) {
		if(vecSet[ID].Face==UP || vecSet[ID].Face==UL_CORN || vecSet[ID].Face==UR_CORN) 
               data[TID].den=0.5*(data[TID].smt_den+data[TID+1].smt_den);
		else if(vecSet[ID].Face==DOWN || vecSet[ID].Face==LR_CORN || vecSet[ID].Face==LL_CORN) 
               data[TID].den=0.5*(data[TID].smt_den+data[TID-1].smt_den);
		else if(vecSet[ID].Face==LEFT || vecSet[ID].Face==RIGHT) {
			if(y==0) 
                    data[TID].den=0.5*(data[TID].smt_den+data[TID+1].smt_den);
			else if(y==ngy-1) 
                    data[TID].den=0.5*(data[TID].smt_den+data[TID-1].smt_den);
			else if(x==0 || x==ngx-1) 
                    data[TID].den=0.25*(data[TID-1].smt_den+2*data[TID].smt_den+data[TID+1].smt_den);
		}
     }else if(vecSet[ID].Boundary==NEUMANN) {
		if(vecSet[ID].Face==NO_FACE && vecSet[ID].CondID==0) {
			if(y==0) 
                    data[TID].den=0.5*(data[TID].smt_den+data[TID+1].smt_den);
			else if(y==ngy-1) 
                    data[TID].den=0.5*(data[TID].smt_den+data[TID-1].smt_den);
			else if(x==0 || x==ngx-1) 
                    data[TID].den=0.25*(data[TID-1].smt_den+2*data[TID].smt_den+data[TID+1].smt_den);
		}else if(vecSet[ID].Face==UP || vecSet[ID].Face==UL_CORN || vecSet[ID].Face==UR_CORN) 
               data[TID].den=0.5*(data[TID].smt_den+data[TID+1].smt_den);
		else if(vecSet[ID].Face==DOWN || vecSet[ID].Face==LR_CORN || vecSet[ID].Face==LL_CORN) 
               data[TID].den=0.5*(data[TID].smt_den+data[TID-1].smt_den);
		else if(vecSet[ID].Face==LEFT || vecSet[ID].Face==RIGHT) {
			if(y==0) 
                    data[TID].den=0.5*(data[TID].smt_den+data[TID+1].smt_den);
			else if(y==ngy-1) 
                    data[TID].den=0.5*(data[TID].smt_den+data[TID-1].smt_den);
			else if(x==0 || x==ngx-1) 
                    data[TID].den=0.25*(data[TID-1].smt_den+2*data[TID].smt_den+data[TID+1].smt_den);
		}
	}
}
__global__ void DepositBoundary(int Gsize, int ngy, int nsp, GGA *vecSet, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize*nsp) return;
    int ID;
    ID = (int)TID%Gsize;

	if(vecSet[ID].Face==LEFT || vecSet[ID].Face==RIGHT || vecSet[ID].Face==UP || vecSet[ID].Face==DOWN) data[TID].den *= 2;
	else if(vecSet[ID].Face==UL_CORN || vecSet[ID].Face==LL_CORN || vecSet[ID].Face==UR_CORN || vecSet[ID].Face==LR_CORN) data[TID].den *= 4;
	if(vecSet[ID].Boundary==NEUMANN) data[TID].den *= 2;
	else if(vecSet[ID].Boundary==DIRICHLET && vecSet[ID].Face!=NO_FACE && vecSet[ID].CondID!=0) data[TID].den *= 2;

}
__global__ void DepositAtom(int Gsize, int ngy, Species *info, GCP *sp, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
	if(TID>=Gsize*info[0].spnum) return;
	int ID,isp;
	isp = (int)TID/Gsize;
    ID = (int)TID%Gsize;

	int i,k,PNC;
    float lx,ly;
	float WS,WN,ES,EN;
   
    PNC = data[TID].PtNumInCell;
    if(PNC==0) return;
    WS=0.0; WN=0.0; ES=0.0; EN=0.0;
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
		lx=sp[i].x; ly=sp[i].y;
		WS+=(1-lx)*(1-ly);
		WN+=(1-lx)*ly;
		ES+=lx*(1-ly);
		EN+=lx*ly;
		i+=Gsize;
	}
	atomicAdd(&data[TID].den,WS);
	atomicAdd(&data[TID+1].den,WN);
	atomicAdd(&data[TID+ngy].den,ES);
	atomicAdd(&data[TID+ngy+1].den,EN); 
	atomicAdd(&info[isp].np,PNC); 
}
__global__ void DepositInitDensity(int Gsize, Species *info, GPG *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=Gsize*info[0].spnum) return;
	int isp = (int)TID/Gsize;
    data[TID].den = 0.0;
	info[isp].np = 0;
}