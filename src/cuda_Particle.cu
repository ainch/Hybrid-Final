#include "cuda_Particle.cuh"

void Set_Particle_cuda(){  
    // This function calculates the following variables :
    // 1. dev_info_sp [Species] - [nsp]
    // 2. dev_sp [GCP] - [nsp * Sum of MAXNP]
    // 3. dev_G_sp [GPG] - [nsp * Gsize]
    int isp;
    printf(" Particle copy CPU to GPU. --> ");
    for(isp=0;isp<nsp;isp++){
        Total_maxnp += SP[isp].MAXNP;
    }
    //printf(" Total Maxnp = %d\n",Total_maxnp);
    // CPU >> CPU
    Host_sp = (GCP *)malloc(Total_maxnp * sizeof(GCP));
    GCPInit(Total_maxnp,Host_sp);
    Copy_HCPtoGCP(SP, PtD, Host_sp, Host_G_sp);
    // CPU >> GPU 
    checkCudaErrors(hipMalloc((void**)&dev_info_sp, nsp * sizeof(Species)));
    checkCudaErrors(hipMemcpy(dev_info_sp, SP, nsp * sizeof(Species), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&dev_sp, Total_maxnp * sizeof(GCP)));
    checkCudaErrors(hipMemcpy(dev_sp, Host_sp, Total_maxnp * sizeof(GCP), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&dev_G_sp, Gsize * nsp * sizeof(GPG)));
    checkCudaErrors(hipMemcpy(dev_G_sp, Host_G_sp, Gsize * nsp * sizeof(GPG), hipMemcpyHostToDevice));
    printf("Complete!\n");
}
void Copy_HCPtoGCP(Species *info, HCP *A, GCP *B, GPG *C){
    // OUTPUT : B[MAXNP*nsp], C[Gsize*nsp]
    int isp,k,PID,GID,XID,YID,index;
    int ID,SPID,SGID,PNC;
    float sum = 0.0;
    int count = 0;
    printf("Gsize=%d GID = %d\n",Gsize,Gsize*(nsp));
    // particle data >> buf data
    for (isp = 0; isp < nsp; isp++){
        SPID = info[isp].St_num;
        SGID = isp * Gsize;
        printf(" - %s : Start address = %d\n",info[isp].name,SPID);
        printf(" - %s : Number of PT = %d\n",info[isp].name,info[isp].np);
        for (k = 0; k < info[isp].np; k++) {
            //printf("[%d].np = %d, k = %d\n",isp,info[isp].np,k);
            XID = A[isp].x[k];
		    YID = A[isp].y[k];
            ID = XID * ngy + YID;
            PID = SPID + ID;
            GID = SGID + ID;
            PNC = C[GID].PtNumInCell;
            index = PID + PNC * Gsize;
            B[index].CellID = GID;
		    B[index].x = A[isp].x[k] - XID;
		    B[index].y = A[isp].y[k] - YID;
		    B[index].vx = A[isp].vx[k];
		    B[index].vy = A[isp].vy[k];
		    B[index].vz = A[isp].vz[k];
            C[GID].PtNumInCell++;
            //if(isp ==1 && XID <8) printf("np[%d] = XID[%d]\n",k,XID);
        }
        sum = 0.0;
        count = 0;
        for(k=0;k<Gsize;k++){
            GID = isp*Gsize + k;
            //printf("[%d]:GID=[%d]\n",k,GID);
            C[GID].den = info[isp].Denscale * C[GID].PtNumInCell;
            sum += C[GID].den;
            if( C[GID].den != 0)  count++;
        }
        printf(" - %s : Average Density = %g\n",info[isp].name,sum/count);
    }
}
void Copy_GCPtoHCP(Species *info, GCP *A, HCP *B, GPG *C){
    // INPUT : A[nsp * MAXNP]
    // OUTPUT : B[isp][NP_LIM], C[Gsize*nsp]
    int isp,i,j,k;
    int CID,AID;
    int XID,YID;
    for(isp=0;isp<nsp;isp++){
        k=0;
        for (i = 0; i < Gsize; i++){
            CID = isp * Gsize + i;
            for(j=0;j<C[CID].PtNumInCell;j++){
                AID = info[isp].St_num + i + j * Gsize;
                B[isp].CellID[k] = A[AID].CellID - isp * Gsize;
                //printf("[%d][%d]=[%d]\n",isp,k,B[isp].CellID[k]);
                XID = (int)(B[isp].CellID[k]/ngy);
                YID = (int)(B[isp].CellID[k]%ngy);
                B[isp].x[k] = XID + A[AID].x;
                B[isp].y[k] = YID + A[AID].y;
                B[isp].vx[k] = A[AID].vx;
                B[isp].vy[k] = A[AID].vy;
                B[isp].vz[k] = A[AID].vz;
                //printf("[%d][%d]=[%g][%g][%g][%g][%g]\n",isp,k,B[isp].x[k],B[isp].y[k],B[isp].vx[k],B[isp].vy[k],B[isp].vz[k]);
                k++;
            }
        }
        info[isp].np = k;
    }
    //exit(1);
}
void GCPInit(int size, GCP *A){
    int i;
    for (i = 0; i < size; i++){
        A[i].CellID = -1;
        A[i].x = 0.0;
        A[i].y = 0.0;
        A[i].vx = 0.0;
        A[i].vy = 0.0;
        A[i].vz = 0.0;
    }
}