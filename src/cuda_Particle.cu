#include "cuda_Particle.cuh"

void Set_Particle_cuda(){  
    // This function calculates the following variables :
    // 1. dev_info_sp [Species] - [nsp]
    // 2. dev_sp [GCP] - [nsp * Sum of MAXNP]
    // 3. dev_G_sp [GPG] - [nsp * Gsize]
    int isp,i;
    int Total_maxnp = 0;
    printf(" Particle copy CPU to GPU. --> ");
    for(isp=0;isp<nsp;isp++){
        Total_maxnp += SP[isp].MAXNP;
    }
    //printf(" Total Maxnp = %d\n",Total_maxnp);
    // CPU >> CPU
    Host_sp = (GCP *)malloc(Total_maxnp * sizeof(GCP));
    GCPInit(Total_maxnp,Host_sp);
    Copy_HCPtoGCP(SP, PtD, Host_sp, Host_G_sp);
    // CPU >> GPU 
    checkCudaErrors(hipMalloc((void**)&dev_info_sp, nsp * sizeof(Species)));
    checkCudaErrors(hipMemcpy(dev_info_sp, SP, nsp * sizeof(Species), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&dev_sp, Total_maxnp * sizeof(GCP)));
    checkCudaErrors(hipMemcpy(dev_sp, Host_sp, Total_maxnp * sizeof(GCP), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&dev_G_sp, Gsize * nsp * sizeof(GPG)));
    checkCudaErrors(hipMemcpy(dev_G_sp, Host_G_sp, Gsize * nsp * sizeof(GPG), hipMemcpyHostToDevice));
    printf("Complete!\n");
}
void Copy_HCPtoGCP(Species *info, HCP *A, GCP *B, GPG *C){
    // OUTPUT : B[MAXNP*nsp], C[Gsize*nsp]
    int isp,k,XID,YID,index;
    // particle data >> buf data
    for (isp = 0; isp < nsp; isp++){
	    for (k = 0; k < info[isp].np; k++) {
            XID = A[isp].x[k];
		    YID = A[isp].y[k];
            // index = nsp? + cellID? + ptNumInCell?
            index = info[isp].St_num + A[isp].CellID[k] + 
                        C[isp*Gsize + A[isp].CellID[k]].PtNumInCell * Gsize;
            //if(k == 0) printf("x=%d y=%d id=%d\n",XID,YID,index);
            B[index].CellID = A[isp].CellID[k];
		    B[index].x = A[isp].x[k] - XID;
		    B[index].y = A[isp].y[k] - YID;
		    B[index].vx = A[isp].vx[k];
		    B[index].vy = A[isp].vy[k];
		    B[index].vz = A[isp].vz[k];
            C[isp*Gsize + A[isp].CellID[k]].PtNumInCell++;
            //if(k<5){
                //printf("ISP[%d][%d]:x[%g]y[%g]>ID[%d]:x[%g]\n",isp,k,A[isp].x[k],A[isp].y[k],A[isp].CellID[k],B[index].x);
                //printf("np[%d] = %d\n",isp*Gsize + A[isp].CellID[k],C[isp*Gsize + A[isp].CellID[k]].PtNumInCell);
            //} 
        }
        for(k=0;k<Gsize;k++){
            C[isp*Gsize + A[isp].CellID[k]].den = C[isp*Gsize + A[isp].CellID[k]].PtNumInCell;
            //printf("ISP[%d],ID[%d] = NP[%d], ",isp,k,C[isp*Gsize + k].PtNumInCell);
            //printf("MaxNP[%d]\n",C[isp*Gsize + k].MaxPtNumInCell);
        }
    }
}
void Copy_GCPtoHCP(Species *info, GCP *A,HCP *B, GPG *C){
    // OUTPUT : B[isp][NP_LIM], C[Gsize*nsp]
    int isp,k,XID,YID,index;
    exit(1);
    for (isp = 0; isp < nsp; isp++){
    }
}
void GCPInit(int size, GCP *A){
    int i;
    for (i = 0; i < size; i++){
        A[i].CellID = -1;
        A[i].x = 0.0;
        A[i].y = 0.0;
        A[i].vx = 0.0;
        A[i].vy = 0.0;
        A[i].vz = 0.0;
    }
}