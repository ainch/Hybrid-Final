#include "hip/hip_runtime.h"
#include "cuda_mcc.cuh"
void MCC_ArO2_cuda(){
	MCC_ArO2_Basic<<<MCC_GRID, MCC_BLOCK>>>(Gsize, Csize, ngy, nsp, dt, DT_MCCn, dt_mcc, idx, idy, h_nvel, dev_vsave, devStates, N_LOGX, idLOGX, dev_SigmaV,
												dev_Coll_Flag, dev_ArO2CX, TnRct, dev_MCC_rate, dev_FG, dev_C_F, dev_GvecSet, dev_info_sp, dev_G_sp, dev_sp);
	hipDeviceSynchronize();
}
void MCC_O2_cuda(){
	MCC_O2_Basic<<<MCC_GRID, MCC_BLOCK>>>(Gsize, Csize, ngy, nsp, dt, DT_MCCn, dt_mcc, idx, idy, h_nvel, dev_vsave, devStates, N_LOGX, idLOGX, dev_SigmaV,
												dev_Coll_Flag, dev_O2CX, TnRct, dev_MCC_rate, dev_FG, dev_C_F, dev_GvecSet, dev_info_sp, dev_G_sp, dev_sp);
	hipDeviceSynchronize();
}
void MCC_Ar_cuda(){
	MCC_Ar_Basic<<<MCC_GRID, MCC_BLOCK>>>(Gsize, Csize, ngy, nsp, dt, DT_MCCn, dt_mcc, idx, idy, h_nvel, dev_vsave, devStates, N_LOGX, idLOGX, dev_SigmaV,
												dev_Coll_Flag, dev_ArCX, TnRct, dev_MCC_rate, dev_FG, dev_C_F, dev_GvecSet, dev_info_sp, dev_G_sp, dev_sp);
	hipDeviceSynchronize();
}
__global__ void MCC_ArO2_Basic(int Gsize, int Csize, int ngy, int nsp, float dt, int MCCn, float dtm, float idx,float idy, int nvel, float *vsave,
											hiprandState *states, int N_LOGX, float idLOGX, MCC_sigmav *sigv, CollF *CollP, ArO2CollD *CX, int TnRct, float*MCCR,
											Fluid *infoF, GFC *Fluid, GGA *BG, Species *info, GPG *data, GCP *sp){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	if(TID>=nsp*Gsize) return;
	int isp = TID/Gsize;
	ArO2_Collision_Check(Gsize, Csize, ngy, TID, dt, MCCn, dtm, idx, idy, states, info, data, sp, sigv, BG, Fluid);
	switch (isp){
	case 0:
		ArO2_Electron(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 1:
		ArO2_Ar_ion(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 2:
		ArO2_O2_ion(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 3:
		ArO2_O_ion(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 4:
		ArO2_O_negative(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	default:
		break;
	}
	/*	
	// Collision check
	//if(TID !=(125*110+55)) return;
	ArO2_Collision_Check_TEST(Gsize, Csize, ngy, TID, dt, MCCn, dtm, idx, idy, states, info, data, sp, sigv, BG, Fluid);
	switch (isp){
	case 0:
		ArO2_Electron_TEST(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 1:
		ArO2_Ar_ion_TEST(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 2:
		ArO2_O2_ion_TEST(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 3:
		ArO2_O_ion_TEST(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 4:
		ArO2_O_negative_TEST(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	default:
		break;
	}	
	*/
}	
__global__ void MCC_O2_Basic(int Gsize, int Csize, int ngy, int nsp, float dt, int MCCn, float dtm, float idx,float idy, int nvel, float *vsave,
											hiprandState *states, int N_LOGX, float idLOGX, MCC_sigmav *sigv, CollF *CollP, O2CollD *CX, int TnRct, float*MCCR,
											Fluid *infoF, GFC *Fluid, GGA *BG, Species *info, GPG *data, GCP *sp){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	if(TID>=nsp*Gsize) return;
	int isp = TID/Gsize;
	// Collision check
	O2Collision_Check(Gsize, Csize, ngy, TID, dt, MCCn, dtm, idx, idy, states, info, data, sp, sigv, BG, Fluid);
	switch (isp){
	case 0:
		O2_Electron(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 1:
		O2_O2_ion(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 2:
		O2_O_ion(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 3:
		O2_O_negative(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	default:
		break;
	}	
}	
__global__ void MCC_Ar_Basic(int Gsize, int Csize, int ngy, int nsp, float dt, int MCCn, float dtm, float idx,float idy, int nvel, float *vsave,
											hiprandState *states, int N_LOGX, float idLOGX, MCC_sigmav *sigv, CollF *CollP, ArCollD *CX, int TnRct, float*MCCR,
											Fluid *infoF, GFC *Fluid, GGA *BG, Species *info, GPG *data, GCP *sp){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	// Direct Method
	//if(TID>=Gsize) return;
	//Direct_Argon_ArIon(Gsize, ngy, TID, MCCn, dt, nvel, vsave, states, info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG, Fluid);
	//Direct_Argon_Electron(Gsize, ngy, TID, MCCn, dtm, nvel, vsave, states, info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG, Fluid);
	// Memory mode
	if(TID>=nsp*Gsize) return;
	int isp = TID/Gsize;
	Ar_Collision_Check(Gsize, Csize, ngy, TID, dt, MCCn, dtm, idx, idy, states, info, data, sp, sigv, BG, Fluid);
	switch (isp){
	case 0:
		Ar_Electron(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	case 1:
		Ar_Ar_ion(Gsize, ngy, TID, nvel, vsave, states,  info, data, sp, N_LOGX, idLOGX, sigv, CollP, CX, TnRct, MCCR, BG);
		break;
	default:
		break;
	}	
}	
__device__ void dev_maxwellv(float *vx_local,float *vy_local,float *vz_local,float vsaven,float vti,float Rphi,float Rthe){
	float aphi,sintheta,costheta;
	aphi=2.0*M_PI*Rphi;
	costheta=1-2*Rthe;
	sintheta=sqrt(1-costheta*costheta);
	*vx_local=vti*vsaven*sintheta*__cosf(aphi);
	*vy_local=vti*vsaven*sintheta*__sinf(aphi);
	*vz_local=vti*vsaven*costheta;
}
__device__ void dev_anewvel(float energy,float vel,float *n_vx,float *n_vy,float *n_vz,int e_flag,float massrate,float rand1,float rand2){
	float phi1,cosphi,sinphi,coschi,sinchi,up1,up2,up3;
	float mag,r11,r12,r13,r21,r22,r23,r31,r32,r33;
	//printf("[] : %g %g %g %g %g %g %g %g\n",energy,vel,*n_vx,*n_vy,*n_vz,massrate,rand1,rand2);
	if(energy<1e-30)coschi=1;
	else coschi=(energy+2.0-2.0*__powf(float(energy+1.0),rand1))/energy;
	sinchi=sqrt(fabs(1.0-coschi*coschi));
	phi1=2*M_PI*rand2;
	cosphi=__cosf(phi1);
	sinphi=__sinf(phi1);
	if(e_flag) vel*=sqrt(1.0-2.0*(1-coschi)*1*massrate);				// hur insert Ver015
	r13=*n_vx; r23=*n_vy; r33=*n_vz;
	if(r33==1.0){up1=0; up2=1; up3=0;}
	else{up1=0; up2=0; up3=1;}
	r12=r23*up3-r33*up2;
	r22=r33*up1-r13*up3;
	r32=r13*up2-r23*up1;
	mag=sqrt(r12*r12+r22*r22+r32*r32);
	r12/=mag; r22/=mag; r32/=mag;
	r11=r22*r33-r32*r23;
	r21=r32*r13-r12*r33;
	r31=r12*r23-r22*r13;
	*n_vx=vel*(r11*sinchi*cosphi+r12*sinchi*sinphi+r13*coschi);
	*n_vy=vel*(r21*sinchi*cosphi+r22*sinchi*sinphi+r23*coschi);
	*n_vz=vel*(r31*sinchi*cosphi+r32*sinchi*sinphi+r33*coschi);
}
__device__ void dev_newvel_IONSC(float *vx_sc,float *vy_sc,float *vz_sc,float vel,float rand1,float rand2){
	float coschi,sinchi,phi1,cosphi,sinphi;
	float r11,r21,r31,r13,r23,r33,r12,r22,r32,up1,up2,up3;
	float mag;
	coschi= sqrt(rand1);
	sinchi= sqrt(fabs(1.0-coschi*coschi));
	phi1=2*M_PI*rand2;
	cosphi=__cosf(phi1);
	sinphi=__sinf(phi1);
	r13=*vx_sc/vel;
	r23=*vy_sc/vel;
	r33=*vz_sc/vel;
	if(r33==1.0){up1=0; up2=1; up3=0;}
	else{up1=0; up2=0; up3=1;}
	r12=r23*up3-r33*up2;
	r22=r33*up1-r13*up3;
	r32=r13*up2-r23*up1;
	mag=sqrt(r12*r12+r22*r22+r32*r32);
	r12/= mag;
	r22/= mag;
	r32/= mag;
	r11=r22*r33-r32*r23;
	r21=r32*r13-r12*r33;
	r31=r12*r23-r22*r13;
	*vx_sc=vel*coschi*(r11*sinchi*cosphi+r12*sinchi*sinphi+r13*coschi);
	*vy_sc=vel*coschi*(r21*sinchi*cosphi+r22*sinchi*sinphi+r23*coschi);
	*vz_sc=vel*coschi*(r31*sinchi*cosphi+r32*sinchi*sinphi+r33*coschi);
}
void Set_NullCollisionTime_cuda(){
    // This function calculates the following variables :
    // 1. int DT_MCCn
    // 2. float dt_mcc
    // 3. dev_Coll_Flag [CollF] - [TnRct]
    // 4. dev_**CX [**CollD] - [N_LOGX]
    // 5. Max_sigma_v[num_a] ,Ttarget[num_a] ColProb[num_b]
    //    Argon - [3][2]
    //    Oxygen - [20][4]
    //    ArO2 - [25][5]
    int i;
    int num_a; // Number of (Projectile + target)
    float engy;
	float *sigma;
    float *Ttarget,*ColProb;
    float ratio, prob_cut = 0.08; // = 5%. Collision rate per time step
    //Cross section Data copy  CPU >> GPU
    checkCudaErrors(hipMalloc((void**)&dev_Coll_Flag, TnRct * sizeof(CollF)));
    checkCudaErrors(hipMemcpy(dev_Coll_Flag, Coll_Flag, TnRct * sizeof(CollF), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**)&dev_MCC_rate, Msize * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_MCC_rate, MCC_rate, Msize * sizeof(float), hipMemcpyHostToDevice));
    if(MainGas == ARGON){
        checkCudaErrors(hipMalloc((void**)&dev_ArCX, N_LOGX * sizeof(ArCollD)));
        checkCudaErrors(hipMemcpy(dev_ArCX, Ar_Data, N_LOGX * sizeof(ArCollD), hipMemcpyHostToDevice));
        num_a = 3; 			
        Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			// Electron
			engy = Ar_Data[i].xee;
			sigma[0] = Ar_Data[i].cx_0+Ar_Data[i].cx_1+Ar_Data[i].cx_2+Ar_Data[i].cx_3; // e + Ar
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = Ar_Data[i].cx_4;    // e + Ar*
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			// Ar ion
			engy = Ar_Data[i].xee;
			sigma[2] = Ar_Data[i].cx_5+Ar_Data[i].cx_6;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[2]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * BG[0].InitDens);
		ColProb[0] = Ttarget[0] + Ttarget[1];
		ColProb[1] = Ttarget[2];
		if(ColProb[0] > prob_cut) {
			DT_MCCn = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCCn;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCCn = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " - Total Collision probability per Time step\n");
		fprintf(stderr, "   Electon - %2.4f %\n",ColProb[0] * 100);
		fprintf(stderr, "   Ar ion  - %2.4f %\n",ColProb[1] * 100);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCCn, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else if(MainGas == OXYGEN){
        checkCudaErrors(hipMalloc((void**)&dev_O2CX, N_LOGX * sizeof(O2CollD)));
        checkCudaErrors(hipMemcpy(dev_O2CX, O2_Data, N_LOGX * sizeof(O2CollD), hipMemcpyHostToDevice));
		num_a = 20; 			
		Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			engy = O2_Data[i].xee;
			sigma[0] = O2_Data[i].cx_0+O2_Data[i].cx_1+O2_Data[i].cx_2+O2_Data[i].cx_3+O2_Data[i].cx_4;
			sigma[0] += O2_Data[i].cx_5+O2_Data[i].cx_6+O2_Data[i].cx_7+O2_Data[i].cx_8+O2_Data[i].cx_9;
			sigma[0] += O2_Data[i].cx_10+O2_Data[i].cx_11+O2_Data[i].cx_12+O2_Data[i].cx_13;
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = O2_Data[i].cx_14+O2_Data[i].cx_15+O2_Data[i].cx_16+O2_Data[i].cx_17+O2_Data[i].cx_18;
			sigma[1] += O2_Data[i].cx_19+O2_Data[i].cx_20+O2_Data[i].cx_21;
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			sigma[2] = O2_Data[i].cx_22+O2_Data[i].cx_23+O2_Data[i].cx_24+O2_Data[i].cx_25+O2_Data[i].cx_26;
			sigma[2] += O2_Data[i].cx_27+O2_Data[i].cx_28+O2_Data[i].cx_29;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[2]);
			sigma[3] = O2_Data[i].cx_30;
			Host_SigmaV[3].val = max(Host_SigmaV[3].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[3]);
			sigma[4] = O2_Data[i].cx_31;
			Host_SigmaV[4].val = max(Host_SigmaV[4].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[4]);
			sigma[5] = O2_Data[i].cx_32+O2_Data[i].cx_33+O2_Data[i].cx_34+O2_Data[i].cx_35+O2_Data[i].cx_36;
			sigma[5] += O2_Data[i].cx_37+O2_Data[i].cx_38;
			Host_SigmaV[5].val = max(Host_SigmaV[5].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[5]);
			sigma[6] = O2_Data[i].cx_39+O2_Data[i].cx_40;
			Host_SigmaV[6].val = max(Host_SigmaV[6].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[6]);
			sigma[7] = O2_Data[i].cx_41 + O2_Data[i].cx_42;
			Host_SigmaV[7].val = max(Host_SigmaV[7].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[7]);
			sigma[8] = O2_Data[i].cx_43;
			Host_SigmaV[8].val = max(Host_SigmaV[8].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[8]);
			sigma[9] = O2_Data[i].cx_44;
			Host_SigmaV[9].val = max(Host_SigmaV[9].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[9]);
			sigma[10] = O2_Data[i].cx_45;
			Host_SigmaV[10].val = max(Host_SigmaV[10].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[10]);
			sigma[11] = O2_Data[i].cx_46;
			Host_SigmaV[11].val = max(Host_SigmaV[11].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[11]);
			sigma[12] = O2_Data[i].cx_47;
			Host_SigmaV[12].val = max(Host_SigmaV[12].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[12]);
			sigma[13] = O2_Data[i].cx_48 + O2_Data[i].cx_49 + O2_Data[i].cx_50;
			Host_SigmaV[13].val = max(Host_SigmaV[13].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[13]);
			sigma[14] = O2_Data[i].cx_51;
			Host_SigmaV[14].val = max(Host_SigmaV[14].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[14]);
			sigma[15] = O2_Data[i].cx_52;
			Host_SigmaV[15].val = max(Host_SigmaV[15].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[15]);
			sigma[16] = O2_Data[i].cx_53 + O2_Data[i].cx_54;
			Host_SigmaV[16].val = max(Host_SigmaV[16].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[16]);
			sigma[17] = O2_Data[i].cx_55;
			Host_SigmaV[17].val = max(Host_SigmaV[17].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[17]);
			sigma[18] = O2_Data[i].cx_56;
			Host_SigmaV[18].val = max(Host_SigmaV[18].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[18]);
			sigma[19] = O2_Data[i].cx_57;
			Host_SigmaV[19].val = max(Host_SigmaV[19].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[19]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);// E + O2
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);// E + O2A
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * FG[1].InitDens);// E + O2B
		Ttarget[3] = 1.0-exp(-1 * dt * Host_SigmaV[3].val * SP[3].InitDens);// E + O-
		Ttarget[4] = 1.0-exp(-1 * dt * Host_SigmaV[4].val * SP[1].InitDens);// E + O2^
		Ttarget[5] = 1.0-exp(-1 * dt * Host_SigmaV[5].val * FG[2].InitDens);// E + OP
		Ttarget[6] = 1.0-exp(-1 * dt * Host_SigmaV[6].val * FG[3].InitDens);// E + OD
		Ttarget[7] = 1.0-exp(-1 * dt * Host_SigmaV[7].val * BG[0].InitDens);// O- + O2
		Ttarget[8] = 1.0-exp(-1 * dt * Host_SigmaV[8].val * FG[2].InitDens);// O- + OP
		Ttarget[9] = 1.0-exp(-1 * dt * Host_SigmaV[9].val* SP[1].InitDens);// O- + O2^
		Ttarget[10] = 1.0-exp(-1 * dt * Host_SigmaV[10].val * SP[2].InitDens);// O- + O^
		Ttarget[11] = 1.0-exp(-1 * dt * Host_SigmaV[11].val * FG[0].InitDens);// O- + O2A
		Ttarget[12] = 1.0-exp(-1 * dt * Host_SigmaV[12].val * FG[2].InitDens);// O2^ + OP
		Ttarget[13] = 1.0-exp(-1 * dt * Host_SigmaV[13].val * BG[0].InitDens);// O2^ + O2
		Ttarget[14] = 1.0-exp(-1 * dt * Host_SigmaV[14].val * FG[0].InitDens);// O2^ + O2A
		Ttarget[15] = 1.0-exp(-1 * dt * Host_SigmaV[15].val * FG[1].InitDens);// O2^ + O2B
		Ttarget[16] = 1.0-exp(-1 * dt * Host_SigmaV[16].val * BG[0].InitDens);// O^ + O2
		Ttarget[17] = 1.0-exp(-1 * dt * Host_SigmaV[17].val * FG[2].InitDens);// O^ + OP
		Ttarget[18] = 1.0-exp(-1 * dt * Host_SigmaV[18].val* FG[0].InitDens);// O^ + O2A
		Ttarget[19] = 1.0-exp(-1 * dt * Host_SigmaV[19].val * FG[1].InitDens);// O^ + O2B
		ColProb[0] = Ttarget[0] + Ttarget[1] + Ttarget[2] + Ttarget[3] + Ttarget[4] + Ttarget[5] + Ttarget[6];// Electron
		ColProb[1] = Ttarget[7] + Ttarget[8] + Ttarget[9] + Ttarget[10] + Ttarget[11];// O-
		ColProb[2] = Ttarget[12] + Ttarget[13] + Ttarget[14] + Ttarget[15];// O2+
		ColProb[3] = Ttarget[16] + Ttarget[17] + Ttarget[18] + Ttarget[19];// O+
		if(ColProb[0] > prob_cut) {
			DT_MCCn = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCCn;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCCn = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " - Total Collision probability per Time step\n");
		fprintf(stderr, "   Electon - %2.4f %\n",ColProb[0] * 100);
		fprintf(stderr, "   O- ion  - %2.4f %\n",ColProb[1] * 100);
		fprintf(stderr, "   O2+ion  - %2.4f %\n",ColProb[2] * 100);
		fprintf(stderr, "   O+ ion  - %2.4f %\n",ColProb[3] * 100);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCCn, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else if(MainGas == ARO2){
        checkCudaErrors(hipMalloc((void**)&dev_ArO2CX, N_LOGX * sizeof(ArO2CollD)));
        checkCudaErrors(hipMemcpy(dev_ArO2CX, ArO2_Data, N_LOGX * sizeof(ArO2CollD), hipMemcpyHostToDevice));
		num_a = 25; 			
		Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			engy = ArO2_Data[i].xee;
			sigma[0] = ArO2_Data[i].cx_0+ArO2_Data[i].cx_1+ArO2_Data[i].cx_2+ArO2_Data[i].cx_3; // e + Ar
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = ArO2_Data[i].cx_4;    // e + Ar*
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			sigma[2] = ArO2_Data[i].cx_5+ArO2_Data[i].cx_6+ArO2_Data[i].cx_7+ArO2_Data[i].cx_8+ArO2_Data[i].cx_9;
			sigma[2] += ArO2_Data[i].cx_10+ArO2_Data[i].cx_11+ArO2_Data[i].cx_12+ArO2_Data[i].cx_13+ArO2_Data[i].cx_14;
			sigma[2] += ArO2_Data[i].cx_15+ArO2_Data[i].cx_16+ArO2_Data[i].cx_17+ArO2_Data[i].cx_18;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[2]);// e + O2
			sigma[3] = ArO2_Data[i].cx_19+ArO2_Data[i].cx_20+ArO2_Data[i].cx_21+ArO2_Data[i].cx_22+ArO2_Data[i].cx_23;
			sigma[3] += ArO2_Data[i].cx_24+ArO2_Data[i].cx_25+ArO2_Data[i].cx_26;
			Host_SigmaV[3].val = max(Host_SigmaV[3].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[3]);// e + O2A
			sigma[4] = ArO2_Data[i].cx_27+ArO2_Data[i].cx_28+ArO2_Data[i].cx_29+ArO2_Data[i].cx_30+ArO2_Data[i].cx_31;
			sigma[4] += ArO2_Data[i].cx_32+ArO2_Data[i].cx_33+ArO2_Data[i].cx_34;
			Host_SigmaV[4].val = max(Host_SigmaV[4].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[4]);// e + O2B
			sigma[5] = ArO2_Data[i].cx_35;
			Host_SigmaV[5].val = max(Host_SigmaV[5].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[5]);
			sigma[6] = ArO2_Data[i].cx_36;
			Host_SigmaV[6].val = max(Host_SigmaV[6].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[6]);
			sigma[7] = ArO2_Data[i].cx_37+ArO2_Data[i].cx_38+ArO2_Data[i].cx_39+ArO2_Data[i].cx_40+ArO2_Data[i].cx_41;
			sigma[7] += ArO2_Data[i].cx_42+ArO2_Data[i].cx_43;
			Host_SigmaV[7].val = max(Host_SigmaV[7].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[7]);
			sigma[8] = ArO2_Data[i].cx_44+ArO2_Data[i].cx_45;
			Host_SigmaV[8].val = max(Host_SigmaV[8].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[8]);
			sigma[9] = ArO2_Data[i].cx_46 + ArO2_Data[i].cx_47;
			Host_SigmaV[9].val = max(Host_SigmaV[9].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[9]);
			sigma[10] = ArO2_Data[i].cx_48;
			Host_SigmaV[10].val = max(Host_SigmaV[10].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[10]);
			sigma[11] = ArO2_Data[i].cx_49;
			Host_SigmaV[11].val = max(Host_SigmaV[11].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[11]);
			sigma[12] = ArO2_Data[i].cx_50;
			Host_SigmaV[12].val = max(Host_SigmaV[12].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[12]);
			sigma[13] = ArO2_Data[i].cx_51;
			Host_SigmaV[13].val = max(Host_SigmaV[13].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[13]);
			sigma[14] = ArO2_Data[i].cx_52;
			Host_SigmaV[14].val = max(Host_SigmaV[14].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[14]);
			sigma[15] = ArO2_Data[i].cx_53 + ArO2_Data[i].cx_54 + ArO2_Data[i].cx_55;
			Host_SigmaV[15].val = max(Host_SigmaV[15].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[15]);
			sigma[16] = ArO2_Data[i].cx_56;
			Host_SigmaV[16].val = max(Host_SigmaV[16].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[16]);
			sigma[17] = ArO2_Data[i].cx_57;
			Host_SigmaV[17].val = max(Host_SigmaV[17].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[17]);
			sigma[18] = ArO2_Data[i].cx_58 + ArO2_Data[i].cx_59;
			Host_SigmaV[18].val = max(Host_SigmaV[18].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[18]);
			sigma[19] = ArO2_Data[i].cx_60 + ArO2_Data[i].cx_61;
			Host_SigmaV[19].val = max(Host_SigmaV[19].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[19]);
			sigma[20] = ArO2_Data[i].cx_62;
			Host_SigmaV[20].val = max(Host_SigmaV[20].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[20]);
			sigma[21] = ArO2_Data[i].cx_63;
			Host_SigmaV[21].val = max(Host_SigmaV[21].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[21]);
			sigma[22] = ArO2_Data[i].cx_64;
			Host_SigmaV[22].val = max(Host_SigmaV[22].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[22]);
			sigma[23] = ArO2_Data[i].cx_65+ArO2_Data[i].cx_66;
			Host_SigmaV[23].val = max(Host_SigmaV[23].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[23]);
			sigma[24] = ArO2_Data[i].cx_67;
			Host_SigmaV[24].val = max(Host_SigmaV[24].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[24]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);// E + Ar
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);// E + Ar*
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * BG[1].InitDens);// E + O2
		Ttarget[3] = 1.0-exp(-1 * dt * Host_SigmaV[3].val * FG[1].InitDens);// E + O2A
		Ttarget[4] = 1.0-exp(-1 * dt * Host_SigmaV[4].val * FG[2].InitDens);// E + O2B
		Ttarget[5] = 1.0-exp(-1 * dt * Host_SigmaV[5].val * SP[4].InitDens);// E + O-
		Ttarget[6] = 1.0-exp(-1 * dt * Host_SigmaV[6].val * SP[2].InitDens);// E + O2^
		Ttarget[7] = 1.0-exp(-1 * dt * Host_SigmaV[7].val * FG[3].InitDens);// E + OP
		Ttarget[8] = 1.0-exp(-1 * dt * Host_SigmaV[8].val * FG[4].InitDens);// E + OD
		Ttarget[9] = 1.0-exp(-1 * dt * Host_SigmaV[9].val * BG[1].InitDens);// O- + O2
		Ttarget[10] = 1.0-exp(-1 * dt * Host_SigmaV[10].val * FG[3].InitDens);// O- + OP
		Ttarget[11] = 1.0-exp(-1 * dt * Host_SigmaV[11].val* SP[2].InitDens);// O- + O2^
		Ttarget[12] = 1.0-exp(-1 * dt * Host_SigmaV[12].val * SP[3].InitDens);// O- + O^
		Ttarget[13] = 1.0-exp(-1 * dt * Host_SigmaV[13].val * FG[1].InitDens);// O- + O2A
		Ttarget[14] = 1.0-exp(-1 * dt * Host_SigmaV[14].val * FG[3].InitDens);// O2^ + OP
		Ttarget[15] = 1.0-exp(-1 * dt * Host_SigmaV[15].val * BG[1].InitDens);// O2^ + O2
		Ttarget[16] = 1.0-exp(-1 * dt * Host_SigmaV[16].val * FG[1].InitDens);// O2^ + O2A
		Ttarget[17] = 1.0-exp(-1 * dt * Host_SigmaV[17].val * FG[2].InitDens);// O2^ + O2B
		Ttarget[18] = 1.0-exp(-1 * dt * Host_SigmaV[18].val * BG[0].InitDens);// O2^ + Ar
		Ttarget[19] = 1.0-exp(-1 * dt * Host_SigmaV[19].val * BG[1].InitDens);// O^ + O2
		Ttarget[20] = 1.0-exp(-1 * dt * Host_SigmaV[20].val * FG[3].InitDens);// O^ + OP
		Ttarget[21] = 1.0-exp(-1 * dt * Host_SigmaV[21].val* FG[1].InitDens);// O^ + O2A
		Ttarget[22] = 1.0-exp(-1 * dt * Host_SigmaV[22].val * FG[2].InitDens);// O^ + O2B
		Ttarget[23] = 1.0-exp(-1 * dt * Host_SigmaV[23].val * BG[0].InitDens);// Ar^ + Ar
		Ttarget[24] = 1.0-exp(-1 * dt * Host_SigmaV[24].val * BG[1].InitDens);// Ar^ + O2
		ColProb[0] = Ttarget[0] + Ttarget[1] + Ttarget[2] + Ttarget[3] + Ttarget[4] + Ttarget[5] + Ttarget[6] + Ttarget[7] + Ttarget[8];// Electron
		ColProb[1] = Ttarget[23] + Ttarget[24];// Ar+
		ColProb[2] = Ttarget[14] + Ttarget[15] + Ttarget[16] + Ttarget[17] + Ttarget[18];// O2+
		ColProb[3] = Ttarget[19] + Ttarget[20] + Ttarget[21] + Ttarget[22];// O+
		ColProb[4] = Ttarget[9] + Ttarget[10] + Ttarget[11] + Ttarget[12] + Ttarget[13];// O-
		if(ColProb[0] > prob_cut) {
			DT_MCCn = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCCn;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCCn = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " - Total Collision probability per Time step\n");
		fprintf(stderr, "   Electon - %2.4f % - [%1.5g][%1.5g][%1.5g][%1.5g][%1.5g][%1.5g][%1.5g][%1.5g][%1.5g]\n"
					,ColProb[0] * 100,Ttarget[0],Ttarget[1],Ttarget[2],Ttarget[3],Ttarget[4],Ttarget[5],Ttarget[6],Ttarget[7],Ttarget[8]);
		fprintf(stderr, "   Ar+ion  - %2.4f % - [%1.5g][%1.5g]\n",ColProb[1] * 100,Ttarget[23],Ttarget[24]);
		fprintf(stderr, "   O2+ion  - %2.4f % - [%1.5g][%1.5g][%1.5g][%1.5g][%1.5g]\n",ColProb[2] * 100,Ttarget[14],Ttarget[15],Ttarget[16],Ttarget[17],Ttarget[18]);
		fprintf(stderr, "   O+ ion  - %2.4f % - [%1.5g][%1.5g][%1.5g][%1.5g]\n",ColProb[3] * 100,Ttarget[19],Ttarget[20],Ttarget[21],Ttarget[22]);
		fprintf(stderr, "   O- ion  - %2.4f % - [%1.5g][%1.5g][%1.5g][%1.5g][%1.5g]\n",ColProb[4] * 100,Ttarget[9],Ttarget[10],Ttarget[11],Ttarget[12],Ttarget[13]);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCCn, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else{
        printf("Error : MainGas = %d\n",MainGas);
        exit(1);
    }
    //Maximum sigma*v copy  CPU >> GPU
    checkCudaErrors(hipMalloc((void**)&dev_SigmaV, num_a * sizeof(MCC_sigmav)));
    checkCudaErrors(hipMemcpy(dev_SigmaV, Host_SigmaV, num_a * sizeof(MCC_sigmav), hipMemcpyHostToDevice));
    printf("MCC Initializing Complete!\n");
}
