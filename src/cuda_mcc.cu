#include "hip/hip_runtime.h"
#include "cuda_mcc.cuh"
void MCC_Ar_cuda(){
	//MCC_Ar_Basic<<<MCC_GRID2, MCC_BLOCK2>>>(Gsize, ngy, devStates, dev_info_sp, dev_sp, dev_G_sp, dev_GvecSet);
    void *kernelArgs[] = {
        (void*)&Gsize,(void*)&ngy,(void*)&DT_MCCn,(void*)&dt_mcc,
		(void*)&idx,(void*)&idy,(void*)&h_nvel,(void*)&dev_vsave,
		(void*)&devStates, (void*)&dev_SigmaV,(void*)&dev_Coll_Flag,(void*)&dev_ArCX,
		(void*)&dev_FG, (void*)&dev_C_F,(void*)&dev_GvecSet, 
		(void*)&dev_info_sp,  (void*)&dev_G_sp, (void*)&dev_sp,
    };
    hipLaunchCooperativeKernel((void *)MCC_Ar_cooper,MCC_GRID,MCC_BLOCK,kernelArgs, sMemSize_MCC, NULL);
    hipDeviceSynchronize();
	exit(1);
}
__global__ void MCC_Ar_cooper(int Gsize, int ngy, float dt, int MCCn, float dtm,float idx,float idy, int nvel, float *vsave,
											hiprandState *states, MCC_sigmav *sigv, CollF *CollP, ArCollD *CX, 
											Fluid *infoF, GFC *Fluid, GGA *BG, Species *info, GPG *data, GCP *sp){
	cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();

	for (int i=grid.thread_rank(); i < 2*Gsize; i+= grid.size()){
		printf("I = %d\n",i);
	}
}
void Set_NullCollisionTime_cuda(){
    // This function calculates the following variables :
    // 1. int DT_MCCn
    // 2. float dt_mcc
    // 3. dev_Coll_Flag [CollF] - [TnRct]
    // 4. dev_**CX [**CollD] - [N_LOGX]
    // 5. Max_sigma_v[num_a] ,Ttarget[num_a] ColProb[num_b]
    //    Argon - [3][2]
    //    Oxygen - [20][4]
    //    ArO2 - [25][5]
    int i;
    int num_a; // Number of (Projectile + target)
    float engy;
	float *sigma;
    float *Ttarget,*ColProb;
    float ratio, prob_cut = 0.05; // = 5%. Collision rate per time step
    //Cross section Data copy  CPU >> GPU
    checkCudaErrors(hipMalloc((void**)&dev_Coll_Flag, TnRct * sizeof(CollF)));
    checkCudaErrors(hipMemcpy(dev_Coll_Flag, Coll_Flag, TnRct * sizeof(CollF), hipMemcpyHostToDevice));
    if(MainGas == ARGON){
        checkCudaErrors(hipMalloc((void**)&dev_ArCX, N_LOGX * sizeof(ArCollD)));
        checkCudaErrors(hipMemcpy(dev_ArCX, Ar_Data, N_LOGX * sizeof(ArCollD), hipMemcpyHostToDevice));
        num_a = 3; 			
        Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			// Electron
			engy = Ar_Data[i].xee;
			sigma[0] = Ar_Data[i].cx_0+Ar_Data[i].cx_1+Ar_Data[i].cx_2+Ar_Data[i].cx_3; // e + Ar
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = Ar_Data[i].cx_4;    // e + Ar*
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			// Ar ion
			engy = Ar_Data[i].xee;
			sigma[2] = Ar_Data[i].cx_5+Ar_Data[i].cx_6;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[2]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * BG[0].InitDens);
		ColProb[0] = Ttarget[0] + Ttarget[1];
		ColProb[1] = Ttarget[2];
		if(ColProb[0] > prob_cut) {
			DT_MCCn = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCCn;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCCn = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " - Total Collision probability per Time step\n");
		fprintf(stderr, "   Electon - %2.4f %\n",ColProb[0] * 100);
		fprintf(stderr, "   Ar ion  - %2.4f %\n",ColProb[1] * 100);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCCn, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else if(MainGas == OXYGEN){
        checkCudaErrors(hipMalloc((void**)&dev_O2CX, N_LOGX * sizeof(O2CollD)));
        checkCudaErrors(hipMemcpy(dev_O2CX, O2_Data, N_LOGX * sizeof(O2CollD), hipMemcpyHostToDevice));
		num_a = 20; 			
		Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			engy = O2_Data[i].xee;
			sigma[0] = O2_Data[i].cx_0+O2_Data[i].cx_1+O2_Data[i].cx_2+O2_Data[i].cx_3+O2_Data[i].cx_4;
			sigma[0] += O2_Data[i].cx_5+O2_Data[i].cx_6+O2_Data[i].cx_7+O2_Data[i].cx_8+O2_Data[i].cx_9;
			sigma[0] += O2_Data[i].cx_10+O2_Data[i].cx_11+O2_Data[i].cx_12+O2_Data[i].cx_13;
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = O2_Data[i].cx_14+O2_Data[i].cx_15+O2_Data[i].cx_16+O2_Data[i].cx_17+O2_Data[i].cx_18;
			sigma[1] += O2_Data[i].cx_19+O2_Data[i].cx_20+O2_Data[i].cx_21;
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			sigma[2] = O2_Data[i].cx_22+O2_Data[i].cx_23+O2_Data[i].cx_24+O2_Data[i].cx_25+O2_Data[i].cx_26;
			sigma[2] += O2_Data[i].cx_27+O2_Data[i].cx_28+O2_Data[i].cx_29;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[2]);
			sigma[3] = O2_Data[i].cx_30;
			Host_SigmaV[3].val = max(Host_SigmaV[3].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[3]);
			sigma[4] = O2_Data[i].cx_31;
			Host_SigmaV[4].val = max(Host_SigmaV[4].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[4]);
			sigma[5] = O2_Data[i].cx_32+O2_Data[i].cx_33+O2_Data[i].cx_34+O2_Data[i].cx_35+O2_Data[i].cx_36;
			sigma[5] += O2_Data[i].cx_37+O2_Data[i].cx_38;
			Host_SigmaV[5].val = max(Host_SigmaV[5].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[5]);
			sigma[6] = O2_Data[i].cx_39+O2_Data[i].cx_40;
			Host_SigmaV[6].val = max(Host_SigmaV[6].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[6]);
			sigma[7] = O2_Data[i].cx_41 + O2_Data[i].cx_42;
			Host_SigmaV[7].val = max(Host_SigmaV[7].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[7]);
			sigma[8] = O2_Data[i].cx_43;
			Host_SigmaV[8].val = max(Host_SigmaV[8].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[8]);
			sigma[9] = O2_Data[i].cx_44;
			Host_SigmaV[9].val = max(Host_SigmaV[9].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[9]);
			sigma[10] = O2_Data[i].cx_45;
			Host_SigmaV[10].val = max(Host_SigmaV[10].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[10]);
			sigma[11] = O2_Data[i].cx_46;
			Host_SigmaV[11].val = max(Host_SigmaV[11].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[11]);
			sigma[12] = O2_Data[i].cx_47;
			Host_SigmaV[12].val = max(Host_SigmaV[12].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[12]);
			sigma[13] = O2_Data[i].cx_48 + O2_Data[i].cx_49 + O2_Data[i].cx_50;
			Host_SigmaV[13].val = max(Host_SigmaV[13].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[13]);
			sigma[14] = O2_Data[i].cx_51;
			Host_SigmaV[14].val = max(Host_SigmaV[14].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[14]);
			sigma[15] = O2_Data[i].cx_52;
			Host_SigmaV[15].val = max(Host_SigmaV[15].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[15]);
			sigma[16] = O2_Data[i].cx_53 + O2_Data[i].cx_54;
			Host_SigmaV[16].val = max(Host_SigmaV[16].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[16]);
			sigma[17] = O2_Data[i].cx_55;
			Host_SigmaV[17].val = max(Host_SigmaV[17].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[17]);
			sigma[18] = O2_Data[i].cx_56;
			Host_SigmaV[18].val = max(Host_SigmaV[18].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[18]);
			sigma[19] = O2_Data[i].cx_57;
			Host_SigmaV[19].val = max(Host_SigmaV[19].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[19]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);// E + O2
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);// E + O2A
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * FG[1].InitDens);// E + O2B
		Ttarget[3] = 1.0-exp(-1 * dt * Host_SigmaV[3].val * SP[3].InitDens);// E + O-
		Ttarget[4] = 1.0-exp(-1 * dt * Host_SigmaV[4].val * SP[1].InitDens);// E + O2^
		Ttarget[5] = 1.0-exp(-1 * dt * Host_SigmaV[5].val * FG[2].InitDens);// E + OP
		Ttarget[6] = 1.0-exp(-1 * dt * Host_SigmaV[6].val * FG[3].InitDens);// E + OD
		Ttarget[7] = 1.0-exp(-1 * dt * Host_SigmaV[7].val * BG[0].InitDens);// O- + O2
		Ttarget[8] = 1.0-exp(-1 * dt * Host_SigmaV[8].val * FG[2].InitDens);// O- + OP
		Ttarget[9] = 1.0-exp(-1 * dt * Host_SigmaV[9].val* SP[1].InitDens);// O- + O2^
		Ttarget[10] = 1.0-exp(-1 * dt * Host_SigmaV[10].val * SP[2].InitDens);// O- + O^
		Ttarget[11] = 1.0-exp(-1 * dt * Host_SigmaV[11].val * FG[0].InitDens);// O- + O2A
		Ttarget[12] = 1.0-exp(-1 * dt * Host_SigmaV[12].val * FG[2].InitDens);// O2^ + OP
		Ttarget[13] = 1.0-exp(-1 * dt * Host_SigmaV[13].val * BG[0].InitDens);// O2^ + O2
		Ttarget[14] = 1.0-exp(-1 * dt * Host_SigmaV[14].val * FG[0].InitDens);// O2^ + O2A
		Ttarget[15] = 1.0-exp(-1 * dt * Host_SigmaV[15].val * FG[1].InitDens);// O2^ + O2B
		Ttarget[16] = 1.0-exp(-1 * dt * Host_SigmaV[16].val * BG[0].InitDens);// O^ + O2
		Ttarget[17] = 1.0-exp(-1 * dt * Host_SigmaV[17].val * FG[2].InitDens);// O^ + OP
		Ttarget[18] = 1.0-exp(-1 * dt * Host_SigmaV[18].val* FG[0].InitDens);// O^ + O2A
		Ttarget[19] = 1.0-exp(-1 * dt * Host_SigmaV[19].val * FG[1].InitDens);// O^ + O2B
		ColProb[0] = Ttarget[0] + Ttarget[1] + Ttarget[2] + Ttarget[3] + Ttarget[4] + Ttarget[5] + Ttarget[6];// Electron
		ColProb[1] = Ttarget[7] + Ttarget[8] + Ttarget[9] + Ttarget[10] + Ttarget[11];// O-
		ColProb[2] = Ttarget[12] + Ttarget[13] + Ttarget[14] + Ttarget[15];// O2+
		ColProb[3] = Ttarget[16] + Ttarget[17] + Ttarget[18] + Ttarget[19];// O+
		if(ColProb[0] > prob_cut) {
			DT_MCCn = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCCn;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCCn = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " - Total Collision probability per Time step\n");
		fprintf(stderr, "   Electon - %2.4f %\n",ColProb[0] * 100);
		fprintf(stderr, "   O- ion  - %2.4f %\n",ColProb[1] * 100);
		fprintf(stderr, "   O2+ion  - %2.4f %\n",ColProb[2] * 100);
		fprintf(stderr, "   O+ ion  - %2.4f %\n",ColProb[3] * 100);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCCn, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else if(MainGas == ARO2){
        checkCudaErrors(hipMalloc((void**)&dev_ArO2CX, N_LOGX * sizeof(ArO2CollD)));
        checkCudaErrors(hipMemcpy(dev_ArO2CX, ArO2_Data, N_LOGX * sizeof(ArO2CollD), hipMemcpyHostToDevice));
		num_a = 25; 			
		Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			engy = ArO2_Data[i].xee;
			sigma[0] = ArO2_Data[i].cx_0+ArO2_Data[i].cx_1+ArO2_Data[i].cx_2+ArO2_Data[i].cx_3; // e + Ar
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = ArO2_Data[i].cx_4;    // e + Ar*
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			sigma[2] = ArO2_Data[i].cx_5+ArO2_Data[i].cx_6+ArO2_Data[i].cx_7+ArO2_Data[i].cx_8+ArO2_Data[i].cx_9;
			sigma[2] += ArO2_Data[i].cx_10+ArO2_Data[i].cx_11+ArO2_Data[i].cx_12+ArO2_Data[i].cx_13+ArO2_Data[i].cx_14;
			sigma[2] += ArO2_Data[i].cx_15+ArO2_Data[i].cx_16+ArO2_Data[i].cx_17+ArO2_Data[i].cx_18;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[2]);// e + O2
			sigma[3] = ArO2_Data[i].cx_19+ArO2_Data[i].cx_20+ArO2_Data[i].cx_21+ArO2_Data[i].cx_22+ArO2_Data[i].cx_23;
			sigma[3] += ArO2_Data[i].cx_24+ArO2_Data[i].cx_25+ArO2_Data[i].cx_26;
			Host_SigmaV[3].val = max(Host_SigmaV[3].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[3]);// e + O2A
			sigma[4] = ArO2_Data[i].cx_27+ArO2_Data[i].cx_28+ArO2_Data[i].cx_29+ArO2_Data[i].cx_30+ArO2_Data[i].cx_31;
			sigma[4] += ArO2_Data[i].cx_32+ArO2_Data[i].cx_33+ArO2_Data[i].cx_34;
			Host_SigmaV[4].val = max(Host_SigmaV[4].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[4]);// e + O2B
			sigma[5] = ArO2_Data[i].cx_35;
			Host_SigmaV[5].val = max(Host_SigmaV[5].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[5]);
			sigma[6] = ArO2_Data[i].cx_36;
			Host_SigmaV[6].val = max(Host_SigmaV[6].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[6]);
			sigma[7] = ArO2_Data[i].cx_37+ArO2_Data[i].cx_38+ArO2_Data[i].cx_39+ArO2_Data[i].cx_40+ArO2_Data[i].cx_41;
			sigma[7] += ArO2_Data[i].cx_42+ArO2_Data[i].cx_43;
			Host_SigmaV[7].val = max(Host_SigmaV[7].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[7]);
			sigma[8] = ArO2_Data[i].cx_44+ArO2_Data[i].cx_45;
			Host_SigmaV[8].val = max(Host_SigmaV[8].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[8]);
			sigma[9] = ArO2_Data[i].cx_46 + ArO2_Data[i].cx_47;
			Host_SigmaV[9].val = max(Host_SigmaV[9].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[9]);
			sigma[10] = ArO2_Data[i].cx_48;
			Host_SigmaV[10].val = max(Host_SigmaV[10].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[10]);
			sigma[11] = ArO2_Data[i].cx_49;
			Host_SigmaV[11].val = max(Host_SigmaV[11].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[11]);
			sigma[12] = ArO2_Data[i].cx_50;
			Host_SigmaV[12].val = max(Host_SigmaV[12].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[12]);
			sigma[13] = ArO2_Data[i].cx_51;
			Host_SigmaV[13].val = max(Host_SigmaV[13].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[13]);
			sigma[14] = ArO2_Data[i].cx_52;
			Host_SigmaV[14].val = max(Host_SigmaV[14].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[14]);
			sigma[15] = ArO2_Data[i].cx_53 + ArO2_Data[i].cx_54 + ArO2_Data[i].cx_55;
			Host_SigmaV[15].val = max(Host_SigmaV[15].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[15]);
			sigma[16] = ArO2_Data[i].cx_56;
			Host_SigmaV[16].val = max(Host_SigmaV[16].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[16]);
			sigma[17] = ArO2_Data[i].cx_57;
			Host_SigmaV[17].val = max(Host_SigmaV[17].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[17]);
			sigma[18] = ArO2_Data[i].cx_58 + ArO2_Data[i].cx_59;
			Host_SigmaV[18].val = max(Host_SigmaV[18].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[18]);
			sigma[19] = ArO2_Data[i].cx_60 + ArO2_Data[i].cx_61;
			Host_SigmaV[19].val = max(Host_SigmaV[19].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[19]);
			sigma[20] = ArO2_Data[i].cx_62;
			Host_SigmaV[20].val = max(Host_SigmaV[20].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[20]);
			sigma[21] = ArO2_Data[i].cx_63;
			Host_SigmaV[21].val = max(Host_SigmaV[21].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[21]);
			sigma[22] = ArO2_Data[i].cx_64;
			Host_SigmaV[22].val = max(Host_SigmaV[22].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[22]);
			sigma[23] = ArO2_Data[i].cx_65+ArO2_Data[i].cx_66;
			Host_SigmaV[23].val = max(Host_SigmaV[23].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[23]);
			sigma[24] = ArO2_Data[i].cx_67;
			Host_SigmaV[24].val = max(Host_SigmaV[24].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[24]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);// E + Ar
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);// E + Ar*
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * BG[1].InitDens);// E + O2
		Ttarget[3] = 1.0-exp(-1 * dt * Host_SigmaV[3].val * FG[1].InitDens);// E + O2A
		Ttarget[4] = 1.0-exp(-1 * dt * Host_SigmaV[4].val * FG[2].InitDens);// E + O2B
		Ttarget[5] = 1.0-exp(-1 * dt * Host_SigmaV[5].val * SP[4].InitDens);// E + O-
		Ttarget[6] = 1.0-exp(-1 * dt * Host_SigmaV[6].val * SP[2].InitDens);// E + O2^
		Ttarget[7] = 1.0-exp(-1 * dt * Host_SigmaV[7].val * FG[3].InitDens);// E + OP
		Ttarget[8] = 1.0-exp(-1 * dt * Host_SigmaV[8].val * FG[4].InitDens);// E + OD
		Ttarget[9] = 1.0-exp(-1 * dt * Host_SigmaV[9].val * BG[1].InitDens);// O- + O2
		Ttarget[10] = 1.0-exp(-1 * dt * Host_SigmaV[10].val * FG[3].InitDens);// O- + OP
		Ttarget[11] = 1.0-exp(-1 * dt * Host_SigmaV[11].val* SP[2].InitDens);// O- + O2^
		Ttarget[12] = 1.0-exp(-1 * dt * Host_SigmaV[12].val * SP[3].InitDens);// O- + O^
		Ttarget[13] = 1.0-exp(-1 * dt * Host_SigmaV[13].val * FG[1].InitDens);// O- + O2A
		Ttarget[14] = 1.0-exp(-1 * dt * Host_SigmaV[14].val * FG[3].InitDens);// O2^ + OP
		Ttarget[15] = 1.0-exp(-1 * dt * Host_SigmaV[15].val * BG[1].InitDens);// O2^ + O2
		Ttarget[16] = 1.0-exp(-1 * dt * Host_SigmaV[16].val * FG[1].InitDens);// O2^ + O2A
		Ttarget[17] = 1.0-exp(-1 * dt * Host_SigmaV[17].val * FG[2].InitDens);// O2^ + O2B
		Ttarget[18] = 1.0-exp(-1 * dt * Host_SigmaV[18].val * BG[0].InitDens);// O2^ + Ar
		Ttarget[19] = 1.0-exp(-1 * dt * Host_SigmaV[19].val * BG[1].InitDens);// O^ + O2
		Ttarget[20] = 1.0-exp(-1 * dt * Host_SigmaV[20].val * FG[3].InitDens);// O^ + OP
		Ttarget[21] = 1.0-exp(-1 * dt * Host_SigmaV[21].val* FG[1].InitDens);// O^ + O2A
		Ttarget[22] = 1.0-exp(-1 * dt * Host_SigmaV[22].val * FG[2].InitDens);// O^ + O2B
		Ttarget[23] = 1.0-exp(-1 * dt * Host_SigmaV[23].val * BG[0].InitDens);// Ar^ + Ar
		Ttarget[24] = 1.0-exp(-1 * dt * Host_SigmaV[24].val * BG[1].InitDens);// Ar^ + O2
		ColProb[0] = Ttarget[0] + Ttarget[1] + Ttarget[2] + Ttarget[3] + Ttarget[4] + Ttarget[5] + Ttarget[6] + Ttarget[7] + Ttarget[8];// Electron
		ColProb[1] = Ttarget[23] + Ttarget[24];// Ar+
		ColProb[2] = Ttarget[14] + Ttarget[15] + Ttarget[16] + Ttarget[17] + Ttarget[18];// O2+
		ColProb[3] = Ttarget[19] + Ttarget[20] + Ttarget[21] + Ttarget[22];// O+
		ColProb[4] = Ttarget[9] + Ttarget[10] + Ttarget[11] + Ttarget[12] + Ttarget[13];// O-
		if(ColProb[0] > prob_cut) {
			DT_MCCn = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCCn;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCCn = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " - Total Collision probability per Time step\n");
		fprintf(stderr, "   Electon - %2.4f %\n",ColProb[0] * 100);
		fprintf(stderr, "   Ar+ion  - %2.4f %\n",ColProb[1] * 100);
		fprintf(stderr, "   O2+ion  - %2.4f %\n",ColProb[2] * 100);
		fprintf(stderr, "   O+ ion  - %2.4f %\n",ColProb[3] * 100);
		fprintf(stderr, "   O- ion  - %2.4f %\n",ColProb[4] * 100);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCCn, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else{
        printf("Error : MainGas = %d\n",MainGas);
        exit(1);
    }
    //Maximum sigma*v copy  CPU >> GPU
    checkCudaErrors(hipMalloc((void**)&dev_SigmaV, num_a * sizeof(MCC_sigmav)));
    checkCudaErrors(hipMemcpy(dev_SigmaV, Host_SigmaV, num_a * sizeof(MCC_sigmav), hipMemcpyHostToDevice));
    printf("MCC Initializing Complete!\n");
}

__global__ void MCC_Ar_Basic(int Gsize, int ngy, hiprandState *states, Species *info, GCP *sp, GPG *data, GGA *Field){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	int isp,ID;
 	isp = (int)TID/Gsize; //species number [< nsp]
    ID = (int)TID%Gsize; // Grid ID [< Gsize]
    if(TID>Gsize*info[isp].spnum) return;
	hiprandState LocalStates;
	LocalStates = states[TID];
	if(TID < 5 ){
		float a,b;
		a = hiprand_uniform(&LocalStates);
		b = hiprand_uniform(&LocalStates);
		printf("[%d] 1 = %g, 2 = %g\n",TID,a,b);
	}
	
	states[TID]=LocalStates;
}	