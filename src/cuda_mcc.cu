#include "hip/hip_runtime.h"
#include "cuda_mcc.cuh"

void Set_NullCollisionTime_cuda(){
    // This function calculates the following variables :
    // 1. int DT_MCC
    // 2. float dt_mcc
    // 3. dev_Coll_Flag [CollF] - [TnRct]
    // 4. dev_**CX [**CollD] - [N_LOGX]
    // 5. Max_sigma_v[num_a] ,Ttarget[num_a] ColProb[num_b]
    //    Argon - [3][2]
    //    Oxygen - [20][4]
    //    ArO2 - [25][5]
    int i;
    int num_a; // Number of (Projectile + target)
    float engy;
	float *sigma;
    float *Ttarget,*ColProb;
    float ratio, prob_cut = 0.05; // = 5%. Collision rate per time step
    //Cross section Data copy  CPU >> GPU
    checkCudaErrors(hipMalloc((void**)&dev_Coll_Flag, TnRct * sizeof(CollF)));
    checkCudaErrors(hipMemcpy(dev_Coll_Flag, Coll_Flag, TnRct * sizeof(CollF), hipMemcpyHostToDevice));
    if(MainGas == ARGON){
        checkCudaErrors(hipMalloc((void**)&dev_ArCX, N_LOGX * sizeof(ArCollD)));
        checkCudaErrors(hipMemcpy(dev_ArCX, Ar_Data, N_LOGX * sizeof(ArCollD), hipMemcpyHostToDevice));
        num_a = 3; 			
        Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			// Electron
			engy = Ar_Data[i].xee;
			sigma[0] = Ar_Data[i].cx_0+Ar_Data[i].cx_1+Ar_Data[i].cx_2+Ar_Data[i].cx_3; // e + Ar
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = Ar_Data[i].cx_4;    // e + Ar*
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			// Ar ion
			engy = Ar_Data[i].xee;
			sigma[2] = Ar_Data[i].cx_5+Ar_Data[i].cx_6;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[2]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * BG[0].InitDens);
		ColProb[0] = Ttarget[0] + Ttarget[1];
		ColProb[1] = Ttarget[2];
		if(ColProb[0] > prob_cut) {
			DT_MCC = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCC;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCC = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " Argon Density = %g\n",BG[0].InitDens);
		fprintf(stderr, " Metastable Argon Density = %g\n",FG[0].InitDens);
		fprintf(stderr, " Calculate Result\n");
		fprintf(stderr, " - Electron sigma*v, Collision Probability\n");
		fprintf(stderr, "   SigmaV Ar - %g, Ar* - %g\n",Host_SigmaV[0].val,Host_SigmaV[1].val);
		fprintf(stderr, "   Prob Ar - %g, Ar* - %g\n",Ttarget[0],Ttarget[1]);
		fprintf(stderr, " - Ar ion \n");
		fprintf(stderr, "   SigV Ar - %g\n",Host_SigmaV[2].val);
		fprintf(stderr, "   Prob Ar - %g\n",Ttarget[2]);
		fprintf(stderr, " - Total Collision probability\n");
		fprintf(stderr, "   Electon - %2.5f \n",ColProb[0]);
		fprintf(stderr, "   Ar ion  - %2.5f \n",ColProb[1]);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCC, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else if(MainGas == OXYGEN){
        checkCudaErrors(hipMalloc((void**)&dev_O2CX, N_LOGX * sizeof(O2CollD)));
        checkCudaErrors(hipMemcpy(dev_O2CX, O2_Data, N_LOGX * sizeof(O2CollD), hipMemcpyHostToDevice));
		num_a = 20; 			
		Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			engy = O2_Data[i].xee;
			sigma[0] = O2_Data[i].cx_0+O2_Data[i].cx_1+O2_Data[i].cx_2+O2_Data[i].cx_3+O2_Data[i].cx_4;
			sigma[0] += O2_Data[i].cx_5+O2_Data[i].cx_6+O2_Data[i].cx_7+O2_Data[i].cx_8+O2_Data[i].cx_9;
			sigma[0] += O2_Data[i].cx_10+O2_Data[i].cx_11+O2_Data[i].cx_12+O2_Data[i].cx_13;
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = O2_Data[i].cx_14+O2_Data[i].cx_15+O2_Data[i].cx_16+O2_Data[i].cx_17+O2_Data[i].cx_18;
			sigma[1] += O2_Data[i].cx_19+O2_Data[i].cx_20+O2_Data[i].cx_21;
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			sigma[2] = O2_Data[i].cx_22+O2_Data[i].cx_23+O2_Data[i].cx_24+O2_Data[i].cx_25+O2_Data[i].cx_26;
			sigma[2] += O2_Data[i].cx_27+O2_Data[i].cx_28+O2_Data[i].cx_29;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[2]);
			sigma[3] = O2_Data[i].cx_30;
			Host_SigmaV[3].val = max(Host_SigmaV[3].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[3]);
			sigma[4] = O2_Data[i].cx_31;
			Host_SigmaV[4].val = max(Host_SigmaV[4].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[4]);
			sigma[5] = O2_Data[i].cx_32+O2_Data[i].cx_33+O2_Data[i].cx_34+O2_Data[i].cx_35+O2_Data[i].cx_36;
			sigma[5] += O2_Data[i].cx_37+O2_Data[i].cx_38;
			Host_SigmaV[5].val = max(Host_SigmaV[5].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[5]);
			sigma[6] = O2_Data[i].cx_39+O2_Data[i].cx_40;
			Host_SigmaV[6].val = max(Host_SigmaV[6].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[6]);
			sigma[7] = O2_Data[i].cx_41 + O2_Data[i].cx_42;
			Host_SigmaV[7].val = max(Host_SigmaV[7].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[7]);
			sigma[8] = O2_Data[i].cx_43;
			Host_SigmaV[8].val = max(Host_SigmaV[8].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[8]);
			sigma[9] = O2_Data[i].cx_44;
			Host_SigmaV[9].val = max(Host_SigmaV[9].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[9]);
			sigma[10] = O2_Data[i].cx_45;
			Host_SigmaV[10].val = max(Host_SigmaV[10].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[10]);
			sigma[11] = O2_Data[i].cx_46;
			Host_SigmaV[11].val = max(Host_SigmaV[11].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[11]);
			sigma[12] = O2_Data[i].cx_47;
			Host_SigmaV[12].val = max(Host_SigmaV[12].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[12]);
			sigma[13] = O2_Data[i].cx_48 + O2_Data[i].cx_49 + O2_Data[i].cx_50;
			Host_SigmaV[13].val = max(Host_SigmaV[13].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[13]);
			sigma[14] = O2_Data[i].cx_51;
			Host_SigmaV[14].val = max(Host_SigmaV[14].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[14]);
			sigma[15] = O2_Data[i].cx_52;
			Host_SigmaV[15].val = max(Host_SigmaV[15].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[15]);
			sigma[16] = O2_Data[i].cx_53 + O2_Data[i].cx_54;
			Host_SigmaV[16].val = max(Host_SigmaV[16].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[16]);
			sigma[17] = O2_Data[i].cx_55;
			Host_SigmaV[17].val = max(Host_SigmaV[17].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[17]);
			sigma[18] = O2_Data[i].cx_56;
			Host_SigmaV[18].val = max(Host_SigmaV[18].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[18]);
			sigma[19] = O2_Data[i].cx_57;
			Host_SigmaV[19].val = max(Host_SigmaV[19].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[19]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);// E + O2
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);// E + O2A
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * FG[1].InitDens);// E + O2B
		Ttarget[3] = 1.0-exp(-1 * dt * Host_SigmaV[3].val * SP[3].InitDens);// E + O-
		Ttarget[4] = 1.0-exp(-1 * dt * Host_SigmaV[4].val * SP[1].InitDens);// E + O2^
		Ttarget[5] = 1.0-exp(-1 * dt * Host_SigmaV[5].val * FG[2].InitDens);// E + OP
		Ttarget[6] = 1.0-exp(-1 * dt * Host_SigmaV[6].val * FG[3].InitDens);// E + OD
		Ttarget[7] = 1.0-exp(-1 * dt * Host_SigmaV[7].val * BG[0].InitDens);// O- + O2
		Ttarget[8] = 1.0-exp(-1 * dt * Host_SigmaV[8].val * FG[2].InitDens);// O- + OP
		Ttarget[9] = 1.0-exp(-1 * dt * Host_SigmaV[9].val* SP[1].InitDens);// O- + O2^
		Ttarget[10] = 1.0-exp(-1 * dt * Host_SigmaV[10].val * SP[2].InitDens);// O- + O^
		Ttarget[11] = 1.0-exp(-1 * dt * Host_SigmaV[11].val * FG[0].InitDens);// O- + O2A
		Ttarget[12] = 1.0-exp(-1 * dt * Host_SigmaV[12].val * FG[2].InitDens);// O2^ + OP
		Ttarget[13] = 1.0-exp(-1 * dt * Host_SigmaV[13].val * BG[0].InitDens);// O2^ + O2
		Ttarget[14] = 1.0-exp(-1 * dt * Host_SigmaV[14].val * FG[0].InitDens);// O2^ + O2A
		Ttarget[15] = 1.0-exp(-1 * dt * Host_SigmaV[15].val * FG[1].InitDens);// O2^ + O2B
		Ttarget[16] = 1.0-exp(-1 * dt * Host_SigmaV[16].val * BG[0].InitDens);// O^ + O2
		Ttarget[17] = 1.0-exp(-1 * dt * Host_SigmaV[17].val * FG[2].InitDens);// O^ + OP
		Ttarget[18] = 1.0-exp(-1 * dt * Host_SigmaV[18].val* FG[0].InitDens);// O^ + O2A
		Ttarget[19] = 1.0-exp(-1 * dt * Host_SigmaV[19].val * FG[1].InitDens);// O^ + O2B
		ColProb[0] = Ttarget[0] + Ttarget[1] + Ttarget[2] + Ttarget[3] + Ttarget[4] + Ttarget[5] + Ttarget[6];// Electron
		ColProb[1] = Ttarget[7] + Ttarget[8] + Ttarget[9] + Ttarget[10] + Ttarget[11];// O-
		ColProb[2] = Ttarget[12] + Ttarget[13] + Ttarget[14] + Ttarget[15];// O2+
		ColProb[3] = Ttarget[16] + Ttarget[17] + Ttarget[18] + Ttarget[19];// O+
		if(ColProb[0] > prob_cut) {
			DT_MCC = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCC;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCC = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " O2  Density = %g\n",BG[0].InitDens);
		fprintf(stderr, " O2A Density = %g\n",FG[0].InitDens);
		fprintf(stderr, " O2B Density = %g\n",FG[1].InitDens);
		fprintf(stderr, " OP  Density = %g\n",FG[2].InitDens);
		fprintf(stderr, " OD  Density = %g\n",FG[3].InitDens);
		fprintf(stderr, " O-  Density = %g\n",SP[3].InitDens);
		fprintf(stderr, " O2+ Density = %g\n",SP[1].InitDens);
		fprintf(stderr, " O+  Density = %g\n",SP[2].InitDens);
		fprintf(stderr, " - Total Collision probability\n");
		fprintf(stderr, "   Electon - %2.5f \n",ColProb[0]);
		fprintf(stderr, "   O- ion  - %2.5f \n",ColProb[1]);
		fprintf(stderr, "   O2+ion  - %2.5f \n",ColProb[2]);
		fprintf(stderr, "   O+ ion  - %2.5f \n",ColProb[3]);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCC, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else if(MainGas == ARO2){
        checkCudaErrors(hipMalloc((void**)&dev_ArO2CX, N_LOGX * sizeof(ArO2CollD)));
        checkCudaErrors(hipMemcpy(dev_ArO2CX, ArO2_Data, N_LOGX * sizeof(ArO2CollD), hipMemcpyHostToDevice));
		num_a = 25; 			
		Host_SigmaV = (MCC_sigmav *)malloc(num_a * sizeof(MCC_sigmav));
        for(i=0;i<num_a;i++) Host_SigmaV[i].val = 0.0;
		sigma = VFMalloc(num_a);
		Ttarget    = VFMalloc(num_a);
		VFInit(Ttarget, 0.0, num_a);
		ColProb    = VFMalloc(nsp);
		VFInit(ColProb, 0.0, nsp);
		for(i=0;i<N_LOGX;i++){
			engy = ArO2_Data[i].xee;
			sigma[0] = ArO2_Data[i].cx_0+ArO2_Data[i].cx_1+ArO2_Data[i].cx_2+ArO2_Data[i].cx_3; // e + Ar
			Host_SigmaV[0].val = max(Host_SigmaV[0].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[0]);
			sigma[1] = ArO2_Data[i].cx_4;    // e + Ar*
			Host_SigmaV[1].val = max(Host_SigmaV[1].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[1]);
			sigma[2] = ArO2_Data[i].cx_5+ArO2_Data[i].cx_6+ArO2_Data[i].cx_7+ArO2_Data[i].cx_8+ArO2_Data[i].cx_9;
			sigma[2] += ArO2_Data[i].cx_10+ArO2_Data[i].cx_11+ArO2_Data[i].cx_12+ArO2_Data[i].cx_13+ArO2_Data[i].cx_14;
			sigma[2] += ArO2_Data[i].cx_15+ArO2_Data[i].cx_16+ArO2_Data[i].cx_17+ArO2_Data[i].cx_18;
			Host_SigmaV[2].val = max(Host_SigmaV[2].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[2]);// e + O2
			sigma[3] = ArO2_Data[i].cx_19+ArO2_Data[i].cx_20+ArO2_Data[i].cx_21+ArO2_Data[i].cx_22+ArO2_Data[i].cx_23;
			sigma[3] += ArO2_Data[i].cx_24+ArO2_Data[i].cx_25+ArO2_Data[i].cx_26;
			Host_SigmaV[3].val = max(Host_SigmaV[3].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[3]);// e + O2A
			sigma[4] = ArO2_Data[i].cx_27+ArO2_Data[i].cx_28+ArO2_Data[i].cx_29+ArO2_Data[i].cx_30+ArO2_Data[i].cx_31;
			sigma[4] += ArO2_Data[i].cx_32+ArO2_Data[i].cx_33+ArO2_Data[i].cx_34;
			Host_SigmaV[4].val = max(Host_SigmaV[4].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[4]);// e + O2B
			sigma[5] = ArO2_Data[i].cx_35;
			Host_SigmaV[5].val = max(Host_SigmaV[5].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[5]);
			sigma[6] = ArO2_Data[i].cx_36;
			Host_SigmaV[6].val = max(Host_SigmaV[6].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[6]);
			sigma[7] = ArO2_Data[i].cx_37+ArO2_Data[i].cx_38+ArO2_Data[i].cx_39+ArO2_Data[i].cx_40+ArO2_Data[i].cx_41;
			sigma[7] += ArO2_Data[i].cx_42+ArO2_Data[i].cx_43;
			Host_SigmaV[7].val = max(Host_SigmaV[7].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[7]);
			sigma[8] = ArO2_Data[i].cx_44+ArO2_Data[i].cx_45;
			Host_SigmaV[8].val = max(Host_SigmaV[8].val,sqrt(2*1.602e-19*engy/SP[0].mass)*sigma[8]);
			sigma[9] = ArO2_Data[i].cx_46 + ArO2_Data[i].cx_47;
			Host_SigmaV[9].val = max(Host_SigmaV[9].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[9]);
			sigma[10] = ArO2_Data[i].cx_48;
			Host_SigmaV[10].val = max(Host_SigmaV[10].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[10]);
			sigma[11] = ArO2_Data[i].cx_49;
			Host_SigmaV[11].val = max(Host_SigmaV[11].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[11]);
			sigma[12] = ArO2_Data[i].cx_50;
			Host_SigmaV[12].val = max(Host_SigmaV[12].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[12]);
			sigma[13] = ArO2_Data[i].cx_51;
			Host_SigmaV[13].val = max(Host_SigmaV[13].val,sqrt(2*1.602e-19*engy/SP[4].mass)*sigma[13]);
			sigma[14] = ArO2_Data[i].cx_52;
			Host_SigmaV[14].val = max(Host_SigmaV[14].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[14]);
			sigma[15] = ArO2_Data[i].cx_53 + ArO2_Data[i].cx_54 + ArO2_Data[i].cx_55;
			Host_SigmaV[15].val = max(Host_SigmaV[15].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[15]);
			sigma[16] = ArO2_Data[i].cx_56;
			Host_SigmaV[16].val = max(Host_SigmaV[16].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[16]);
			sigma[17] = ArO2_Data[i].cx_57;
			Host_SigmaV[17].val = max(Host_SigmaV[17].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[17]);
			sigma[18] = ArO2_Data[i].cx_58 + ArO2_Data[i].cx_59;
			Host_SigmaV[18].val = max(Host_SigmaV[18].val,sqrt(2*1.602e-19*engy/SP[2].mass)*sigma[18]);
			sigma[19] = ArO2_Data[i].cx_60 + ArO2_Data[i].cx_61;
			Host_SigmaV[19].val = max(Host_SigmaV[19].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[19]);
			sigma[20] = ArO2_Data[i].cx_62;
			Host_SigmaV[20].val = max(Host_SigmaV[20].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[20]);
			sigma[21] = ArO2_Data[i].cx_63;
			Host_SigmaV[21].val = max(Host_SigmaV[21].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[21]);
			sigma[22] = ArO2_Data[i].cx_64;
			Host_SigmaV[22].val = max(Host_SigmaV[22].val,sqrt(2*1.602e-19*engy/SP[3].mass)*sigma[22]);
			sigma[23] = ArO2_Data[i].cx_65+ArO2_Data[i].cx_66;
			Host_SigmaV[23].val = max(Host_SigmaV[23].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[23]);
			sigma[24] = ArO2_Data[i].cx_67;
			Host_SigmaV[24].val = max(Host_SigmaV[24].val,sqrt(2*1.602e-19*engy/SP[1].mass)*sigma[24]);
		}
		Ttarget[0] = 1.0-exp(-1 * dt * Host_SigmaV[0].val * BG[0].InitDens);// E + Ar
		Ttarget[1] = 1.0-exp(-1 * dt * Host_SigmaV[1].val * FG[0].InitDens);// E + Ar*
		Ttarget[2] = 1.0-exp(-1 * dt * Host_SigmaV[2].val * BG[1].InitDens);// E + O2
		Ttarget[3] = 1.0-exp(-1 * dt * Host_SigmaV[3].val * FG[1].InitDens);// E + O2A
		Ttarget[4] = 1.0-exp(-1 * dt * Host_SigmaV[4].val * FG[2].InitDens);// E + O2B
		Ttarget[5] = 1.0-exp(-1 * dt * Host_SigmaV[5].val * SP[4].InitDens);// E + O-
		Ttarget[6] = 1.0-exp(-1 * dt * Host_SigmaV[6].val * SP[2].InitDens);// E + O2^
		Ttarget[7] = 1.0-exp(-1 * dt * Host_SigmaV[7].val * FG[3].InitDens);// E + OP
		Ttarget[8] = 1.0-exp(-1 * dt * Host_SigmaV[8].val * FG[4].InitDens);// E + OD
		Ttarget[9] = 1.0-exp(-1 * dt * Host_SigmaV[9].val * BG[1].InitDens);// O- + O2
		Ttarget[10] = 1.0-exp(-1 * dt * Host_SigmaV[10].val * FG[3].InitDens);// O- + OP
		Ttarget[11] = 1.0-exp(-1 * dt * Host_SigmaV[11].val* SP[2].InitDens);// O- + O2^
		Ttarget[12] = 1.0-exp(-1 * dt * Host_SigmaV[12].val * SP[3].InitDens);// O- + O^
		Ttarget[13] = 1.0-exp(-1 * dt * Host_SigmaV[13].val * FG[1].InitDens);// O- + O2A
		Ttarget[14] = 1.0-exp(-1 * dt * Host_SigmaV[14].val * FG[3].InitDens);// O2^ + OP
		Ttarget[15] = 1.0-exp(-1 * dt * Host_SigmaV[15].val * BG[1].InitDens);// O2^ + O2
		Ttarget[16] = 1.0-exp(-1 * dt * Host_SigmaV[16].val * FG[1].InitDens);// O2^ + O2A
		Ttarget[17] = 1.0-exp(-1 * dt * Host_SigmaV[17].val * FG[2].InitDens);// O2^ + O2B
		Ttarget[18] = 1.0-exp(-1 * dt * Host_SigmaV[18].val * BG[0].InitDens);// O2^ + Ar
		Ttarget[19] = 1.0-exp(-1 * dt * Host_SigmaV[19].val * BG[1].InitDens);// O^ + O2
		Ttarget[20] = 1.0-exp(-1 * dt * Host_SigmaV[20].val * FG[3].InitDens);// O^ + OP
		Ttarget[21] = 1.0-exp(-1 * dt * Host_SigmaV[21].val* FG[1].InitDens);// O^ + O2A
		Ttarget[22] = 1.0-exp(-1 * dt * Host_SigmaV[22].val * FG[2].InitDens);// O^ + O2B
		Ttarget[23] = 1.0-exp(-1 * dt * Host_SigmaV[23].val * BG[0].InitDens);// Ar^ + Ar
		Ttarget[24] = 1.0-exp(-1 * dt * Host_SigmaV[24].val * BG[1].InitDens);// Ar^ + O2
		ColProb[0] = Ttarget[0] + Ttarget[1] + Ttarget[2] + Ttarget[3] + Ttarget[4] + Ttarget[5] + Ttarget[6] + Ttarget[7] + Ttarget[8];// Electron
		ColProb[1] = Ttarget[23] + Ttarget[24];// Ar+
		ColProb[2] = Ttarget[14] + Ttarget[15] + Ttarget[16] + Ttarget[17] + Ttarget[18];// O2+
		ColProb[3] = Ttarget[19] + Ttarget[20] + Ttarget[21] + Ttarget[22];// O+
		ColProb[4] = Ttarget[9] + Ttarget[10] + Ttarget[11] + Ttarget[12] + Ttarget[13];// O-
		if(ColProb[0] > prob_cut) {
			DT_MCC = (int)ceil(ColProb[0]/prob_cut);
			ratio = 1/(float)DT_MCC;
			dt_mcc = dt*ratio;
		}
		else {
			DT_MCC = 1;
			dt_mcc = dt;
		}
		// Null Method Information
		fprintf(stderr,"--------------<Null Collision Information>-------------\n");
		fprintf(stderr, " AR  Density = %g\n",BG[0].InitDens);
		fprintf(stderr, " O2  Density = %g\n",BG[1].InitDens);
		fprintf(stderr, " Ar* Density = %g\n",FG[0].InitDens);
		fprintf(stderr, " O2A Density = %g\n",FG[1].InitDens);
		fprintf(stderr, " O2B Density = %g\n",FG[2].InitDens);
		fprintf(stderr, " OP  Density = %g\n",FG[3].InitDens);
		fprintf(stderr, " OD  Density = %g\n",FG[4].InitDens);
		fprintf(stderr, " Ar+ Density = %g\n",SP[1].InitDens);
		fprintf(stderr, " O2+ Density = %g\n",SP[2].InitDens);
		fprintf(stderr, " O+  Density = %g\n",SP[3].InitDens);
		fprintf(stderr, " O-  Density = %g\n",SP[4].InitDens);
		fprintf(stderr, " - Total Collision probability\n");
		fprintf(stderr, "   Electon - %2.5f \n",ColProb[0]);
		fprintf(stderr, "   Ar+ion  - %2.5f \n",ColProb[1]);
		fprintf(stderr, "   O2+ion  - %2.5f \n",ColProb[2]);
		fprintf(stderr, "   O+ ion  - %2.5f \n",ColProb[3]);
		fprintf(stderr, "   O- ion  - %2.5f \n",ColProb[4]);
		fprintf(stderr, " - Number of Electron MCC Cycle\n");
		fprintf(stderr, "   Cycle : %d, dt_mcc : %g \n",DT_MCC, dt_mcc);
		fprintf(stderr,"-------------------------------------------------------\n");
    }else{
        printf("Error : MainGas = %d\n",MainGas);
        exit(1);
    }
    //Maximum sigma*v copy  CPU >> GPU
    checkCudaErrors(hipMalloc((void**)&dev_SigmaV, num_a * sizeof(MCC_sigmav)));
    checkCudaErrors(hipMemcpy(dev_SigmaV, Host_SigmaV, num_a * sizeof(MCC_sigmav), hipMemcpyHostToDevice));
    printf("MCC Initializing Complete!\n");
}