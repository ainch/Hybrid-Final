#include "cuda_Tecplot.cuh"

void Tecplot_save(){
    int isp,i;
    int sum;
    static int Movie_Init = 1;
    static int PTMovie_Init = 1;
    if(TecplotS_2D_Flag){
        if(cstep == 1){
            Tecplot_2D();
        }else if(cstep >= 1 &&(cstep/TecplotS_2D_Ncycle == 1.0f)){
            Tecplot_2D();
        }
    }
    if(TecplotS_Movie_Flag){
        if((cstep%TecplotS_Movie_Ncycle == 0)){
            TecplotS_Movie_Count++;
            if(TecplotS_Movie_SCYCLE == TecplotS_Movie_Count){
                hipMemcpy(Host_G_sp, dev_G_sp, nsp * Gsize * sizeof(GPG),hipMemcpyDeviceToHost);
                hipMemcpy(vec_G, dev_GvecSet, Gsize * sizeof(GGA),hipMemcpyDeviceToHost);
                if(Movie_Init){
                    Tecplot_Gsize_Movie(Movie_Init);
                    Movie_Init--;
                }
                Tecplot_Gsize_Movie(Movie_Init);
                TecplotS_Movie_Count = 0;
            }
        }
    }
    if(TecplotS_PT_Movie_Flag){
        if((cstep%TecplotS_PT_Movie_Ncycle == 0)){
            TecplotS_PT_Movie_Count++;
            if(TecplotS_PT_Movie_SCYCLE == TecplotS_PT_Movie_Count){
                hipMemcpy(Host_G_sp, dev_G_sp, nsp * Gsize * sizeof(GPG),hipMemcpyDeviceToHost);
                for(isp=0;isp<nsp;isp++){
                    sum = 0;
                    for(i=0;i<Gsize;i++)
                        sum += Host_G_sp[isp*Gsize + i].PtNumInCell;
                    SP[isp].np = sum;
                }
                hipMemcpy(Host_sp, dev_sp, Total_maxnp * sizeof(GCP),hipMemcpyDeviceToHost);
                checkCudaErrors(hipMemcpy(dev_info_sp, SP, nsp * sizeof(Species), hipMemcpyHostToDevice));
                Copy_GCPtoHCP(SP, Host_sp, PtD, Host_G_sp);
                if(PTMovie_Init){
                    for(isp=0;isp<nsp;isp++) Tecplot_PT_Movie(PTMovie_Init,isp);
                    PTMovie_Init--;
                }
                for(isp=0;isp<nsp;isp++) Tecplot_PT_Movie(PTMovie_Init,isp);
                TecplotS_PT_Movie_Count = 0;
            }
        }  
    }
}
void Tecplot_PT_Movie(int Init,int isp){
    char filename[512];
    FILE *fp;
    int i,j,k;
    sprintf(filename,"%s_PT%d_Movie.dat",InputFile,isp);
    if(Init){
        PT_Movie_S_count = 0;
        fp = fopen(filename,"w");
        fprintf(fp, "TITLE = \"2D PIC Movie\"\n");
	    fprintf(fp, "VARIABLES = \"X(m)\",\"Y(m)\",\"Vx(m/s)\",\"Vy(m/s)\",\"Vz(m/s)\"\n");
        // GEOMETRY
        fprintf(fp, "GEOMETRY\nF=POINT\nCS=GRID\nX=0.00,Y=0.00,Z=0.00\nC=BLACK\nS=GLOBAL\nL=SOLID\nPL=4\nLT=0.1\n");
        fprintf(fp, "CLIPPING=CLIPTOVIEWPORT\nDRAWORDER=AFTERDATA\nMFC=\"\"\n");
        fprintf(fp, "T=RECTANGLE %g %g\n",xlength,ylength);
        for(i=0;i<CondNUM;i++){
            fprintf(fp, "GEOMETRY\nF=POINT\nCS=GRID\n");
            fprintf(fp, "X=%g,Y=%g,Z=0.00\n",CondX0[i]*dx,CondY0[i]*dy);
            fprintf(fp, "C=BLACK\nS=GLOBAL\nL=SOLID\nPL=4\nLT=0.1\n");
            fprintf(fp, "CLIPPING=CLIPTOVIEWPORT\nDRAWORDER=AFTERDATA\nMFC=\"\"\n");
            fprintf(fp, "T=RECTANGLE %g %g\n",CondX1[i]*dx-CondX0[i]*dx,CondY1[i]*dy-CondY0[i]*dy);
        }
        for(i=0;i<DielNUM;i++){
            fprintf(fp, "GEOMETRY\nF=POINT\nCS=GRID\n");
            fprintf(fp, "X=%g,Y=%g,Z=0.00\n",DielX0[i]*dx,DielY0[i]*dy);
            fprintf(fp, "C=BLACK\nS=GLOBAL\nL=SOLID\nPL=4\nLT=0.1\n");
            fprintf(fp, "CLIPPING=CLIPTOVIEWPORT\nDRAWORDER=AFTERDATA\nMFC=\"\"\n");
            fprintf(fp, "T=RECTANGLE %g %g\n",DielX1[i]*dx-DielX0[i]*dx,DielY1[i]*dy-DielY0[i]*dy);
        }
    }else{
        PT_Movie_S_count++;
        fp = fopen(filename,"a");
    }
    fprintf(fp, "ZONE T=\"ZONE %d\"\n",PT_Movie_S_count);
	fprintf(fp, " STRANDID=0, SOLUTIONTIME=1\n");
	fprintf(fp, " I=%d, J=1, K=1, ZONETYPE=Ordered\n",SP[isp].np);
	fprintf(fp, " DATAPACKING=POINT\n");
	fprintf(fp, " DT=(SINGLE SINGLE SINGLE SINGLE SINGLE)\n");
    for (k = 0; k < SP[isp].np; k++) {
		fprintf(fp,"%3.5g %3.5g %3.5g %3.5g %3.5g\n",dx*PtD[isp].x[k],dy*PtD[isp].y[k],PtD[isp].vx[k],PtD[isp].vy[k],PtD[isp].vz[k]);
	}
	fclose(fp);				
}
void Tecplot_Gsize_Movie(int Init){
    char filename[512];
    FILE *fp;
    int isp,i,j,k;
    int Flag1 = 1;
    int Flag2 = 1;
    int Flag3 = 1;
    sprintf(filename,"%s_Movie.dat",InputFile);
    if(Init){
        fp = fopen(filename,"w");
        fprintf(fp, "TITLE = \"2D PIC Movie\"\n");
	    fprintf(fp, "VARIABLES = \"X (m)\", \"Y (m)\", ");
        if(Flag1) fprintf(fp, "\"Ex\", ");
        if(Flag2) fprintf(fp, "\"Ey\", ");
        if(Flag3){
            for(isp=0;isp<nsp;isp++){
                fprintf(fp, "\"Den%d\", ",isp);
            }
        } 
        fprintf(fp, "\n");
        // GEOMETRY
        fprintf(fp, "GEOMETRY\nF=POINT\nCS=GRID\nX=0.00,Y=0.00,Z=0.00\nC=BLACK\nS=GLOBAL\nL=SOLID\nPL=4\nLT=0.1\n");
        fprintf(fp, "CLIPPING=CLIPTOVIEWPORT\nDRAWORDER=AFTERDATA\nMFC=\"\"\n");
        fprintf(fp, "T=RECTANGLE %g %g\n",xlength,ylength);
        for(i=0;i<CondNUM;i++){
            fprintf(fp, "GEOMETRY\nF=POINT\nCS=GRID\n");
            fprintf(fp, "X=%g,Y=%g,Z=0.00\n",CondX0[i]*dx,CondY0[i]*dy);
            fprintf(fp, "C=BLACK\nS=GLOBAL\nL=SOLID\nPL=4\nLT=0.1\n");
            fprintf(fp, "CLIPPING=CLIPTOVIEWPORT\nDRAWORDER=AFTERDATA\nMFC=\"\"\n");
            fprintf(fp, "T=RECTANGLE %g %g\n",CondX1[i]*dx-CondX0[i]*dx,CondY1[i]*dy-CondY0[i]*dy);
        }
        for(i=0;i<DielNUM;i++){
            fprintf(fp, "GEOMETRY\nF=POINT\nCS=GRID\n");
            fprintf(fp, "X=%g,Y=%g,Z=0.00\n",DielX0[i]*dx,DielY0[i]*dy);
            fprintf(fp, "C=BLACK\nS=GLOBAL\nL=SOLID\nPL=4\nLT=0.1\n");
            fprintf(fp, "CLIPPING=CLIPTOVIEWPORT\nDRAWORDER=AFTERDATA\nMFC=\"\"\n");
            fprintf(fp, "T=RECTANGLE %g %g\n",DielX1[i]*dx-DielX0[i]*dx,DielY1[i]*dy-DielY0[i]*dy);
        }
    }else{
        fp = fopen(filename,"a");
    }
    fprintf(fp, "ZONE I = %d, J = %d\n", ngy, ngx);
	fprintf(fp, "ZONETYPE = Ordered, DATAPACKING = POINT\n");
		for(i=0;i<ngx;i++) {
			for(j=0;j<ngy;j++) {
                fprintf(fp,"%g %g ",i*dx,j*dy);
                if(Flag1) fprintf(fp,"%g ",vec_G[i*ngy+j].Ex);
                if(Flag2) fprintf(fp,"%g ",vec_G[i*ngy+j].Ey);
                if(Flag3){
                    for(isp=0;isp<nsp;isp++){
                        fprintf(fp,"%g ",Host_G_sp[isp*Gsize + i*ngy+j].den);
                    }
                } 
				fprintf(fp,"\n");
			}
		}
	fclose(fp);				
}
void Tecplot_2D(){

}