#include "cuda_Fluid.cuh"

void Set_Fluid_cuda(){

    checkCudaErrors(hipMalloc((void**)&dev_FG, nfsp * sizeof(Fluid)));
    checkCudaErrors(hipMemcpy(dev_FG, FG, nfsp * sizeof(Fluid), hipMemcpyHostToDevice));
    
    checkCudaErrors(hipMalloc((void**)&dev_C_F, nfsp * Csize *  sizeof(GFC)));
    checkCudaErrors(hipMemcpy(dev_C_F, Host_C_F, nfsp * Csize *  sizeof(GFC), hipMemcpyHostToDevice));
}