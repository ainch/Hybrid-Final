#include "hip/hip_runtime.h"
#include "cuda_Fluid.cuh"

void Set_Fluid_cuda(){
    int isp,i,j,k,GID;
    int size,grid,block,mingrid; 

    checkCudaErrors(hipMalloc((void**)&dev_FG, nfsp * sizeof(Fluid)));
    checkCudaErrors(hipMemcpy(dev_FG, FG, nfsp * sizeof(Fluid), hipMemcpyHostToDevice));

    Sync_Fluid_GFCtoGFG_forDen(Fluid_sp, Fluid_Den); 
    checkCudaErrors(hipMalloc((void**)&dev_FG_Den, nfsp * Gsize * sizeof(GFG)));
    checkCudaErrors(hipMemcpy(dev_FG_Den, Fluid_Den, nfsp * Gsize * sizeof(GFG), hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc((void**)&dev_FG_Src, nfsp * Gsize * sizeof(GFG)));
    checkCudaErrors(hipMemcpy(dev_FG_Src, Fluid_Src, nfsp * Gsize * sizeof(GFG), hipMemcpyHostToDevice));
   
    // Calculate Diffusion coefficient
    if(MainGas == ARGON) 
        Cal_D_Argon(vec_C,vec_G,Fluid_sp);
    else if(MainGas == OXYGEN) 
        Cal_D_Oxygen(vec_C,vec_G,Fluid_sp);
    else if(MainGas == ARO2) 
        Cal_D_ArO2(vec_C,vec_G,Fluid_sp);

    // Gummel coefficient
    calculate_gummel_coef_x();
    calculate_gummel_coef_y();

    for(isp=0;isp<nfsp;isp++) {
        Calculate_Flux_x(isp);
        Calculate_Flux_y(isp);
    }
}
void Solve_Continuity_eqn(){
    int isp;
    for(isp=0;isp<nfsp;isp++) {
		Solve_Density_x(isp);
		Calculate_Flux_x(isp);
		Solve_Density_y(isp);
		Calculate_Flux_y(isp);
	}
}
void Solve_Continuity_eqn_check(){
    int isp;
    for(isp=0;isp<nfsp;isp++) {
        if(FG[isp].CSS_Flag){
		    Solve_Density_x(isp);
		    Calculate_Flux_x(isp);
		    Solve_Density_y(isp);
		    Calculate_Flux_y(isp);
        }
	}
}
void Calculate_Flux_x(int isp){
    int i, k, x1, x2, yy;
	float left_den, right_den;
    GFC *pointer;
    Con_RegionX *pointer2;
    pointer = &(Fluid_sp[isp]);
    pointer2 = &(Conti_x[isp]);

	for(k=0;k<Conti_xnum;k++){
		x1=pointer2->x1[k];
		x2=pointer2->x2[k];
		yy=pointer2->yy[k];
		pointer->flux_x[x1][yy] = -Conti_x[isp].fg1[k] * pointer->gummel_bx[x1][yy] * pointer->den[x1][yy];
		for(i=x1+1;i<=x2;i++) {
			left_den = pointer->den[i-1][yy];
			right_den = pointer->den[i][yy];
			pointer->flux_x[i][yy] = pointer->gummel_ax[i][yy]*left_den - pointer->gummel_bx[i][yy]*right_den;
		}
		pointer->flux_x[x2+1][yy] = Conti_x[isp].fg2[k]*pointer->gummel_ax[x2+1][yy]*pointer->den[x2][yy];
	}
}
void Calculate_Flux_y(int isp){
    int j, k, y1, y2, xx;
	float lower_den, upper_den;
    GFC *pointer;
    Con_RegionY * pointer2;
    pointer = &(Fluid_sp[isp]);
    pointer2 = &(Conti_y[isp]);

	for(k=0;k<Conti_ynum;k++){
		xx=pointer2->xx[k];
		y1=pointer2->y1[k];
		y2=pointer2->y2[k];
		pointer->flux_y[xx][y1] = -Conti_y[isp].fg1[k]*pointer->gummel_by[xx][y1]*pointer->den[xx][y1];
		for(j=y1+1;j<=y2;j++) {
			lower_den = pointer->den[xx][j-1];
			upper_den = pointer->den[xx][j];
			pointer->flux_y[xx][j] = pointer->gummel_ay[xx][j]*lower_den - pointer->gummel_by[xx][j]*upper_den;
		}
		pointer->flux_y[xx][y2+1] = Conti_y[isp].fg2[k]*pointer->gummel_ay[xx][y2+1]*pointer->den[xx][y2];
	}
}
void Solve_Density_x(int isp){
    int k,i,x1,x2,yy,num,xx;
    float src, ga, gb, gc;
    float subd[ngx], diag[ngx], superd[ngx], rhs[ngx], density[ngx], gam[ngx];
    float del_f;
    GFC *pointer;
    Con_RegionX *pointer2;
    pointer = &(Fluid_sp[isp]);
    pointer2 = &(Conti_x[isp]);
    del_f = dtc/dx;
    for(k=0;k<Conti_xnum;k++){
        x1=pointer2->x1[k];
		x2=pointer2->x2[k];
		yy=pointer2->yy[k];
		num = x2-x1+1;
		for(i=0;i<num;i++) {
			xx = i + x1;
            ga =  -pointer->gummel_ax[xx][yy]*del_f;
            gb =  1+(pointer->gummel_ax[xx+1][yy]+pointer->gummel_bx[xx][yy])*del_f;
            gc =  -pointer->gummel_bx[xx+1][yy]*del_f;
            src = pointer->den[xx][yy] + pointer->Source[xx][yy]*dtc - (pointer->flux_y[xx][yy+1]-pointer->flux_y[xx][yy])*dtc/dy;
            subd[i] =  ga;
			diag[i] = gb;
			superd[i] = gc;
			rhs[i] =  src;
		}
        tridiag(subd,diag,superd,rhs,density,gam,0,num-1);
        for(i=0;i<num;i++) {
			if(vec_C[(i+x1)*ncy+yy].PlasmaRegion == 1 && density[i] > 1e9){
				pointer->den[i+x1][yy] = density[i];
            }else{
				pointer->den[i+x1][yy] = 0.0f;
			}
		}
    }
}
void Solve_Density_y(int isp){
    int k,j,xx,y1,y2,num,yy;
    float src, ga, gb, gc;
    float subd[ngy], diag[ngy], superd[ngy], rhs[ngy], density[ngy], gam[ngy];
    float del_f;
    GFC *pointer;
    Con_RegionY * pointer2;
    pointer = &(Fluid_sp[isp]);
    pointer2 = &(Conti_y[isp]);
    del_f = dtc/dy;
    for(k=0;k<Conti_ynum;k++){
		xx=pointer2->xx[k];
		y1=pointer2->y1[k];
		y2=pointer2->y2[k];
		num = y2-y1+1;
		for(j=0;j<num;j++) {
			yy = j + y1;
            ga =  -pointer->gummel_ay[xx][yy]*del_f;
            gb =  1+(pointer->gummel_ay[xx][yy+1]+pointer->gummel_by[xx][yy])*del_f;
            gc =  -pointer->gummel_by[xx][yy+1]*del_f;
            src = pointer->den[xx][yy] + pointer->Source[xx][yy]*dtc - (pointer->flux_x[xx+1][yy]-pointer->flux_x[xx][yy])*dtc/dx;
            subd[j] = ga;
			diag[j] = gb;
			superd[j] = gc;
			rhs[j] = src;
		}
		tridiag(subd,diag,superd,rhs,density,gam,0,num-1);
        for(j=0;j<num;j++) {
			if(vec_C[xx*ncy+j+y1].PlasmaRegion == 1 && density[j] > 1e9){
				pointer->den[xx][j+y1]=density[j];
            }else{
				pointer->den[xx][j+y1]=0.0f;
			}
		}
	}
}
int tridiag(float *a, float *b, float *c, float *d, float *x, float *gam, int min, int max)
{
	int j;
	float bet;

	x[min]=d[min]/(bet=b[min]);
	for(j=min+1;j<=max;j++) {
		gam[j-1]=c[j-1]/bet;
		bet=b[j]-a[j]*c[j-1]/bet;
		x[j]=(d[j]-a[j]*x[j-1])/bet;
		if(bet==0) return 0;
	}

	for(j=max-1;j>=min;j--) x[j] -= gam[j]*x[j+1];

	return 1;
}
void calculate_gummel_coef_x(){
    int isp,k,i;
    int x1,x2,yy;
    for(isp=0;isp<nfsp;isp++){
        for(k=0;k<Conti_xnum;k++){
		    x1=Conti_x[isp].x1[k];
		    x2=Conti_x[isp].x2[k];
		    yy=Conti_x[isp].yy[k];
		    Fluid_sp[isp].gummel_bx[x1][yy]=-Conti_x[isp].fg1[k] * (-0.25*sqrt(vec_G[x1*ngy+yy].Temp*1.38e-23/FG[isp].mass));
		    Fluid_sp[isp].gummel_ax[x1][yy]=0;
		    for(i=x1+1;i<=x2;i++){
			    Fluid_sp[isp].gummel_ax[i][yy] = (Fluid_sp[isp].D[i-1][yy] + Fluid_sp[isp].D[i-1][yy])/2/dx;
			    Fluid_sp[isp].gummel_bx[i][yy] = Fluid_sp[isp].gummel_ax[i][yy];
		    }
		    Fluid_sp[isp].gummel_ax[x2+1][yy]=Conti_x[isp].fg2[k] * (0.25*sqrt(vec_G[x2*ngy+yy].Temp*1.38e-23/FG[isp].mass));
		    Fluid_sp[isp].gummel_bx[x2+1][yy]=0;
	    }
    }
}
void calculate_gummel_coef_y(){
    int isp,k,j;
    int xx,y1,y2;
    float aa=0,bb;
    for(isp=0;isp<nfsp;isp++){
        for(k=0;k<Conti_ynum;k++){
		    xx=Conti_y[isp].xx[k];
		    y1=Conti_y[isp].y1[k];
		    y2=Conti_y[isp].y2[k];
            Fluid_sp[isp].gummel_by[xx][y1]=-Conti_y[isp].fg1[k] * (-0.25*sqrt(vec_G[xx*ngy+y1].Temp*1.38e-23/FG[isp].mass));
		    Fluid_sp[isp].gummel_ay[xx][y1]=0;
		    for(j=y1+1;j<=y2;j++){
			    Fluid_sp[isp].gummel_ay[xx][j] = (Fluid_sp[isp].D[xx][j-1] + Fluid_sp[isp].D[xx][j])/2/dy;
			    Fluid_sp[isp].gummel_by[xx][j] = Fluid_sp[isp].gummel_ay[xx][j];
		    }
		    Fluid_sp[isp].gummel_ay[xx][y2+1]=Conti_y[isp].fg2[k] * (0.25*sqrt(vec_G[xx*ngy+y2].Temp*1.38e-23/FG[isp].mass));
		    Fluid_sp[isp].gummel_by[xx][y2+1]=0;
        }
    }
}
void Cal_D_Argon(GCA *vecC, GGA *vecG, GFC *data){
    int isp,i,j,CID,GID;
    float T = 0.0f;
    float epsi_k = 124.0f; 	// epsilon/K from Viscosity
	float sigmasq = 11.682724f; 	// r0, A. from Viscosity (3.418)^2
    float omega;
    for(isp=0;isp<nfsp;isp++){
        for(i=0;i<ncx;i++){
		    for(j=0;j<ncy;j++){
                CID = i * ncy + j;
                GID = i * ngy + j;
                T = 0.25*(vecG[GID].Temp + vecG[GID+1].Temp + vecG[GID+ngy].Temp + vecG[GID+ngy+1].Temp);
                omega = Ar_meta_omega(T/epsi_k);
                data[isp].D[i][j] = 1.997279e-4*sqrt(T/39.948)*T/(Total_Pressure*sigmasq*omega)*20;
            }
        }
    }
    //printf("D[%d] = %g\n",TID,data[TID].D);
}
void Cal_D_Oxygen(GCA *vecC, GGA *vecG, GFC *data){
    int isp,i,j,CID,GID;
    float T = 0.0f;
	float sigmasq1 = 0.10892f; 	// r0, A. from Viscosity 1/(3.03)^2  for OP  OD
    float sigmasq2 = 0.08650f; 	// r0, A. from Viscosity 1/(3.40)^2  for O2A O2B
    float omega;
    for(isp=0;isp<nfsp;isp++){
        for(i=0;i<ncx;i++){
		    for(j=0;j<ncy;j++){
                CID = i * ncy + j;
                GID = i * ngy + j;
                T = 0.25*(vecG[GID].Temp + vecG[GID+1].Temp + vecG[GID+ngy].Temp + vecG[GID+ngy+1].Temp);
                if(isp <= 1){
                    // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
                    data[isp].D[i][j] = 2.63e-3*sqrt(T*T*T*24/32/32)/Total_Pressure*sigmasq2;
                }else{
                     // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
                    data[isp].D[i][j] = 2.63e-3*sqrt(T*T*T*24/16/16)/Total_Pressure*sigmasq1;  
                }
            }
        }
    }
}
void Cal_D_ArO2(GCA *vecC, GGA *vecG, GFC *data){   
    int isp,i,j,CID,GID;
    float T = 0.0f;
	float sigmasq0 = 0.08559f;  // r0, A. from Viscosity 1/(3.418)^2 for AR meta
	float sigmasq1 = 0.10892f; 	// r0, A. from Viscosity 1/(3.03)^2  for OP  OD
    float sigmasq2 = 0.08650f; 	// r0, A. from Viscosity 1/(3.40)^2  for O2A O2B
    float omega;
    for(isp=0;isp<nfsp;isp++){
        for(i=0;i<ncx;i++){
		    for(j=0;j<ncy;j++){
                CID = i * ncy + j;
                GID = i * ngy + j;
                T = 0.25*(vecG[GID].Temp + vecG[GID+1].Temp + vecG[GID+ngy].Temp + vecG[GID+ngy+1].Temp);
                if(isp == 0){
                    //data[TID].D = 1.997279e-4*sqrt(T/39.948)*T/(press*sigmasq*omega)*20;
                    data[isp].D[i][j] = 2.63e-3*sqrt(T*T*T*24/39.948/39.948)/Total_Pressure*sigmasq0;
                }else if(isp == 1){
                    // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
                    data[isp].D[i][j] = 2.63e-3*sqrt(T*T*T*24/32/32)/Total_Pressure*sigmasq2;
                }else if(isp == 2){
                    // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
                    data[isp].D[i][j] = 2.63e-3*sqrt(T*T*T*24/32/32)/Total_Pressure*sigmasq2;
                }else if(isp == 3){
                    // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
                    data[isp].D[i][j] = 2.63e-3*sqrt(T*T*T*24/16/16)/Total_Pressure*sigmasq1;
                }else if(isp == 4){
                    // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
                    data[isp].D[i][j] = 2.63e-3*sqrt(T*T*T*24/16/16)/Total_Pressure*sigmasq1;
                }
            }
        }
    }
}
float Ar_meta_omega(float value){
    int   i;
	float tstar[27] = {0.250, 0.267, 0.286, 0.308, 0.333, 0.364, 0.400, 0.444,
	                   0.500, 0.571, 0.667, 0.800, 1.000, 1.330, 2.000, 2.500,
	                   2.860, 3.330, 4.000, 5.000, 6.670, 10.00, 12.50, 16.67,
	                   25.00, 50.00, 100.0},
	      omega[27] = {1.8714, 1.8344, 1.7962, 1.7560, 1.7142, 1.6702, 1.6238,
	                   1.5748, 1.5228, 1.4676, 1.4090, 1.3466, 1.2802, 1.2096,
	                   1.1348, 1.1042, 1.0888, 1.0738, 1.0592, 1.0450, 1.0318,
	                   1.0198, 1.0154, 1.0112, 1.0072, 1.0034, 1.000};
	float temp1;
    float temp2 = DBL_MIN;
	if      (value<tstar[0])  temp2=omega[0];
	else if (value>tstar[26]) temp2=omega[26];
	else
		for (i=1;i<27;i++)
			if (value<tstar[i]) {
				temp1=(value-tstar[i-1])/(tstar[i]-tstar[i-1]);
				temp2=omega[i]*temp1+omega[i-1]*(1-temp1);
			}
	return temp2;
}
void Sync_Fluid_GFCtoGFG_forDen(GFC *A, GFG *B){ // [ncx][ncy] --> [Gsize], cpu DATA > GPU
    int isp,i,j,GID;
    float buf;
    for(isp=0;isp<nfsp;isp++){
        buf = 0.0f;
        for(i=0;i<ncx;i++){
		    for(j=0;j<ncy;j++){
                GID = isp * Gsize + i * ngy + j;
                B[GID].n = A[isp].den[i][j];
                if(i == ncx-1 || j == ncy-1){
                    GID = isp * Gsize + (i+1) * ngy + j+1;
                    B[GID].n = A[isp].den[i][j];
                }
                buf += A[isp].den[i][j];
            }
        }
        FG[isp].ave_Den = buf/Csize;
    }
}
void Sync_Fluid_GFGtoGFC_forSource(GFG *A, GFC *B){ // [nsp*Gsize+Gsize] --> [ncx][ncy], GPU data > CPU
    int isp,i,j,GID;
    for(isp=0;isp<nfsp;isp++){
        for(i=0;i<ncx;i++){
		    for(j=0;j<ncy;j++){
                GID = isp * Gsize + i * ngy + j;
                B[isp].Source[i][j] = A[GID].n;
            }
        }
    }
}