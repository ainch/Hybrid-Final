#include "hip/hip_runtime.h"
#include "cuda_Fluid.cuh"

void Set_Fluid_cuda(){
    int isp,i,j,GID;
    int size,grid,block,mingrid; 
    Conti_Flag = 0;
    
    checkCudaErrors(hipMalloc((void**)&dev_FG, nfsp * sizeof(Fluid)));
    checkCudaErrors(hipMemcpy(dev_FG, FG, nfsp * sizeof(Fluid), hipMemcpyHostToDevice));

    // Density, D, nfsp * Csize
    checkCudaErrors(hipMalloc((void**)&dev_C_F, nfsp * Csize *  sizeof(GFC)));
    checkCudaErrors(hipMemcpy(dev_C_F, Host_C_F, nfsp * Csize *  sizeof(GFC), hipMemcpyHostToDevice));

    // Flux_x, Flux_y,  Gummel_ax,bx,ay,by : [nfsp] * [Gsize]
    Host_G_F = (GFG *) malloc(nfsp * Gsize * sizeof(GFG)); //__Global_Fluid_Gsize_Data
    for(isp=0;isp<nfsp;isp++){
        for(i=0;i<Gsize;i++){
            GID = isp*Gsize + i;
            Host_G_F[GID].Flux_x = 0.0f;
            Host_G_F[GID].Gummel_ax = 0.0f;
            Host_G_F[GID].Gummel_bx = 0.0f;
            Host_G_F[GID].Flux_y = 0.0f;
            Host_G_F[GID].Gummel_ay = 0.0f;
            Host_G_F[GID].Gummel_by = 0.0f;
        }
    }
    checkCudaErrors(hipMalloc((void**)&dev_G_F, nfsp * Gsize *  sizeof(GFG)));
    checkCudaErrors(hipMemcpy(dev_G_F, Host_G_F, nfsp * Gsize *  sizeof(GFG), hipMemcpyHostToDevice));

    // Calculate Diffusion coefficient
    if(MainGas == ARGON) Cal_D_Argon<<<CONTI_GRID,CONTI_BLOCK>>>(nfsp,ncy,Csize,Total_Pressure,dev_CvecSet,dev_GvecSet,dev_C_F);
    if(MainGas == OXYGEN) Cal_D_Oxygen<<<CONTI_GRID,CONTI_BLOCK>>>(nfsp,ncy,Csize,Total_Pressure,dev_CvecSet,dev_GvecSet,dev_C_F);
    if(MainGas == ARO2) Cal_D_ArO2<<<CONTI_GRID,CONTI_BLOCK>>>(nfsp,ncy,Csize,Total_Pressure,dev_CvecSet,dev_GvecSet,dev_C_F);
    checkCudaErrors(hipMemcpy(Host_C_F, dev_C_F, nfsp * Csize *  sizeof(GFC), hipMemcpyDeviceToHost));

    // Region check
    Conti_xnum = Cal_XRegion_check();
    Conti_ynum = Cal_YRegion_check();
    // number of tri cal = nfsp * Conti_xnum + nfsp * Conti_ynum
    // tridiag data size = nfsp * (3 * ncx * Conti_ynum)
    // tridiag data size = nfsp * (3 * ncy * Conti_xnum)
    size = nfsp * Conti_xnum;
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Cal_D_Argon,0,size); 
    grid = (size + block - 1) / block;
    CONTIx_GRID = dim3(grid, 1, 1);
    CONTIx_BLOCK = dim3(block, 1, 1);
    size = nfsp * Conti_ynum;
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Cal_D_Argon,0,size); 
    grid = (size + block - 1) / block;
    CONTIy_GRID = dim3(grid, 1, 1);
    CONTIy_BLOCK = dim3(block, 1, 1);
    //
    Conti_x = (Con_RegionX *) malloc(nfsp * Conti_xnum * sizeof(Con_RegionX));
    Conti_y = (Con_RegionY *) malloc(nfsp * Conti_ynum * sizeof(Con_RegionY));
    for(isp=0;isp<nfsp;isp++){
        Set_Con_Region(isp, Conti_x, Conti_y);
        Set_Con_Boundary(isp, Conti_x, Conti_y);
    } 
    checkCudaErrors(hipMalloc((void**)&dev_Conti_x, nfsp * Conti_xnum *  sizeof(Con_RegionX)));
    checkCudaErrors(hipMemcpy(dev_Conti_x, Conti_x, nfsp * Conti_xnum *  sizeof(Con_RegionX), hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void**)&dev_Conti_y, nfsp * Conti_ynum *  sizeof(Con_RegionY)));
    checkCudaErrors(hipMemcpy(dev_Conti_y, Conti_y, nfsp * Conti_ynum *  sizeof(Con_RegionY), hipMemcpyHostToDevice));

    // Calculate Gummel Coef 
    calculate_gummel_coef_x<<<CONTIx_GRID,CONTIx_BLOCK>>>(nfsp, Conti_xnum, ngy, Gsize, Csize, dx, dev_Conti_x, dev_FG, dev_C_F, dev_G_F);
    calculate_gummel_coef_y<<<CONTIy_GRID,CONTIy_BLOCK>>>(nfsp, Conti_ynum, ngy, Gsize, Csize, dy, dev_Conti_y, dev_FG, dev_C_F, dev_G_F);

    // Calculate Tridiagonal A matrix 
    // tridiag size 3 * ncx * Conti_xnum
    // tridiag size 3 * ncy * Conti_ynum


    hipDeviceSynchronize();
    exit(1);
}
__global__ void calculate_gummel_coef_x(int nfsp, int size, int ngy, int Gsize,int Csize, float dx, Con_RegionX *val, Fluid *info, GFC *Cdata, GFG *Gdata){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nfsp*size) return;
    int i,isp,ID;
    ID = TID%size;
    isp = TID/size;
    int x1,x2,yy;
    float aa=0,bb;
    
    x1=val[TID].x1;
	x2=val[TID].x2;
	yy=val[TID].yy;
    Gdata[isp*Gsize + x1*ngy+yy].Gummel_bx =- val[TID].fg1*(-0.25*info[isp].Vel);
    Gdata[isp*Gsize + x1*ngy+yy].Gummel_ax =  0;
	for(i=x1+1;i<=x2;i++){
		bb=(Cdata[isp*Csize + (i-1)*(ngy-1)+yy].D + Cdata[isp*Csize + i*(ngy-1)+yy].D)/2;
		Gdata[isp*Gsize + i*ngy+yy].Gummel_ax=bb/dx;
		Gdata[isp*Gsize + i*ngy+yy].Gummel_bx=bb/dx;
        printf("%g ",bb/dx);
	}
    Gdata[isp*Gsize + (x2+1)*ngy+yy].Gummel_bx = 0;
    Gdata[isp*Gsize + (x2+1)*ngy+yy].Gummel_ax =- val[TID].fg2*(0.25*info[isp].Vel);

}
__global__ void calculate_gummel_coef_y(int nfsp, int size, int ngy, int Gsize,int Csize, float dy, Con_RegionY *val, Fluid *info, GFC *Cdata, GFG *Gdata){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nfsp*size) return;
    int j,isp,ID;
    ID = TID%size;
    isp = TID/size;
    int xx,y1,y2;
    float aa=0,bb;

    xx=val[TID].xx;
	y1=val[TID].y1;
	y2=val[TID].y2;
    Gdata[isp*Gsize + xx*ngy+y1].Gummel_by =- val[TID].fg1*(-0.25*info[isp].Vel);
    Gdata[isp*Gsize + xx*ngy+y1].Gummel_ay =  0;
	for(j=y1+1;j<=y2;j++){
		bb=(Cdata[isp*Csize + xx*(ngy-1)+j-1].D + Cdata[isp*Csize + xx*(ngy-1)+j].D)/2;
		Gdata[isp*Gsize + xx*ngy+j].Gummel_ay=bb/dy;
		Gdata[isp*Gsize + xx*ngy+j].Gummel_by=bb/dy;
	}
    Gdata[isp*Gsize + xx*ngy+y2+1].Gummel_by = 0;
    Gdata[isp*Gsize + xx*ngy+y2+1].Gummel_ay =- val[TID].fg2*(0.25*info[isp].Vel);

}
void Set_Con_Boundary(int isp, Con_RegionX *Cx,Con_RegionY *Cy){
    int i, j ,k,GID,GID1,GID2,GID3;
	int x1, x2, y1, y2, xx, yy;
	for(i=0;i<Conti_xnum;i++) {
		x1=Cx[i].x1;
		x2=Cx[i].x2;
		yy=Cx[i].yy;
        GID = x1*ngy + yy;
        GID1 = x1*ngy + yy+1;
        GID2 = (x2+1)*ngy + yy;
        GID3 = (x2+1)*ngy + yy + 1;
		if(vec_G[GID].Boundary==NEUMANN || vec_G[GID1].Boundary==NEUMANN) {
			if(x1==0) 
                Cx[isp*Conti_xnum+i].fg1=0;
			else 
                Cx[isp*Conti_xnum+i].fg1=FG[isp].Gamma1;
		}
		else if(vec_G[GID].Boundary==DIELECTRIC) {
			Cx[isp*Conti_xnum+i].fg1=FG[isp].Gamma1;
		}
		else if(vec_G[GID].Boundary==CONDUCTOR) {
			Cx[isp*Conti_xnum+i].fg1=FG[isp].Gamma1;
		}
		else Cx[isp*Conti_xnum+i].fg1=FG[isp].Gamma1;

		if(vec_G[GID2].Boundary==NEUMANN || vec_G[GID3].Boundary==NEUMANN) {
			if(x2==ncx-1) Cx[isp*Conti_xnum+i].fg2=0;
			else Cx[isp*Conti_xnum+i].fg2=FG[isp].Gamma1;
		}
		else if(vec_G[GID2].Boundary==DIELECTRIC) {
			Cx[isp*Conti_xnum+i].fg2=FG[isp].Gamma1;
		}
		else if(vec_G[GID2].Boundary==CONDUCTOR) {
			Cx[isp*Conti_xnum+i].fg2=FG[isp].Gamma1;
		}
		else Cx[isp*Conti_xnum+i].fg2=FG[isp].Gamma1;

	}
	for(i=0;i<Conti_ynum;i++) {
        xx=Cy[i].xx;
		y1=Cy[i].y1;
		y2=Cy[i].y2;
        GID = xx*ngy + y1;
        GID1 = (xx+1)*ngy + y1;
        GID2 = xx*ngy + y2 + 1;
        GID3 = (xx+1)*ngy + y2+1;
		if(vec_G[GID].Boundary==NEUMANN || vec_G[GID1].Boundary==NEUMANN) {
			if(y1==0) Cy[isp*Conti_ynum+i].fg1=0;
			else Cy[isp*Conti_ynum+i].fg1=FG[isp].Gamma1;
		}
		else if(vec_G[GID].Boundary==DIELECTRIC) {
			Cy[isp*Conti_ynum+i].fg1=FG[isp].Gamma1;
		}
		else if(vec_G[GID].Boundary==CONDUCTOR) {
			Cy[isp*Conti_ynum+i].fg1=FG[isp].Gamma1;
		}
		else Cy[isp*Conti_ynum+i].fg1=FG[isp].Gamma1;
		if(vec_G[GID2].Boundary==NEUMANN || vec_G[GID3].Boundary==NEUMANN) {
			if(y2==ncy-1) Cy[isp*Conti_xnum+i].fg2=0;
			else Cy[isp*Conti_ynum+i].fg2=FG[isp].Gamma1;
		}
		else if(vec_G[GID2].Boundary==DIELECTRIC) {
			Cy[isp*Conti_ynum+i].fg2=FG[isp].Gamma1;
		}
		else if(vec_G[GID2].Boundary==CONDUCTOR) {
			Cy[isp*Conti_ynum+i].fg2=FG[isp].Gamma1;
		}
		else Cy[isp*Conti_ynum+i].fg2=FG[isp].Gamma1;
	}
}
void Set_Con_Region(int isp, Con_RegionX *Cx,Con_RegionY *Cy){
    int i, j ,k,add,CID,CID1,CID2;
	int x1, x2, y1, y2, xx, yy;
	k=0;
	for(j=0;j<ncy;j++) {
		for(i=0;i<ncx;i++) {
            CID = i*ncy + j;
            CID1 = (i-1)*ncy + j;
            CID2 = (i+1)*ncy + j;
			if(vec_C[CID].PlasmaRegion == 1) { // plasma
				if(i == 0) { //left side wallCycle of check
					Cx[isp * Conti_xnum + k].x1 = i;
					Cx[isp * Conti_xnum + k].yy = j;
				}
				else if(vec_C[CID1].PlasmaRegion != 1) { // if left side is not plasma region
					Cx[isp * Conti_xnum + k].x1 = i;
					Cx[isp * Conti_xnum + k].yy = j;
				}
				if(i == ncx-1) { //right side wall
					Cx[isp * Conti_xnum + k].x2 = i;
					k++;
				}
				else if(vec_C[CID2].PlasmaRegion != 1) { // if Right side is not plasma region
					Cx[isp * Conti_xnum + k].x2 = i;
					k++;
				}
			}
		}
	}
	k=0;
	for(i=0;i<ncx;i++) {
		for(j=0;j<ncy;j++) {
            CID = i*ncy + j;
            CID1 = i*ncy + j-1;
            CID2 = i*ncy + j+1;
			if(vec_C[CID].PlasmaRegion == 1) {
				if(j == 0) {
					Cy[isp * Conti_ynum + k].xx = i;
					Cy[isp * Conti_ynum + k].y1 = j;
				}
				else if(vec_C[CID1].PlasmaRegion != 1) {
					Cy[isp * Conti_ynum + k].xx = i;
					Cy[isp * Conti_ynum + k].y1 = j;
				}

				if(j == ncy-1) {
					Cy[isp * Conti_ynum + k].y2 = j;
					k++;
				}
				else if(vec_C[CID2].PlasmaRegion != 1) {
					Cy[isp * Conti_ynum + k].y2 = j;
					k++;
				}
			}
		}
	}
}
int Cal_XRegion_check(){
    int i,j,k;
    k=0;
    for(j=0;j<ncy;j++){
        for(i=0;i<ncx;i++){
            if(vec_C[i*ncy+j].PlasmaRegion == 1){
                if(i == ncx-1) {
                    k++;
                }else if(vec_C[(i+1)*ncy+j].PlasmaRegion != 1) {
                    k++;
                }
            }
        }
    }
    return k;
}
int Cal_YRegion_check(){
    int i,j,k;
    k=0;
    for(i=0;i<ncx;i++){
        for(j=0;j<ncy;j++){
            if(vec_C[i*ncy+j].PlasmaRegion == 1){
                if(j == ncy-1) {
                    k++;
                }else if(vec_C[i*ncy+j+1].PlasmaRegion != 1) {
                    k++;
                }
            }
        }
    }
    return k;
}
__global__ void Cal_D_Argon(int nfsp, int ncy, int Csize, float press, GCA *vec_C, GGA *vec_G, GFC *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nfsp*Csize) return;
    int ID = TID%Csize;
    int xi = ID/ncy;
    int yi = ID%ncy;
    int GID;
    float T = 0.0f;
    float epsi_k = 124.0f; 	// epsilon/K from Viscosity
	float sigmasq = 11.682724f; 	// r0, A. from Viscosity (3.418)^2
    float omega;

    GID = xi * (ncy+1) + yi;
    T = 0.25*(vec_G[GID].Temp + vec_G[GID+1].Temp + vec_G[GID+ncy+1].Temp + vec_G[GID+ncy+2].Temp);
    omega = Ar_meta_omega(T/epsi_k);
    data[TID].D = 1.997279e-4*sqrt(T/39.948)*T/(press*sigmasq*omega)*20;
    //printf("D[%d] = %g\n",TID,data[TID].D);
}
__global__ void Cal_D_Oxygen(int nfsp, int ncy, int Csize, float press, GCA *vec_C, GGA *vec_G, GFC *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nfsp*Csize) return;
    int ID = TID%Csize;
    int isp = TID/Csize;
    int xi = ID/ncy;
    int yi = ID%ncy;
    int GID;
    float T = 0.0f;
	float sigmasq1 = 0.10892f; 	// r0, A. from Viscosity 1/(3.03)^2  for OP  OD
    float sigmasq2 = 0.08650f; 	// r0, A. from Viscosity 1/(3.40)^2  for O2A O2B

    GID = xi * (ncy+1) + yi;
    T = 0.25*(vec_G[GID].Temp + vec_G[GID+1].Temp + vec_G[GID+ncy+1].Temp + vec_G[GID+ncy+2].Temp);
    if(isp <= 1){
        // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
        data[TID].D = 2.63e-3*sqrt(T*T*T*24/32/32)/press*sigmasq2;
        //if(ID == 510) printf("D2[%d] = %g\n",TID,data[TID].D);
    }else{
        // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
        data[TID].D = 2.63e-3*sqrt(T*T*T*24/16/16)/press*sigmasq1;
        //if(ID == 510) printf("D1[%d] = %g\n",TID,data[TID].D);
    }
}
__global__ void Cal_D_ArO2(int nfsp, int ncy, int Csize, float press, GCA *vec_C, GGA *vec_G, GFC *data){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
    if(TID>=nfsp*Csize) return;
    int ID = TID%Csize;
    int isp = TID/Csize;
    int xi = ID/ncy;
    int yi = ID%ncy;
    int GID;
    float T = 0.0f;
    //float epsi_k = 124.0f; 	// epsilon/K from Viscosity
	//float sigmasq = 11.682724f; // r0, A. from Viscosity (3.418)^2  for AR meta
    float sigmasq0 = 0.08559f;  // r0, A. from Viscosity 1/(3.418)^2 for AR meta
	float sigmasq1 = 0.10892f; 	// r0, A. from Viscosity 1/(3.03)^2  for OP  OD
    float sigmasq2 = 0.08650f; 	// r0, A. from Viscosity 1/(3.40)^2  for O2A O2B
    //float omega;

    GID = xi * (ncy+1) + yi;
    T = 0.25*(vec_G[GID].Temp + vec_G[GID+1].Temp + vec_G[GID+ncy+1].Temp + vec_G[GID+ncy+2].Temp);
    if(isp == 0){
        //omega = Ar_meta_omega(T/epsi_k);
        //data[TID].D = 1.997279e-4*sqrt(T/39.948)*T/(press*sigmasq*omega)*20;
        data[TID].D = 2.63e-3*sqrt(T*T*T*24/39.948/39.948)/press*sigmasq0;
        //if(ID == 510) printf("D0[%d] = %g\n",TID,data[TID].D);
    }else if(isp == 1){
        // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
        data[TID].D = 2.63e-3*sqrt(T*T*T*24/32/32)/press*sigmasq2;
        //if(ID == 510) printf("D1[%d] = %g\n",TID,data[TID].D);
    }else if(isp == 2){
        // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
        data[TID].D = 2.63e-3*sqrt(T*T*T*24/32/32)/press*sigmasq2;
        //if(ID == 510) printf("D2[%d] = %g\n",TID,data[TID].D);
    }else if(isp == 3){
        // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
        data[TID].D = 2.63e-3*sqrt(T*T*T*24/16/16)/press*sigmasq1;
        //if(ID == 510) printf("D3[%d] = %g\n",TID,data[TID].D);
    }else if(isp == 4){
        // D = [(2.63X10^3)/P/(SIGMA)^2] * sqrt( T^3 *(Ma+Mb)/2/Ma/Mb);
        data[TID].D = 2.63e-3*sqrt(T*T*T*24/16/16)/press*sigmasq1;
        //if(ID == 510) printf("D4[%d] = %g\n",TID,data[TID].D);
    }
}
__device__ float Ar_meta_omega(float value){
    int   i;
	float tstar[27] = {0.250, 0.267, 0.286, 0.308, 0.333, 0.364, 0.400, 0.444,
	                   0.500, 0.571, 0.667, 0.800, 1.000, 1.330, 2.000, 2.500,
	                   2.860, 3.330, 4.000, 5.000, 6.670, 10.00, 12.50, 16.67,
	                   25.00, 50.00, 100.0},
	      omega[27] = {1.8714, 1.8344, 1.7962, 1.7560, 1.7142, 1.6702, 1.6238,
	                   1.5748, 1.5228, 1.4676, 1.4090, 1.3466, 1.2802, 1.2096,
	                   1.1348, 1.1042, 1.0888, 1.0738, 1.0592, 1.0450, 1.0318,
	                   1.0198, 1.0154, 1.0112, 1.0072, 1.0034, 1.000};
	float temp1;
    float temp2 = DBL_MIN;
	if      (value<tstar[0])  temp2=omega[0];
	else if (value>tstar[26]) temp2=omega[26];
	else
		for (i=1;i<27;i++)
			if (value<tstar[i]) {
				temp1=(value-tstar[i-1])/(tstar[i]-tstar[i-1]);
				temp2=omega[i]*temp1+omega[i-1]*(1-temp1);
			}
	return temp2;
}