#include "hip/hip_runtime.h"
#include "cuda_mccO2.cuh"
__device__ void O2_Electron(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, O2CollD *CX, int TnRct, float*MCCR,GGA *BG){
	int i,j,k,n,index,index2,index3;
	int PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;
	int Iz_isp1,Iz_isp2;		
	float SumSigma,SumEngyLoss;

    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[0].St_num + TID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 1;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Energy loss
        // 2 : Attachment using maxwellv
		// 3 : ionization 1 -> 3 charged
		// 4 : dissociative  recombination just delete
		// 5 : Detachment 
        switch(Flag){
			case 0:{
				mofm = info_CX[0].mofM;
                R1 = hiprand_uniform(&LocalStates) * sigv[0].val / vel;
				if(engy > info_CX[0].Th_e &&R1<=(SumSigma=O2_CrossSection(0, engy, N_LOGX, idLOGX, CX))){
                    // R0 Elastic
					MCCR[TID*TnRct]++;
				}else if(engy > info_CX[1].Th_e && R1<=(SumSigma += O2_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
                    //"1.e+O2>e+O2*");
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+1]++;
				}else if(engy > info_CX[2].Th_e && R1<=(SumSigma += O2_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
                    //"2.e+O2>e+O2*");
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+2]++;
				}else if(engy > info_CX[3].Th_e && R1<=(SumSigma += O2_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
                    //"3.e+O2>e+O2A");
					engy-=info_CX[3].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+3]++;
				}else if(engy > info_CX[4].Th_e && R1<=(SumSigma += O2_CrossSection(4, engy, N_LOGX, idLOGX, CX))){
					//"4.e+O2>e+O2B");
                    engy-=info_CX[4].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+4]++;
				}else if(engy > info_CX[5].Th_e && R1<=(SumSigma += O2_CrossSection(5, engy, N_LOGX, idLOGX, CX))){
                    //"5.e+O2>e+O2*");
					engy-=info_CX[5].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+5]++;
				}else if(engy > info_CX[6].Th_e && R1<=(SumSigma += O2_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
                    //"6.e+O2>OP+O-"
                    Colltype = 2;
					MCCR[TID*TnRct+6]++;
                }else if(engy > info_CX[7].Th_e && R1<=(SumSigma += O2_CrossSection(7, engy, N_LOGX, idLOGX, CX))){
                    //"7.e+O2>e+2OP");
					engy-=info_CX[7].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+7]++;
                }else if(engy > info_CX[8].Th_e && R1<=(SumSigma += O2_CrossSection(8, engy, N_LOGX, idLOGX, CX))){
                    //"8.e+O2>e+OP+OD");
					engy-=info_CX[8].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+8]++;
                }else if(engy > info_CX[9].Th_e && R1<=(SumSigma += O2_CrossSection(9, engy, N_LOGX, idLOGX, CX))){
                    //"9.e+O2>e+2OD");
					engy-=info_CX[9].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+9]++;
                }else if(engy > info_CX[10].Th_e && R1<=(SumSigma += O2_CrossSection(10, engy, N_LOGX, idLOGX, CX))){
                    //"10.e+O2>2e+O2^");
                    Colltype = 3;
					engy-=info_CX[10].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+10]++;
                }else if(engy > info_CX[11].Th_e && R1<=(SumSigma += O2_CrossSection(11, engy, N_LOGX, idLOGX, CX))){
                    //"11.e+O2>e+OP+O*");
					engy-=info_CX[11].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+11]++;
                }else if(engy > info_CX[12].Th_e && R1<=(SumSigma += O2_CrossSection(12, engy, N_LOGX, idLOGX, CX))){
                    //"12.e+O2>e+O^+O-");
                    Colltype = 3;
					engy-=info_CX[12].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 2;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+12]++;
                }else if(engy > info_CX[13].Th_e && R1<=(SumSigma += O2_CrossSection(13, engy, N_LOGX, idLOGX, CX))){
                    //"13.e+O2>2e+O^+OP");  
                    Colltype = 3;
					engy-=info_CX[13].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+13]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[14].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[1].val / vel;
				if(engy > info_CX[14].Th_e &&R1<=(SumSigma=O2_CrossSection(14, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"14.e+O2A>2e+O2+");
					engy-=info_CX[14].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+14]++;
				}else if(engy > info_CX[15].Th_e && R1<=(SumSigma += O2_CrossSection(15, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; //"15.e+O2A>OP+O-");
					MCCR[TID*TnRct+15]++;
				}else if(engy > info_CX[16].Th_e && R1<=(SumSigma += O2_CrossSection(16, engy, N_LOGX, idLOGX, CX))){
					engy+=0.977f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+16]++;
				}else if(engy > info_CX[17].Th_e && R1<=(SumSigma += O2_CrossSection(17, engy, N_LOGX, idLOGX, CX))){
					engy+=0.977f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+17]++;
				}else if(engy > info_CX[18].Th_e && R1<=(SumSigma += O2_CrossSection(18, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[18].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+18]++;
				}else if(engy > info_CX[19].Th_e && R1<=(SumSigma += O2_CrossSection(19, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[19].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+19]++;
				}else if(engy > info_CX[20].Th_e && R1<=(SumSigma += O2_CrossSection(20, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[20].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+20]++;
				}else if(engy > info_CX[21].Th_e && R1<=(SumSigma += O2_CrossSection(21, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"21.e+O2A>2e+O^+OP");
					engy-=info_CX[21].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+21]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[22].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[2].val / vel;
                if(engy > info_CX[22].Th_e &&R1<=(SumSigma=O2_CrossSection(22, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"22.e+O2B>2e+O2^");
					engy-=info_CX[22].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+22]++;
				}else if(engy > info_CX[23].Th_e && R1<=(SumSigma += O2_CrossSection(23, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; //"23.e+O2B>OP+O-");
					MCCR[TID*TnRct+23]++;
				}else if(engy > info_CX[24].Th_e && R1<=(SumSigma += O2_CrossSection(24, engy, N_LOGX, idLOGX, CX))){
					engy+=1.627f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+24]++;
				}else if(engy > info_CX[25].Th_e && R1<=(SumSigma += O2_CrossSection(25, engy, N_LOGX, idLOGX, CX))){
					engy+=1.627f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+25]++;
				}else if(engy > info_CX[26].Th_e && R1<=(SumSigma += O2_CrossSection(26, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[26].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+26]++;
				}else if(engy > info_CX[27].Th_e && R1<=(SumSigma += O2_CrossSection(27, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[27].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+27]++;
				}else if(engy > info_CX[28].Th_e && R1<=(SumSigma += O2_CrossSection(28, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[28].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+28]++;
				}else if(engy > info_CX[29].Th_e && R1<=(SumSigma += O2_CrossSection(29, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"29.e+O2B>2e+O^+OP");
					engy-=info_CX[29].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+29]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[30].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[3].val / vel;
                if(engy > info_CX[30].Th_e && R1<=(SumSigma=O2_CrossSection(30, engy, N_LOGX, idLOGX, CX))){
					Colltype = 5; //"30.e+O->2e+OP");
					engy-=info_CX[30].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+30]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 4:{
				mofm = info_CX[31].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[4].val / vel;
                if(engy > info_CX[31].Th_e &&R1<=(SumSigma=O2_CrossSection(31, engy, N_LOGX, idLOGX, CX))){
					Colltype = 4; //"31.e+O2^>OP+OD");
					MCCR[TID*TnRct+31]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 5:{
				mofm = info_CX[32].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[5].val / vel;
                if(engy > info_CX[32].Th_e &&R1<=(SumSigma=O2_CrossSection(32, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[32].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+32]++;
				}else if(engy > info_CX[33].Th_e && R1<=(SumSigma += O2_CrossSection(33, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[33].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+33]++;
				}else if(engy > info_CX[34].Th_e && R1<=(SumSigma += O2_CrossSection(34, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[34].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+34]++;
				}else if(engy > info_CX[35].Th_e && R1<=(SumSigma += O2_CrossSection(35, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[35].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+35]++;
				}else if(engy > info_CX[36].Th_e && R1<=(SumSigma += O2_CrossSection(36, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[36].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+36]++;
				}else if(engy > info_CX[37].Th_e && R1<=(SumSigma += O2_CrossSection(37, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"37.e+OP>2e+O^");
					engy-=info_CX[37].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+37]++;
				}else if(engy > info_CX[38].Th_e && R1<=(SumSigma += O2_CrossSection(38, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[38].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+38]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 6:{
				mofm = info_CX[39].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[6].val / vel;
                if(engy > info_CX[39].Th_e &&R1<=(SumSigma=O2_CrossSection(39, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"39.e+OD>2e+O^");
					engy-=info_CX[39].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+39]++;
				}else if(engy > info_CX[40].Th_e && R1<=(SumSigma += O2_CrossSection(40, engy, N_LOGX, idLOGX, CX))){
					engy+=1.96f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+40]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		} 
        switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Energy loss
            dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : Attachment using maxwellv
			oldPNC = atomicAdd(&data[TID+3*Gsize].PtNumInCell,1);
			index = info[3].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+3*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            break;
        case 3: // 3 : ionization 1 -> 3 charged
			// second charged create
			oldPNC = atomicAdd(&data[TID+Iz_isp1*Gsize].PtNumInCell,1);
			index = info[Iz_isp1].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp1*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp1 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}
			// Third charged create
			oldPNC = atomicAdd(&data[TID+Iz_isp2*Gsize].PtNumInCell,1);
			index = info[Iz_isp2].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp2*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp2 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}
			// energy loss electron 
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 4: // 4 : dissociative  recombination just delete
            break;
		case 5: // 5 : // Detachment 
			//"30.e+O->2e+OP");
			// new electron
			//printf("Ecollision case 5\n");
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			// energy loss
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			// delete O-
			oldPNC = atomicAdd(&data[TID+3*Gsize].PtNumInCell,0);
			if(oldPNC>1){
				R2 = hiprand_uniform(&LocalStates);
				index = info[3].St_num + TID + oldPNC*Gsize;
				index2 = (int)((float)oldPNC * R2);
				index3 = info[3].St_num + TID + index2*Gsize;
				sp[index3].CellID = sp[index].CellID;
				sp[index3].x = sp[index].x;
				sp[index3].y = sp[index].y;
				sp[index3].vx = sp[index].vx;
				sp[index3].vy = sp[index].vy;
				sp[index3].vz = sp[index].vz;
				atomicAdd(&data[TID+3*Gsize].PtNumInCell,-1);
				//printf("2[%d][%d]: %g,%g,%g,%g,%g,\n",TID,sp[index].x,sp[index].y,sp[index].vx,sp[index].vy,sp[i].vz);
			}else if(oldPNC == 1){
				atomicAdd(&data[TID+3*Gsize].PtNumInCell,-1);
			}
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void O2_O2_ion(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, O2CollD *CX, int TnRct, float*MCCR,GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[1].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[1].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange O2+
		// 3 : Charge exchange O+
        switch(Flag){
			case 0:{
				mofm = info_CX[47].mofM;
                R1 = hiprand_uniform(&LocalStates) * sigv[12].val / vel;
				if(engy > info_CX[47].Th_e &&R1<=(SumSigma=O2_CrossSection(47, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+47]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[48].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[13].val / vel;
				if(engy > info_CX[48].Th_e &&R1<=(SumSigma=O2_CrossSection(48, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+48]++;
				}else if(engy > info_CX[49].Th_e &&R1<=(SumSigma=O2_CrossSection(49, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+49]++;
				}else if(engy > info_CX[50].Th_e &&R1<=(SumSigma=O2_CrossSection(50, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+50]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[51].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[14].val / vel;
                if(engy > info_CX[51].Th_e &&R1<=(SumSigma=O2_CrossSection(51, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+51]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[52].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[15].val / vel;
                if(engy > info_CX[52].Th_e && R1<=(SumSigma=O2_CrossSection(52, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+52]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        case 3: // 3 : Charge exchange o+
			oldPNC = atomicAdd(&data[TID+Gsize].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID+Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void O2_O_ion(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, O2CollD *CX, int TnRct, float*MCCR,GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[2].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[2].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange O2+
		// 3 : Charge exchange O+
        switch(Flag){
			case 0:{
				mofm = info_CX[53].mofM;
                R1 = hiprand_uniform(&LocalStates) * sigv[16].val / vel;
				if(engy > info_CX[53].Th_e &&R1<=(SumSigma=O2_CrossSection(53, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+53]++;
				}else if(engy > info_CX[54].Th_e &&R1<=(SumSigma=O2_CrossSection(54, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+54]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[55].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[17].val / vel;
				if(engy > info_CX[55].Th_e &&R1<=(SumSigma=O2_CrossSection(55, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+55]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[56].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[18].val / vel;
                if(engy > info_CX[56].Th_e &&R1<=(SumSigma=O2_CrossSection(56, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+56]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[57].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[19].val / vel;
                if(engy > info_CX[57].Th_e && R1<=(SumSigma=O2_CrossSection(57, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+57]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID-Gsize].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID-Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        case 3: // 3 : Charge exchange o+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void O2_O_negative(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, O2CollD *CX, int TnRct, float*MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;
	int Iz_isp1,Iz_isp2;		
	float SumSigma,SumEngyLoss;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[3].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
		//if(Flag !=0) printf("\n[%d] : Flag = %d \n\n",TID,Flag);
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[3].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Detachment using maxwellv
		// 3 : dissociative  recombination just delete
        switch(Flag){
			case 0:{
				mofm = info_CX[41].mofM;
                R1 = hiprand_uniform(&LocalStates) * sigv[9].val / vel;
				if(engy > info_CX[41].Th_e &&R1<=(SumSigma=O2_CrossSection(41, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+41]++;
				}else if(engy > info_CX[42].Th_e && R1<=(SumSigma += O2_CrossSection(42, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+42]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[43].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[10].val / vel;
				if(engy > info_CX[43].Th_e &&R1<=(SumSigma=O2_CrossSection(43, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+43]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[44].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[11].val / vel;
                if(engy > info_CX[44].Th_e &&R1<=(SumSigma=O2_CrossSection(44, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+44]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[45].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[12].val / vel;
                if(engy > info_CX[45].Th_e && R1<=(SumSigma=O2_CrossSection(45, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+45]++;
				}else{
					Null++;
				}
                break;
			}
			case 4:{
				mofm = info_CX[46].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[13].val / vel;
                if(engy > info_CX[46].Th_e &&R1<=(SumSigma=O2_CrossSection(46, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+46]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			//printf("\n[%d][%d] : oldPNC = %d \n\n",TID,Gsize,oldPNC);
			index = info[3].St_num + ID + oldPNC*Gsize;
			//printf("0[%d][%d]: %g,%g,%g,%g,%g,\n",TID,ID,sp[i].x,sp[i].y,sp[i].vx,sp[i].vy,sp[i].vz);
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			//printf("1[%d][%d]: %g,%g,%g,%g,%g,\n",TID,ID,sp[i].x,sp[i].y,sp[i].vx,sp[i].vy,sp[i].vz);
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : Detachment using maxwellv
			//printf("2[%d][%d]: %g,%g,%g,%g,%g,\n",TID,ID,sp[i].x,sp[i].y,sp[i].vx,sp[i].vy,sp[i].vz);
			oldPNC = atomicAdd(&data[ID].PtNumInCell,1);
			index = info[0].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = ID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			break;
        case 3: // 3 : dissociative  recombination just delete
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void  O2Collision_Check(int Gsize, int Csize, int ngy, int TID, float dt, int MCCn, float dtm, float dx, float dy,
                                        hiprandState *states, Species *info, GPG *data, GCP *sp, MCC_sigmav *sigv, GGA *BG, GFC *Fluid){
	int i,j,k,index,Randn;
	int ID,isp,CID,PNMC,MPNC;
    int PNC,Flag;
	int nx,ny,ngx;
	float Tprob,Prob1,Prob2,Prob3,Prob4,Prob5,Prob6,Prob7;
	float R1;
	ID = TID%Gsize;
    isp = TID/Gsize;
	nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	hiprandState LocalStates = states[TID];
	PNC = data[TID].PtNumInCell;
	MPNC = data[TID].MaxPtNumInCell;
	PNMC = 0;
	// Calculate total Collision probability.
    switch (isp){
    case 0: // Electron
        Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);
	    Prob2 = Prob1 + 1.0f - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);
        Prob3 = Prob2 + 1.0f - exp(-1*dtm*sigv[2].val*Fluid[CID+Csize].ave_den);
        Prob4 = Prob3 + 1.0f - exp(-1*dtm*sigv[3].val*data[ID+3*Gsize].den*info[3].np2c*dx*dy);
        Prob5 = Prob4 + 1.0f - exp(-1*dtm*sigv[4].val*data[ID+Gsize].den*info[1].np2c*dx*dy);
        Prob6 = Prob5 + 1.0f - exp(-1*dtm*sigv[5].val*Fluid[CID+2*Csize].ave_den);
        Prob7 = Prob6 + 1.0f - exp(-1*dtm*sigv[6].val*Fluid[CID+3*Csize].ave_den);
	    Tprob = Prob7;
		Randn = MCCn;
        break;
    case 1: // O2+
        Prob1 = 1.0 - exp(-1*dtm*sigv[12].val*Fluid[CID+2*Csize].ave_den);
	    Prob2 = Prob1 + 1.0 - exp(-1*dtm*sigv[13].val*BG[ID].BackDen1);
	    Prob3 = Prob2 + 1.0 - exp(-1*dtm*sigv[14].val*Fluid[CID].ave_den);
	    Prob4 = Prob3 + 1.0 - exp(-1*dtm*sigv[15].val*Fluid[CID+Csize].ave_den);
        Tprob = Prob4;
		Randn = MCCn;
        break;
    case 2: // O+
        Prob1 = 1.0 - exp(-1*dt*sigv[16].val*BG[ID].BackDen1);
	    Prob2 = Prob1 + 1.0 - exp(-1*dt*sigv[17].val*Fluid[CID+2*Csize].ave_den);
	    Prob3 = Prob2 + 1.0 - exp(-1*dt*sigv[18].val*Fluid[CID].ave_den);
	    Prob4 = Prob3 + 1.0 - exp(-1*dt*sigv[19].val*Fluid[CID+Csize].ave_den);
	    Tprob = Prob4;
		Randn = 1;
        break;
    case 3: // O-
        Prob1 = 1.0 - exp(-1*dt*sigv[7].val*BG[ID].BackDen1);
	    Prob2 = Prob1 + 1.0 - exp(-1*dt*sigv[8].val*Fluid[CID+2*Csize].ave_den);
	    Prob3 = Prob2 + 1.0 - exp(-1*dt*sigv[9].val*data[ID+Gsize].den*info[1].np2c*dx*dy);
	    Prob4 = Prob3 + 1.0 - exp(-1*dt*sigv[10].val*data[ID+2*Gsize].den*info[2].np2c*dx*dy);
	    Prob5 = Prob4 + 1.0 - exp(-1*dt*sigv[11].val*Fluid[CID].ave_den);
	    Tprob = Prob5;
		Randn = 1;
        break;
    default:
        break;
    }
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
        for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob) break;
		}
		if(R1 >= Tprob){ // no collision
			index = i - PNMC*Gsize;
            Flag = sp[i].CellID;
		}else{ // collision
			PNMC++;
			index = info[isp].St_num + ID + (MPNC-PNMC)*Gsize;
            switch (isp){
            case 0:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
                else if(R1 <= Prob4)	Flag = (int)3;
                else if(R1 <= Prob5)	Flag = (int)4;
                else if(R1 <= Prob6)	Flag = (int)5;
		        else			        Flag = (int)6;
                break;
            case 1:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
		        else			        Flag = (int)3;
                break;
            case 2:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
		        else			        Flag = (int)3;
                break;
            case 3:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
                else if(R1 <= Prob4)	Flag = (int)3;
		        else			        Flag = (int)4;
                break;
            default:
                break;
            }
		}
		sp[index].CellID = Flag;
		sp[index].vx=sp[i].vx;
		sp[index].vy=sp[i].vy;
		sp[index].vz=sp[i].vz;
        sp[index].x=sp[i].x;
		sp[index].y=sp[i].y;
		i+=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNumMCCInCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}
__device__ float O2_CrossSection(int R, float engy, int N_LOGX, float idLOGX, O2CollD *data){
	if(engy == 0) return 0.0;
	float lengy = log10(engy);
	float ee1, a1, a2;
	int ee2;
	ee1 = idLOGX * (lengy - data[0].xe);
	ee2 = (int)ee1;
	a1 = ee1 - ee2;
	a2 = 1 - a1;
    switch (R) {
        case 0 : 
            if(lengy < data[0].xe){
			    return data[0].cx_0;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_0 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_0+a1*data[ee2+1].cx_0;
            break;
        case 1 :
            if(lengy < data[0].xe){
			    return data[0].cx_1;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_1 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_1+a1*data[ee2+1].cx_1;
            break;
        case 2 :
            if(lengy < data[0].xe){
			    return data[0].cx_2;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_2 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_2+a1*data[ee2+1].cx_2;
            break;
        case 3 :
            if(lengy < data[0].xe){
			    return data[0].cx_3;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_3 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_3+a1*data[ee2+1].cx_3;
            break;
        case 4 :
            if(lengy < data[0].xe){
			    return data[0].cx_4;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_4 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_4+a1*data[ee2+1].cx_4;
            break;
        case 5 :
            if(lengy < data[0].xe){
			    return data[0].cx_5;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_5 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_5+a1*data[ee2+1].cx_5;
            break;
        case 6 :
            if(lengy < data[0].xe){
			    return data[0].cx_6;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_6 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_6+a1*data[ee2+1].cx_6;
            break;
        case 7 :
            if(lengy < data[0].xe){
			    return data[0].cx_7;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_7 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_7+a1*data[ee2+1].cx_7;
            break;
        case 8 :
            if(lengy < data[0].xe){
			    return data[0].cx_8;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_8 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_8+a1*data[ee2+1].cx_8;
            break;
        case 9 :
            if(lengy < data[0].xe){
			    return data[0].cx_9;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_9 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_9+a1*data[ee2+1].cx_9;
            break;
        case 10 :
            if(lengy < data[0].xe){
			    return data[0].cx_10;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_10 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_10+a1*data[ee2+1].cx_10;
            break;
        case 11 :
            if(lengy < data[0].xe){
			    return data[0].cx_11;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_11 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_11+a1*data[ee2+1].cx_11;
            break;
        case 12 :
            if(lengy < data[0].xe){
			    return data[0].cx_12;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_12 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_12+a1*data[ee2+1].cx_12;
            break;
        case 13 :
            if(lengy < data[0].xe){
			    return data[0].cx_13;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_13 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_13+a1*data[ee2+1].cx_13;
            break;
        case 14 :
            if(lengy < data[0].xe){
			    return data[0].cx_14;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_14 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_14+a1*data[ee2+1].cx_14;
            break;
        case 15 :
            if(lengy < data[0].xe){
			    return data[0].cx_15;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_15 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_15+a1*data[ee2+1].cx_15;
            break;
        case 16 :
            if(lengy < data[0].xe){
			    return data[0].cx_16;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_16 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_16+a1*data[ee2+1].cx_16;
            break;
        case 17 :
            if(lengy < data[0].xe){
			    return data[0].cx_17;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_17 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_17+a1*data[ee2+1].cx_17;
            break;
        case 18 :
            if(lengy < data[0].xe){
			    return data[0].cx_18;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_18 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_18+a1*data[ee2+1].cx_18;
            break;
        case 19 :
            if(lengy < data[0].xe){
			    return data[0].cx_19;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_19 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_19+a1*data[ee2+1].cx_19;
            break;
        case 20 :
            if(lengy < data[0].xe){
			    return data[0].cx_20;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_20 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_20+a1*data[ee2+1].cx_20;
            break;
        case 21 :
            if(lengy < data[0].xe){
			    return data[0].cx_21;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_21 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_21+a1*data[ee2+1].cx_21;
            break;
        case 22 :
            if(lengy < data[0].xe){
			    return data[0].cx_22;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_22 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_22+a1*data[ee2+1].cx_22;
            break;
        case 23 :
            if(lengy < data[0].xe){
			    return data[0].cx_23;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_23 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_23+a1*data[ee2+1].cx_23;
            break;
        case 24 :
            if(lengy < data[0].xe){
			    return data[0].cx_24;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_24 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_24+a1*data[ee2+1].cx_24;
            break;
        case 25 :
            if(lengy < data[0].xe){
			    return data[0].cx_25;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_25 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_25+a1*data[ee2+1].cx_25;
            break;
        case 26 :
            if(lengy < data[0].xe){
			    return data[0].cx_26;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_26 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_26+a1*data[ee2+1].cx_26;
            break;
        case 27 :
            if(lengy < data[0].xe){
			    return data[0].cx_27;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_27 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_27+a1*data[ee2+1].cx_27;
            break;
        case 28 :
            if(lengy < data[0].xe){
			    return data[0].cx_28;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_28 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_28+a1*data[ee2+1].cx_28;
            break;
        case 29 :
            if(lengy < data[0].xe){
			    return data[0].cx_29;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_29 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_29+a1*data[ee2+1].cx_29;
            break;
        case 30 :
            if(lengy < data[0].xe){
			    return data[0].cx_30;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_30 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_30+a1*data[ee2+1].cx_30;
            break;
        case 31 :
            if(lengy < data[0].xe){
			    return data[0].cx_31;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_31 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_31+a1*data[ee2+1].cx_31;
            break;
        case 32 :
            if(lengy < data[0].xe){
			    return data[0].cx_32;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_32 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_32+a1*data[ee2+1].cx_32;
            break;
        case 33 :
            if(lengy < data[0].xe){
			    return data[0].cx_33;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_33 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_33+a1*data[ee2+1].cx_33;
            break;
        case 34 :
            if(lengy < data[0].xe){
			    return data[0].cx_34;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_34 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_34+a1*data[ee2+1].cx_34;
            break;
        case 35 :
            if(lengy < data[0].xe){
			    return data[0].cx_35;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_35 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_35+a1*data[ee2+1].cx_35;
            break;
        case 36 :
            if(lengy < data[0].xe){
			    return data[0].cx_36;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_36 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_36+a1*data[ee2+1].cx_36;
            break;
        case 37 :
            if(lengy < data[0].xe){
			    return data[0].cx_37;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_37 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_37+a1*data[ee2+1].cx_37;
            break;
        case 38 :
            if(lengy < data[0].xe){
			    return data[0].cx_38;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_38 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_38+a1*data[ee2+1].cx_38;
            break;
        case 39 :
            if(lengy < data[0].xe){
			    return data[0].cx_39;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_39 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_39+a1*data[ee2+1].cx_39;
            break;
        case 40 :
            if(lengy < data[0].xe){
			    return data[0].cx_40;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_40 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_40+a1*data[ee2+1].cx_40;
            break;
        case 41 :
            if(lengy < data[0].xe){
			    return data[0].cx_41;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_41 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_41+a1*data[ee2+1].cx_41;
            break;
        case 42 :
            if(lengy < data[0].xe){
			    return data[0].cx_42;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_42 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_42+a1*data[ee2+1].cx_42;
            break;
        case 43 :
            if(lengy < data[0].xe){
			    return data[0].cx_43;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_43 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_43+a1*data[ee2+1].cx_43;
            break;
        case 44 :
            if(lengy < data[0].xe){
			    return data[0].cx_44;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_44 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_44+a1*data[ee2+1].cx_44;
            break;
        case 45 :
            if(lengy < data[0].xe){
			    return data[0].cx_45;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_45 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_45+a1*data[ee2+1].cx_45;
            break;
        case 46 :
            if(lengy < data[0].xe){
			    return data[0].cx_46;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_46 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_46+a1*data[ee2+1].cx_46;
            break;
        case 47 :
            if(lengy < data[0].xe){
			    return data[0].cx_47;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_47 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_47+a1*data[ee2+1].cx_47;
            break;
        case 48 :
            if(lengy < data[0].xe){
			    return data[0].cx_48;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_48 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_48+a1*data[ee2+1].cx_48;
            break;
        case 49 :
            if(lengy < data[0].xe){
			    return data[0].cx_49;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_49 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_49+a1*data[ee2+1].cx_49;
            break;
        case 50 :
            if(lengy < data[0].xe){
			    return data[0].cx_50;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_50 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_50+a1*data[ee2+1].cx_50;
            break;
        case 51 :
            if(lengy < data[0].xe){
			    return data[0].cx_51;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_51 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_51+a1*data[ee2+1].cx_51;
            break;
        case 52 :
            if(lengy < data[0].xe){
			    return data[0].cx_52;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_52 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_52+a1*data[ee2+1].cx_52;
            break;
        case 53 :
            if(lengy < data[0].xe){
			    return data[0].cx_53;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_53 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_53+a1*data[ee2+1].cx_53;
            break;
        case 54 :
            if(lengy < data[0].xe){
			    return data[0].cx_54;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_54 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_54+a1*data[ee2+1].cx_54;
            break;
        case 55 :
            if(lengy < data[0].xe){
			    return data[0].cx_55;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_55 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_55+a1*data[ee2+1].cx_55;
            break;
        case 56 :
            if(lengy < data[0].xe){
			    return data[0].cx_56;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_56 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_56+a1*data[ee2+1].cx_56;
            break;
        case 57 :
            if(lengy < data[0].xe){
			    return data[0].cx_57;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_57 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_57+a1*data[ee2+1].cx_57;
            break;
        default :
            printf("\nError : Call about cross section data in O2MCC.\n\n");
            return 0.0;
    }
}