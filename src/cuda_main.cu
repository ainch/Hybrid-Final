#include "hip/hip_runtime.h"
#include "cuda_main.cuh"

extern "C" void main_cuda()
{
    int isp,i,k,sum,dsum,dsum2;
    float dsum3,dsum4;
    hipEvent_t start, stop;
    float gputime;
    printf("-------------GPU_CUDA START-------------\n");
    info_Device();
    start_cuda();
    Set_Device_Parameter();
    Set_Particle_cuda();
    Set_NullCollisionTime_cuda();
	Set_DiagParameter_cuda();
    Set_SortBoundary_cuda();
	Set_MatrixPCG_cuda();
	if(Lap_Field_Solver_Test) PCG_Laplace_TEST();
    PCG_SOLVER_Laplace();
    Set_Fluid_cuda();
	Deposit_cuda();
    /*
    gputime_field	=0.0;
	gputime_efield	=0.0;
	gputime_diag	=0.0;
	gputime_move	=0.0;
	gputime_mcc		=0.0;
	gputime_continue=0.0;
	gputime_deposit	=0.0;
	gputime_sort	=0.0;
	gputime_trace	=0.0;
	gputime_dump	=0.0;
	totaltime		=0.0;
	TotalT_D		=0;
	TotalT_H		=0;
	TotalT_M		=0;
	TotalT_S		=0;
    */
    int np[5],np2[5],np3[5];
    float np4[5];
    np[0] = 0;np[1] = 0;np[2] = 0;np[3] = 0;np[4] = 0;
    np2[0] = 0;np2[1] = 0;np2[2] = 0;np2[3] = 0;np2[4] = 0;
    np3[0] = 0;np3[1] = 0;np3[2] = 0;np3[3] = 0;np3[4] = 0;
    np4[0] = 0.0f;np4[1] = 0.0f;np4[2] = 0.0f;np4[3] = 0.0f;np4[4] = 0.0f;
    while(1){
        //hipEventCreate(&start); hipEventCreate(&stop);
	    //hipEventRecord( start, 0 );
		PCG_SOLVER();
        //hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    //hipEventElapsedTime( &gputime, start, stop );
	    //hipEventDestroy( start );hipEventDestroy( stop );
        //gputime_field+=gputime;
		//totaltime+=gputime;
        //hipEventCreate(&start); hipEventCreate(&stop);
	    //hipEventRecord( start, 0 );
		Efield_cuda();
        //hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    //hipEventElapsedTime( &gputime, start, stop );
	    //hipEventDestroy( start );hipEventDestroy( stop );
        //gputime_efield+=gputime;
		//totaltime+=gputime;
        //hipEventCreate(&start); hipEventCreate(&stop);
	    //hipEventRecord( start, 0 );
        Move_cuda();
        //hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    //hipEventElapsedTime( &gputime, start, stop );
	    //hipEventDestroy( start );hipEventDestroy( stop );
        //gputime_move+=gputime;
		//totaltime+=gputime;
        //hipEventCreate(&start); hipEventCreate(&stop);
	    //hipEventRecord( start, 0 );
        SortBounndary_cuda();
        //hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    //hipEventElapsedTime( &gputime, start, stop );
	    //hipEventDestroy( start );hipEventDestroy( stop );
        //gputime_sort+=gputime;
		//totaltime+=gputime;
        //hipEventCreate(&start); hipEventCreate(&stop);
	    //hipEventRecord( start, 0 );
        if(MainGas == ARGON) MCC_Ar_cuda();
        if(MainGas == OXYGEN) MCC_O2_cuda();
        if(MainGas == ARO2) MCC_ArO2_cuda();
        //hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    //hipEventElapsedTime( &gputime, start, stop );
	    //hipEventDestroy( start );hipEventDestroy( stop );
        //gputime_mcc+=gputime;
		//totaltime+=gputime;
        //hipEventCreate(&start); hipEventCreate(&stop);
	    //hipEventRecord( start, 0 );
		Deposit_cuda();
        //hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    //hipEventElapsedTime( &gputime, start, stop );
	    //hipEventDestroy( start );hipEventDestroy( stop );
        //gputime_deposit+=gputime;
		//totaltime+=gputime;
        Tecplot_save();
        //
        t+=dt; // real time
        tstep++; // step
        //if(tstep == 1){
        if((tstep%CYCLE_NUM) == 0){
            cstep++; // Number of Cycle step
            hipMemcpy(Host_G_sp, dev_G_sp, nsp * Gsize * sizeof(GPG),hipMemcpyDeviceToHost);
            for(isp=0;isp<nsp;isp++){
                sum = 0;
                dsum = 0;
                dsum2 = 0;
                for(i=0;i<Gsize;i++){
                    if(vec_G[i].DensRegion){
                        sum +=Host_G_sp[isp*Gsize+i].PtNumInCell;
                        dsum +=Host_G_sp[isp*Gsize+i].PtNumMCCInCell;
                        dsum2 +=Host_G_sp[isp*Gsize+i].PtNullMCCInCell;   
                    }
                }
                np[isp] = sum;
                np2[isp] = dsum;
                np3[isp] = dsum2;
            }
            printf("Np = [%d],[%d],[%d],[%d],[%d]\n",np[0],np[1],np[2],np[3],np[4]);
            printf("mcc = [%d],[%d],[%d],[%d],[%d]\n",np2[0],np2[1],np2[2],np2[3],np2[4]);
            printf("null = [%d],[%d],[%d],[%d],[%d]\n",np3[0],np3[1],np3[2],np3[3],np3[4]);
            //printf("[%] = [%3.2g %],[%3.2g %],[%3.2g %],[%3.2g %],[%3.2g %]\n"
            //                        ,100*(float)np2[0]/(np[0]+np2[0]),100*(float)np2[1]/(np[1]+np2[1])
            //                        ,100*(float)np2[2]/(np[2]+np2[2]),100*(float)np2[3]/(np[3]+np2[3])
            //                        ,100*(float)np2[4]/(np[4]+np2[4]));
        }
        printf("TIME = %1.4e (s),[%3d][%3d], Iter = %3d, res = %1.3e\n",t,tstep,cstep,*FIter,*dot_result);
        //if(t>1e-3) break;    
        //if(tstep == 1){
        //if(cstep==200){
            hipMemcpy(Host_G_sp, dev_G_sp, nsp * Gsize * sizeof(GPG),hipMemcpyDeviceToHost);
            for(isp=0;isp<nsp;isp++){
                sum = 0;
                dsum = 0;
                dsum2 = 0;
                for(i=0;i<Gsize;i++){
                    if(vec_G[i].DensRegion){
                        sum +=Host_G_sp[isp*Gsize+i].PtNumInCell; 
                        dsum +=Host_G_sp[isp*Gsize+i].PtNumMCCInCell;
                        dsum2 +=Host_G_sp[isp*Gsize+i].PtNullMCCInCell;   
                    }
                }
                np[isp] = sum;
                np2[isp] = dsum;
                np3[isp] = dsum2;
            }
            if(MainGas == ARGON){
                printf("Np = [%d],[%d]\n",np[0],np[1]);
                printf("mcc = [%d],[%d]\n",np2[0],np2[1]);
                printf("null = [%d],[%d]\n",np3[0],np3[1]);
                printf("Coll = [%d],[%d]\n",np2[0]-np3[0],np2[1]-np3[1]);
                printf("[%] = [%3.2g %],[%3.2g %]\n",100*(float)np2[0]/(np[0]+np2[0]),100*(float)np2[1]/(np[1]+np2[1]));
                hipMemcpy(MCC_rate, dev_MCC_rate, Msize * sizeof(float), hipMemcpyDeviceToHost);
                for(isp=0;isp<TnRct;isp++){
                    for(i=0;i<Gsize;i++){
                        if(vec_G[i].DensRegion){
                            if(isp<=4){
                                np4[0] += MCC_rate[i*TnRct+isp];
                            }else if(isp>4 && isp<=6){
                                np4[1] += MCC_rate[i*TnRct+isp];
                            }
                        }
                    }
                    printf("R[%d] = [%g],[%g]\n",isp,np4[0],np4[1]);
                    np4[0] = 0.0f;np4[1] = 0.0f;
                }
            }
            if(MainGas == OXYGEN){
                printf("Np = [%d],[%d],[%d],[%d]\n",np[0],np[1],np[2],np[3]);
                printf("mcc = [%d],[%d],[%d],[%d]\n",np2[0],np2[1],np2[2],np2[3]);
                printf("null = [%d],[%d],[%d],[%d]\n",np3[0],np3[1],np3[2],np3[3]);
                printf("Coll = [%d],[%d],[%d],[%d]\n",np2[0]-np3[0],np2[1]-np3[1],np2[2]-np3[2],np2[3]-np3[3]);
                printf("[%] = [%3.2g %],[%3.2g %],[%3.2g %],[%3.2g %]\n"
                                    ,100*(float)np2[0]/(np[0]+np2[0]),100*(float)np2[1]/(np[1]+np2[1])
                                    ,100*(float)np2[2]/(np[2]+np2[2]),100*(float)np2[3]/(np[3]+np2[3]));
                hipMemcpy(MCC_rate, dev_MCC_rate, Msize * sizeof(float), hipMemcpyDeviceToHost);
                for(isp=0;isp<TnRct;isp++){
                    for(i=0;i<Gsize;i++){
                        if(vec_G[i].DensRegion){
                            if(isp<=40){
                                np4[0] += MCC_rate[i*TnRct+isp];
                            }else if(isp>40 && isp<=46){
                                np4[3] += MCC_rate[i*TnRct+isp];
                            }else if(isp>46 && isp<=52){
                                np4[1] += MCC_rate[i*TnRct+isp];
                            }else if(isp>52 && isp<=57){
                                np4[2] += MCC_rate[i*TnRct+isp];
                            }
                        }
                    }
                    printf("R[%d] = [%g],[%g],[%g],[%g]\n",isp,np4[0],np4[1],np4[2],np4[3]);
                    np4[0] = 0.0f;np4[1] = 0.0f;np4[2] = 0.0f;np4[3] = 0.0f;
                }
            }
            if(MainGas == ARO2){
                printf("Np = [%d],[%d],[%d],[%d],[%d]\n",np[0],np[1],np[2],np[3],np[4]);
                //printf("mcc = [%d],[%d],[%d],[%d],[%d]\n",np2[0],np2[1],np2[2],np2[3],np2[4]);
                //printf("null = [%d],[%d],[%d],[%d],[%d]\n",np3[0],np3[1],np3[2],np3[3],np3[4]);
                printf("   Coll = [%d],[%d],[%d],[%d],[%d]\n",np2[0]-np3[0],np2[1]-np3[1],np2[2]-np3[2],np2[3]-np3[3],np2[4]-np3[4]);
                printf("[%] = [%3.2g %],[%3.2g %],[%3.2g %],[%3.2g %],[%3.2g %]\n"
                                    ,100*(float)np2[0]/(np[0]+np2[0]),100*(float)np2[1]/(np[1]+np2[1])
                                    ,100*(float)np2[2]/(np[2]+np2[2]),100*(float)np2[3]/(np[3]+np2[3])
                                    ,100*(float)np2[4]/(np[4]+np2[4]));
                hipMemcpy(MCC_rate, dev_MCC_rate, Msize * sizeof(float), hipMemcpyDeviceToHost);
                for(isp=0;isp<TnRct;isp++){
                    for(i=0;i<Gsize;i++){
                        if(vec_G[i].DensRegion){
                            if(isp<=45){
                                np4[0] += MCC_rate[i*TnRct+isp];
                            }else if(isp>45 && isp<=51){
                                np4[4] += MCC_rate[i*TnRct+isp];
                            }else if(isp>51 && isp<=59){
                                np4[2] += MCC_rate[i*TnRct+isp];
                            }else if(isp>59 && isp<=64){
                                np4[3] += MCC_rate[i*TnRct+isp];
                            }else if(isp>64 && isp<=67){
                                np4[1] += MCC_rate[i*TnRct+isp];
                            }
                        }
                    }
                    //printf("R[%d] = [%g],[%g],[%g],[%g],[%g]\n",isp,np4[0],np4[1],np4[2],np4[3],np4[4]);
                    np4[0] = 0.0f;np4[1] = 0.0f;np4[2] = 0.0f;np4[3] = 0.0f;np4[4] = 0.0f;
                }
            }
            //time_sum = gputime_field+gputime_efield+gputime_move+gputime_sort+gputime_mcc+gputime_continue+gputime_deposit+gputime_diag+gputime_trace+gputime_dump;
            //fprintf(stderr, "\n");
	        //fprintf(stderr, "Total : time = %2.8f	(s)\n", time_sum * 0.001);
	        //fprintf(stderr, "Field	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_field * 0.001, gputime_field * 100 / time_sum);
	        //fprintf(stderr, "Efield	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_efield * 0.001, gputime_efield * 100 / time_sum);
	        //fprintf(stderr, "Move	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_move * 0.001, gputime_move * 100 / time_sum);
	        //fprintf(stderr, "Sort	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_sort * 0.001, gputime_sort * 100 / time_sum);
	        //fprintf(stderr, "Mcc	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_mcc * 0.001, gputime_mcc * 100 / time_sum);
	        //fprintf(stderr, "Depo	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_deposit * 0.001, gputime_deposit * 100 / time_sum);
	        //fprintf(stderr, "------------------------------------------------------------------------------\n");
            //break; 
            //exit(1);
        //}    
        if(isnan(*dot_result) || isinf(*dot_result)){
            printf("\n");
            hipMemcpy(Host_G_sp, dev_G_sp, nsp * Gsize * sizeof(GPG),hipMemcpyDeviceToHost);
            for(isp=0;isp<nsp;isp++){
                sum = 0;
                dsum3 = 0.0;
                dsum4 = 0.0;
                k = 0;
                for(i=0;i<Gsize;i++){
                    if(vec_G[i].DensRegion){
                        k++;
                        sum += Host_G_sp[isp*Gsize+i].PtNumInCell;
                        dsum3 += Host_G_sp[isp*Gsize+i].den * SP[isp].np2c/dx/dy;
                        dsum4 += Host_G_sp[isp*Gsize+i].sigma;
                        if(isnan(Host_G_sp[isp*Gsize+i].sigma)) printf("\tsigma[%d].[%d] = [%g]\n",isp,i,Host_G_sp[isp*Gsize+i].sigma);
                    }
                }
                printf("\tNP - %s : %d, %g, %g\n",SP[isp].name,sum,dsum3/k,dsum4/k);
                if(dsum3/k == 0) exit(1);
                //if(isp !=0 && isnan(dsum)) exit(1);
            }
            exit(1);
        }
        
        /*
        if((tstep%CYCLE_NUM) == 0){
            cstep++; // Number of Cycle step
            // time calculate
	        while(totaltime > 1000){
		        TotalT_S++;
		        totaltime = totaltime - 1000;
	        }
	        while(TotalT_S >= 60){
			        TotalT_M++;
			        TotalT_S -= 60;
	        }
	        while(TotalT_M >= 60){
			        TotalT_H++;
			        TotalT_M -= 60;
	        }
	        while(TotalT_H >= 24){
			        TotalT_D++;
			        TotalT_H -= 24;
	        }
	        fprintf(stderr, "\nDump at t=%1.5e(s)\n",t);
	        for (isp = 0; isp < nsp; isp++){
		        fprintf(stderr, "%s : %d, ", SP[isp].name, SP[isp].np);
	        }fprintf(stderr, "\n");
	        fprintf(stderr, "Domain size : %d X %d =%d,  ", ngx, ngy, Gsize);
	        fprintf(stderr, "Time: %d(d), %d(h), %d(m), %d(s)\n",TotalT_D,TotalT_H,TotalT_M,TotalT_S);
	        time_sum = gputime_field+gputime_efield+gputime_move+gputime_sort+gputime_mcc+gputime_continue+gputime_deposit+gputime_diag+gputime_trace+gputime_dump;
	        fprintf(stderr, "Total : time = %2.8f	(s)\n", time_sum * 0.001);
	        fprintf(stderr, "Field	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_field * 0.001, gputime_field * 100 / time_sum);
	        fprintf(stderr, "Efield	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_efield * 0.001, gputime_efield * 100 / time_sum);
	        fprintf(stderr, "Move	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_move * 0.001, gputime_move * 100 / time_sum);
	        fprintf(stderr, "Sort	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_sort * 0.001, gputime_sort * 100 / time_sum);
	        fprintf(stderr, "Mcc	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_mcc * 0.001, gputime_mcc * 100 / time_sum);
	        fprintf(stderr, "CONTI	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_continue * 0.001, gputime_continue * 100 / time_sum);
	        fprintf(stderr, "Depo	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_deposit * 0.001, gputime_deposit * 100 / time_sum);
	        fprintf(stderr, "Diag	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_diag * 0.001, gputime_diag * 100 / time_sum);
	        fprintf(stderr, "Dump	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_dump * 0.001, gputime_dump * 100 / time_sum);
	        fprintf(stderr, "------------------------------------------------------------------------------\n");
	
        } 
        */
    }
}


__global__ void testKernel(point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("B data[%d] %g, %g\n",i,p[i].a,p[i].b);
    p[i].a = 1.1;
    p[i].b = 2.2;
    //printf("A data[%d] %g, %g\n",i,p[i].a,p[i].b);
    
}
__global__ void MakeVectorForMoveKernel(int ngx,int ngy,point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].a = (float)ngx;
    p[i].b = (float)ngy;
	printf("point.a: %g, point.b: %g\n",p[i].a,p[i].b);
}
int test(void)
{
    // set number of points 
    int numPoints    = 16;
    int gpuBlockSize = 4;
    int pointSize    = sizeof(point);
    int numBytes     = numPoints * pointSize;
    int gpuGridSize  = numPoints / gpuBlockSize;
    point *cpuPointArray;
    point *gpuPointArray;
        // allocate memory
    cpuPointArray = (point*)malloc(numBytes);
    hipMalloc((void**)&gpuPointArray, numBytes);

    // launch kernel
    testKernel<<<gpuGridSize,gpuBlockSize>>>(gpuPointArray);

    // retrieve the results
    checkCudaErrors(hipMemcpy(cpuPointArray, gpuPointArray, numBytes, hipMemcpyDeviceToHost));
    //printf("testKernel results:\n");
    for(int i = 0; i < numPoints; ++i)
    {
        //printf("point.a: %g, point.b: %g\n",cpuPointArray[i].a,cpuPointArray[i].b);
    }
        // deallocate memory
    free(cpuPointArray);
    hipFree(gpuPointArray);

    return 0;
}
