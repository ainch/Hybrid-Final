#include "hip/hip_runtime.h"
#include "cuda_main.cuh"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
extern "C" void main_cuda()
{
    int isp,i,k,sum;
    float dsum,dsum2;
    int KEY0, KEY1, KEY2;
    printf("-------------GPU_CUDA START-------------\n");
    info_Device();
    start_cuda();
    Set_Device_Parameter();
    Set_Particle_cuda();
    Set_NullCollisionTime_cuda();
	Set_Diagnostic_cuda();
    Set_SortBoundary_cuda();
	Set_MatrixPCG_cuda();
	if(Lap_Field_Solver_Test) PCG_Laplace_TEST();
    //else{
    PCG_SOLVER_Laplace();
    Set_Fluid_cuda();
	Deposit_Basic();
    while(cstep<Basic_Flag || Basic_Flag < 0){
        t+=dt; // real time
        tstep++; // step
        if((tstep%CYCLE_NUM) == 0) cstep++;
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        PCG_SOLVER();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        (*EFIELD)();
        Move_cuda();
        SortBounndary_cuda();
        (*MCC_Basic)();
        Deposit_Basic();
        Diagnostic_Basic();
        SaveDumpFile(0,0,0);
        fprintf(stderr,"TIME = %1.4e (s),[%3d][%3d], Iter = %3d, Field Solve time=%2.4f (ms)\r",t,tstep,cstep,*FIter,gputime);
        if(isnan(*dot_result) || isinf(*dot_result)){
            printf("\nField solver Error!\n");
            exit(1);
        }
    }
    KEY2 = 0, KEY1 = 0, KEY0 = 0; // Save DumpFile version setting; 
    while(t<1e-3){
        t+=dt; // real time
        tstep++; // step
        if((tstep%CYCLE_NUM) == 0) cstep++;
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
		PCG_SOLVER();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_field+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        fprintf(stderr,"TIME = %1.4e (s),[%3d][%3d], Iter = %3d, Field Solve time=%2.4f (ms)\r",t,tstep,cstep,*FIter,gputime);
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
		(*EFIELD)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_efield+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*MOVE)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_move+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*SORT_BOUNDARY)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_sort+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*MCC)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_mcc+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
		(*DEPOSIT)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_deposit+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
		hipEventRecord( start, 0 );
		//if(Conti_Flag) (*CONTIEQ)();
		hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
		hipEventElapsedTime( &gputime, start, stop );
		hipEventDestroy( start );hipEventDestroy( stop );
		gputime_continue+=gputime;
		totaltime+=gputime;
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*DIAG)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_diag+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        Tecplot_save();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_Tec+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
		hipEventRecord( start, 0 );
		SaveDumpFile(KEY2,KEY1,KEY0);
		hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
		hipEventElapsedTime( &gputime, start, stop );
		hipEventDestroy( start );hipEventDestroy( stop );
		gputime_dump+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////  
        if(isnan(*dot_result) || isinf(*dot_result)){
            printf("\nField solver Error!\n");
            exit(1);
        }
        //if(tstep > 5) break;
        //exit(1);
        //if(tstep > 3) exit(1);
    }
    //}
}