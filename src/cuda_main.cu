#include "hip/hip_runtime.h"
#include "cuda_main.cuh"

extern "C" void main_cuda()
{
    int isp;
    hipEvent_t start, stop;
    printf("-------------GPU_CUDA START-------------\n");
    info_Device();
    start_cuda();
    Set_Device_Parameter();
    Set_Particle_cuda();
    Set_NullCollisionTime_cuda();
	Set_DiagParameter_cuda();
	Set_MatrixPCG_cuda();
	if(Lap_Field_Solver_Test) PCG_Laplace_TEST();
    PCG_SOLVER_Laplace();
	Deposit_cuda();
    while(1){
		PCG_SOLVER();
		Efield_cuda();
        //START
		//(*MOVE)();
		//(*SORT_BOUNDARY)();
		//(*MCC)();
		//if(add_izrate_flag==1) AddIzRate_cuda(); // ADD IONIZATION RATE instead of ICP SOURCE
		//(*DEPOSIT)();
		//if(Meta_flag){// && t>5e-7)
		//	(*CONTIEQ)();
		//}
		//(*DIAG)();
		//SaveDumpFile();
        //
        
        t+=dt; // real time
        tstep++; // step
        if((tstep%CYCLE_NUM) == 0) cstep++; // Number of Cycle step
        printf("TIME = %2.4g (s), STEP = %d (#), CYCLE = %d\n",t,tstep,cstep);
        //if(t>1e-3) break;    
        if(tstep>=1) break;    
        /*
        if((tstep%CYCLE_NUM) == 0){
            cstep++; // Number of Cycle step
            // time calculate
	        while(totaltime > 1000){
		        TotalT_S++;
		        totaltime = totaltime - 1000;
	        }
	        while(TotalT_S >= 60){
			        TotalT_M++;
			        TotalT_S -= 60;
	        }
	        while(TotalT_M >= 60){
			        TotalT_H++;
			        TotalT_M -= 60;
	        }
	        while(TotalT_H >= 24){
			        TotalT_D++;
			        TotalT_H -= 24;
	        }
	        fprintf(stderr, "\nDump at t=%1.5e(s)\n",t);
	        for (isp = 0; isp < nsp; isp++){
		        fprintf(stderr, "%s : %d, ", SP[isp].name, SP[isp].np);
	        }fprintf(stderr, "\n");
	        fprintf(stderr, "Domain size : %d X %d =%d,  ", ngx, ngy, Gsize);
	        fprintf(stderr, "Time: %d(d), %d(h), %d(m), %d(s)\n",TotalT_D,TotalT_H,TotalT_M,TotalT_S);
	        time_sum = gputime_field+gputime_efield+gputime_move+gputime_sort+gputime_mcc+gputime_continue+gputime_deposit+gputime_diag+gputime_trace+gputime_dump;
	        fprintf(stderr, "Total : time = %2.8f	(s)\n", time_sum * 0.001);
	        fprintf(stderr, "Field	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_field * 0.001, gputime_field * 100 / time_sum);
	        fprintf(stderr, "Efield	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_efield * 0.001, gputime_efield * 100 / time_sum);
	        fprintf(stderr, "Move	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_move * 0.001, gputime_move * 100 / time_sum);
	        fprintf(stderr, "Sort	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_sort * 0.001, gputime_sort * 100 / time_sum);
	        fprintf(stderr, "Mcc	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_mcc * 0.001, gputime_mcc * 100 / time_sum);
	        fprintf(stderr, "CONTI	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_continue * 0.001, gputime_continue * 100 / time_sum);
	        fprintf(stderr, "Depo	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_deposit * 0.001, gputime_deposit * 100 / time_sum);
	        fprintf(stderr, "Diag	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_diag * 0.001, gputime_diag * 100 / time_sum);
	        fprintf(stderr, "Dump	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_dump * 0.001, gputime_dump * 100 / time_sum);
	        fprintf(stderr, "------------------------------------------------------------------------------\n");
	
        } 
        */
    }
}


__global__ void testKernel(point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //printf("B data[%d] %g, %g\n",i,p[i].a,p[i].b);
    p[i].a = 1.1;
    p[i].b = 2.2;
    //printf("A data[%d] %g, %g\n",i,p[i].a,p[i].b);
    
}
__global__ void MakeVectorForMoveKernel(int ngx,int ngy,point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].a = (float)ngx;
    p[i].b = (float)ngy;
	printf("point.a: %g, point.b: %g\n",p[i].a,p[i].b);
}
int test(void)
{
    // set number of points 
    int numPoints    = 16;
    int gpuBlockSize = 4;
    int pointSize    = sizeof(point);
    int numBytes     = numPoints * pointSize;
    int gpuGridSize  = numPoints / gpuBlockSize;
    point *cpuPointArray;
    point *gpuPointArray;
        // allocate memory
    cpuPointArray = (point*)malloc(numBytes);
    hipMalloc((void**)&gpuPointArray, numBytes);

    // launch kernel
    testKernel<<<gpuGridSize,gpuBlockSize>>>(gpuPointArray);

    // retrieve the results
    checkCudaErrors(hipMemcpy(cpuPointArray, gpuPointArray, numBytes, hipMemcpyDeviceToHost));
    //printf("testKernel results:\n");
    for(int i = 0; i < numPoints; ++i)
    {
        //printf("point.a: %g, point.b: %g\n",cpuPointArray[i].a,cpuPointArray[i].b);
    }
        // deallocate memory
    free(cpuPointArray);
    hipFree(gpuPointArray);

    return 0;
}
