#include "hip/hip_runtime.h"
#include "cuda_main.cuh"

extern "C" void main_cuda()
{
    info_Device();
    start_cuda();
    test();
}


__global__ void testKernel(point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("B data[%d] %g, %g\n",i,p[i].a,p[i].b);
    p[i].a = 1.1;
    p[i].b = 2.2;
    printf("A data[%d] %g, %g\n",i,p[i].a,p[i].b);
}
__global__ void MakeVectorForMoveKernel(int ngx,int ngy,point *p)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].a = ngx;
    p[i].b = ngy;
}
int test(void)
{
    // set number of points 
    int numPoints    = 1;
    int gpuBlockSize = 1;
    int pointSize    = sizeof(point);
    int numBytes     = numPoints * pointSize;
    int gpuGridSize  = numPoints / gpuBlockSize;
    point *cpuPointArray;
    point *gpuPointArray;
        // allocate memory
    cpuPointArray = (point*)malloc(numBytes);
    gpuPointArray = (point*)malloc(numBytes);
    hipMalloc((void**)&gpuPointArray, numBytes);

    // launch kernel
    MakeVectorForMoveKernel<<<gpuGridSize,gpuBlockSize>>>(ngx,ngy,gpuPointArray);
    testKernel<<<gpuGridSize,gpuBlockSize>>>(gpuPointArray);

    // retrieve the results
    checkCudaErrors(hipMemcpy(cpuPointArray, gpuPointArray, numBytes, hipMemcpyDeviceToHost));
    printf("testKernel results:\n");
    for(int i = 0; i < numPoints; ++i)
    {
        printf("point.a: %g, point.b: %g\n",cpuPointArray[i].a,cpuPointArray[i].b);
    }
        // deallocate memory
    free(cpuPointArray);
    hipFree(gpuPointArray);

    return 0;
}
