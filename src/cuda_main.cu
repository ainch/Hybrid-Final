#include "hip/hip_runtime.h"
#include "cuda_main.cuh"
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
extern "C" void main_cuda()
{
    int isp,i,k,sum;
    float dsum,dsum2;
    int KEY0, KEY1, KEY2;
    printf("-------------GPU_CUDA START-------------\n");
    info_Device();
    start_cuda();
    Set_Device_Parameter();
    Set_Particle_cuda();
    Set_NullCollisionTime_cuda();
	Set_Diagnostic_cuda();
    Set_SortBoundary_cuda();
	Set_MatrixPCG_cuda();
	if(Lap_Field_Solver_Test) PCG_Laplace_TEST();
    PCG_SOLVER_Laplace();
    Set_Fluid_cuda();
	Deposit_cuda();
    while(cstep<Basic_Flag){
        t+=dt; // real time
        tstep++; // step
        if((tstep%CYCLE_NUM) == 0) cstep++;
        PCG_SOLVER();
        Efield_cuda();
        Move_cuda();
        SortBounndary_cuda();
        if(MainGas == ARGON) MCC_Ar_cuda();
        else if(MainGas == OXYGEN) MCC_O2_cuda();
        else if(MainGas == ARO2) MCC_ArO2_cuda();
        Deposit_cuda();
        Diagnostic();
        SaveDumpFile(0,0,0);
        printf("TIME = %1.4e (s), Iter = %3d, res = %1.3e\r",t,*FIter,*dot_result);
        if(isnan(*dot_result) || isinf(*dot_result)){
            printf("\nField solver Error!\n");
            exit(1);
        }
    }
    KEY2 = 0, KEY1 = 0, KEY0 = 0; // Save DumpFile version setting; 
    while(t<1e-3){
        t+=dt; // real time
        tstep++; // step
        if((tstep%CYCLE_NUM) == 0) cstep++;
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
		PCG_SOLVER();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_field+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        fprintf(stderr,"TIME = %1.4e (s),[%3d][%3d], Iter = %3d, res = %1.3e\r",t,tstep,cstep,*FIter,*dot_result);
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
		Efield_cuda();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_efield+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*MOVE)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_move+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*SORT_BOUNDARY)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_sort+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*MCC)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_mcc+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
		(*DEPOSIT)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_deposit+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
		hipEventRecord( start, 0 );
		//if(Conti_Flag) (*CONTIEQ)();
		hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
		hipEventElapsedTime( &gputime, start, stop );
		hipEventDestroy( start );hipEventDestroy( stop );
		gputime_continue+=gputime;
		totaltime+=gputime;
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        (*DIAG)();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_diag+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        Tecplot_save();
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        gputime_Tec+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////
        hipEventCreate(&start); hipEventCreate(&stop);
		hipEventRecord( start, 0 );
		SaveDumpFile(KEY2,KEY1,KEY0);
		hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
		hipEventElapsedTime( &gputime, start, stop );
		hipEventDestroy( start );hipEventDestroy( stop );
		gputime_dump+=gputime;
		totaltime+=gputime;        
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        ///////////////////////////////////////////////////////////////////////////  
        if(isnan(*dot_result) || isinf(*dot_result)){
            printf("\nField solver Error!\n");
            exit(1);
        }
        //if(tstep > 3) exit(1);
    }
}