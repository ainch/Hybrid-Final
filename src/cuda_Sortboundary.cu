#include "hip/hip_runtime.h"
#include "cuda_Sortboundary.cuh"


void SortBounndary_cuda(){
	SortBoundary_Basic<<<SORT_GRID, SORT_BLOCK>>>(Gsize,ngy,CondNUMR,dt_dx,dt_dy, dev_StructureIndex, dev_info_sp, dev_sp, dev_G_sp, dev_GvecSet, dev_CondVec, dev_ReArgFlag);
	hipDeviceSynchronize();
}
void Set_SortBoundary_cuda(){
	int size;
	size = (ncx + 2) * (ncy + 2);
	checkCudaErrors(hipMalloc((void**) &dev_StructureIndex, size * sizeof(int)));
    checkCudaErrors(hipMemset((void *) dev_StructureIndex, 0.0, size * sizeof(int)));
	checkCudaErrors(hipMemcpy(dev_StructureIndex, vec_StructureIndex , size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**) &dev_ReArgFlag, sizeof(int)));
	checkCudaErrors(hipMemset((void *) dev_ReArgFlag, 0, sizeof(int)));
}	
__global__ void SortBoundary_Basic(int Gsize,int ngy, int CondNum, float dt_dx,float dt_dy,int *StructureIndex, Species *info, GCP *sp, GPG *data, GGA *Field, GCondA *Cond, int *ReArgFlag){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	int isp,ID;
    if(TID>=Gsize*info[0].spnum) return;
    isp = (int)TID/Gsize; //species number [< nsp]
    ID = (int)TID%Gsize; // Grid ID [< Gsize]
	
	int MPNC,PNMIC,oldPNC;
	int CID,MCID,SMCID,DID;
	int PNC;
	int SIndex;
	int index,i,x,k;
	int del_pa,del_pb;
	float delta,del_a,del_b;
	int Flag_type,Flag_x,Flag_y,Flag_vx,Flag_vy;
	int Sum_CID;

	PNMIC = data[TID].PtNumMoveInterCell;
	if(PNMIC==0) return;

	MPNC = data[TID].MaxPtNumInCell;
	PNC = data[TID].PtNumInCell;
	if(PNC+PNMIC>0.9*MPNC) {
		*ReArgFlag=1;
	}
	x = ID/ngy;
	i = info[isp].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMIC;k++){
		// Where do the particles go?
		MCID = sp[i].CellID;
		if(MCID>1) // ngy, ngy+1, ngy-1
			SMCID = (ID + ngy + 2 + x) + (MCID+1);	
		else if(MCID<-1) // -ngy, -ngy+1, -ngy-1
			SMCID = (ID + ngy + 2 + x) + (MCID-1);
		else // +1, -1
			SMCID = (ID + ngy + 2 + x) + (MCID);	
		
		CID = TID + MCID; // Destination
		SIndex=StructureIndex[SMCID]; // Destination information
		// 0: PLASMA
		// 1 ~ : Dielectric >> del_a, del_b
		// 100 ~ : Conductor >> charge
		// -2, -4, -6, -8 : Neumann >>  position velocity update
		// -12, -16, -20, -24 : Neumann edge >>  position velocity update

		del_pa = 0; del_pb = 0;
		del_a = 0.0; del_b = 0.0;

		Flag_type = 1;  
		if(SIndex==0){ // Plasma
			Sum_CID = 0;
			Flag_x = 1;	Flag_y = 1;	Flag_vx = 1; Flag_vy = 1;
		}else if(SIndex>=1 && SIndex<100){ // Dielectric
			Flag_type = 0; 
			if(MCID == 1){ // top
				delta = sp[i].x - sp[i].vx/sp[i].vy * sp[i].y;
 				del_pa = CID;		del_a = 1-delta;
				del_pb = CID+ngy;	del_b = delta;
			}else if(MCID == -1){ //bottom
				delta = sp[i].x+sp[i].vx/sp[i].vy*(1-sp[i].y);
				del_pa = TID;		del_a = 1 - delta;
				del_pb = TID+ngy;	del_b = delta;
			}else if(MCID==ngy){		 	// right
				delta = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
				del_pa = CID;		del_a = 1 - delta;
				del_pb = CID+1;		del_b = delta;
			}else if(MCID==-ngy){	    // left
				delta = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
				del_pa = TID;		del_a = 1 - delta;
				del_pb = TID+1;		del_b = delta;
			}else if(MCID==ngy+1){
				delta = sp[i].x-sp[i].vx/sp[i].vy*sp[i].y;
				if(delta>=0) {
					if(StructureIndex[SMCID-1]) { // right surf --> right cell  --> ngy+1
						delta = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_pa = CID - 1;	del_a = -delta;
						del_pb = CID;		del_b = 1 + delta;						
					}else {
						del_pa = CID;		del_a = 1 - delta;
						del_pb = CID+ngy;	del_b = delta;		
					}
				}else { 							// top surf --> top cell --> ngy+1
					if(StructureIndex[SMCID - ngy - 1]) {
						del_pa = TID + 1;	del_a = -delta;
						del_pb = CID;		del_b = 1 + delta;	
					}else {
						delta = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_pa = CID;		del_a = 1 - delta;
						del_pb = CID+1;		del_b = delta;	
					}
				}
			}
			else if(MCID==ngy-1){
				delta = sp[i].x+sp[i].vx/sp[i].vy*(1-sp[i].y);
				if(delta>=0) { // right surf --> right cell  --> ngy-1
					if(StructureIndex[SMCID+1]) {
						delta = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_pa = CID + 1;	del_a = 2 - delta;
						del_pb = CID + 2;	del_b = delta - 1;	
					}
					else {
						del_pa = CID + 1;	del_a = 1 - delta;
						del_pb = CID+1+ngy;	del_b = delta;	
					}
				}
				else {		 // Bottom surf --> Bottom cell  --> ngy-1
					if(StructureIndex[SMCID-ngy-1]) {
						del_pa = TID;		del_a = -delta;
						del_pb = CID + 1;	del_b = 1 + delta;	
					}
					else {
						delta = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_pa = CID;		del_a = 1 - delta;
						del_pb = CID+1;		del_b = delta;	
					}
				}
			}
			else if(MCID==-ngy+1){
				delta = sp[i].x-sp[i].vx/sp[i].vy*sp[i].y;
				if(delta<1) { // Left surf --> Left cell  --> -ngy+1
					if(StructureIndex[SMCID-1]) {
						delta = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_pa = CID+ngy-1;	del_a = -delta;
						del_pb = CID+ngy;	del_b = 1 + delta;	
					}
					else {
						del_pa = CID;		del_a = 1-delta;
						del_pb = CID+ngy;	del_b = delta;	
					}
				}
				else {  // top surf --> top cell  --> -ngy+1
					if(StructureIndex[SMCID+ngy+1]) {
						del_pa = TID + 1;	del_a = 2-delta;
						del_pb = TID+1+ngy;	del_b = delta-1;	
					}
					else {
						delta = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_pa = TID+1;		del_a = 1-delta;
						del_pb = TID+2;		del_b = delta;
					}
				}
			}
			else if(MCID==-ngy-1){
				delta = sp[i].x+sp[i].vx/sp[i].vy*(1-sp[i].y);
				if(delta<1) { // Left surf --> Left cell  --> -ngy-11
					if(StructureIndex[SMCID+1]) {
						delta = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_pa = TID;		del_a = 2-delta;
						del_pb = TID+1;		del_b = delta-1;
					}
					else {
						del_pa = CID;		del_a = 1-delta;
						del_pb = TID;		del_b = delta;	
					}
				}
				else { // Bottom surf --> Bottom cell  --> -ngy-1
					if(StructureIndex[SMCID+ngy+1]) {
						del_pa = TID;		del_a = 2 - delta;
						del_pb = TID+ngy;	del_b = delta - 1;	
					}
					else {
						delta = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_pa = TID-1;		del_a = 1 - delta;
						del_pb = TID;		del_b = delta;	
					}
				}
			}
		}else if(SIndex>=100){ // Conductor
			Flag_type = 2;
			index = SIndex - 101;
		}else if(SIndex==-1){ // Dirichlet B.C
			Flag_type = 3; 
		}else if(SIndex==-2){ // Neumann B.C 
			Sum_CID = ngy;
			Flag_x = 0;	Flag_y = 1;	Flag_vx = -1; Flag_vy = 1;
		}else if(SIndex==-6){ // Neumann B.C 
			Sum_CID = -ngy;
			Flag_x = 0;	Flag_y = 1;	Flag_vx = -1; Flag_vy = 1;
		}else if(SIndex==-4){ // Neumann B.C 
			Sum_CID = 1;
			Flag_x = 1;	Flag_y = 0;	Flag_vx = 1; Flag_vy = -1;
		}else if(SIndex==-8){ // Neumann B.C 
			Sum_CID = -1;
			Flag_x = 1;	Flag_y = 0;	Flag_vx = 1; Flag_vy = -1;
		}else if(SIndex==-12){ // Neumann B.C 
			Sum_CID = ngy+1;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}else if(SIndex==-16){ // Neumann B.C 
			Sum_CID = 1-ngy;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}else if(SIndex==-20){ // Neumann B.C 
			Sum_CID = ngy-1;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}else if(SIndex==-24){ // Neumann B.C 
			Sum_CID = -ngy-1;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}
		if(Flag_type==0){
			atomicAdd(&data[del_pa].sigma,del_a);
			atomicAdd(&data[del_pb].sigma,del_b);
		}else if(Flag_type==1){
			CID+=Sum_CID;
			oldPNC = atomicAdd(&data[CID].PtNumInCell,1);
			DID = (int)CID%Gsize;
			index = info[isp].St_num + DID + oldPNC * Gsize; // ??
			sp[index].CellID = CID;
			if(Flag_x==1) sp[index].x = sp[i].x;
			else sp[index].x = 1-sp[i].x;
			if(Flag_y==1) sp[index].y = sp[i].y;
			else sp[index].y = 1-sp[i].y;
			sp[index].vx = Flag_vx * sp[i].vx;
			sp[index].vy = Flag_vy * sp[i].vy;
			sp[index].vz = sp[i].vz;
		}else if(Flag_type==2){
			atomicAdd(&Cond[isp*CondNum + index].Charge,1.0);
		}
		i-=Gsize;
	}		
}