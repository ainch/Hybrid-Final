#include "hip/hip_runtime.h"
#include "cuda_Sortboundary.cuh"


void SortBounndary_cuda(){
	SortBoundary_Basic<<<SORT_GRID, SORT_BLOCK>>>(Gsize,ngy,dt_dx,dt_dy,dev_StructureIndex, SP,dev_sp, dev_G_sp, dev_GvecSet, dev_CondVec, dev_ReArgFlag);
}
void Set_SortBoundary_cuda(){
	int size;
	size = (ncx + 2) * (ncy + 2);
	checkCudaErrors(hipMalloc((void**) &dev_StructureIndex, size * sizeof(int)));
    checkCudaErrors(hipMemset((void *) dev_StructureIndex, 0.0, size * sizeof(int)));
	checkCudaErrors(hipMemcpy(dev_StructureIndex, vec_StructureIndex , size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMalloc((void**) &dev_ReArgFlag, sizeof(int)));
	checkCudaErrors(hipMemset((void *) dev_ReArgFlag, 0, sizeof(int)));
}	
__global__ void SortBoundary_Basic(int Gsize,int ngy,float dt_dx,float dt_dy,int *StructureIndex, Species *info, GCP *sp, GPG *data, GGA *Field, GCondA *Cond, int *ReArgFlag){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	int isp,ID;
    if(TID>Gsize*info[0].spnum) return;
    isp = (int)TID/Gsize; //species number [< nsp]
    ID = (int)TID%Gsize; // Grid ID [< Gsize]

	int MPNC,PNMIC,oldPNC;
	int CID,MCID,SMCID;
	int PNC;
	int SIndex;
	int index,i,x,k;
	int del_pa,del_pb;
	float del_a,del_b;
	int Flag_type,Flag_x,Flag_y,Flag_vx,Flag_vy;
	int Sum_CID, Sum_Charge;
	// Flag_type = 0 : Particle into Meterial or Dirichlet B.C

	PNMIC = data[TID].PtNumMoveInterCell;
	if(PNMIC==0) return;

	MPNC = data[TID].MaxPtNumInCell;
	PNC = data[TID].PtNumInCell;
	if(PNC+PNMIC>0.9*MPNC) {
		*ReArgFlag=1;
	}
	x=ID/ngy;
	i = info[isp].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMIC;k++){
		// Where do the particles go?
		MCID = sp[i].CellID;
		if(MCID>1) // ngy, ngy+1, ngy-1
			SMCID = (ID + ngy + 2 + x) + (MCID+1);	
		else if(MCID<-1) // -ngy, -ngy+1, -ngy-1
			SMCID = (ID + ngy + 2 + x) + (MCID-1);
		else // +1, -1
			SMCID = (ID + ngy + 2 + x) + (MCID);	
		
		CID = TID + MCID; // Destination
		SIndex=StructureIndex[SMCID]; // Destination information
		// 0: PLASMA
		// 1 ~ : Dielectric >> del_a, del_b
		// 100 ~ : Conductor >> charge
		// -2, -4, -6, -8 : Neumann >>  position velocity update
		// -12, -16, -20, -24 : Neumann edge >>  position velocity update

		Sum_Charge = 0;
		del_pa = 0; del_pb = 0;
		del_a = 0.0; del_b = 0.0;

		Flag_type = 1;  
		if(SIndex==0){ // Plasma
			Sum_CID = 0;
			Flag_x = 1;	Flag_y = 1;	Flag_vx = 1; Flag_vy = 1;
		}else if(SIndex>=1 && SIndex<100){ // Dielectric
			Flag_type = 0; 
			if(MCID == 1){ // top
 				del_pb = CID+ngy;
				del_b = sp[i].x-sp[i].vx/sp[i].vy*sp[i].y;
				del_pa = CID;
				del_a = 1-del_b;
			}else if(MCID == -1){ //bottom
				del_pb = TID+ngy;
				del_b = sp[i].x+sp[i].vx/sp[i].vy*(1-sp[i].y);
				del_pa = TID;
				del_a = 1-del_b;
			}else if(MCID==ngy){		 	// right
				del_pb = CID+1;
				del_b = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
				del_pa = CID;
				del_a = 1-del_b;
			}else if(MCID==-ngy){	    // left
				del_pb = TID+1;
				del_b = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
				del_pa = TID;
				del_a = 1-del_b;
			}else if(MCID==ngy+1){
				del_b = sp[i].x-sp[i].vx/sp[i].vy*sp[i].y;
				if(del_b>=0) {
					if(StructureIndex[SMCID-1]) { // right surf --> right cell  --> ngy+1
						del_b = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_b = 1+del_b;
						del_pb = CID;
						del_pa = CID-1;
						del_a = -1*del_b;
					}else {
						del_pb = CID + ngy;
						del_pa = CID;
						del_a = 1-del_b;
					}
				}else { 							// top surf --> top cell --> ngy+1
					if(StructureIndex[SMCID-ngy-1]) {
						del_pb = CID;
						del_pa = TID+1;
						del_a = -1  *del_b;
						del_b = 1 + del_b;
					}else {
						del_b = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_pb = CID+1;
						del_pa = CID;
						del_a = 1-del_b;
					}
				}
			}
			else if(MCID==ngy-1){
				del_b = sp[i].x+sp[i].vx/sp[i].vy*(1-sp[i].y);
				if(del_b>=0) { // right surf --> right cell  --> ngy-1
					if(StructureIndex[SMCID+1]) {
						del_b = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_b = del_b - 1;
						del_pb = CID + 2;
						del_pa = CID + 1;
						del_a = 2 - del_b;
					}
					else {
						del_pb = CID+1+ngy;
						del_pa = CID + 1;
						del_a = 1 - del_b;
					}
				}
				else {		 // Bottom surf --> Bottom cell  --> ngy-1
					if(StructureIndex[SMCID-ngy-1]) {
						del_b = 1+del_b;
						del_pb = CID + 1;
						del_pa = TID;
						del_a = -1*del_b;
					}
					else {
						del_b = sp[i].y-sp[i].vy/sp[i].vx*sp[i].x;
						del_pb = CID+1;
						del_pa = CID;
						del_a = 1 - del_b;
					}
				}
			}
			else if(MCID==-ngy+1){
				del_b = sp[i].x-sp[i].vx/sp[i].vy*sp[i].y;
				if(del_b<1) { // Left surf --> Left cell  --> -ngy+1
					if(StructureIndex[SMCID-1]) {
						del_b = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_b = 1+del_b;
						del_pb = CID+ngy;
						del_pa = CID+ngy-1;
						del_a = -1*del_b;
					}
					else {
						del_pb = CID+ngy;
						del_pa = CID;
						del_a = 1-del_b;
					}
				}
				else {  // top surf --> top cell  --> -ngy+1
					if(StructureIndex[SMCID+ngy+1]) {
						del_b = del_b-1;
						del_pb = TID+1+ngy;
						del_pa = TID+1;
						del_a = 2-del_b;
					}
					else {
						del_b = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_pb = TID+2;
						del_pa = TID+1;
						del_a = 1-del_b;
					}
				}
			}
			else if(MCID==-ngy-1){
				del_b = sp[i].x+sp[i].vx/sp[i].vy*(1-sp[i].y);
				if(del_b<1) { // Left surf --> Left cell  --> -ngy-11
					if(StructureIndex[SMCID+1]) {
						del_b = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_b = del_b-1;
						del_pb = TID+1;
						del_pa = TID;
						del_a = 2-del_b;
					}
					else {
						del_pb = TID;
						del_pa = CID;
						del_a = 1-del_b;
					}
				}
				else { // Bottom surf --> Bottom cell  --> -ngy-1
					if(StructureIndex[SMCID+ngy+1]) {
						del_b = del_b-1;
						del_pb = TID+ngy;
						del_pa = TID;
						del_a = 2-del_b;
					}
					else {
						del_b = sp[i].y+sp[i].vy/sp[i].vx*(1-sp[i].x);
						del_pb = TID;
						del_pa = TID-1;
						del_a = 1-del_b;
					}
				}
			}
		}else if(SIndex>=100){ // Conductor
			Flag_type = 2;
			index = SIndex - 100;
		}else if(SIndex==-1){ // Dirichlet B.C
			Flag_type = 3; 
		}else if(SIndex==-2){ // Neumann B.C 
			Sum_CID = ngy;
			Flag_x = 0;	Flag_y = 1;	Flag_vx = -1; Flag_vy = 1;
		}else if(SIndex==-6){ // Neumann B.C 
			Sum_CID = -ngy;
			Flag_x = 0;	Flag_y = 1;	Flag_vx = -1; Flag_vy = 1;
		}else if(SIndex==-4){ // Neumann B.C 
			Sum_CID = 1;
			Flag_x = 1;	Flag_y = 0;	Flag_vx = 1; Flag_vy = -1;
		}else if(SIndex==-8){ // Neumann B.C 
			Sum_CID = -1;
			Flag_x = 1;	Flag_y = 0;	Flag_vx = 1; Flag_vy = -1;
		}else if(SIndex==-12){ // Neumann B.C 
			Sum_CID = ngy+1;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}else if(SIndex==-16){ // Neumann B.C 
			Sum_CID = 1-ngy;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}else if(SIndex==-20){ // Neumann B.C 
			Sum_CID = ngy-1;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}else if(SIndex==-24){ // Neumann B.C 
			Sum_CID = -ngy-1;
			Flag_x = 0;	Flag_y = 0;	Flag_vx = -1; Flag_vy = -1;
		}
		if(Flag_type==1){
			CID+=Sum_CID;
			oldPNC = atomicAdd(&data[CID].PtNumInCell,1);
			index = info[isp].St_num + ID + oldPNC * Gsize; // ??
			sp[index].CellID = CID;
			if(Flag_x==1) sp[index].x = sp[i].x;
			else sp[index].x = 1-sp[i].x;
			if(Flag_y==1) sp[index].y = sp[i].y;
			else sp[index].y = 1-sp[i].y;
			sp[index].vx= Flag_vx * sp[i].vx;
			sp[index].vy= Flag_vy * sp[i].vy;
			sp[index].vz=sp[i].vz;
		}else if(Flag_type==2){
			atomicAdd(&Cond[isp*index].Charge,1.0);
		}else if(Flag_type==3){

		}else{
			atomicAdd(&data[del_pa].sigma,del_a);
			atomicAdd(&data[del_pb].sigma,del_b);
		}
		i-=Gsize;
	}		
}