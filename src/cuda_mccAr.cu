#include "hip/hip_runtime.h"
#include "cuda_mccAr.cuh"
__device__ void Direct_Argon_Electron(int Gsize, int ngy, int ID, int MCCn, float dtm, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct, float*MCCR, GGA *BG, GFC *Fluid){
	int i,j,k,n,index;
	int CID,PNC,Null,PNC2;
	int nx,ny,ngx;
	int Target,oldPNC;
	int Colltype;
	float Tprob,Prob1,Prob2;
    int Randn,AddPt1;
	float R1,R2;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,vel2,engy,rengy;
	float SumSigma,SumEngyLoss;
	PNC = data[ID].PtNumInCell;
    PNC2 = data[ID+Gsize].PtNumInCell;
	Null = 0;
	hiprandState LocalStates = states[ID];
    nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);
	Prob2 = 1.0f - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);
	Tprob = Prob1 + Prob2;
    // Calculate total Collision probability.
	Randn = MCCn;
    AddPt1 = 0;
	i = info[0].St_num + ID;
	for(k=0;k<PNC;k++){
        Colltype = 0;
		for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob){
                Colltype = 1;
                break;
            }
		}
        if(Colltype == 0){
            i+=Gsize;
			continue;
        }
        if(R1 <= Prob1)	Target = (int)0;
		else			Target = (int)1;
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 2;
        switch(Target){
			case 0:{
                R2 = hiprand_uniform(&LocalStates) * sigv[0].val / vel;
				// 0. e + Ar > e + Ar 			Elastic Scattering
				SumSigma = Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX);
				if(R2<=SumSigma){
				// 1. e + Ar > e + Ar* 			Excitation to Total Excited state
					MCCR[ID*TnRct]++;
				}else if(engy > info_CX[1].Th_e && R2<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[ID*TnRct+1]++;
				// 2. e + Ar > e + Ar* 			Excitation to AR4SM
				}else if(engy > info_CX[2].Th_e && R2<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[ID*TnRct+2]++;
				// 3. e + Ar > e + e + Ar^		Direct ionization
				}else if(engy > info_CX[3].Th_e && R2<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					AddPt1++;
					MCCR[ID*TnRct+3]++;
				}else{
					Colltype = 1;
					Null++;
				}
				break;
			}
			case 1:{
                R2 = hiprand_uniform(&LocalStates)*sigv[1].val / vel;
				// 4. e + Ar* > e + e + Ar^		step ionization
				SumSigma = Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX);
				if(engy > info_CX[4].Th_e && R2<=SumSigma){
					Colltype = 3;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					AddPt1++;
					MCCR[ID*TnRct+4]++;
				}else{
					Colltype = 1;
					Null++;
				}
				break;
			}
		}
        if(Colltype == 2){ // Just energy loss
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			sp[i].vx = VX;
			sp[i].vy = VY;
			sp[i].vz = VZ;
		}else if(Colltype == 3){ //ionization 
			//printf("Ionization 1 ! \n");
            ///// scatter the created electron
			index = info[0].St_num + ID + (PNC + AddPt1 - 1) * Gsize; 
			sp[index].CellID = sp[i].CellID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            ///// assign velocities to the created ion
            index = info[1].St_num + ID + (PNC2 + AddPt1 - 1) * Gsize; 
            sp[index].CellID = sp[i].CellID + Gsize;
            sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
            sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			///// scatter the incident electron
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            sp[i].vx = VX;
			sp[i].vy = VY;
			sp[i].vz = VZ;
			//AddPt1--;
		}
		i+=Gsize;
	}
    data[ID].PtNumInCell = PNC + AddPt1;
    data[ID+Gsize].PtNumInCell = PNC2 + AddPt1;
	states[ID]=LocalStates;
	data[ID].PtNullMCCInCell = Null;
}
__device__ void Direct_Argon_ArIon(int Gsize, int ngy, int ID, int MCCn, float dt, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct, float*MCCR, GGA *BG, GFC *Fluid){
	int i,j,k,n,index;
	int PNC,Null;
	float Prob;
	float R1;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,engy;
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

    PNC = data[ID+Gsize].PtNumInCell;
	hiprandState LocalStates = states[ID];
	Prob = 1.0f - exp(-1*dt*sigv[2].val*BG[ID].BackDen1);
	Null = 0;
    // Calculate total Collision probability.
	i = info[1].St_num + ID;
	for(k=0;k<PNC;k++){
		R1 = hiprand_uniform(&LocalStates);
		if(R1>Prob){
			i+=Gsize;
			continue;
		} 
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX=sp[i].vx-vneutx;
	  	VY=sp[i].vy-vneutz;
	  	VZ=sp[i].vz-vneuty;
		dum=VX*VX+VY*VY+VZ*VZ;
		engy=info[1].Escale*dum;
	  	vel=sqrt(dum);
		R1 = hiprand_uniform(&LocalStates) * sigv[2].val / vel;
		SumSigma = Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX);
		if(R1<=SumSigma){
			// 5. Ar + Ar^ > Ar + Ar^		Charge Exchange
			VX_buf = vneutx;
			VY_buf = vneuty;
			VZ_buf = vneutz;
			MCCR[ID*TnRct+5]++;
		}else if(R1<=(SumSigma += Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
			// 6. AR + AR^ > AR + AR^		ELASTIC SCATTERING
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			VX_buf = VX+vneutx;
			VY_buf = VY+vneuty;
			VZ_buf = VZ+vneutz;
			MCCR[ID*TnRct+6]++;
		}else{
			VX_buf = sp[i].vx;
			VY_buf = sp[i].vy;
			VZ_buf = sp[i].vz;
			Null++;
		}
		sp[i].vx = VX_buf;
		sp[i].vy = VY_buf;
		sp[i].vz = VZ_buf;	
		i+=Gsize;
	}
	states[ID]=LocalStates;
	data[ID].PtNullMCCInCell = Null;
}
__device__ void Ar_Collision_Check(int Gsize, int Csize, int ngy, int TID, float dt, int MCCn, float dtm, float dx, float dy,
                                        hiprandState *states, Species *info, GPG *data, GCP *sp, MCC_sigmav *sigv, GGA *BG, GFC *Fluid){
	int i,j,k,index,Randn;
	int ID,isp,CID,PNMC,MPNC;
    int PNC,Flag;
	int nx,ny,ngx;
	float Tprob,Prob1,Prob2;
	float R1;
	ID = TID%Gsize;
    isp = TID/Gsize;
	nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	hiprandState LocalStates = states[TID];
	PNC = data[TID].PtNumInCell;
	MPNC = data[TID].MaxPtNumInCell;
	PNMC = 0;
	// Calculate total Collision probability.
    switch (isp){
    case 0: // Electron
		Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);  // E + Ar
		Prob2 = 1.0f - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);  // E + Ar*
	    Tprob = Prob1 + Prob2; 
		Randn = MCCn;
        break;
	case 1: // Ar+
		Tprob = 1.0 - exp(-1*dt*sigv[2].val*BG[ID].BackDen1);
		Randn = 1;
		break;
    default:
        break;
    }
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
        for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob) break;
		}
		if(R1 >= Tprob){ // no collision
			index = i - PNMC*Gsize;
            Flag = sp[i].CellID;
		}else{ // collision
			PNMC++;
			index = info[isp].St_num + ID + (MPNC-PNMC)*Gsize;
            switch (isp){
            case 0:
                if(R1 <= Prob1)	        Flag = (int)0;
		        else			        Flag = (int)1;
                break;
            case 1:
				Flag = sp[i].CellID;
                break;
            default:
                break;
            }
		}
		sp[index].CellID = Flag;
		sp[index].vx=sp[i].vx;
		sp[index].vy=sp[i].vy;
		sp[index].vz=sp[i].vz;
        sp[index].x=sp[i].x;
		sp[index].y=sp[i].y;
		i+=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNumMCCInCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}
__device__ void Ar_Electron(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float rengy,engy,dum,vel,vel2;
	int Iz_isp1,Iz_isp2;		
	float SumSigma,SumEngyLoss;

    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
	// Calculate total Collision probability
	i = info[0].St_num + TID + (MPNC-1)*Gsize;
	//printf("PNMC = %d\n",PNMC);
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 1;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Energy loss
		// 2 : ionization 1 -> 3 charged
		R1 = hiprand_uniform(&LocalStates);
        switch(Flag){
			case 0:{ // E + Ar
				mofm = info_CX[0].mofM;
				R1 *= sigv[0].val / vel;
				if(engy > info_CX[0].Th_e && R1<=(SumSigma=Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX))){
					MCCR[TID*TnRct]++;
				}else if(engy > info_CX[1].Th_e && R1<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+1]++;
				}else if(engy > info_CX[2].Th_e && R1<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+2]++;
				}else if(engy > info_CX[3].Th_e && R1<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+3]++;
				}else{
					Colltype = 0;
					Null++;
				}
				
				break;
			}
			case 1:{ // E + Ar*
				mofm = info_CX[4].mofM;
				R1 *= sigv[1].val / vel;
				if(engy > info_CX[4].Th_e && R1<=(SumSigma=Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+4]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			default:{
            	break;
        	}
		} 
        switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Energy loss
            dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : ionization 1 -> 3 charged
			// second charged create
			oldPNC = atomicAdd(&data[TID+Iz_isp1*Gsize].PtNumInCell,1);
			index = info[Iz_isp1].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp1*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp1 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
				dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}
			// Third charged create
			oldPNC = atomicAdd(&data[TID+Iz_isp2*Gsize].PtNumInCell,1);
			index = info[Iz_isp2].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp2*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp2 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
				dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}
			// energy loss electron 
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void Ar_Ar_ion(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[1].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[1].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange 
		mofm = info_CX[5].mofM;
        R1 = hiprand_uniform(&LocalStates)*sigv[2].val / vel;
		if(engy > info_CX[5].Th_e &&R1<=(SumSigma=Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX))){
			Colltype = 2; 
			MCCR[ID*TnRct+5]++;
		}else if(engy > info_CX[6].Th_e &&R1<=(SumSigma=Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
			Colltype = 1; 
			MCCR[ID*TnRct+6]++;
		}else{
			Null++;
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}

__device__ float Argon_CrossSection(int R, float engy, int N_LOGX, float idLOGX, ArCollD *data){
	if(engy == 0) return 0.0;
	float lengy = log10(engy);
	float ee1, a1, a2;
	int ee2;
	ee1 = idLOGX * (lengy - data[0].xe);
	ee2 = (int)ee1;
	a1 = ee1 - ee2;
	a2 = 1 - a1;
	switch (R) {
        case 0 : 
			if(lengy < data[0].xe){
				return data[0].cx_0;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_0 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_0+a1*data[ee2+1].cx_0;
			break;
        case 1 :
			if(lengy < data[0].xe){
				return data[0].cx_1;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_1 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_1+a1*data[ee2+1].cx_1;
			break;
        case 2 :
			if(lengy < data[0].xe){
				return data[0].cx_2;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_2 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_2+a1*data[ee2+1].cx_2;
			break;
        case 3 :
			if(lengy < data[0].xe){
				return data[0].cx_3;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_3 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_3+a1*data[ee2+1].cx_3;
			break;
        case 4 :
			if(lengy < data[0].xe){
				return data[0].cx_4;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_4 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_4+a1*data[ee2+1].cx_4;
			break;
        case 5 :
			if(lengy < data[0].xe){
				return data[0].cx_5;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_5 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_5+a1*data[ee2+1].cx_5;
			break;
        case 6 :
			if(lengy < data[0].xe){
				return data[0].cx_6;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_6 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_6+a1*data[ee2+1].cx_6;
			break;
        default :
            printf("\nError : Call about cross section data in ARMCC.\n\n");
            return 0.0;
    }
}