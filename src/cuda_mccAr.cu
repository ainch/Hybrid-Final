#include "hip/hip_runtime.h"
#include "cuda_mccAr.cuh"
__device__ void Direct_Argon_Electron(int Gsize, int ngy, int ID, int MCCn, float dtm, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, GGA *BG, GFC *Fluid){
	int i,j,k,n,index;
	int CID,PNC,PNC2;
	int nx,ny,ngx;
	int Target,oldPNC;
	int Colltype;
	float Tprob,Prob1,Prob2;
    int Randn,AddPt1;
	float R1,R2;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,vel2,engy,rengy;
	float SumSigma,SumEngyLoss;
	PNC = data[ID].PtNumInCell;
    PNC2 = data[ID+Gsize].PtNumInCell;
	hiprandState LocalStates = states[ID];
    nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);
	Prob2 = 1.0f - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);
	Tprob = Prob1 + Prob2;
    // Calculate total Collision probability.
	Randn = MCCn;
    AddPt1 = 0;
	i = info[0].St_num + ID;
	for(k=0;k<PNC;k++){
        Colltype = 0;
		for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob){
                Colltype = 1;
                break;
            }
		}
        if(Colltype == 0){
            i+=Gsize;
			continue;
        }
		R1 = Tprob * R1;
        if(R1 <= Prob1)	Target = (int)0;
		else			Target = (int)1;
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 2;
        switch(Target){
			case 0:{
                R2 = hiprand_uniform(&LocalStates) * sigv[0].val / vel;
				// 0. e + Ar > e + Ar 			Elastic Scattering
				SumSigma = Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX);
				if(R2<=SumSigma){
				// 1. e + Ar > e + Ar* 			Excitation to Total Excited state
				}else if(engy > info_CX[1].Th_e && R2<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
				// 2. e + Ar > e + Ar* 			Excitation to AR4SM
				}else if(engy > info_CX[2].Th_e && R2<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
				// 3. e + Ar > e + e + Ar^		Direct ionization
				}else if(engy > info_CX[3].Th_e && R2<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					AddPt1++;
				}else{
					Colltype = 1;
				}
				break;
			}
			case 1:{
                R2 = hiprand_uniform(&LocalStates)*sigv[1].val / vel;
				// 4. e + Ar* > e + e + Ar^		step ionization
				SumSigma = Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX);
				if(engy > info_CX[4].Th_e && R2<=SumSigma){
					Colltype = 3;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					AddPt1++;
				}else{
					Colltype = 1;
				}
				break;
			}
		}
        if(Colltype == 2){ // Just energy loss
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			sp[i].vx = VX;
			sp[i].vy = VY;
			sp[i].vz = VZ;
		}else if(Colltype == 3){ //ionization 
			//printf("Ionization 1 ! \n");
            ///// scatter the created electron
			index = info[0].St_num + ID + (PNC + AddPt1 - 1) * Gsize; 
			sp[index].CellID = sp[i].CellID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            ///// assign velocities to the created ion
            index = info[1].St_num + ID + (PNC2 + AddPt1 - 1) * Gsize; 
            sp[index].CellID = sp[i].CellID + Gsize;
            sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
            sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			///// scatter the incident electron
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            sp[i].vx = VX;
			sp[i].vy = VY;
			sp[i].vz = VZ;
			//AddPt1--;
		}
		i+=Gsize;
	}
    data[ID].PtNumInCell = PNC + AddPt1;
    data[ID+Gsize].PtNumInCell = PNC2 + AddPt1;
	states[ID]=LocalStates;
}
__device__ void Direct_Argon_ArIon(int Gsize, int ngy, int ID, int MCCn, float dt, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, GGA *BG, GFC *Fluid){
	int i,j,k,n,index;
	int PNC;
	float Prob;
	float R1;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,engy;
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

    PNC = data[ID+Gsize].PtNumInCell;
	hiprandState LocalStates = states[ID];
	Prob = 1.0f - exp(-1*dt*sigv[2].val*BG[ID].BackDen1);

    // Calculate total Collision probability.
	i = info[1].St_num + ID;
	for(k=0;k<PNC;k++){
		R1 = hiprand_uniform(&LocalStates);
		if(R1>Prob){
			i+=Gsize;
			continue;
		} 
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX=sp[i].vx-vneutx;
	  	VY=sp[i].vy-vneutz;
	  	VZ=sp[i].vz-vneuty;
		dum=VX*VX+VY*VY+VZ*VZ;
		engy=info[1].Escale*dum;
	  	vel=sqrt(dum);
		R1 = hiprand_uniform(&LocalStates) * sigv[2].val / vel;
		SumSigma = Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX);
		if(R1<=SumSigma){
			// 5. Ar + Ar^ > Ar + Ar^		Charge Exchange
			VX_buf = vneutx;
			VY_buf = vneuty;
			VZ_buf = vneutz;
		}else if(R1<=(SumSigma += Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
			// 6. AR + AR^ > AR + AR^		ELASTIC SCATTERING
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			VX_buf = VX+vneutx;
			VY_buf = VY+vneuty;
			VZ_buf = VZ+vneutz;
		}else{
			VX_buf = sp[i].vx;
			VY_buf = sp[i].vy;
			VZ_buf = sp[i].vz;
		}
		sp[i].vx = VX_buf;
		sp[i].vy = VY_buf;
		sp[i].vz = VZ_buf;	
		i+=Gsize;
	}
	states[ID]=LocalStates;
}
__device__ void Collision_Check(int Gsize, int Csize, int ngy, int ID, int isp, float dt, int MCCn, float dtm, hiprandState *states, Species *info, GPG *data, GCP *sp, MCC_sigmav *sigv, GGA *BG, GFC *Fluid){
	int i,j,k,index,Randn;
	int TID,CID,PNC,PNMC,MPNC;
	int nx,ny,ngx;
	float Prob = 0.0f;
	float R1;
	TID = isp*Gsize + ID;
	nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	hiprandState LocalStates = states[TID];
	PNC = data[TID].PtNumInCell;
	MPNC = data[TID].MaxPtNumInCell;
	PNMC = 0;
	// Calculate total Collision probability.
	if(isp == 0){ // Electron
		Prob = 2.0 - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1) - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);
		Randn = MCCn;
	}else{ // ion
		Prob = 1.0 - exp(-1*dt*sigv[2].val*BG[ID].BackDen1);
		Randn = 1;
	}
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
		for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Prob) break;
		}
		if(R1 > Prob){ // no collision
			index = i - PNMC*Gsize;
		}else{ // collision
			PNMC++;
			index = info[isp].St_num + ID + (MPNC-PNMC)*Gsize;
		}
		sp[index].CellID = sp[i].CellID;
		sp[index].vx=sp[i].vx;
		sp[index].vy=sp[i].vy;
		sp[index].vz=sp[i].vz;
        sp[index].x=sp[i].x;
		sp[index].y=sp[i].y;
		i+=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNumMCCInCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}

__device__ void Argon_Ar_Collision(int Gsize, int TID, float dt, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, GGA *BG, GFC *Fluid){
	int i,k,n,index;
	int ID,PNMC,MPNC,PNC,AddPt;
	int Target,oldPNC;
	float prob;
	float cut1;
	float R1,R2;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,vel2,engy,rengy;
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;
	ID = Gsize + TID;
	PNC = data[ID].PtNumInCell;
	PNMC = data[ID].PtNumMCCInCell;
	MPNC = data[ID].MaxPtNumInCell;
	AddPt = 0;
	hiprandState LocalStates = states[ID];
	prob = 1.0f - exp(-1*dt*sigv[2].val*BG[TID].BackDen1);
	i = info[1].St_num + TID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
		// Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[1].Escale * dum;
		R1 = hiprand_uniform(&LocalStates) * sigv[2].val / vel;
		SumSigma = Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX);
		if(R1<=SumSigma){
			// 5. Ar + Ar^ > Ar + Ar^		Charge Exchange
			VX_buf = vneutx;
			VY_buf = vneuty;
			VZ_buf = vneutz;
		}else if(R1<=(SumSigma += Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
			// 6. AR + AR^ > AR + AR^		ELASTIC SCATTERING
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			VX_buf = VX+vneutx;
			VY_buf = VY+vneuty;
			VZ_buf = VZ+vneutz;
		}else{
			VX_buf = sp[i].vx;
			VY_buf = sp[i].vy;
			VZ_buf = sp[i].vz;
		}
		index = info[1].St_num + TID + (PNC+AddPt) * Gsize; 
		sp[index].CellID = ID;
		sp[index].x = sp[i].x;
		sp[index].y = sp[i].y;
		sp[index].vx = VX_buf;
		sp[index].vy = VY_buf;
		sp[index].vz = VZ_buf;		
		i-=Gsize;
		AddPt++;
	}
	data[ID].PtNumInCell = PNC + AddPt;
	states[TID]=LocalStates;
}
__device__ void Argon_E_Collision(int Gsize, int ngy, int TID, int MCCn, float dtm, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, GGA *BG, GFC *Fluid){
	int i,k,n,index;
	int CID,PNMC,MPNC;
	int PNC,PNC_ion,AddPt,AddPt_ion;
	int nx,ny,ngx;
	int Target,oldPNC;
	int Colltype;
	float Tprob,Prob1,Prob2;
	float R1,R2;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,vel2,engy,rengy;
	float SumSigma,SumEngyLoss;
	PNC = data[TID].PtNumInCell;
	PNC_ion = data[TID+Gsize].PtNumInCell;
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	AddPt = 0;
	AddPt_ion = 0;
	hiprandState LocalStates = states[TID];
	nx = TID/ngy;
	ny = TID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[TID].BackDen1);
	Prob2 = 1.0f - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);
	Tprob = Prob1 + Prob2;
	i = info[0].St_num + TID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
		// Collision target setting
		R1 = Tprob * hiprand_uniform(&LocalStates);
		if(R1 <= Prob1)	Target = (int)0;
		else			Target = (int)1;
		// Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
		Colltype = 2;
		switch(Target){
			case 0:{ // Target == Argon
				R2 = hiprand_uniform(&LocalStates) * sigv[0].val / vel;
				SumSigma = Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX);
				if(R2<=SumSigma){ 
					// 0. e + Ar > e + Ar 	 	Elastic Scattering
				}else if(engy > info_CX[1].Th_e && R2<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					// 1. e + Ar > e + Ar* 		Excitation to Total Excited state
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
				}else if(engy > info_CX[2].Th_e && R2<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					// 2. e + Ar > e + Ar* 		Excitation to AR4SM
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
				}else if(engy > info_CX[3].Th_e && R2<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					// 3. e + Ar > e + e + Ar^	Direct ionization
					Colltype = 3;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
				}else{
					Colltype = 1;
				}
				break;
			}
			case 1:{ // Target == Metastable argon
				R2 = hiprand_uniform(&LocalStates)*sigv[1].val / vel;
				SumSigma = Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX);
				if(engy > info_CX[4].Th_e && R2<=SumSigma){
					// 4. e + Ar* > e + e + Ar^		step ionization
					Colltype = 3;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
				}else{
					Colltype = 1;
				}
				break;
			}
		}
		//PNC,AddPt
		if(Colltype == 1){ // null Collision
			index = info[0].St_num + TID + (PNC+AddPt) * Gsize; 
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
			AddPt++;
		}else if(Colltype == 2){ // Just energy loss
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			index = info[0].St_num + TID + (PNC+AddPt) * Gsize; 
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			AddPt++;
		}else if(Colltype == 3){ //ionization 
			index = info[0].St_num + TID + (PNC+AddPt) * Gsize; 
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			dev_anewvel(rengy,vel2,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			AddPt++;
			// New Electron create
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			index = info[0].St_num + TID + (PNC+AddPt) * Gsize; 
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			AddPt++;
			// ion create
			index = info[1].St_num + TID + (PNC_ion+AddPt_ion) * Gsize; 
			n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			sp[index].CellID = TID + Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			AddPt_ion++;
		}
		i-=Gsize;
	}
	data[TID].PtNumInCell = PNC + AddPt;
	data[TID+Gsize].PtNumInCell = PNC_ion + AddPt_ion;
	states[TID]=LocalStates;
}

__device__ float Argon_CrossSection(int R, float engy, int N_LOGX, float idLOGX, ArCollD *data){
	if(engy == 0) return 0.0;
	float lengy = log10(engy);
	float ee1, a1, a2;
	int ee2;
	lengy = lengy - data[0].xe;
	ee1 = idLOGX * lengy;
	ee2 = (int)ee1;
	a1 = ee1 - ee2;
	a2 = 1 - a1;
	switch (R) {
        case 0 : 
			if(lengy < data[0].xe){
				return data[0].cx_0;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_0 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_0+a1*data[ee2+1].cx_0;
			break;
        case 1 :
			if(lengy < data[0].xe){
				return data[0].cx_1;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_1 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_1+a1*data[ee2+1].cx_1;
			break;
        case 2 :
			if(lengy < data[0].xe){
				return data[0].cx_2;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_2 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_2+a1*data[ee2+1].cx_2;
			break;
        case 3 :
			if(lengy < data[0].xe){
				return data[0].cx_3;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_3 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_3+a1*data[ee2+1].cx_3;
			break;
        case 4 :
			if(lengy < data[0].xe){
				return data[0].cx_4;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_4 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_4+a1*data[ee2+1].cx_4;
			break;
        case 5 :
			if(lengy < data[0].xe){
				return data[0].cx_5;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_5 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_5+a1*data[ee2+1].cx_5;
			break;
        case 6 :
			if(lengy < data[0].xe){
				return data[0].cx_6;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_6 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_6+a1*data[ee2+1].cx_6;
			break;
        default :
            printf("\nError : Call about cross section data in ARMCC.\n\n");
            return 0.0;
    }
}