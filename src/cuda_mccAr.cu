#include "hip/hip_runtime.h"
#include "cuda_mccAr.cuh"
__device__ void MCC_Argon_RC(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
	int TnRct, float *MCCR, float *Stack, CollF *info_CX, Species *info, GPG *data, GCP *sp, GFG *FG){
	float PNC,M1;
	int oldPNC,index,index2,n;
	int xi = TID/ngy;
	int yi = TID%ngy;
	if(yi>ngy-2) return;
	if(xi>Gsize/ngy-2) return;
	hiprandState LocalStates = states[TID];
	M1 = 0;
	PNC = Stack[TID];
	PNC += info_CX[8].RR*FG[TID].n*FG[TID].n * info[0].MCCscale;
	while(PNC >= 1.0f){
		//printf("[%d]=[%d][%d]PNC = %g %g %g %g \n",TID,xi,yi,PNC,info_CX[8].RR,FG[CID].den,info[isp].MCCscale);
		oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
		index = info[0].St_num + TID + oldPNC*Gsize;
		sp[index].CellID = TID;
		sp[index].x = hiprand_uniform(&LocalStates);
		sp[index].y = hiprand_uniform(&LocalStates);
		n = (nvel-1) * hiprand_uniform(&LocalStates);
		dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],info[0].vti,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));	
		oldPNC = atomicAdd(&data[TID+Gsize].PtNumInCell,1);
		index2 = info[1].St_num + TID + oldPNC*Gsize;
		sp[index2].CellID = TID + Gsize;
		sp[index2].x = sp[index].x;
		sp[index2].y = sp[index].y;
		n = (nvel-1) * hiprand_uniform(&LocalStates);
		dev_maxwellv(&sp[index2].vx,&sp[index2].vy,&sp[index2].vz,vsave[n],info[1].vti,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));	
		PNC--;
		M1 += 0.25;
		break;
	}
	atomicAdd(&MCCR[TID*TnRct + 8] ,M1);
	atomicAdd(&MCCR[(TID+1)*TnRct + 8] ,M1);
	atomicAdd(&MCCR[(TID+ngy)*TnRct + 8] ,M1);
	atomicAdd(&MCCR[(TID+ngy+1)*TnRct + 8] ,M1);
	Stack[TID] = PNC;
    states[TID] = LocalStates;
}
__device__ void Direct_Argon_Electron(int Gsize, int ngy, int ID, int MCCn, float dtm, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct, float*MCCR, GGA *BG, GFG *Fluid){
	int i,j,k,n,index;
	int PNC,PNC2;
	int Target;
	int Colltype;
	float Prob1,Prob2;
    int AddPt;
	float R1,R2;
	float VX,VY,VZ;
	float dum,vel,vel2,engy,rengy,massRatio;
	float SumSigma,SumEngyLoss;
	PNC = data[ID].PtNumInCell;
    PNC2 = data[ID+Gsize].PtNumInCell;
	hiprandState LocalStates = states[ID];
	Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);
	Prob2 = Prob1 + 1.0f - exp(-1*dtm*sigv[1].val*Fluid[ID].n);
    // Calculate total Collision probability.
    AddPt = 0;
	massRatio = info_CX[4].mofM;
	i = info[0].St_num + ID;


	int x = PNC;
	int y = 0;
	
	if(true)
	{
		while(x != 0)
		{
			int mask0 = 0;
			int mask1 = 0;
			int mask;
			for(int j=0;j<30;j++)
			{
				Colltype = 0;
				for(int k=0;k<MCCn;k++){
					R1 = hiprand_uniform(&LocalStates);
					if(R1<Prob2){
						Colltype = 1;
						break;
					}
				}
				if(Colltype == 1)
				{				
					if(R1 <= Prob1)
					{
						//Target = (int)0;
						mask0 = mask0 | (1<<j);
					}	
					else
					{
						//Target = (int)1;
						mask1 = mask1 | (1<<j);
					}
				}
				--x;
				if(x == 0)break;
			}
			mask = mask0 | mask1;
			
			int st = 0;
			while(mask0 != 0)
			{
				Colltype = 1;
				int lb = __ffs(mask0);
				i = (y + st + lb - 1) * Gsize + info[0].St_num + ID;
				Target = (int)0;
				
				{
					// Calculate energy
					VX = sp[i].vx;
					VY = sp[i].vy;
					VZ = sp[i].vz;
					dum = VX*VX+VY*VY+VZ*VZ;
					vel = sqrt(dum);
					VX/=vel; VY/=vel; VZ/=vel;
					engy = info[0].Escale * dum;
					{
						R2 = hiprand_uniform(&LocalStates) * sigv[0].val / vel;
						// 0. e + Ar > e + Ar 			Elastic Scattering
						SumSigma = Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX);
						if(R2<=SumSigma){
						// 1. e + Ar > e + Ar* 			Excitation to Total Excited state
							Colltype = 0;
							MCCR[ID*TnRct]++;
						}else if(engy > info_CX[1].Th_e && R2<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
							Colltype = 0;
							engy-=info_CX[1].Th_e;
							vel=sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+1]++;
						// 2. e + Ar > e + Ar* 			Excitation to AR4SM
						}else if(engy > info_CX[2].Th_e && R2<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
							Colltype = 0;
							engy-=info_CX[2].Th_e;
							vel=sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+2]++;
						// 3. e + Ar > e + e + Ar^		Direct ionization
						}else if(engy > info_CX[3].Th_e && R2<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
							Colltype = 2;
							engy-=info_CX[3].Th_e;
							rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
							engy-=rengy;
							vel = sqrt(fabs(rengy)/info[0].Escale);
							vel2 = sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+3]++;
						}
					}
					if(Colltype == 0){ // Just energy loss
						dev_anewvel(engy,vel,&VX,&VY,&VZ,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						sp[i].vx = VX;
						sp[i].vy = VY;
						sp[i].vz = VZ;
					}else if(Colltype == 2){ //ionization 
						//printf("Ionization 1 ! \n");
						///// scatter the created electron
						index = info[0].St_num + ID + (PNC + AddPt) * Gsize; 
						sp[index].CellID = ID;
						sp[index].x = sp[i].x;
						sp[index].y = sp[i].y;
						sp[index].vx = VX;
						sp[index].vy = VY;
						sp[index].vz = VZ;
						dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						///// assign velocities to the created ion
						index = info[1].St_num + ID + (PNC2 + AddPt) * Gsize; 
						sp[index].CellID = ID + Gsize;
						sp[index].x = sp[i].x;
						sp[index].y = sp[i].y;
						sp[index].vx = VX;
						sp[index].vy = VY;
						sp[index].vz = VZ;
						//printf("\n[%d][%d] ionization \n",ID,ID+Gsize);
						n = (nvel-1)*hiprand_uniform(&LocalStates);
						dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						///// scatter the incident electron
						dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						sp[i].vx = VX;
						sp[i].vy = VY;
						sp[i].vz = VZ;
						AddPt++;
					}
				}
	
				mask0 = mask0 >> lb;
				st += lb;
			}
			
			st = 0;
			while(mask1 != 0)
			{
				Colltype = 1;
				int lb = __ffs(mask);
				i = (y + st + lb - 1) * Gsize + info[0].St_num + ID;
				
				Target = (int)1;
				
				{
					// Calculate energy
					VX = sp[i].vx;
					VY = sp[i].vy;
					VZ = sp[i].vz;
					dum = VX*VX+VY*VY+VZ*VZ;
					vel = sqrt(dum);
					VX/=vel; VY/=vel; VZ/=vel;
					engy = info[0].Escale * dum;
					{
						R2 = hiprand_uniform(&LocalStates)*sigv[1].val / vel;
						// 4. e + Ar* > e + e + Ar^		step ionization
						SumSigma = Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX);
						if(engy > info_CX[4].Th_e && R2<=SumSigma){
							Colltype = 2;
							engy-=info_CX[4].Th_e;
							rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
							engy-=rengy;
							vel = sqrt(fabs(rengy)/info[0].Escale);
							vel2 = sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+4]++;
						}
					}
					if(Colltype == 0){ // Just energy loss
						dev_anewvel(engy,vel,&VX,&VY,&VZ,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						sp[i].vx = VX;
						sp[i].vy = VY;
						sp[i].vz = VZ;
					}else if(Colltype == 2){ //ionization 
						//printf("Ionization 1 ! \n");
						///// scatter the created electron
						index = info[0].St_num + ID + (PNC + AddPt) * Gsize; 
						sp[index].CellID = ID;
						sp[index].x = sp[i].x;
						sp[index].y = sp[i].y;
						sp[index].vx = VX;
						sp[index].vy = VY;
						sp[index].vz = VZ;
						dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						///// assign velocities to the created ion
						index = info[1].St_num + ID + (PNC2 + AddPt) * Gsize; 
						sp[index].CellID = ID + Gsize;
						sp[index].x = sp[i].x;
						sp[index].y = sp[i].y;
						sp[index].vx = VX;
						sp[index].vy = VY;
						sp[index].vz = VZ;
						//printf("\n[%d][%d] ionization \n",ID,ID+Gsize);
						n = (nvel-1)*hiprand_uniform(&LocalStates);
						dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						///// scatter the incident electron
						dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						sp[i].vx = VX;
						sp[i].vy = VY;
						sp[i].vz = VZ;
						AddPt++;
					}
				}
	
				mask1 = mask1 >> lb;
				st += lb;
			}

			y += 30;
		}
	}

	if(!true)
	{
		for(k=0;k<PNC;k++){
			Colltype = 0;
			for(j=0;j<MCCn;j++){
				R1 = hiprand_uniform(&LocalStates);
				if(R1<Prob2){
					Colltype = 1;
					break;
				}
			}
			
			if(Colltype == 1)
			{
				if(R1 <= Prob1)	Target = (int)0;
				else			Target = (int)1;
				// Calculate energy
				VX = sp[i].vx;
				VY = sp[i].vy;
				VZ = sp[i].vz;
				dum = VX*VX+VY*VY+VZ*VZ;
				vel = sqrt(dum);
				VX/=vel; VY/=vel; VZ/=vel;
				engy = info[0].Escale * dum;
				switch(Target){
					case 0:{
						R2 = hiprand_uniform(&LocalStates) * sigv[0].val / vel;
						// 0. e + Ar > e + Ar 			Elastic Scattering
						SumSigma = Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX);
						if(R2<=SumSigma){
						// 1. e + Ar > e + Ar* 			Excitation to Total Excited state
							Colltype = 0;
							MCCR[ID*TnRct]++;
						}else if(engy > info_CX[1].Th_e && R2<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
							Colltype = 0;
							engy-=info_CX[1].Th_e;
							vel=sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+1]++;
						// 2. e + Ar > e + Ar* 			Excitation to AR4SM
						}else if(engy > info_CX[2].Th_e && R2<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
							Colltype = 0;
							engy-=info_CX[2].Th_e;
							vel=sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+2]++;
						// 3. e + Ar > e + e + Ar^		Direct ionization
						}else if(engy > info_CX[3].Th_e && R2<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
							Colltype = 2;
							engy-=info_CX[3].Th_e;
							rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
							engy-=rengy;
							vel = sqrt(fabs(rengy)/info[0].Escale);
							vel2 = sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+3]++;
						}
						break;
					}
					case 1:{
						R2 = hiprand_uniform(&LocalStates)*sigv[1].val / vel;
						// 4. e + Ar* > e + e + Ar^		step ionization
						SumSigma = Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX);
						if(engy > info_CX[4].Th_e && R2<=SumSigma){
							Colltype = 2;
							engy-=info_CX[4].Th_e;
							rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
							engy-=rengy;
							vel = sqrt(fabs(rengy)/info[0].Escale);
							vel2 = sqrt(fabs(engy)/info[0].Escale);
							MCCR[ID*TnRct+4]++;
						}
						break;
					}
				}
				if(Colltype == 0){ // Just energy loss
					dev_anewvel(engy,vel,&VX,&VY,&VZ,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
					sp[i].vx = VX;
					sp[i].vy = VY;
					sp[i].vz = VZ;
				}else if(Colltype == 2){ //ionization 
					//printf("Ionization 1 ! \n");
					///// scatter the created electron
					index = info[0].St_num + ID + (PNC + AddPt) * Gsize; 
					sp[index].CellID = ID;
					sp[index].x = sp[i].x;
					sp[index].y = sp[i].y;
					sp[index].vx = VX;
					sp[index].vy = VY;
					sp[index].vz = VZ;
					dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
					///// assign velocities to the created ion
					index = info[1].St_num + ID + (PNC2 + AddPt) * Gsize; 
					sp[index].CellID = ID + Gsize;
					sp[index].x = sp[i].x;
					sp[index].y = sp[i].y;
					sp[index].vx = VX;
					sp[index].vy = VY;
					sp[index].vz = VZ;
					//printf("\n[%d][%d] ionization \n",ID,ID+Gsize);
					n = (nvel-1)*hiprand_uniform(&LocalStates);
					dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
					///// scatter the incident electron
					dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,massRatio,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
					sp[i].vx = VX;
					sp[i].vy = VY;
					sp[i].vz = VZ;
					AddPt++;
				}
			}
			i+=Gsize;
		}
	}



	//if(AddPt != 0) printf("[%d] PNC[%d] addpt[%d]\n",ID,PNC,AddPt);
    data[ID].PtNumInCell = PNC + AddPt;
    data[ID+Gsize].PtNumInCell = PNC2+AddPt;
	states[ID]=LocalStates;
}
__device__ void Direct_Argon_ArIon(int Gsize, int ngy, int ID, int MCCn, float dt, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct, float*MCCR, GGA *BG, GFG *Fluid){
	int i,k,n;
	int PNC,Null;
	float Prob;
	float R1;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,engy;
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

    PNC = data[ID+Gsize].PtNumInCell;
	hiprandState LocalStates = states[ID];
	Prob = 1.0f - exp(-1*dt*sigv[2].val*BG[ID].BackDen1);
    // Calculate total Collision probability.
	i = info[1].St_num + ID;

	int x = PNC;
	int y = 0;
	
	if(true)
	{
		while(x != 0)
		{
			int mask = 0;
			for(int j=0;j<30;j++)
			{
				R1 = hiprand_uniform(&LocalStates);
				if(R1 <= Prob)
				{
					mask = (1<<j) | mask;
				}
				--x;
				if(x == 0)break;
			}
			
			int st = 0;
			while(mask != 0)
			{
				int lb = __ffs(mask);
				i = (y + st + lb - 1) * Gsize + info[1].St_num + ID;
				
				{
					n = (nvel-1)*hiprand_uniform(&LocalStates);
					dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
					VX=sp[i].vx-vneutx;
					VY=sp[i].vy-vneutz;
					VZ=sp[i].vz-vneuty;
					dum=VX*VX+VY*VY+VZ*VZ;
					engy=info[1].Escale*dum;
					vel=sqrt(dum);
					R1 = hiprand_uniform(&LocalStates) * sigv[2].val / vel;
					SumSigma = Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX);
					if(R1<=SumSigma){
						// 5. Ar + Ar^ > Ar + Ar^		Charge Exchange
						VX_buf = vneutx;
						VY_buf = vneuty;
						VZ_buf = vneutz;
						MCCR[ID*TnRct+5]++;
					}else if(R1<=(SumSigma += Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
						// 6. AR + AR^ > AR + AR^		ELASTIC SCATTERING
						dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
						VX_buf = VX+vneutx;
						VY_buf = VY+vneuty;
						VZ_buf = VZ+vneutz;
						MCCR[ID*TnRct+6]++;
					}else{
						VX_buf = sp[i].vx;
						VY_buf = sp[i].vy;
						VZ_buf = sp[i].vz;
					}
					sp[i].vx = VX_buf;
					sp[i].vy = VY_buf;
					sp[i].vz = VZ_buf;
				}
	
				mask = mask >> lb;
				st += lb;
			}
			
			y += 30;
		}
	}

	if(!true)
	{
		for(k=0;k<PNC;k++)
		{
			R1 = hiprand_uniform(&LocalStates);
			if(R1<=Prob)
			{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
				VX=sp[i].vx-vneutx;
				VY=sp[i].vy-vneutz;
				VZ=sp[i].vz-vneuty;
				dum=VX*VX+VY*VY+VZ*VZ;
				engy=info[1].Escale*dum;
				vel=sqrt(dum);
				R1 = hiprand_uniform(&LocalStates) * sigv[2].val / vel;
				SumSigma = Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX);
				if(R1<=SumSigma){
					// 5. Ar + Ar^ > Ar + Ar^		Charge Exchange
					VX_buf = vneutx;
					VY_buf = vneuty;
					VZ_buf = vneutz;
					MCCR[ID*TnRct+5]++;
				}else if(R1<=(SumSigma += Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
					// 6. AR + AR^ > AR + AR^		ELASTIC SCATTERING
					dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
					VX_buf = VX+vneutx;
					VY_buf = VY+vneuty;
					VZ_buf = VZ+vneutz;
					MCCR[ID*TnRct+6]++;
				}else{
					VX_buf = sp[i].vx;
					VY_buf = sp[i].vy;
					VZ_buf = sp[i].vz;
				}
				sp[i].vx = VX_buf;
				sp[i].vy = VY_buf;
				sp[i].vz = VZ_buf;
			}
			i+=Gsize;
		}
	}
	states[ID]=LocalStates;
}
__device__ void Ar_Collision_Check(int Gsize, int Csize, int ngy, int TID, float dt, int MCCn, float dtm, float dx, float dy,
                                        hiprandState *states, Species *info, GPG *data, GCP *sp, MCC_sigmav *sigv, GGA *BG, GFG *Fluid){
	int i,j,k,index,Randn;
	int ID,isp,PNMC,MPNC;
    int PNC,Flag;
	float Tprob,Prob1,Prob2;
	float R1;
	ID = TID%Gsize;
    isp = TID/Gsize;
	hiprandState LocalStates = states[TID];
	PNC = data[TID].PtNumInCell;
	MPNC = data[TID].MaxPtNumInCell;
	PNMC = 0;
	// Calculate total Collision probability.
    switch (isp){
    case 0: // Electron
		Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);  // E + Ar
		Prob2 = Prob1 + 1.0f - exp(-1*dtm*sigv[1].val*Fluid[ID].n);  // E + Ar*
	    Tprob = Prob2; 
		Randn = MCCn;
        break;
	case 1: // Ar+
		Tprob = 1.0 - exp(-1*dt*sigv[2].val*BG[ID].BackDen1);
		Randn = 1;
		break;
    default:
        break;
    }
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
        for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob) break;
		}
		if(R1 >= Tprob){ // no collision
			index = i - PNMC*Gsize;
            Flag = sp[i].CellID;
		}else{ // collision
			PNMC++;
			index = info[isp].St_num + ID + (MPNC-PNMC)*Gsize;
            switch (isp){
            case 0:
                if(R1 <= Prob1)	        Flag = (int)0;
		        else			        Flag = (int)1;
                break;
            case 1:
				Flag = sp[i].CellID;
                break;
            default:
                break;
            }
		}
		sp[index].CellID = Flag;
		sp[index].vx=sp[i].vx;
		sp[index].vy=sp[i].vy;
		sp[index].vz=sp[i].vz;
        sp[index].x=sp[i].x;
		sp[index].y=sp[i].y;
		i+=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNumMCCInCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}
__device__ void Ar_Collision_Check_v2(int Gsize, int Csize, int ngy, int TID, float dt, int MCCn, float dtm, float dx, float dy,
                                        hiprandState *states, Species *info, GPG *data, GCP *sp, MCC_sigmav *sigv, GGA *BG, GFG *Fluid){
	int i,j,k,index,Randn;
	int PNMC,MPNC;
    int PNC,Flag;
	float Prob1,Prob2;
	float R1;
	hiprandState LocalStates = states[TID];
	PNC = data[TID].PtNumInCell;
	MPNC = data[TID].MaxPtNumInCell;
	PNMC = 0;
	// Calculate total Collision probability.
	Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[TID].BackDen1);  // E + Ar
	Prob2 = Prob1 + 1.0f - exp(-1*dtm*sigv[1].val*Fluid[TID].n);  // E + Ar*
	Randn = MCCn;
	i = TID;
	for(k=0;k<PNC;k++){
        for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Prob2) break;
		}
		if(R1 >= Prob2){ // no collision
			index = i - PNMC*Gsize;
            Flag = sp[i].CellID;
		}else{ // collision
			PNMC++;
			index = TID + (MPNC-PNMC)*Gsize;
            if(R1 <= Prob1)	        Flag = (int)0;
		    else			        Flag = (int)1;            
		}
		sp[index].CellID = Flag;
		sp[index].vx=sp[i].vx;
		sp[index].vy=sp[i].vy;
		sp[index].vz=sp[i].vz;
        sp[index].x=sp[i].x;
		sp[index].y=sp[i].y;
		i+=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNumMCCInCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}
__device__ void Ar_Electron_v2(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,k,n,index;
	int PNMC,MPNC,Flag;
	int Colltype;
	int PNC,PNC1,Addpt,Addpt1;
	float mofm,R1;
	float VX,VY,VZ;
	float rengy,engy,dum,vel,vel2;		
	float SumSigma,SumEngyLoss;

    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	// Calculate total Collision probability
	i = TID + (MPNC-1)*Gsize;
	Addpt = 0; Addpt1 = 0;
	PNC = data[TID].PtNumInCell;
	PNC1 = data[Gsize + TID].PtNumInCell;
	//printf("PNMC = %d\n",PNMC);
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Energy loss
		// 2 : ionization 1 -> 3 charged
		R1 = hiprand_uniform(&LocalStates);
        switch(Flag){
			case 0:{ // E + Ar
				mofm = info_CX[0].mofM;
				R1 *= sigv[0].val / vel;
				if(engy > info_CX[0].Th_e && R1<=(SumSigma=Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1;
					MCCR[TID*TnRct]++;
				}else if(engy > info_CX[1].Th_e && R1<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1;
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+1]++;
				}else if(engy > info_CX[2].Th_e && R1<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1;
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+2]++;
				}else if(engy > info_CX[3].Th_e && R1<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+3]++;
				}				
				break;
			}
			case 1:{ // E + Ar*
				mofm = info_CX[4].mofM;
				R1 *= sigv[1].val / vel;
				if(engy > info_CX[4].Th_e && R1<=(SumSigma=Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+4]++;
				}
				break;
			}
			default:{
            	break;
        	}
		} 
        switch (Colltype){
        case 0: // 0 : Null collision
			index = TID + (PNC+Addpt)*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
			Addpt++;
            break;
        case 1: // 1 : Energy loss
            dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			index = TID + (PNC+Addpt)*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			Addpt++;
            break;
        case 2: // 2 : ionization 1 -> 3 charged
			// second charged create
			index = TID + (PNC+Addpt)*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			Addpt++;
			// Third charged create
			index = info[1].St_num + TID + (PNC1+Addpt1)*Gsize;
			sp[index].CellID = TID+Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			Addpt1++;
			// energy loss electron 
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			index = TID + (PNC+Addpt)*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			Addpt++;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	data[TID].PtNumInCell = PNC + Addpt;
	data[Gsize + TID].PtNumInCell = PNC1 + Addpt1;
	states[TID]=LocalStates;
}
__device__ void Ar_Electron(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,k,n,index;
	int PNMC,MPNC,Flag;
	int oldPNC;
	int Colltype;
	float mofm,R1;
	float VX,VY,VZ;
	float rengy,engy,dum,vel,vel2;		
	float SumSigma,SumEngyLoss;

    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	// Calculate total Collision probability
	i = TID + (MPNC-1)*Gsize;
	//printf("PNMC = %d\n",PNMC);
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Energy loss
		// 2 : ionization 1 -> 3 charged
		R1 = hiprand_uniform(&LocalStates);
        switch(Flag){
			case 0:{ // E + Ar
				mofm = info_CX[0].mofM;
				R1 *= sigv[0].val / vel;
				if(engy > info_CX[0].Th_e && R1<=(SumSigma=Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1;
					MCCR[TID*TnRct]++;
				}else if(engy > info_CX[1].Th_e && R1<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1;
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+1]++;
				}else if(engy > info_CX[2].Th_e && R1<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1;
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+2]++;
				}else if(engy > info_CX[3].Th_e && R1<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+3]++;
				}				
				break;
			}
			case 1:{ // E + Ar*
				mofm = info_CX[4].mofM;
				R1 *= sigv[1].val / vel;
				if(engy > info_CX[4].Th_e && R1<=(SumSigma=Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+4]++;
				}
				break;
			}
			default:{
            	break;
        	}
		} 
        switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Energy loss
            dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : ionization 1 -> 3 charged
			// second charged create
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			// Third charged create
			oldPNC = atomicAdd(&data[TID+Gsize].PtNumInCell,1);
			index = info[1].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			// energy loss electron 
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
}
__device__ void Ar_Ar_ion(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArCollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,k,n,index;
	int ID,PNMC,MPNC;
	int oldPNC;
	int Colltype;
	float R1;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;

    // Calculate total Collision probability
	i = info[1].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[1].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange 
        R1 = hiprand_uniform(&LocalStates)*sigv[2].val / vel;
		if(engy > info_CX[5].Th_e &&R1<=(SumSigma=Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX))){
			Colltype = 2; 
			MCCR[ID*TnRct+5]++;
		}else if(engy > info_CX[6].Th_e &&R1<=(SumSigma=Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
			Colltype = 1; 
			MCCR[ID*TnRct+6]++;
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
}
__device__ float Argon_CrossSection(int R, float engy, int N_LOGX, float idLOGX, ArCollD *data){
	if(engy == 0) return 0.0;
	float lengy = log10(engy);
	float ee1, a1, a2;
	int ee2;
	ee1 = idLOGX * (lengy - data[0].xe);
	ee2 = (int)ee1;
	a1 = ee1 - ee2;
	a2 = 1 - a1;
	switch (R) {
        case 0 : 
			if(lengy < data[0].xe){
				return data[0].cx_0;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_0 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_0+a1*data[ee2+1].cx_0;
			break;
        case 1 :
			if(lengy < data[0].xe){
				return data[0].cx_1;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_1 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_1+a1*data[ee2+1].cx_1;
			break;
        case 2 :
			if(lengy < data[0].xe){
				return data[0].cx_2;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_2 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_2+a1*data[ee2+1].cx_2;
			break;
        case 3 :
			if(lengy < data[0].xe){
				return data[0].cx_3;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_3 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_3+a1*data[ee2+1].cx_3;
			break;
        case 4 :
			if(lengy < data[0].xe){
				return data[0].cx_4;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_4 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_4+a1*data[ee2+1].cx_4;
			break;
        case 5 :
			if(lengy < data[0].xe){
				return data[0].cx_5;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_5 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_5+a1*data[ee2+1].cx_5;
			break;
        case 6 :
			if(lengy < data[0].xe){
				return data[0].cx_6;
			}else if(lengy > data[N_LOGX-1].xe){
				return data[N_LOGX-1].cx_6 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
			}
			return a2*data[ee2].cx_6+a1*data[ee2+1].cx_6;
			break;
        default :
            printf("\nError : Call about cross section data in ARMCC.\n\n");
            return 0.0;
    }
}