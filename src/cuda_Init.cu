#include "hip/hip_runtime.h"
#include "cuda_Init.cuh"
#define THREADS_PER_BLOCK 512   

void Set_Device_Parameter(){
    int grid,block;
    int mingrid;
    int numBlocksPerSm;
    int numThreads;
    int numSms;
    int size;
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SetSeed,0,nsp*Gsize); 
    grid = (nsp*Gsize + block - 1) / block;
    hipMalloc((void**) &devStates, nsp*Gsize*sizeof(hiprandState));
    SetSeed<<<grid,block>>>(devStates,seed,nsp*Gsize); // Each thread gets same seed
    printf(" Find good grids and blocks. \n");
    hipMalloc((void**) &dev_vsave, h_nvel * sizeof(float));
    hipMemcpy(dev_vsave, vsave, h_nvel * sizeof(float),hipMemcpyHostToDevice);
    // Field Solver 
    sMemSize = sizeof(double) * THREADS_PER_BLOCK;
    numBlocksPerSm = 0;
    numThreads = THREADS_PER_BLOCK;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, PCG, numThreads, sMemSize));
    numSms = prop.multiProcessorCount;
    FIELD_GRID = dim3(numSms*numBlocksPerSm, 1, 1);
    FIELD_BLOCK = dim3(THREADS_PER_BLOCK, 1, 1);
    printf(" - Field Solver : [%d][%d]\n",numSms*numBlocksPerSm,THREADS_PER_BLOCK);
    printf("   Cooperative_groups = [%d]\n",numSms*numBlocksPerSm*THREADS_PER_BLOCK);
    // Field2
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Cond_Sigma_Lap,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Field module : [%d][%d]\n",grid,block);
    FIELD_GRID2 = dim3(grid, 1, 1);
    FIELD_BLOCK2 = dim3(block, 1, 1);
    // Deposit 
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)DepositAtom,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Deposit module : [%d][%d]\n",grid,block);
    DEPOSIT_GRID = dim3(grid, 1, 1);
    DEPOSIT_BLOCK = dim3(block, 1, 1);
    // Efield
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)GCondAInit,0,CondNUMR); 
    grid = (CondNUMR*nsp + block - 1) / block;
    printf(" - Efield module : [%d][%d]\n",grid,block);
    EFIELD_GRID = dim3(grid, 1, 1);
    EFIELD_BLOCK = dim3(block, 1, 1);
    // Move 
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)MoveE_Basic,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Move module : [%d][%d]\n",grid,block);
    MOVE_GRID = dim3(grid, 1, 1);
    MOVE_BLOCK = dim3(block, 1, 1);
    // SortBoundary
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SortBoundary_Basic,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - SORT module : [%d][%d]\n",grid,block);
    SORT_GRID = dim3(grid, 1, 1);
    SORT_BLOCK = dim3(block, 1, 1);
    // MCC
    size = nsp * Gsize;
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)MCC_Ar_Basic,0,size); 
    grid = (size + block - 1) / block;
    printf(" - MCC module : [%d][%d]\n",grid,block);
    MCC_GRID = dim3(grid, 1, 1);
    MCC_BLOCK = dim3(block, 1, 1);
    // time setting
    gputime_field	=0.0;	gputime_efield	=0.0;	gputime_diag	=0.0;	gputime_move	=0.0;
	gputime_mcc		=0.0;	gputime_continue=0.0;	gputime_deposit	=0.0;	gputime_sort	=0.0;
	gputime_Tec	    =0.0;	gputime_dump	=0.0;	totaltime		=0.0;	TotalT_D		=0;
	TotalT_H		=0;	TotalT_M		=0;	TotalT_S		=0;
}
__global__ void SetSeed(hiprandState *state,long int seed,int num)
{
	int TID = blockDim.x * blockIdx.x + threadIdx.x;
	if(TID>=num) return;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, TID, 0, &state[TID]);
}
