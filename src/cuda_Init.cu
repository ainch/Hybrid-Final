#include "hip/hip_runtime.h"
#include "cuda_Init.cuh"
#define THREADS_PER_BLOCK 512   

void Set_Device_Parameter(){
    int grid,block;
    int mingrid;
    int numBlocksPerSm;
    int numThreads;
    int numSms;
    int size;
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SetSeed,0,nsp*Gsize); 
    grid = (nsp*Gsize + block - 1) / block;
    hipMalloc((void**) &devStates, nsp*Gsize*sizeof(hiprandState));
    SetSeed<<<grid,block>>>(devStates,seed,nsp*Gsize); // Each thread gets same seed
    printf(" Find good grids and blocks. \n");
    hipMalloc((void**) &dev_vsave, h_nvel * sizeof(float));
    hipMemcpy(dev_vsave, vsave, h_nvel * sizeof(float),hipMemcpyHostToDevice);
    // Field Solver 
    sMemSize = sizeof(float) * THREADS_PER_BLOCK;
    numBlocksPerSm = 0;
    numThreads = THREADS_PER_BLOCK;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, PCG_float, numThreads, sMemSize));
    numSms = prop.multiProcessorCount;
    FIELD_GRID = dim3(numSms*numBlocksPerSm, 1, 1);
    FIELD_BLOCK = dim3(THREADS_PER_BLOCK, 1, 1);
    printf(" - Field Solver : [%d][%d]\n",numSms*numBlocksPerSm,THREADS_PER_BLOCK);
    printf("   Cooperative_groups = [%d]\n",numSms*numBlocksPerSm*THREADS_PER_BLOCK);
    // Field2
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Cond_Sigma_Lap,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Field module : [%d][%d]\n",grid,block);
    FIELD_GRID2 = dim3(grid, 1, 1);
    FIELD_BLOCK2 = dim3(block, 1, 1);
    // Deposit 
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)DepositAtom,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Deposit module : [%d][%d]\n",grid,block);
    DEPOSIT_GRID = dim3(grid, 1, 1);
    DEPOSIT_BLOCK = dim3(block, 1, 1);
    // Move 
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)MoveE_Basic,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Move module : [%d][%d]\n",grid,block);
    MOVE_GRID = dim3(grid, 1, 1);
    MOVE_BLOCK = dim3(block, 1, 1);
    // SortBoundary
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SortBoundary_Basic,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - SORT module : [%d][%d]\n",grid,block);
    SORT_GRID = dim3(grid, 1, 1);
    SORT_BLOCK = dim3(block, 1, 1);
    // MCC
    size = nsp*Gsize;
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)MCC_Ar_Basic,0,size); 
    grid = (size + block - 1) / block;
    printf(" - MCC module : [%d][%d]\n",grid,block);
    MCC_GRID = dim3(grid, 1, 1);
    MCC_BLOCK = dim3(block, 1, 1);
    // Diagnostics
    size = Gsize;
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Average_Particle_Density,0,size); 
    grid = (size + block - 1) / block;
    printf(" - Diagnostics 1 module : [%d][%d]\n",grid,block);
    DIAG_G_GRID = dim3(grid, 1, 1);
    DIAG_G_BLOCK = dim3(block, 1, 1);
    size = nsp * Gsize;
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Average_Argon_MCC_rate,0,size); 
    grid = (size + block - 1) / block;
    printf(" - Diagnostics 2 module : [%d][%d]\n",grid,block);
    DIAG_NSPG_GRID = dim3(grid, 1, 1);
    DIAG_NSPG_BLOCK = dim3(block, 1, 1);
    
    // time setting
    gputime_field	=0.0;	gputime_efield	=0.0;	gputime_diag	=0.0;	gputime_move	=0.0;
	gputime_mcc		=0.0;	gputime_continue=0.0;	gputime_deposit	=0.0;	gputime_sort	=0.0;
	gputime_Tec	    =0.0;	gputime_dump	=0.0;	totaltime		=0.0;	TotalT_D		=0;
	TotalT_H		=0;	TotalT_M		=0;	TotalT_S		=0;
}
__global__ void SetSeed(hiprandState *state,long int seed,int num)
{
	int TID = blockDim.x * blockIdx.x + threadIdx.x;
	if(TID>=num) return;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, TID, 0, &state[TID]);
}
