#include "hip/hip_runtime.h"
#include "cuda_Init.cuh"
#define THREADS_PER_BLOCK 512   

void Set_DiagParameter_cuda(){
    
    // Host BUF VECTOR
    Host_G_buf = VFMalloc(Gsize);
    Host_C_buf = VFMalloc(Csize);
    VFInit(Host_G_buf,0.0,Gsize);
    VFInit(Host_C_buf,0.0,Csize);
}
void Set_Device_Parameter(){
    int grid,block;
    int mingrid;
    int numBlocksPerSm;
    int numThreads;
    int numSms;
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SetSeed,0,nsp*Gsize); 
    grid = (nsp*Gsize + block - 1) / block;
    hipMalloc((void**) &devStates, nsp*Gsize*sizeof(hiprandState));
    SetSeed<<<grid,block>>>(devStates,seed,nsp*Gsize); // Each thread gets same seed
    printf(" Find good grids and blocks. \n");
    hipMalloc((void**) &dev_vsave, h_nvel * sizeof(float));
    hipMemcpy(dev_vsave, vsave, h_nvel * sizeof(float),hipMemcpyHostToDevice);
    // Field Solver 
    sMemSize = sizeof(double) * THREADS_PER_BLOCK;
    numBlocksPerSm = 0;
    numThreads = THREADS_PER_BLOCK;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, PCG, numThreads, sMemSize));
    numSms = prop.multiProcessorCount;
    FIELD_GRID = dim3(numSms*numBlocksPerSm, 1, 1);
    FIELD_BLOCK = dim3(THREADS_PER_BLOCK, 1, 1);
    printf(" - Field Solver : [%d][%d]\n",numSms*numBlocksPerSm,THREADS_PER_BLOCK);
    printf("   Cooperative_groups = [%d]\n",numSms*numBlocksPerSm*THREADS_PER_BLOCK);
    // Field2
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Cond_Sigma_Lap,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Field module : [%d][%d]\n",grid,block);
    FIELD_GRID2 = dim3(grid, 1, 1);
    FIELD_BLOCK2 = dim3(block, 1, 1);
    // Deposit 
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)DepositAtom,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Deposit module : [%d][%d]\n",grid,block);
    DEPOSIT_GRID = dim3(grid, 1, 1);
    DEPOSIT_BLOCK = dim3(block, 1, 1);
    // Efield
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)GCondAInit,0,CondNUMR); 
    grid = (CondNUMR*nsp + block - 1) / block;
    printf(" - Efield module : [%d][%d]\n",grid,block);
    EFIELD_GRID = dim3(grid, 1, 1);
    EFIELD_BLOCK = dim3(block, 1, 1);
    // Move 
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)MoveE_Basic,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Move module : [%d][%d]\n",grid,block);
    MOVE_GRID = dim3(grid, 1, 1);
    MOVE_BLOCK = dim3(block, 1, 1);
    // SortBoundary
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SortBoundary_Basic,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - SORT module : [%d][%d]\n",grid,block);
    SORT_GRID = dim3(grid, 1, 1);
    SORT_BLOCK = dim3(block, 1, 1);
    // MCC
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)MCC_Ar_Basic,0,Gsize); 
    grid = (Gsize + block - 1) / block;
    printf(" - MCC module : [%d][%d]\n",grid,block);
    MCC_GRID = dim3(grid, 1, 1);
    MCC_BLOCK = dim3(block, 1, 1);

    // Example : Find good grid and block size
    int Search_Occupancy_Flag = 0;
    if(Search_Occupancy_Flag){
        //Example
        int *array;
        int blockSize;      // The launch configurator returned block size
        int minGridSize;    // The minimum grid size needed to achieve the
                            // maximum occupancy for a full device
                            // launch
        int gridSize;       // The actual grid size needed, based on input
                            // size
        hipDeviceProp_t prop;
        int numBlocks;       // Occupancy in terms of active blocks
        int activeWarps;
        int maxWarps;
        hipGetDevice(&device_num);
        hipGetDeviceProperties(&prop, device_num);
        hipMalloc((void**) &array, Gsize * sizeof(int));
        hipMemset((void *) array, 0, Gsize * sizeof(int));
        // Get MiniGridSize and blockSize
        hipOccupancyMaxPotentialBlockSize(&minGridSize,&blockSize,(void*)MyKernel,0,Gsize); 
        // Get Occupancy
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,MyKernel,blockSize,0);
        activeWarps = numBlocks * blockSize / prop.warpSize;
        maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
        printf("Occupancy: = %3.0f %\n",(double)activeWarps / maxWarps * 100);
        printf("maxWarps = %d\n",maxWarps);
        // Round up according to array size
        gridSize = (Gsize + blockSize - 1) / blockSize;
        printf("minGridSize = %d\n",minGridSize);
        printf("blockSize = %d\n",blockSize);
        printf("gridSize = %d\n",gridSize);
        MyKernel<<<gridSize, blockSize>>>(array, Gsize);
        hipDeviceSynchronize();
    }
}
__global__ void SetSeed(hiprandState *state,long int seed,int num)
{
	int TID = blockDim.x * blockIdx.x + threadIdx.x;
	if(TID>=num) return;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, TID, 0, &state[TID]);
}
/*
__global__ void SetSeed_Coorperative(hiprandState *state,long int seed,int num){
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    Curand_initialized(seed,num,state,grid);
    cg::sync(grid);
}
__device__ void Curand_initialized(long int seed, int num, hiprandState *state, const cg::grid_group &grid){
	for (int i=grid.thread_rank(); i < num; i+= grid.size()){
        hiprand_init(seed, i, 0, &state[i]);
	} 
}
*/
__global__ void MyKernel(int *array, int arrayCount)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arrayCount) {
        array[idx] *= array[idx];
    }
}