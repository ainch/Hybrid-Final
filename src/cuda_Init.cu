#include "hip/hip_runtime.h"
#include "cuda_Init.cuh"
#define THREADS_PER_BLOCK 512   


void Set_DiagParameter_cuda(){
    // Host BUF VECTOR
    Host_G_buf = VFMalloc(Gsize);
    Host_C_buf = VFMalloc(Csize);
    VFInit(Host_G_buf,0.0,Gsize);
    VFInit(Host_C_buf,0.0,Csize);
}
void Set_Device_Parameter(){
    int grid,block;
    int mingrid;
    int numBlocksPerSm;
    int numThreads;
    int numSms;
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SetSeed,0,Gsize); 
    grid = (Gsize + block - 1) / block;
    hipMalloc((void**) &devStates, Gsize * sizeof(hiprandState));
    SetSeed<<<grid,block>>>(devStates,seed,Gsize); // Each thread gets same seed
    //
    printf(" Find good grids and blocks. \n");
    // Field Solver 
    sMemSize = sizeof(double) * THREADS_PER_BLOCK;
    numBlocksPerSm = 0;
    numThreads = THREADS_PER_BLOCK;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, PCG, numThreads, sMemSize));
    numSms = prop.multiProcessorCount;
    FIELD_GRID = dim3(numSms*numBlocksPerSm, 1, 1);
    FIELD_BLOCK = dim3(THREADS_PER_BLOCK, 1, 1);
    printf(" - Field Solver : [%d][%d]\n",numSms*numBlocksPerSm,THREADS_PER_BLOCK);
    printf("   Cooperative_groups = [%d]\n",numSms*numBlocksPerSm*THREADS_PER_BLOCK);
    // Field2
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)Cond_Sigma_Lap,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Field module : [%d][%d]\n",grid,block);
    FIELD_GRID2 = dim3(grid, 1, 1);
    FIELD_BLOCK2 = dim3(block, 1, 1);
    // Deposit 
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)DepositAtom,0,Gsize*nsp); 
    grid = (Gsize*nsp + block - 1) / block;
    printf(" - Deposit module : [%d][%d]\n",grid,block);
    DEPOSIT_GRID = dim3(grid, 1, 1);
    DEPOSIT_BLOCK = dim3(block, 1, 1);
    // Efield
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)GCondAInit,0,CondNUMR); 
    grid = (CondNUMR*nsp + block - 1) / block;
    printf(" - Efield module : [%d][%d]\n",grid,block);
    EFIELD_GRID = dim3(grid, 1, 1);
    EFIELD_BLOCK = dim3(block, 1, 1);

    // Example : Find good grid and block size
    int Search_Occupancy_Flag = 0;
    if(Search_Occupancy_Flag){
        //Example
        int *array;
        int blockSize;      // The launch configurator returned block size
        int minGridSize;    // The minimum grid size needed to achieve the
                            // maximum occupancy for a full device
                            // launch
        int gridSize;       // The actual grid size needed, based on input
                            // size
        hipDeviceProp_t prop;
        int numBlocks;       // Occupancy in terms of active blocks
        int activeWarps;
        int maxWarps;
        hipGetDevice(&device_num);
        hipGetDeviceProperties(&prop, device_num);
        hipMalloc((void**) &array, Gsize * sizeof(int));
        hipMemset((void *) array, 0, Gsize * sizeof(int));
        // Get MiniGridSize and blockSize
        hipOccupancyMaxPotentialBlockSize(&minGridSize,&blockSize,(void*)MyKernel,0,Gsize); 
        // Get Occupancy
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,MyKernel,blockSize,0);
        activeWarps = numBlocks * blockSize / prop.warpSize;
        maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
        printf("Occupancy: = %3.0f %\n",(double)activeWarps / maxWarps * 100);
        printf("maxWarps = %d\n",maxWarps);
        // Round up according to array size
        gridSize = (Gsize + blockSize - 1) / blockSize;
        printf("minGridSize = %d\n",minGridSize);
        printf("blockSize = %d\n",blockSize);
        printf("gridSize = %d\n",gridSize);
        MyKernel<<<gridSize, blockSize>>>(array, Gsize);
        hipDeviceSynchronize();
    }
}
__global__ void SetSeed(hiprandState *state,long int seed,int num)
{
		int TID = blockDim.x * blockIdx.x + threadIdx.x;
		if(TID>num) return;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, TID, 0, &state[TID]);
}
__global__ void MyKernel(int *array, int arrayCount)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arrayCount) {
        array[idx] *= array[idx];
    }
}