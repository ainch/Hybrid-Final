#include "hip/hip_runtime.h"
#include "cuda_Init.cuh"

void Set_NullCollisionTime_cuda(){
    
}
void Set_DiagParameter_cuda(){
    
}
void Set_Particle_cuda(){
    
}
void Set_Device_Parameter(){
    int grid,block;
    int mingrid;
    
    int   *vec_cond_Garray;
    int   *vec_boundary_Garray;
    int   *vec_face_Garray;
    float *vec_area_Garray;
    float *vec_eps_Carray;
    float *dev_Sigma;
    int   *dev_face_Garray;
    float *dev_area_Garray;
    float *dev_eps_Carray;
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)SetSeed,0,Gsize); 
    grid = (Gsize + block - 1) / block;
    

    hipMalloc((void**) &devStates, Gsize * sizeof(hiprandState));
    SetSeed<<<grid,block>>>(devStates,seed,Gsize); // Each thread gets same seed

    // Example : Find good grid and block size
    int Search_Occupancy_Flag = 0;
    if(Search_Occupancy_Flag){
        //Example
        int *array;
        int blockSize;      // The launch configurator returned block size
        int minGridSize;    // The minimum grid size needed to achieve the
                            // maximum occupancy for a full device
                            // launch
        int gridSize;       // The actual grid size needed, based on input
                            // size
        hipDeviceProp_t prop;
        int numBlocks;       // Occupancy in terms of active blocks
        int activeWarps;
        int maxWarps;
        hipGetDevice(&device_num);
        hipGetDeviceProperties(&prop, device_num);
        hipMalloc((void**) &array, Gsize * sizeof(int));
        hipMemset((void *) array, 0, Gsize * sizeof(int));
        // Get MiniGridSize and blockSize
        hipOccupancyMaxPotentialBlockSize(&minGridSize,&blockSize,(void*)MyKernel,0,Gsize); 
        // Get Occupancy
        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks,MyKernel,blockSize,0);
        activeWarps = numBlocks * blockSize / prop.warpSize;
        maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;
        printf("Occupancy: = %3.0f %\n",(double)activeWarps / maxWarps * 100);
        printf("maxWarps = %d\n",maxWarps);
        // Round up according to array size
        gridSize = (Gsize + blockSize - 1) / blockSize;
        printf("minGridSize = %d\n",minGridSize);
        printf("blockSize = %d\n",blockSize);
        printf("gridSize = %d\n",gridSize);
        MyKernel<<<gridSize, blockSize>>>(array, Gsize);
        hipDeviceSynchronize();
    }
}
__global__ void SetSeed(hiprandState *state,long int seed,int num)
{
		int TID=blockDim.x*blockIdx.x+threadIdx.x;
		if(TID>num) return;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, TID, 0, &state[TID]);
}
__global__ void MyKernel(int *array, int arrayCount)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < arrayCount) {
        array[idx] *= array[idx];
    }
}