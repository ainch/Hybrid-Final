#include "cuda_start.cuh"

void info_Device()
{
	int count,i;
	hipDeviceProp_t prop;
	// Determine the number of CUDA capable GPUs
	checkCudaErrors(hipGetDeviceCount(&count));
	if(count<1){
		printf("No CUDA Capable GPU(s) Detected \n");
		return;
	}
    // Display the GPU processor specification
	printf("<CUDA Parallel Progaraming>\n");
	printf("- Number of CUDA devices:\t%d\n", count);
	for(i=0;i<count;i++){
		hipDeviceProp_t dprop;
		checkCudaErrors(hipGetDeviceProperties(&dprop, i));
		printf(">>>>>>>> Device %d is a %s\n", i, dprop.name);
	}
    printf("GPU selected Device ID = %d \n", device_num);
	checkCudaErrors(hipGetDeviceProperties (&prop, device_num));
	checkCudaErrors(hipSetDevice(device_num));
    // This will pick the best possible CUDA capable device
	printf("-------------------- Device%d --------------------\n", device_num);
	printf("Name: %s \n", prop.name);
	printf("Computer capability: %d.%d \n", prop.major, prop.minor);
	//printf("Clock rate: %d\n", prop.clockRate);
	printf("Total Global Mem.: %u Mbytes\n", prop.totalGlobalMem/1024/1024);
	//printf("Total constant Mem.: %u Kbytes\n", prop.totalConstMem/1024);
	printf("Shared Mem. per block: %u bytes \n", prop.sharedMemPerBlock);
	//printf("Registers available per block.: %u #\n", prop.regsPerBlock);
	//printf("Max Mem. pitch: %ld\n", prop.memPitch);
	printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
	printf("Threads in warp: %d \n", prop.warpSize);
	printf("Max thread dimensions: (%d, %d, %d) \n",prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("Max grid dimensions: (%d, %d, %d) \n",prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("---------------------------------------------------\n");	
}
void start_cuda(){

	/*** Field solver ***/
	if(PCG_Method==0){
		
	}else if(PCG_Method==1){

	}else if(PCG_Method==2){

	}else if(PCG_Method==3){

	}
	//FieldSolver = PCG_SOLVER;
	/*** Move ***/
	if(ConstB_Flag){
		//MOVE = MoveB_cuda;
	}else {
		//MOVE = Move_cuda;
	}
	switch(MainGas){
	case ARGON:
		//SORT_BOUNDARY = AR_SortAndBoundary_cuda;
		//MCC		= ArMccDiag_cuda;
		//CONTIEQ = Ar_solve_continuity_eqn;
		//DIAG    = Diagnostic;
		break;
	case OXYGEN:
		//SORT_BOUNDARY = Oxy_SortAndBoundary_cuda;
		//MCC		= OxyMccDiag_cuda;
		//CONTIEQ = Oxy_solve_continuity_eqn;
		//DIAG    = Diagnostic;
		break;
	case ARO2:
		//SORT_BOUNDARY = ARO2_SortAndBoundary_cuda;
		//MCC		= ARO2_MccDiag_cuda;
		//CONTIEQ = ARO2_solve_continuity_eqn;
		//DIAG    = Diagnostic;
		break;
	}
	/*** Deposit ***/
	//DEPOSIT = Deposit_cuda;

}
