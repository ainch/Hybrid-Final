#include "cuda_start.cuh"
void start_cuda(){
	if(External_Flag){
		EFIELD = Efield_cuda;
	}else{
		EFIELD = Efield_cuda_Basic;
	}
	/*** Move ***/
	if(ConstB_Flag){
		//MOVE = MoveB_cuda;
	}else {
		MOVE = Move_cuda;
	}
	SORT_BOUNDARY = SortBounndary_cuda;

	switch(MainGas){
	case ARGON:
		MCC	= MCC_Ar_cuda;
		MCC_Basic = MCC_Ar_cuda;
		break;
	case OXYGEN:
		MCC		= MCC_O2_cuda;
		MCC_Basic = MCC_O2_cuda;
		break;
	case ARO2:
		MCC		= MCC_ArO2_cuda;
		MCC_Basic = MCC_ArO2_cuda;
		break;
	}
	if(CSS_Flag) CONTIEQ = Solve_Continuity_eqn_check;
	else CONTIEQ = Solve_Continuity_eqn;
	DEPOSIT = Deposit_cuda;
	DIAG    = Diagnostic;
}

void info_Device()
{
	int count,i,isp;
	int NeedMemory = 0;
	// Determine the number of CUDA capable GPUs
	checkCudaErrors(hipGetDeviceCount(&count));
	if(count<1){
		printf("No CUDA Capable GPU(s) Detected \n");
		return;
	}
    // Display the GPU processor specification
	printf("<CUDA Parallel Progaraming>\n");
	printf("- Number of CUDA devices:\t%d\n", count);
	for(i=0;i<count;i++){
		hipDeviceProp_t dprop;
		checkCudaErrors(hipGetDeviceProperties(&dprop, i));
		printf(">>>>>>>> Device %d is a %s\n", i, dprop.name);
	}
    printf("GPU selected Device ID = %d \n", device_num);
	checkCudaErrors(hipGetDeviceProperties (&prop, device_num));
	checkCudaErrors(hipSetDevice(device_num));
    // This will pick the best possible CUDA capable device
	printf("-------------------- Device%d --------------------\n", device_num);
	printf("Name: %s \n", prop.name);
	printf("Computer capability: %d.%d \n", prop.major, prop.minor);
	//printf("Clock rate: %d\n", prop.clockRate);
	printf("Total Global Mem.: %u Mbytes\n", prop.totalGlobalMem/1024/1024);
	printf("Total constant Mem.: %u Kbytes\n", prop.totalConstMem/1024);
	printf("Shared Mem. per block: %u bytes \n", prop.sharedMemPerBlock);
	printf("Registers available per block.: %u #\n", prop.regsPerBlock);
	printf("Max Mem. pitch: %ld\n", prop.memPitch);
	printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
	printf("Threads in warp: %d \n", prop.warpSize);
	printf("Max thread dimensions: (%d, %d, %d) \n",prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
	printf("Max grid dimensions: (%d, %d, %d) \n",prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
	printf("---------------------------------------------------\n");	
	checkCudaErrors(hipGetDeviceProperties(&prop, device_num));
    if (!prop.managedMemory){
        // This sample requires being run on a device that supports Unified Memory
        fprintf(stderr, "Unified Memory not supported on this device\n");
        exit(EXIT_WAIVED);
    }
    // This sample requires being run on a device that supports Cooperative Kernel Launch
    if (!prop.cooperativeLaunch){
        printf("\nSelected GPU (%d) does not support Cooperative Kernel Launch, Waiving the run\n", device_num);
        exit(EXIT_WAIVED);
    }
	// Aprroximation
    for(isp=0;isp<nsp;isp++){
        NeedMemory += 6 * SP[isp].MAXNP*sizeof(float)/1024/1024;
    }
	//eedMemory += 1.1 * Gsize * sizeof(float);
	if(prop.totalGlobalMem/1024/1024<NeedMemory){
		printf("Error : Insufficient GPU memory.\n");
		printf(" GPU Global: %u Mbytes\n", prop.totalGlobalMem/1024/1024);
		printf(" Need Memory: %u Mbytes (Approx.)\n", NeedMemory);
		printf(" You should reduce MaxNP.\n");
		exit(1);
	}
}
