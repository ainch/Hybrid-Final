#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"

void PCG_SOLVER_Laplace(){
    // Solve Laplace Equation. (To use every time step.)
    // Goal
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    int i,j,k,TID; 

    //////////////////////////////////////////////////////////////////////////////
    int grid,block,mingrid;
    int IIter;
    float *buf;
    float **CPUsol;
    CPUsol = MFMalloc(CondNUMR,Gsize);
    buf = VFMalloc(A_size);

    printf("<FIELD SOVER>\n");
	printf(" Laplace eq. using PCG\n");
	printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)PCG_LAP,0,Gsize); 
    grid = (Gsize + block - 1) / block;
    printf("minGridSize = %d\n",mingrid);
    printf("blockSize = %d\n",block);
    printf("gridSize = %d\n",grid);
  
    for (k = 0; k < CondNUMR-1; k++) {
        checkCudaErrors(hipMemcpy(dev_b, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
        checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
        //  Method 1
        //PCG_LAP<<<grid,block>>>(dev_A,dev_Ai,dev_Aj,dev_PCG_const,dev_PCG_DATA,dev_X,dev_b);
        //  Method 2
        FieldIter = PCG_LAP_Divide(grid,block);
        //
        checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
        //checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
        //checkCudaErrors(hipMemcpy(Host_PCG_DATA, dev_PCG_DATA, A_size*sizeof(DPS_Data), hipMemcpyDeviceToHost));
        for(j=ngy-1;j>=0;j--){
            for(i=0;i<ngx;i++){
                TID = i*ngy+j;
                if((vec_G[TID].CondID-1)==k){
                    CPUsol[k][TID] = 1.0;
                }
                if(vec_A_idx[TID]){
                    CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                }
            }
        }
        /*
        for(j=ngy-1;j>=0;j--){
            for(i=0;i<ngx;i++){
                TID = i*ngy+j;
                printf("%6.2g",CPUsol[k][TID]);
            }printf("\n");
        }printf("\n");
        */
    }
    exit(1);
}
void Set_MatrixPCG_cuda(){
    int TID;
    int i,j,k; 
    int PCG_Laplace_SINGLECPU_Flag=1;
    float **CPUsol;
	//vec_cond_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_boundary_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_face_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_area_Garray = (float *) malloc(ngx * ngy * sizeof(float));
	//vec_eps_Carray = (float *) malloc(ncx * ncy * sizeof(float));

    vec_A_idx = (int *) malloc(ngx * ngy * sizeof(int));
    for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
			//vec_cond_Garray[j + i * ngy] = cond_Garray[i][j];
			//vec_boundary_Garray[j + i * ngy] = boundary_Garray[i][j];
			//vec_face_Garray[j + i * ngy] = face_Garray[i][j];
			//vec_area_Garray[j + i * ngy] = area_Garray[i][j];
		} // matrix save direction ^ >
	}
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  A_size * sizeof(float)));
    // Initialize
    checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
    //Copy
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, A_size * sizeof(int), hipMemcpyHostToDevice));
    // Laplace Solution
    hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
    // hipMemset((void *) array, 0, Gsize * sizeof(int));
    
    //Make a Field constant set  
    Host_PCG_const = (DPS_Const*)malloc(sizeof(DPS_Const));
    checkCudaErrors(hipMalloc((void**)&dev_PCG_const,sizeof(DPS_Const)));
    Make_PCG_Const_Init<<<1,1>>>(dev_PCG_const,A_size,PCGtol);
    checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));

    //Make a Field DATA set  
    Host_PCG_DATA = (DPS_Data*)malloc(A_size*sizeof(DPS_Data));
    checkCudaErrors(hipMalloc((void**)&dev_PCG_DATA, A_size*sizeof(DPS_Data)));
    Make_PCG_DATA_Init<<<A_size/4,4>>>(dev_PCG_DATA,A_size,dev_M);
    
    if(PCG_Laplace_SINGLECPU_Flag==1){
		printf(" Preconditioner[Jacovi]\n"); 
        printf(" Main Library set[Single CPU PCG]\n"); 
        printf(" Laplace Equation TEST\n"); 
        X = VFMalloc(A_size);
        B = VFMalloc(A_size);
        R0 = VFMalloc(A_size);
        AX = VFMalloc(A_size);
        Z0 = VFMalloc(A_size);
        P0 = VFMalloc(A_size);
        AP = VFMalloc(A_size);
        PAP = VFMalloc(A_size);
        VFInit(X,0.0,A_size);
        VFInit(B,0.0,A_size);
        VFInit(AX,0.0,A_size);
        VFInit(R0,0.0,A_size);
        VFInit(Z0,0.0,A_size);
        VFInit(P0,0.0,A_size);
        VFInit(AP,0.0,A_size);
        VFInit(PAP,0.0,A_size);
        CPUsol = MFMalloc(CondNUMR,Gsize);
        for (k = 0; k < CondNUMR; k++) {
            VFCopy(B,cond_b[k],A_size);
            VFInit(X,0.0,A_size);
            FieldIter = PCG_SINGLECPU();
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Save
            VFInit(CPUsol[k],0.0,Gsize);
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = X[vec_A_idx[TID]-1];
                    }
                }
            }
            /*
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    printf("%6.2g",CPUsol[k][TID]);
                }printf("\n");
            }printf("\n");
            */
            CPU_PCG_Laplace_Solution_Save(CPUsol);
        }
        //exit(1);
	}
}
int PCG_LAP_Divide(int grid,int block){
    int Iter = 0;
    float *dev_rsold;
    float *dev_Temp;
    float *dev_rnew;
    float rsold,rnew,Temp;
    float alpha,beta;
    
    checkCudaErrors(hipMalloc((void**) &dev_rsold,sizeof(dev_rsold)));
    checkCudaErrors(hipMemset((void *) dev_rsold, 0,  sizeof(dev_rsold)));
    checkCudaErrors(hipMalloc((void**) &dev_Temp,sizeof(dev_Temp)));
    checkCudaErrors(hipMemset((void *) dev_Temp, 0,  sizeof(dev_Temp)));
    checkCudaErrors(hipMalloc((void**) &dev_rnew,sizeof(dev_rnew)));
    checkCudaErrors(hipMemset((void *) dev_rnew, 0,  sizeof(dev_rnew)));

    Make_PCG_DATA_Init<<<A_size/4,4>>>(dev_PCG_DATA,A_size,dev_M);
    PCG_LAP_Part0<<<grid,block>>>(dev_A,dev_Ai,dev_Aj,dev_PCG_const,dev_PCG_DATA,dev_X,dev_b,dev_rsold);
    hipMemcpy(&rsold, dev_rsold, sizeof(dev_rsold), hipMemcpyDeviceToHost);
    printf("Iter = %d, rsold = %g\n",Iter,rsold);
    while(rsold>Host_PCG_const[0].tol2){
        Iter++;
        PCG_LAP_Part1<<<grid,block>>>(dev_A,dev_Ai,dev_Aj,dev_PCG_const,dev_PCG_DATA,dev_Temp);
        hipMemcpy(&Temp, dev_Temp, sizeof(dev_Temp), hipMemcpyDeviceToHost);
        alpha = (Temp) ? rsold/Temp : 0.0f;
        printf("Iter = %d, alpha = %g\n",Iter,alpha);
        PCG_LAP_Part2<<<grid,block>>>(dev_PCG_const,dev_PCG_DATA,dev_X,alpha, dev_rnew);
        hipMemcpy(&rnew, dev_rnew, sizeof(dev_rnew), hipMemcpyDeviceToHost);
        beta = (rsold) ? rnew/rsold : 0.0f;
        printf("Iter = %d, beta = %g\n",Iter,beta);
        PCG_LAP_Part3<<<grid,block>>>(dev_PCG_const,dev_PCG_DATA,beta);
        rsold = rnew;
        hipMemset((void *) dev_Temp, 0,  sizeof(dev_Temp));
        hipMemset((void *) dev_rnew, 0,  sizeof(dev_rnew));
        if(Iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",Iter,Temp,alpha,beta,rsold);
    }
    return Iter;
}
__global__ void Make_PCG_DATA_Init(DPS_Data *p, int size,float *MatrixM){
    int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if(TID>=size) return;
    //printf("TID = %d, M = %g\n",TID,MatrixM[TID]);
    p[TID].R = 0.0;
    p[TID].Z = 0.0;
    p[TID].P = 0.0;
    p[TID].AP = 0.0;
    p[TID].M = MatrixM[TID];
}
__global__ void Make_PCG_Const_Init(DPS_Const *p,int Asize, float tol){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].A_size = Asize;
    p[i].Iter = 0;
    p[i].tol = tol;
    p[i].tol2 = tol*tol;
    p[i].rsold = 0.0;
    p[i].Temp = 0.0;
    p[i].rnew = 0.0;
    p[i].alpha = 0.0;
    p[i].beta = 0.0;    
}
int PCG_SINGLECPU(){
    int TID,i,Iter=0;
    float tol2;
    float rsold,rnew,Temp;
    float alpha,beta;

    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        Z0[TID] = MatM[TID] * R0[TID];
        P0[TID] = Z0[TID];
        rsold += R0[TID]*Z0[TID]; //AtomicAdd!!
    }
    tol2 = PCGtol*PCGtol;
    while(rsold>tol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
            }
            //printf("AP[%d] = %g\n",TID,P0[TID]);
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //AtomicAdd!!
        }
        alpha = (Temp)? rsold/Temp:0.0f ;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            Z0[TID] = MatM[TID] * R0[TID];
            rnew += R0[TID]*Z0[TID];  //AtomicAdd!!
        }
        beta = (rsold) ? rnew/rsold: 0.0f;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = Z0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
        if(Iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",Iter,Temp,alpha,beta,rsold);
    }
    return Iter;
}
__global__ void PCG_LAP(float *A,int *Ai,int *Aj,DPS_Const *PCG_C,DPS_Data *PCG_D,float *X,float *b){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=PCG_C[0].A_size) return;
    int i;
    float sum;
    float *rs1,*rs2,*rs3;
    int MAXITER = 20;
    //Initial
    if(TID==0){
        PCG_C[0].Iter = 0;
        PCG_C[0].rsold = 0;
        PCG_C[0].Temp = 0;
        PCG_C[0].rnew = 0;
    } 
    __syncthreads();
    // cal  AP = A * P
    for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
        PCG_D[TID].AP += A[i] * X[Aj[i]-1];
    }
    PCG_D[TID].R = b[TID] - PCG_D[TID].AP;
    PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
    PCG_D[TID].P = PCG_D[TID].Z;
    sum = PCG_D[TID].R*PCG_D[TID].Z;
    atomicAdd(&PCG_C[0].rsold,sum);
    //if(TID==0) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
    //if(TID==10) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
    //if(TID==0) printf(" [%d]Initial rsold = %g, tol2 = %g\n",TID,PCG_C[0].rsold,PCG_C[0].tol2);
    while(PCG_C[0].rsold > PCG_C[0].tol2){
        if(TID==0){
            PCG_C[0].Iter++;
            PCG_C[0].Temp = 0;
            PCG_C[0].rnew = 0;
        } 
        __syncthreads();
        //if(TID==0) printf(" [%d]Iter %d start!\n",TID,PCG_C[0].Iter);
        PCG_D[TID].AP = 0;
        for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
            PCG_D[TID].AP += A[i] * PCG_D[Aj[i]-1].P;
        }
        //printf("[%d] AP = %g\n",TID,PCG_D[TID].P);
        sum = PCG_D[TID].P * PCG_D[TID].AP;
        atomicAdd(&PCG_C[0].Temp,sum);
        //if(TID==0) printf(" [%d]Temp = %g\n",TID,PCG_C[0].Temp);       
        if(PCG_C[0].Iter>MAXITER){
            //if(TID==0) printf("Iteration[%d] = %g\n",TID,PCG_C[0].Iter);    
            break;
        } 
        if(TID==0){
            PCG_C[0].alpha = (PCG_C[0].Temp)? PCG_C[0].rsold/PCG_C[0].Temp : 0.0f; 
        } 
        __syncthreads();     
        X[TID] = X[TID] + PCG_C[0].alpha * PCG_D[TID].P;
        PCG_D[TID].R = PCG_D[TID].R - PCG_C[0].alpha * PCG_D[TID].AP;
        PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
        sum = PCG_D[TID].R * PCG_D[TID].Z;
        atomicAdd(&PCG_C[0].rnew,sum);
       //if(TID==0) printf(" [%d]rnew = %g\n",TID,PCG_C[0].rnew);     
        if(TID==0){
             PCG_C[0].beta = (PCG_C[0].rsold)? PCG_C[0].rnew/PCG_C[0].rsold : 0.0f;
        }
        __syncthreads();
        PCG_D[TID].P = PCG_D[TID].Z + PCG_C[0].beta*PCG_D[TID].P;    
        if(TID==0) PCG_C[0].rsold = PCG_C[0].rnew;
        __syncthreads();
        if(TID==0) printf(" [%d]Iter %d,Temp = %g,alpha = %g,beat = %g, rsold = %g\n",TID,PCG_C[0].Iter,PCG_C[0].Temp,PCG_C[0].alpha,PCG_C[0].beta,PCG_C[0].rsold);
    }
    if(TID==0) printf("Iter [%d]= %d, ",TID,PCG_C[0].Iter);  
    if(TID==0) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
}
__global__ void PCG_LAP_Part0(float *A,int *Ai,int *Aj,DPS_Const *PCG_C,DPS_Data *PCG_D,float *X,float *b,float *rsold){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=PCG_C[0].A_size) return;
    int i;
    float sum;
    // cal  AP = A * P
    for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
        PCG_D[TID].AP += A[i] * X[Aj[i]-1];
    }
    PCG_D[TID].R = b[TID] - PCG_D[TID].AP;
    PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
    PCG_D[TID].P = PCG_D[TID].Z;
    sum = PCG_D[TID].R*PCG_D[TID].Z;
    atomicAdd(rsold,sum); 
}
__global__ void PCG_LAP_Part1(float *A,int *Ai,int *Aj,DPS_Const *PCG_C,DPS_Data *PCG_D,float *Temp){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=PCG_C[0].A_size) return;
    int i;
    float sum;
    PCG_D[TID].AP = 0;
    for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
        PCG_D[TID].AP += A[i] * PCG_D[Aj[i]-1].P;
    }
    sum = PCG_D[TID].P * PCG_D[TID].AP;
    atomicAdd(Temp,sum);
}
__global__ void PCG_LAP_Part2(DPS_Const *PCG_C,DPS_Data *PCG_D,float *X,float alpha,float *rnew){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=PCG_C[0].A_size) return;
    int i;
    float sum;
    printf("alpha = %g\n",alpha);
    X[TID] = X[TID] + alpha * PCG_D[TID].P;
    PCG_D[TID].R = PCG_D[TID].R - alpha * PCG_D[TID].AP;
    PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
    sum = PCG_D[TID].R * PCG_D[TID].Z;
    atomicAdd(rnew,sum);
}
__global__ void PCG_LAP_Part3(DPS_Const *PCG_C,DPS_Data *PCG_D,float beta){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=PCG_C[0].A_size) return;
    PCG_D[TID].P = PCG_D[TID].Z + beta * PCG_D[TID].P;
}

__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}
