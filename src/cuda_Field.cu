#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"
int PCG_SINGLECPU(){
    int TID,i,Iter=0;
    float tol2;
    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        Z0[TID] = MatM[TID] * R0[TID];
        P0[TID] = Z0[TID];
        rsold += R0[TID]*Z0[TID];
    }
    tol2 = PCGtol*PCGtol;
    //printf("rsold=%g\n",rsold);
    while(rsold>tol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
                //printf("[%d],[%d] %g, %g\n",TID,Aj[i],A_val[i],P0[TID]);
            }
            //printf("PAP[%d] = %g\n",TID,AP[TID]);
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //Reduction
        }
        alpha = rsold/Temp;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            Z0[TID] = MatM[TID] * R0[TID];
            rnew += R0[TID]*Z0[TID];  //Reduction
        }
        beta = rnew/rsold;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = Z0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
        //printf("alpha %d = %2g, pap = %g, Beta %d = %2g, rsold=%2g\n",FieldIter-1,alpha,pap,FieldIter-1,beta,rsold);
    }
    return Iter;
}
void PCG_SOLVER_Laplace(){
    // Solve Laplace Equation. (To use every time step.)
    // Goal
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    int i,j;
    float *dev_A, *dev_b, *dev_R, *dev_P;	// PCG device parameter
    int *dev_Aj,*dev_Ai;			
    float *dev_AP, *dev_M, *dev_Z, *dev_X, *dev_Tmp;	// PCG device parameter
    int   *vec_A_idx;
    int   *vec_cond_Garray;
    int   *vec_boundary_Garray;
    int   *vec_face_Garray;
    float *vec_area_Garray;
    float *vec_eps_Carray;
    float *dev_Sigma;
    int   *dev_face_Garray;
    float *dev_area_Garray;
    float *dev_eps_Carray;
    hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float));
	hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int));
	hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int));
	hipMalloc((void**) &dev_b, A_size * sizeof(float));
	hipMalloc((void**) &dev_R, A_size * sizeof(float));
	hipMalloc((void**) &dev_Tmp, A_size * sizeof(float));
	hipMalloc((void**) &dev_Z, A_size * sizeof(float));
	hipMalloc((void**) &dev_P, A_size * sizeof(float));
	hipMalloc((void**) &dev_AP, A_size * sizeof(float));
	hipMalloc((void**) &dev_X, A_size * sizeof(float));
	hipMalloc((void**) &dev_M, A_size * sizeof(float));
    hipMalloc((void**) &dev_b, A_size * sizeof(float));
    // Initialize
    hipMemset((void *) dev_X, 0, A_size * sizeof(float));
	hipMemset((void *) dev_AP, 0, A_size * sizeof(float));
    //Copy
    hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_M, MatM, A_size * sizeof(float), hipMemcpyHostToDevice);
	vec_A_idx = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_cond_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_boundary_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_face_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_area_Garray = (float *) malloc(ngx * ngy * sizeof(float));
	//vec_eps_Carray = (float *) malloc(ncx * ncy * sizeof(float));
	for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
			//vec_cond_Garray[j + i * ngy] = cond_Garray[i][j];
			//vec_boundary_Garray[j + i * ngy] = boundary_Garray[i][j];
			//vec_face_Garray[j + i * ngy] = face_Garray[i][j];
			//vec_area_Garray[j + i * ngy] = area_Garray[i][j];
		} // matrix save direction ^ >
	}
    //////////////////////////////////////////////////////////////////////////////
	
    int grid,block,mingrid,TID;
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)PCG,0,Gsize); 
    grid = (Gsize + block - 1) / block;
    printf("minGridSize = %d\n",mingrid);
    printf("blockSize = %d\n",block);
    printf("gridSize = %d\n",grid);
    int k; 
    float **sol;
    sol = MFMalloc(CondNUMR,Gsize);
    MFInit(sol,0.0,CondNUMR,Gsize);
    for (k = 0; k < CondNUMR; k++) {

        

        
        printf("Solution %d\n",k);
        for(j=ngy-1;j>=0;j--){
            for(i=0;i<ngx;i++){
                TID = i*ngy+j;
                printf("%6.2g", sol[k][TID]);
            }printf("\n");
        }printf("\n");
    }
    exit(1);
    PCGtol *= 1e-3;
    for (i = 0; i < CondNUMR; i++) {
        hipMemcpy(dev_b, cond_b[i], A_size * sizeof(float),hipMemcpyHostToDevice);
        for(j=0;j<A_size;j++){
			//if(cond_b[i][j] !=0 )
				//printf("cond_b[%d][%d] = %g\n",i,j,cond_b[i][j]);
		} 
        PCG<<<grid,block>>>(FieldIter,Gsize,A_size,dev_A,dev_Ai,dev_Aj,dev_X,dev_b);
        printf("FieldIter = %d\n",FieldIter);
    }
    PCGtol *= 1e3;
    
}
void Set_MatrixPCG_cuda(){
    int PCG_Laplace_SINGLECPU_Flag=0;
    float *dev_phi_dw;
    float *dev_phi_u;
    printf("<FIELD SOVER>\n");
	printf(" Laplace eq. using PCG\n");
	printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    if(PCG_Laplace_SINGLECPU_Flag){
		printf(" Preconditioner[Jacovi]\n"); 
        printf(" Main Library set[Single CPU PCG]\n"); 
        X = VFMalloc(A_size);
        B = VFMalloc(A_size);
        R0 = VFMalloc(A_size);
        AX = VFMalloc(A_size);
        Z0 = VFMalloc(A_size);
        P0 = VFMalloc(A_size);
        AP = VFMalloc(A_size);
        PAP = VFMalloc(A_size);
        VFInit(X,0.0,A_size);
        VFInit(B,0.0,A_size);
        VFInit(AX,0.0,A_size);
        VFInit(R0,0.0,A_size);
        VFInit(Z0,0.0,A_size);
        VFInit(P0,0.0,A_size);
        VFInit(AP,0.0,A_size);
        VFInit(PAP,0.0,A_size);
	}
    // Laplace Solution
    hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
   // hipMemset((void *) array, 0, Gsize * sizeof(int));
}
__global__ void PCG(int Iter,int Gsize,int Asize,float *A,int *Ai,int *Aj,float *X,float *b){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=Asize) return;
    int i;
    float AP;
    // r0 = b-AX
    // Data access
    // TID = 0~(Asize-1)
    // St_ID = Ai[0];
    // ID = Ai[TID] - 1;
    // Dn = Ai[TID]-Ai[TID-1]

    
    Iter = TID;
    //printf("Iter = %d\n",Iter);
}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}