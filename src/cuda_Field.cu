#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"


void PCG_SOLVER_Laplace(){
    // Solve Laplace Equation. (To use every time step.)
    // Goal
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit

    float *dev_A, *dev_b, *dev_R, *dev_P;				// PCG device parameter
    float *dev_AP, *dev_M, *dev_Z, *dev_X, *dev_Tmp;	// PCG device parameter
    float *dev_phi_dw;
    float *dev_phi_u;

    hipMalloc((void**) &dev_b, A_size * sizeof(float));
    hipMalloc((void**) &dev_phi_dw, ngx * sizeof(float));
	hipMalloc((void**) &dev_phi_u, ngx * sizeof(float));

	int i;
    fprintf(stderr, "<FIELD SOVER>\n");
	fprintf(stderr, " Laplace eq. using PCG\n");
	fprintf(stderr, " Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);

    PCGtol *= 1e-3;
    for (i = 0; i < CondNUMR; i++) {
        hipMemcpy(dev_b, cond_b[i], A_size * sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(dev_phi_dw, phi_dw[i], ngx * sizeof(float),hipMemcpyHostToDevice);
		//hipMemcpy(dev_phi_u, phi_u[i], ngx * sizeof(float),hipMemcpyHostToDevice);
    }
    PCGtol *= 1e3;
}
void Set_MatrixPCG_cuda(){
    int   *vec_A_idx;
    int   *vec_cond_Garray;
    int   *vec_boundary_Garray;
    int   *vec_face_Garray;
    float *vec_area_Garray;
    float *vec_eps_Carray;
    float *dev_Sigma;
    int   *dev_face_Garray;
    float *dev_area_Garray;
    float *dev_eps_Carray;
    float *dev_phi_dw;
    float *dev_phi_u;

    // Laplace Solution
    hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
   // hipMemset((void *) array, 0, Gsize * sizeof(int));
}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}