#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"
// FOR Field method 4
#define THREADS_PER_BLOCK 512   
//
void PCG_SOLVER_Laplace(){
    // OUTPUT
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit

    //Make a Field DATA set  
    //Host_PCG_DATA = (DPS_Data*)malloc(A_size*sizeof(DPS_Data));
    //checkCudaErrors(hipMalloc((void**)&dev_PCG_DATA, A_size*sizeof(DPS_Data)));
    //Make_PCG_DATA_Init<<<A_size/4,4>>>(dev_PCG_DATA,A_size,dev_M);
    //checkCudaErrors(hipMemcpy(Host_PCG_DATA, dev_PCG_DATA, A_size*sizeof(DPS_Data), hipMemcpyDeviceToHost));
    // Laplace Solution
    //hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
    // hipMemset((void *) array, 0, Gsize * sizeof(int));
    

           // Find good grid and block size
        //hipOccupancyMaxPotentialBlockSize(&mingrid,&FIELD_BLOCK,(void*)hipsparseSpMV,0,Gsize); 
        //FIELD_GRID = (Gsize + FIELD_BLOCK - 1) / FIELD_BLOCK;
        //printf("blockSize = %d\n",FIELD_BLOCK);
        //printf("gridSize = %d\n",FIELD_GRID);
    // For test

    //if (matA       ) { checkCudaErrors(hipsparseDestroySpMat(matA)); }
    //if (vecx       ) { checkCudaErrors(hipsparseDestroyDnVec(vecx)); }
    //if (vecAP      ) { checkCudaErrors(hipsparseDestroyDnVec(vecAP)); }
    //if (vecp       ) { checkCudaErrors(hipsparseDestroyDnVec(vecp)); }
}
void Set_MatrixPCG_cuda(){

}
void Field_Method0_Initial(){
    // CPU Conjugate Gradient
    printf(" Field Solver : [CPU] Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    X = VFMalloc(A_size);
    B = VFMalloc(A_size);
    R0 = VFMalloc(A_size);
    AX = VFMalloc(A_size);
    P0 = VFMalloc(A_size);
    AP = VFMalloc(A_size);
    PAP = VFMalloc(A_size);
    VFInit(B,0.0,A_size);
    VFInit(AX,0.0,A_size);
    VFInit(R0,0.0,A_size);
    VFInit(P0,0.0,A_size);
    VFInit(AP,0.0,A_size);
    VFInit(PAP,0.0,A_size);
}
void Field_Method1_Initial(){
    // CPU Preconditioned Conjugate Gradient [Jacovi]
    printf(" Field Solver : [CPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Preconditioner[Jacovi]\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    X = VFMalloc(A_size);
    B = VFMalloc(A_size);
    R0 = VFMalloc(A_size);
    AX = VFMalloc(A_size);
    Z0 = VFMalloc(A_size);
    P0 = VFMalloc(A_size);
    AP = VFMalloc(A_size);
    PAP = VFMalloc(A_size);
    VFInit(B,0.0,A_size);
    VFInit(AX,0.0,A_size);
    VFInit(R0,0.0,A_size);
    VFInit(Z0,0.0,A_size);
    VFInit(P0,0.0,A_size);
    VFInit(AP,0.0,A_size);
    VFInit(PAP,0.0,A_size);
}
void Field_Method2_Initial(){
    printf(" Field Solver : [GPU] Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Cuda Handle setting
    cublasHandle = 0;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    checkCudaErrors(hipblasStatus_t);
    cusparseHandle = 0;
    checkCudaErrors(hipsparseCreate(&cusparseHandle));
    // Data cpu > gpu
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_AP,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    /* Wrap raw data into cuSPARSE generic API objects */
    matA = NULL;
    vecx = NULL;
    vecp = NULL;
    vecAP = NULL;
    checkCudaErrors(hipsparseCreateCsr(
        &matA, A_size, A_size, 5*A_size, dev_Ai, dev_Aj, dev_A, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecx, A_size, dev_X, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecp, A_size, dev_P, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecAP, A_size, dev_AP, HIP_R_32F));
}
void Field_Method3_Initial(){
    printf(" Field Solver : [GPU] Conjugate Gradient\n"); 
    printf(" Cuda Function : Cuda Graphs launch\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);

    // Data cpu > gpu
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_AP,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    /* Wrap raw data into cuSPARSE generic API objects */
    matA = NULL;
    vecx = NULL;
    vecp = NULL;
    vecAP = NULL;
    checkCudaErrors(hipsparseCreateCsr(
        &matA, A_size, A_size, 5*A_size, dev_Ai, dev_Aj, dev_A, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecx, A_size, dev_X, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecp, A_size, dev_P, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecAP, A_size, dev_AP, HIP_R_32F));
}
void Field_Method4_Initial(){
    printf(" Field Solver : [GPU] Conjugate Gradient\n"); 
    printf(" Cuda Function : Multi Block\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Data cpu > gpu
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_AP,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
}
void Field_Method5_Initial(){
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Cuda Function : Multi Block\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Preconditioner[Jacovi]\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Data cpu > gpu
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_AP,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Z,  A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, A_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset((void *) dev_Z, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_P, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_AP, 0, A_size * sizeof(float)));
}
void Field_Method6_Initial(){
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Cuda Function : Multi GPU\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Preconditioner[Jacovi]\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Data cpu > gpu
    checkCudaErrors(hipMallocManaged((void **)&man_I, sizeof(int) * (A_size + 1)));
    checkCudaErrors(hipMallocManaged((void **)&man_J, sizeof(int) * 5 * A_size));
    checkCudaErrors(hipMallocManaged((void **)&man_A, sizeof(float) * 5 * A_size));
    checkCudaErrors(hipMemcpy(man_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(man_J, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(man_I, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemAdvise(man_I, sizeof(int) * (A_size + 1), hipMemAdviseSetReadMostly, 0));
    checkCudaErrors(hipMemAdvise(man_J, sizeof(int) * 5 * A_size, hipMemAdviseSetReadMostly, 0));
    checkCudaErrors(hipMemAdvise(man_A, sizeof(float) * 5 * A_size, hipMemAdviseSetReadMostly, 0));
    // temp memory for ConjugateGradient
    checkCudaErrors(hipMallocManaged((void **)&man_R, A_size * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&man_P, A_size * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&man_AP, A_size * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&man_X, A_size * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&man_Z, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) man_Z, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) man_P, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) man_AP, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) man_X, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) man_R, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMallocManaged((void **)&man_M, A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(man_M, MatM, A_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemAdvise(man_M, A_size * sizeof(float), hipMemAdviseSetReadMostly, 0));
}
void Field_Method7_Initial(){
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Cuda Function : Multi Block\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Preconditioner[IChol]\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Data cpu > gpu
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_AP,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Z,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Y,  A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, A_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset((void *) dev_Z, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_P, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_AP, 0, A_size * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_Y, 0, A_size * sizeof(float)));
}
void PCG_Laplace_TEST(){
    // Solve Laplace Equation. 
    // INPUT
    // Lap_Field_Solver_Flag 0 - [CPU] Conjugate Gradient 
    // Lap_Field_Solver_Flag 1 - [CPU] Preconditioned Conjugate Gradient 
    // Lap_Field_Solver_Flag 2 - [GPU] Conjugate Gradient 
    // Lap_Field_Solver_Flag 3 - [GPU] Conjugate Gradient + Cuda Graphs launch
    // Lap_Field_Solver_Flag 4 - [GPU] Conjugate Gradient + Multi Block
    // Lap_Field_Solver_Flag 5 - [GPU] [Jacovi] Preconditioned Conjugate Gradient + Multi Block
    // Lap_Field_Solver_Flag 6 - [GPU] [Jacovi] Preconditioned Conjugate Gradient + Multi GPU 
    // Lap_Field_Solver_Flag 7 - [GPU] [IChol or ILU] Preconditioned Conjugate Gradient + Multi Block 
    // OUTPUT
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    int i,j,k,TID; 
    int mingrid;
    int IIter;
    char Namebuf[256];
    float *buf;
    float **CPUsol;
    buf = VFMalloc(A_size);
    CPUsol = MFMalloc(CondNUMR,Gsize);
    vec_A_idx = (int *) malloc(ngx * ngy * sizeof(int));
    for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
		} 
	}
    // SPEED TEST
    hipEvent_t start, stop;
    float gputime;
    //
    if(Lap_Field_Solver_Flag == 0){// [CPU] Conjugate Gradient 
        Field_Method0_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {
            VFCopy(B,cond_b[k],A_size);
            VFInit(X,0.0,A_size);
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord( start, 0 );
            FieldIter = CG_CPU();
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d, time = %2.8f (ms)\n",FieldIter,gputime);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = X[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"CPU_CG");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Lap_Field_Solver_Flag == 1){// [CPU] Preconditioned Conjugate Gradient 
        Field_Method1_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {
            VFCopy(B,cond_b[k],A_size);
            VFInit(X,0.0,A_size);
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord( start, 0 );
            FieldIter = PCG_CPU();
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d, time = %2.8f (ms)\n",FieldIter,gputime);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = X[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"CPU_PCG");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Lap_Field_Solver_Flag == 2){// [GPU] Conjugate Gradient 
		Field_Method2_Initial(); // Initial Setting 
        for (k = 0; k < CondNUMR; k++) {
            checkCudaErrors(hipMemcpy(dev_R, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord( start, 0 );
            FieldIter = CG_GPU();
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d, time = %2.8f (ms)\n",FieldIter,gputime);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_CG");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Lap_Field_Solver_Flag == 3){// [GPU] Conjugate Gradient + Cuda Graphs launch
		Field_Method3_Initial(); // Initial Setting
        // stream
        for (k = 0; k < CondNUMR; k++) {
            // Cuda Handle setting
            checkCudaErrors(hipStreamCreate(&stream1));
            cublasHandle = 0;
            hipblasStatus_t = hipblasCreate(&cublasHandle);
            checkCudaErrors(hipblasStatus_t);
            cusparseHandle = 0;
            checkCudaErrors(hipsparseCreate(&cusparseHandle));
            cusparseStatus = hipsparseCreate(&cusparseHandle);
            checkCudaErrors(cusparseStatus);
            checkCudaErrors(hipMemcpy(dev_R, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord( start, 0 );
            FieldIter = CG_GPU_CudaGraphs();
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d, time = %2.8f (ms)\n",FieldIter,gputime);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpyAsync(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost, streamForGraph));
            checkCudaErrors(hipStreamSynchronize(streamForGraph));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
            checkCudaErrors(hipGraphExecDestroy(graphExec));
            checkCudaErrors(hipGraphDestroy(initGraph));
            checkCudaErrors(hipStreamDestroy(streamForGraph));
            checkCudaErrors(hipStreamDestroy(stream1));
            checkCudaErrors(hipsparseDestroy(cusparseHandle));
            checkCudaErrors(hipblasDestroy(cublasHandle));
        }
        sprintf(Namebuf,"GPU_CG_Graph");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);        
    }else if(Lap_Field_Solver_Flag == 4){// [GPU] Conjugate Gradient + Multi Block
        Field_Method4_Initial(); // Initial Setting
        //Make a Field constant set  
        Host_PCG_const = (DPS_Const*)malloc(sizeof(DPS_Const));
        checkCudaErrors(hipMalloc((void**)&dev_PCG_const,sizeof(DPS_Const)));
        Make_PCG_Const_Init<<<1,1>>>(dev_PCG_const,A_size,PCGtol);
        checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
        //checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));

        hipDeviceProp_t deviceProp;
        int sMemSize = sizeof(double) * THREADS_PER_BLOCK;
        int numBlocksPerSm = 0;
        int numThreads = THREADS_PER_BLOCK;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, device_num));
        if (!deviceProp.managedMemory) {
            // This sample requires being run on a device that supports Unified Memory
            fprintf(stderr, "Unified Memory not supported on this device\n");
            exit(EXIT_WAIVED);
        }
        // This sample requires being run on a device that supports Cooperative Kernel Launch
        if (!deviceProp.cooperativeLaunch)
        {
            printf("\nSelected GPU (%d) does not support Cooperative Kernel Launch, Waiving the run\n", device_num);
            exit(EXIT_WAIVED);
        }
        // Statistics about the GPU device
        printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);
        
        checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, gpuConjugateGradient, numThreads, sMemSize));
        int numSms = deviceProp.multiProcessorCount;
        dim3 dimGrid(numSms*numBlocksPerSm, 1, 1), dimBlock(THREADS_PER_BLOCK, 1, 1);
        float nz = 5*A_size;
        //
        double *dot_result;
        hipMallocManaged((void **)&dot_result, sizeof(double));
        *dot_result = 0.0;
        //
        checkCudaErrors(hipMalloc((void**)&dot_result,sizeof(float)));
        void *kernelArgs[] = {
            (void*)&dev_Ai,
            (void*)&dev_Aj,
            (void*)&dev_A,
            (void*)&dev_X,
            (void*)&dev_AP,
            (void*)&dev_P,
            (void*)&dev_R,
            (void*)&dev_PCG_const,
            (void*)&dot_result,
        };
        for (k = 0; k < CondNUMR; k++) {
            checkCudaErrors(hipMemcpy(dev_PCG_const, Host_PCG_const,sizeof(DPS_Const), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dev_b, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dev_R, dev_b, A_size * sizeof(float),hipMemcpyDeviceToDevice));
            checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) dev_AP, 0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) dev_P, 0, A_size * sizeof(float)));
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord( start, 0 );
            checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuConjugateGradient, dimGrid, dimBlock, kernelArgs, sMemSize, NULL));
            checkCudaErrors(hipDeviceSynchronize());
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf(" time = %2.8f (ms)\n",gputime);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_CG_MultiBlock");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Lap_Field_Solver_Flag == 5){// [GPU] [Jacovi] Preconditioned Conjugate Gradient + Multi Block
		Field_Method5_Initial(); // Initial Setting
        //Make a Field constant set  
        Host_PCG_const = (DPS_Const*)malloc(sizeof(DPS_Const));
        checkCudaErrors(hipMalloc((void**)&dev_PCG_const,sizeof(DPS_Const)));
        Make_PCG_Const_Init<<<1,1>>>(dev_PCG_const,A_size,PCGtol);
        checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
        //checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));

        hipDeviceProp_t deviceProp;
        int sMemSize = sizeof(double) * THREADS_PER_BLOCK;
        int numBlocksPerSm = 0;
        int numThreads = THREADS_PER_BLOCK;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, device_num));
        if (!deviceProp.managedMemory) {
            // This sample requires being run on a device that supports Unified Memory
            fprintf(stderr, "Unified Memory not supported on this device\n");
            exit(EXIT_WAIVED);
        }
        // This sample requires being run on a device that supports Cooperative Kernel Launch
        if (!deviceProp.cooperativeLaunch)
        {
            printf("\nSelected GPU (%d) does not support Cooperative Kernel Launch, Waiving the run\n", device_num);
            exit(EXIT_WAIVED);
        }
        // Statistics about the GPU device
        printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);
        
        checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, gpuPreConjugateGradient, numThreads, sMemSize));
        int numSms = deviceProp.multiProcessorCount;
        dim3 dimGrid(numSms*numBlocksPerSm, 1, 1), dimBlock(THREADS_PER_BLOCK, 1, 1);
        float nz = 5*A_size;
        //
        double *dot_result;
        hipMallocManaged((void **)&dot_result, sizeof(double));
        *dot_result = 0.0;
        //
        void *kernelArgs[] = {
            (void*)&dev_Ai,
            (void*)&dev_Aj,
            (void*)&dev_A,
            (void*)&dev_M,
            (void*)&dev_X,
            (void*)&dev_AP,
            (void*)&dev_P,
            (void*)&dev_R,
            (void*)&dev_Z,
            (void*)&dev_PCG_const,
            (void*)&dot_result,
        };
        for (k = 0; k < CondNUMR; k++) {
            checkCudaErrors(hipMemcpy(dev_PCG_const, Host_PCG_const,sizeof(DPS_Const), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dev_b, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dev_R, dev_b, A_size * sizeof(float),hipMemcpyDeviceToDevice));
            checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) dev_AP, 0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) dev_P, 0, A_size * sizeof(float)));
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord( start, 0 );
            checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuPreConjugateGradient, dimGrid, dimBlock, kernelArgs, sMemSize, NULL));
            checkCudaErrors(hipDeviceSynchronize());
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf(" time = %2.8f (ms)\n",gputime);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_PCG_Jacobi_MB");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Lap_Field_Solver_Flag == 6){// [GPU] [Jacovi] Preconditioned Conjugate Gradient + Multi GPU
		Field_Method6_Initial(); // Initial Setting
        //Make a Field constant set  
        Host_PCG_const = (DPS_Const*)malloc(sizeof(DPS_Const));
        checkCudaErrors(hipMalloc((void**)&dev_PCG_const,sizeof(DPS_Const)));
        Make_PCG_Const_Init<<<1,1>>>(dev_PCG_const,A_size,PCGtol);
        checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
        //checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));

        hipDeviceProp_t deviceProp;
        int num_of_gpus = 0;
        int num_buf = device_num;
        GPUn = 2; 
        deviceN = VIMalloc(GPUn);
        checkCudaErrors(hipGetDeviceCount(&num_of_gpus));
        if (num_of_gpus <= 1 || num_of_gpus < device_num + GPUn) {
            printf("No. of GPU on node %d\n", num_of_gpus);
            printf("Minimum Two or more GPUs are required to run this code\n");
            exit(EXIT_WAIVED);
        }
        printf("Using GPU list : %d\n",GPUn);
        for(i=0;i<GPUn;i++){
            deviceN[i] = num_buf;
            num_buf++;
            hipGetDeviceProperties(&deviceProp, deviceN[i]); 
            printf("Name %d : %s \n",deviceN[i], deviceProp.name);
            if (!deviceProp.managedMemory) {
            // This sample requires being run on a device that supports Unified Memory
                fprintf(stderr, "Unified Memory not supported on this device\n");
                exit(EXIT_WAIVED);
            }
            // This sample requires being run on a device that supports Cooperative Kernel Launch
            if (!deviceProp.cooperativeLaunch)
            {
                printf("\nSelected GPU (%d) does not support Cooperative Kernel Launch, Waiving the run\n", device_num);
                exit(EXIT_WAIVED);
            }
        }
        //
        double *dot_result;
        hipMallocManaged((void **)&dot_result, sizeof(double));
        checkCudaErrors(hipMemset(dot_result, 0.0, sizeof(double)));
        //
        hipStream_t *nStreams = (hipStream_t *)malloc(GPUn * sizeof(hipStream_t));
        int NNZ = 5*A_size;
        void *kernelArgs[] = {
            (void*)&man_I,
            (void*)&man_J,
            (void*)&man_A,
            (void*)&man_M,
            (void*)&man_X,
            (void*)&man_AP,
            (void*)&man_P,
            (void*)&man_R,
            (void*)&man_Z,
            (void*)&NNZ,
            (void*)&A_size,
            (void*)&PCGtol,
            (void*)&dot_result,
        };
        int sMemSize = sizeof(double) * THREADS_PER_BLOCK;
        int numBlocksPerSm = 0;
        int numThreads = THREADS_PER_BLOCK;
        num_buf = device_num;
        checkCudaErrors(hipSetDevice(num_buf));
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, num_buf));                    
        checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &numBlocksPerSm, multiGpuPreConjugateGradient, numThreads, sMemSize));
        int numSms = deviceProp.multiProcessorCount;
        dim3 dimGrid(numSms * numBlocksPerSm, 1, 1), dimBlock(THREADS_PER_BLOCK, 1, 1);
        
        int device_count = 0;
        int totalThreadsPerGPU = numSms * numBlocksPerSm * THREADS_PER_BLOCK;
        num_buf = device_num;
        // Data Divide
        for(device_count = 0;device_count<GPUn;device_count++){
            num_buf = deviceN[device_count];
            checkCudaErrors(hipSetDevice(num_buf));
            checkCudaErrors(hipGetDeviceProperties(&deviceProp, num_buf));
            checkCudaErrors(hipStreamCreate(&nStreams[device_count]));
            if (deviceProp.concurrentManagedAccess) {
                int perGPUIter = A_size / (totalThreadsPerGPU * GPUn);
                int offset_Ax = device_count * totalThreadsPerGPU;
                int offset_r = device_count * totalThreadsPerGPU;
                int offset_p = device_count * totalThreadsPerGPU;
                int offset_x = device_count * totalThreadsPerGPU;
                checkCudaErrors(hipMemPrefetchAsync(man_I, sizeof(int) * (A_size+1), num_buf,nStreams[device_count]));
                checkCudaErrors(hipMemPrefetchAsync(man_A, sizeof(float) * 5*A_size, num_buf,nStreams[device_count]));
                checkCudaErrors(hipMemPrefetchAsync(man_J, sizeof(int) * 5*A_size, num_buf,nStreams[device_count]));
                if (offset_Ax <= A_size) {
                    for (i = 0; i < perGPUIter; i++) {
                        hipMemAdvise(man_AP + offset_Ax, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetPreferredLocation, num_buf);
                        hipMemAdvise(man_R + offset_r, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetPreferredLocation, num_buf);
                        hipMemAdvise(man_X + offset_x, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetPreferredLocation, num_buf);
                        hipMemAdvise(man_P + offset_p, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetPreferredLocation, num_buf);
                        hipMemAdvise(man_AP + offset_Ax, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetAccessedBy, num_buf);
                        hipMemAdvise(man_R + offset_r, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetAccessedBy, num_buf);
                        hipMemAdvise(man_P + offset_p, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetAccessedBy, num_buf);
                        hipMemAdvise(man_X + offset_x, sizeof(float) * totalThreadsPerGPU,hipMemAdviseSetAccessedBy, num_buf);
                        offset_Ax += totalThreadsPerGPU * GPUn;
                        offset_r += totalThreadsPerGPU * GPUn;
                        offset_p += totalThreadsPerGPU * GPUn;
                        offset_x += totalThreadsPerGPU * GPUn;
                        if (offset_Ax >= A_size) {
                            break;
                        }
                    }
                }
            }
        }
        printf("Total threads per GPU = %d numBlocksPerSm  = %d\n",numSms * numBlocksPerSm * THREADS_PER_BLOCK, numBlocksPerSm);
        launchParamsList = (hipLaunchParams *)malloc(GPUn * sizeof(hipLaunchParams));
        for (i = 0; i < GPUn; i++) {
            launchParamsList[i].func = (void *)multiGpuPreConjugateGradient;
            launchParamsList[i].gridDim = dimGrid;
            launchParamsList[i].blockDim = dimBlock;
            launchParamsList[i].sharedMem = sMemSize;
            launchParamsList[i].stream = nStreams[i];
            launchParamsList[i].args = kernelArgs;
        }
        for (k = 0; k < CondNUMR; k++) {
            checkCudaErrors(hipMemcpy(dev_PCG_const, Host_PCG_const,sizeof(DPS_Const), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(man_R, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset((void *) man_X, 0.0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) man_AP, 0.0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) man_P, 0.0, A_size * sizeof(float)));
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord(start,0);
            checkCudaErrors(hipLaunchCooperativeKernelMultiDevice(
                launchParamsList, GPUn,
                hipCooperativeLaunchMultiDeviceNoPreSync |
                hipCooperativeLaunchMultiDeviceNoPostSync));
            checkCudaErrors(hipMemPrefetchAsync(man_X, sizeof(float) * A_size, hipCpuDeviceId));
            checkCudaErrors(hipMemPrefetchAsync(dot_result, sizeof(double), hipCpuDeviceId));
            for(device_count = 0;device_count<GPUn;device_count++){
                num_buf = deviceN[device_count];
                checkCudaErrors(hipSetDevice(num_buf));
                checkCudaErrors(hipStreamSynchronize(nStreams[device_count]));
            }
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf(" time = %2.8f (ms)",gputime);
            float r1 = *dot_result;
            printf(", residual = %e \n", r1);

            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = man_X[vec_A_idx[TID]-1];
                        //if(CPUsol[k][TID]!=0) printf("CHECK %g \n  ", CPUsol[k][TID]);
                    }
                }
            }

        }
        sprintf(Namebuf,"GPU_PCG_MultiGPU");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Lap_Field_Solver_Flag == 7){// [GPU] [IChol] Preconditioned Conjugate Gradient + Multi Block
		Field_Method7_Initial(); // Initial Setting
        //Make a Field constant set  
        Host_PCG_const = (DPS_Const*)malloc(sizeof(DPS_Const));
        checkCudaErrors(hipMalloc((void**)&dev_PCG_const,sizeof(DPS_Const)));
        Make_PCG_Const_Init<<<1,1>>>(dev_PCG_const,A_size,PCGtol);
        checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
        //checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));

        hipDeviceProp_t deviceProp;
        int sMemSize = sizeof(double) * THREADS_PER_BLOCK;
        int numBlocksPerSm = 0;
        int numThreads = THREADS_PER_BLOCK;
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, device_num));
        if (!deviceProp.managedMemory) {
            // This sample requires being run on a device that supports Unified Memory
            fprintf(stderr, "Unified Memory not supported on this device\n");
            exit(EXIT_WAIVED);
        }
        // This sample requires being run on a device that supports Cooperative Kernel Launch
        if (!deviceProp.cooperativeLaunch)
        {
            printf("\nSelected GPU (%d) does not support Cooperative Kernel Launch, Waiving the run\n", device_num);
            exit(EXIT_WAIVED);
        }
        // Statistics about the GPU device
        printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);
        
        checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, gpuPreConjugateGradient, numThreads, sMemSize));
        int numSms = deviceProp.multiProcessorCount;
        dim3 dimGrid(numSms*numBlocksPerSm, 1, 1), dimBlock(THREADS_PER_BLOCK, 1, 1);
        float nz = 5*A_size;
        //
        double *dot_result;
        hipMallocManaged((void **)&dot_result, sizeof(double));
        *dot_result = 0.0;
        // Make a Preconditioner matrix
        checkCudaErrors(hipMalloc((void**) &dev_L, 5 * A_size * sizeof(float)));
	    checkCudaErrors(hipMalloc((void**) &dev_Lj, 5 * A_size * sizeof(int)));
	    checkCudaErrors(hipMalloc((void**) &dev_Li, (A_size + 1) * sizeof(int)));
        checkCudaErrors(hipMemset((void *) dev_L, 0, 5 * A_size * sizeof(float)));
        checkCudaErrors(hipMemset((void *) dev_Lj, 0, 5 * A_size * sizeof(int)));
        checkCudaErrors(hipMemset((void *) dev_Li, 0, (A_size + 1) * sizeof(int)));
        checkCudaErrors(hipMalloc((void**) &dev_U, 5 * A_size * sizeof(float)));
	    checkCudaErrors(hipMalloc((void**) &dev_Uj, 5 * A_size * sizeof(int)));
	    checkCudaErrors(hipMalloc((void**) &dev_Ui, (A_size + 1) * sizeof(int)));
        checkCudaErrors(hipMemset((void *) dev_U, 0, 5 * A_size * sizeof(float)));
        checkCudaErrors(hipMemset((void *) dev_Uj, 0, 5 * A_size * sizeof(int)));
        checkCudaErrors(hipMemset((void *) dev_Ui, 0, (A_size + 1) * sizeof(int)));
        if(Preconditioner_Flag==0){
            // Incomplete Cholesky Preconditioner
            printf("Make a preconditioner : [I Cholesky]\n");
            float *L_val;
            int *Li,Lj;
            int row_elem,next_row_elem;
            L_val = VFMalloc(5*A_size);VFInit(L_val,0.0,5*A_size);
            Li = VIMalloc(A_size+1);VIInit(Li,0,A_size+1);
            Lj = VIMalloc(5*A_size);VIInit(Lj,0,5*A_size);
            k = 0;
            for (i=0; i < A_size; i++){ //ROW
                row_elem = Ai[i];
                next_row_elem = Ai[i+1];
                for (j=row_elem-1; j < next_row_elem-1; j++){ // Column
                    if(A_val[j] != 0){
                        if(j=i){
                            L_val[k] = sqrt(A_val[j]);
                            k++;
                        }else if(j<i){
                            L_val[k] = sqrt(A_val[j]); // start
                            k++;
                        }
                    }
                }
            }
 
	        exit(1);
        }else if(Preconditioner_Flag==1){
            // Incomplete LU Preconditioner
            printf("Make a preconditioner : [I LU]\n");

        }else{
            //Jacovi

        }
        printf("Complete!!\n");
        //
        void *kernelArgs[] = {
            (void*)&dev_Ai,
            (void*)&dev_Aj,
            (void*)&dev_A,
            (void*)&dev_M,
            (void*)&dev_X,
            (void*)&dev_AP,
            (void*)&dev_P,
            (void*)&dev_R,
            (void*)&dev_Z,
            (void*)&dev_PCG_const,
            (void*)&dot_result,
        };
        for (k = 0; k < CondNUMR; k++) {
            checkCudaErrors(hipMemcpy(dev_PCG_const, Host_PCG_const,sizeof(DPS_Const), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dev_b, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemcpy(dev_R, dev_b, A_size * sizeof(float),hipMemcpyDeviceToDevice));
            checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) dev_AP, 0, A_size * sizeof(float)));
            checkCudaErrors(hipMemset((void *) dev_P, 0, A_size * sizeof(float)));
            hipEventCreate(&start); hipEventCreate(&stop);
	        hipEventRecord( start, 0 );
            checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuPreConjugateGradient, dimGrid, dimBlock, kernelArgs, sMemSize, NULL));
            checkCudaErrors(hipDeviceSynchronize());
            hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	        hipEventElapsedTime( &gputime, start, stop );
	        hipEventDestroy( start );hipEventDestroy( stop );
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf(" time = %2.8f (ms)\n",gputime);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_PCG_IChol_MB");
        if(Lap_Field_Solver_Save) Field_Laplace_Solution_Save(Namebuf,CPUsol);
    
    }else if(Lap_Field_Solver_Flag >= 9){
        printf("Empty Test Laplace Field Solver\n");
        exit(1);
    }
}
int CG_CPU(){
    int TID,i,Iter=0;
    float tol2;
    float rsold,rnew,Temp;
    float alpha,beta;

    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        P0[TID] = R0[TID];
        rsold += R0[TID]*R0[TID]; //AtomicAdd!!
    }
    printf("First:result[0].rsold = %g\n",rsold);
    tol2 = PCGtol*PCGtol;
    while(rsold>tol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
            }
            //if(P0[TID]!=0) printf("p[%d] = %g\n",TID,P0[TID]);
            //printf("AP[%d] = %g\n",TID,P0[TID]);
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //AtomicAdd!!
            
        }
        //printf("Temp = %g\n",Temp);
        //exit(1);
        alpha = (Temp)? rsold/Temp:0.0f ;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            rnew += R0[TID]*R0[TID];  //AtomicAdd!!
        }
        beta = (rsold) ? rnew/rsold: 0.0f;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = R0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
        //if(Iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",Iter,Temp,alpha,beta,rsold);
    }
    return Iter;
}
int PCG_CPU(){
    int TID,i,Iter=0;
    float tol2;
    float rsold,rnew,Temp;
    float alpha,beta;

    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        Z0[TID] = MatM[TID] * R0[TID];
        P0[TID] = Z0[TID];
        rsold += R0[TID]*Z0[TID]; //AtomicAdd!!
    }
    while(rsold>PCGtol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
            }
            //printf("AP[%d] = %g\n",TID,P0[TID]);
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //AtomicAdd!!
        }
        alpha = (Temp)? rsold/Temp:0.0f ;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            Z0[TID] = MatM[TID] * R0[TID];
            rnew += R0[TID]*Z0[TID];  //AtomicAdd!!
        }
        beta = (rsold) ? rnew/rsold: 0.0f;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = Z0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
        //if(Iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",Iter,Temp,alpha,beta,rsold);
    }
    return Iter;
}
int CG_GPU(){
    int iter;
    int max_iter = 1000000;
    float a = 1.0;
    float b = 0.0;
    float na = -1.0;
    float rnew = 0.;
    float rsold,Temp;
    float alpha,beta;
    float nalpha;
    /* Allocate workspace for cuSPARSE */
    size_t bufferSize = 0;
    checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, &bufferSize));
    void *buffer = NULL;
    checkCudaErrors(hipMalloc(&buffer, bufferSize));
    /* Begin CG */
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, buffer));
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, &na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, &rsold));
    iter = 1;
    while (rsold > PCGtol2 && iter <= max_iter){
         if (iter > 1)
        {
            beta = rnew / rsold;
            rsold = rnew;
            hipblasSscal(cublasHandle, A_size, &beta, dev_P, 1);
            hipblasSaxpy(cublasHandle, A_size, &a, dev_R, 1, dev_P, 1);
        }
        else
        {
            hipblasScopy(cublasHandle, A_size, dev_R, 1, dev_P, 1);
        }
        checkCudaErrors(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA,
            vecp, &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, buffer));
        hipblasSdot(cublasHandle, A_size, dev_P, 1, dev_AP, 1, &Temp);
        alpha = (Temp) ? rsold / Temp: 0.0f;
        hipblasSaxpy(cublasHandle, A_size, &alpha, dev_P, 1, dev_X, 1);
        nalpha = -alpha;
        hipblasSaxpy(cublasHandle, A_size, &nalpha, dev_AP, 1, dev_R, 1);
        hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, &rnew);
        hipDeviceSynchronize();
        //if(iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",iter,Temp,a,b,rsold);
        iter++;
    }
    return iter;

}
int CG_GPU_CudaGraphs(){
    int iter;
    int max_iter = 1000000;
    static int init_Flag = 0;
    float a = 1.0;
    float b = 0.0;
    float na = -1.0;
    float r1;
    float *d_r1, *d_r0, *d_dot, *d_a, *d_na, *d_b;
    // SPEED TEST
    hipEvent_t start, stop;
    float gputime;
    //
    /* Allocate workspace for cuSPARSE */
    size_t bufferSize = 0;
    checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, &bufferSize));
    void *buffer = NULL;
    checkCudaErrors(hipMalloc(&buffer, bufferSize));
    // DEVICE variable
    checkCudaErrors(hipMalloc((void **)&d_r1, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r0, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_dot, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_a, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_na, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(float)));
    //
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, &na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, d_r1));

    iter = 1;
    // First Iteration when iter=1 starts
    checkCudaErrors(hipblasScopy(cublasHandle, A_size, dev_R, 1, dev_P, 1));
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecp,
        &b, vecAP, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_P, 1, dev_AP, 1, d_dot));
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_a, dev_P, 1, dev_X, 1));
    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),hipMemcpyDeviceToDevice, stream1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, d_r1));
    checkCudaErrors(hipMemcpyAsync(&r1, d_r1, sizeof(float),hipMemcpyDeviceToHost, stream1));
    checkCudaErrors(hipStreamSynchronize(stream1));
    printf("iteration = %3d, residual = %e\n", iter, sqrt(r1));
    // First Iteration when k=1 ends
    iter++;
    checkCudaErrors(hipStreamCreate(&streamForGraph));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
   // if(init_Flag == 0){
    // Capture start
    checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    checkCudaErrors(hipblasSscal(cublasHandle, A_size, d_b, dev_P, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, &a, dev_R, 1, dev_P, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    checkCudaErrors(hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST));
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecp,
        &b, vecAP, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
    checkCudaErrors(hipMemsetAsync(d_dot, 0, sizeof(float), stream1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_P, 1, dev_AP, 1, d_dot));
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_a, dev_P, 1, dev_X, 1));
    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),hipMemcpyDeviceToDevice, stream1));
    checkCudaErrors(hipMemsetAsync(d_r1, 0, sizeof(float), stream1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, d_r1));
    checkCudaErrors(hipMemcpyAsync((float *)&r1, d_r1, sizeof(float),hipMemcpyDeviceToHost, stream1));
    checkCudaErrors(hipStreamEndCapture(stream1, &initGraph));
    // Capture End
    checkCudaErrors(hipGraphInstantiate(&graphExec, initGraph, NULL, NULL, 0));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
        init_Flag++;
    //}

    hipEventCreate(&start); hipEventCreate(&stop);
	hipEventRecord( start, 0 );
    while (r1 > PCGtol2 && iter <= max_iter) {
        checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
        checkCudaErrors(hipStreamSynchronize(streamForGraph));
        //printf("iteration = %3d, residual = %e\n", iter, sqrt(r1));
    iter++;
    }
    hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
    hipEventElapsedTime( &gputime, start, stop );
	hipEventDestroy( start );hipEventDestroy( stop );
    printf("time = %2.8f (ms)\n",gputime);

    return iter;
}
__global__ void PCG_LAP(float *A,int *Ai,int *Aj,DPS_Const *PCG_C,DPS_Data *PCG_D,float *X,float *b){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=PCG_C[0].A_size) return;
    int i;
    float sum;
    int MAXITER = 20;
    //Initial
    if(TID==0){
        PCG_C[0].Iter = 0;
        PCG_C[0].rsold = 0;
        PCG_C[0].Temp = 0;
        PCG_C[0].rnew = 0;
    } 
    __syncthreads();
    // cal  AP = A * P
    for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
        PCG_D[TID].AP += A[i] * X[Aj[i]-1];
    }
    PCG_D[TID].R = b[TID] - PCG_D[TID].AP;
    PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
    PCG_D[TID].P = PCG_D[TID].Z;
    sum = PCG_D[TID].R*PCG_D[TID].Z;
    atomicAdd(&PCG_C[0].rsold,sum);
    //if(TID==0) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
    //if(TID==10) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
    //if(TID==0) printf(" [%d]Initial rsold = %g, tol2 = %g\n",TID,PCG_C[0].rsold,PCG_C[0].tol2);
    while(PCG_C[0].rsold > PCG_C[0].tol2){
        if(TID==0){
            PCG_C[0].Iter++;
            PCG_C[0].Temp = 0;
            PCG_C[0].rnew = 0;
        } 
        __syncthreads();
        //if(TID==0) printf(" [%d]Iter %d start!\n",TID,PCG_C[0].Iter);
        PCG_D[TID].AP = 0;
        for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
            PCG_D[TID].AP += A[i] * PCG_D[Aj[i]-1].P;
        }
        //printf("[%d] AP = %g\n",TID,PCG_D[TID].P);
        sum = PCG_D[TID].P * PCG_D[TID].AP;
        atomicAdd(&PCG_C[0].Temp,sum);
        //if(TID==0) printf(" [%d]Temp = %g\n",TID,PCG_C[0].Temp);       
        if(PCG_C[0].Iter>MAXITER){
            //if(TID==0) printf("Iteration[%d] = %g\n",TID,PCG_C[0].Iter);    
            break;
        } 
        if(TID==0){
            PCG_C[0].alpha = (PCG_C[0].Temp)? PCG_C[0].rsold/PCG_C[0].Temp : 0.0f; 
        } 
        __syncthreads();     
        X[TID] = X[TID] + PCG_C[0].alpha * PCG_D[TID].P;
        PCG_D[TID].R = PCG_D[TID].R - PCG_C[0].alpha * PCG_D[TID].AP;
        PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
        sum = PCG_D[TID].R * PCG_D[TID].Z;
        atomicAdd(&PCG_C[0].rnew,sum);
       //if(TID==0) printf(" [%d]rnew = %g\n",TID,PCG_C[0].rnew);     
        if(TID==0){
             PCG_C[0].beta = (PCG_C[0].rsold)? PCG_C[0].rnew/PCG_C[0].rsold : 0.0f;
        }
        __syncthreads();
        PCG_D[TID].P = PCG_D[TID].Z + PCG_C[0].beta*PCG_D[TID].P;    
        if(TID==0) PCG_C[0].rsold = PCG_C[0].rnew;
        __syncthreads();
        if(TID==0) printf(" [%d]Iter %d,Temp = %g,alpha = %g,beat = %g, rsold = %g\n",TID,PCG_C[0].Iter,PCG_C[0].Temp,PCG_C[0].alpha,PCG_C[0].beta,PCG_C[0].rsold);
    }
    if(TID==0) printf("Iter [%d]= %d, ",TID,PCG_C[0].Iter);  
    if(TID==0) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
}
__global__ void Make_PCG_DATA_Init(DPS_Data *p, int size,float *MatrixM){
    int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if(TID>=size) return;
    //printf("TID = %d, M = %g\n",TID,MatrixM[TID]);
    p[TID].R = 0.0;
    p[TID].Z = 0.0;
    p[TID].P = 0.0;
    p[TID].AP = 0.0;
    p[TID].M = MatrixM[TID];
}
__global__ void Make_PCG_Const_Init(DPS_Const *p,int Asize, float tol){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].A_size = Asize;
    p[i].Iter = 0;
    p[i].tol = tol;
    p[i].tol2 = tol*tol;
    p[i].rsold = 0.0;
    p[i].Temp = 0.0;
    p[i].rnew = 0.0;
    p[i].alpha = 0.0;
    p[i].beta = 0.0;    
}
__global__ void initVectors(float *rhs, float *x, int N) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  for (size_t i = gid; i < N; i += gridDim.x * blockDim.x) {
    rhs[i] = 1.0;
    x[i] = 0.0;
  }
}
__global__ void r1_div_x(float *r1, float *r0, float *b) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    b[0] = r1[0] / r0[0];
  }
}
__global__ void a_minus(float *a, float *na) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    na[0] = -(a[0]);
  }
}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}
__device__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha, float *inputVecX, 
                        float *outputVecY, cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < num_rows; i+= grid.size())    {
        // i = 0 ~ A_size-1; 
        //printf("val[%d][]\n",i);
        //for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
        //    AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        //}
        int row_elem = I[i];
        int next_row_elem = I[i+1];
        int num_elems_this_row = next_row_elem - row_elem;
        float output = 0.0;
        for (int j=row_elem-1; j < next_row_elem-1; j++){
            //if(i==0) printf("val[%d][]\n",j);
            // I or J or val arrays - can be put in shared memory 
            // as the access is random and reused in next calls of gpuSpMV function.
            output +=  alpha*val[j] * inputVecX[J[j]-1];
            //if(i==0) printf("val[%d][%d] = %g, %g, %g\n",j,J[j]-1,val[j],inputVecX[J[j]-1],output);
        }
        outputVecY[i] = output;
    }
}
__device__ void gpuSaxpy(float *x, float *y, float a, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()){        
        y[i] = a*x[i] + y[i];
    }
}
__device__ void gpuRSaxpy(float *x, float *y, float a, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()){        
        y[i] = a*y[i] + x[i];
    }
}
__device__ void gpuDotProduct(float *vecA, float *vecB, double *result, int size, const cg::thread_block &cta, const cg::grid_group &grid)
{
   __shared__ double tmp[THREADS_PER_BLOCK];
    double temp_sum = 0.0;
    for (int i=grid.thread_rank(); i < size; i+=grid.size()){
        temp_sum += (double) (vecA[i] * vecB[i]);
    }
    tmp[cta.thread_rank()] = temp_sum;
    cg::sync(cta);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
    double beta  = temp_sum;
    double temp;
    for (int i = tile32.size() / 2; i > 0; i >>= 1) {
        if (tile32.thread_rank() < i) {
            temp       = tmp[cta.thread_rank() + i];
            beta       += temp;
            tmp[cta.thread_rank()] = beta;
        }
        cg::sync(tile32);
    }
    cg::sync(cta);
    if (cta.thread_rank() == 0) {
        beta  = 0.0;
        for (int i = 0; i < cta.size(); i += tile32.size()) {
            beta  += tmp[i];
        }
        atomicAdd(result, beta);
    }
}
__device__ void gpuCopyVector(float *srcA, float *destB, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()){
        destB[i] = srcA[i];
    }
}
__device__ void gpuScaleVector(float *vec, float alpha, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()){
        vec[i] = alpha*vec[i];
    }
}
__global__ void gpuConjugateGradient(int *I, int *J, float *val, float *x,  float *Ax, float *p, float *r, 
            DPS_Const *result,double *d_result)
{
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    //int TID = blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    int k = 0;
    int max_iter = 10000;
    float a = 1.0;
    float na = -1.0;
    int nnz = 5 * result[0].A_size;
    int N = result[0].A_size;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;

    rsold = 0.0;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *d_result = 0.0;  
    } 
    gpuSpMV(I, J, val, nnz, N, a, x, Ax, cta, grid); 
    gpuSaxpy(Ax, r, na, N, grid); 
    gpuCopyVector(r, p, N, grid);
    //if(r[TID] !=0) printf("r[%d] = %g\n",TID,r[TID]);
    cg::sync(grid);
    gpuDotProduct(r, r, d_result, N, cta, grid); 
    cg::sync(grid);
    rsold = *d_result;
    //if(threadIdx.x == 0 && blockIdx.x == 0) printf("First:result[0].rsold = %g\n",rsold);
    //return;
    while (rsold > result[0].tol2 && k <= max_iter){
        k++;
        gpuSpMV(I, J, val, nnz, N, a, p, Ax, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        //if(Ax[TID] !=0) printf("Ax[%d] = %g\n",TID,Ax[TID]);
        gpuDotProduct(p, Ax, d_result, N, cta, grid);
        cg::sync(grid);
        Temp = *d_result;
        //if(threadIdx.x == 0 && blockIdx.x == 0) printf("Temp = %g\n",Temp);
        //return;
        alpha = (Temp)? rsold/Temp:0.0f;
        gpuSaxpy(p, x, alpha, N, grid);
        nalpha = -alpha;
        gpuSaxpy(Ax, r, nalpha, N, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        gpuDotProduct(r, r, d_result, N, cta, grid);
        cg::sync(grid);
        rnew = *d_result;
        beta = (rsold) ? rnew/rsold: 0.0f;
        gpuRSaxpy(r, p, beta, N, grid);
        rsold = rnew;
        rnew = 0.0;
        //if(threadIdx.x == 0 && blockIdx.x == 0 && k<20) printf("Iter = %d, temp = %g,  AL = %g, BE = %g Res = %g\n",k,Temp,alpha,beta,rsold);
    }
    if(threadIdx.x == 0 && blockIdx.x == 0 ) printf("End Iter = %d, Res = %g, b = %g, a = %g\n",k,Temp,alpha,beta,rsold);
}
__device__ void gpuProductVector(float *vecA, float *vecB, float *vecC, int size, const cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+=grid.size()){
        vecC[i] = (vecA[i] * vecB[i]);
    }
}
__global__ void gpuPreConjugateGradient(int *I, int *J, float *val, float *M, float *x,  float *Ax, float *p, float *r, float *Z, 
            DPS_Const *result,double *d_result){
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    //int TID = blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    int k = 0;
    int max_iter = 100000;
    float a = 1.0;
    float na = -1.0;
    int nnz = 5 * result[0].A_size;
    int N = result[0].A_size;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;

    rsold = 0.0;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *d_result = 0.0;  
    } 
    gpuSpMV(I, J, val, nnz, N, a, x, Ax, cta, grid); 
    gpuSaxpy(Ax, r, na, N, grid); 
    gpuProductVector(M, r, Z, N, cta, grid);
    gpuCopyVector(Z, p, N, grid);
    //if(r[TID] !=0) printf("r[%d] = %g\n",TID,r[TID]);
    cg::sync(grid);
    gpuDotProduct(r, Z, d_result, N, cta, grid); 
    cg::sync(grid);
    rsold = *d_result;
    //if(threadIdx.x == 0 && blockIdx.x == 0) printf("First:result[0].rsold = %g\n",rsold);
    //return;
    while (rsold > result[0].tol2 && k <= max_iter){
        k++;
        gpuSpMV(I, J, val, nnz, N, a, p, Ax, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        //if(Ax[TID] !=0) printf("Ax[%d] = %g\n",TID,Ax[TID]);
        gpuDotProduct(p, Ax, d_result, N, cta, grid);
        cg::sync(grid);
        Temp = *d_result;
        //if(threadIdx.x == 0 && blockIdx.x == 0) printf("Temp = %g\n",Temp);
        //return;
        alpha = (Temp)? rsold/Temp:0.0f;
        gpuSaxpy(p, x, alpha, N, grid);
        nalpha = -alpha;
        gpuSaxpy(Ax, r, nalpha, N, grid);
        gpuProductVector(M, r, Z, N, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        gpuDotProduct(r, Z, d_result, N, cta, grid);
        cg::sync(grid);
        rnew = *d_result;
        beta = (rsold) ? rnew/rsold: 0.0f;
        gpuRSaxpy(Z, p, beta, N, grid);
        rsold = rnew;
        rnew = 0.0;
        //if(threadIdx.x == 0 && blockIdx.x == 0 && k<20) printf("Iter = %d, temp = %g,  AL = %g, BE = %g Res = %g\n",k,Temp,alpha,beta,rsold);
    }
    if(threadIdx.x == 0 && blockIdx.x == 0 ) printf("End Iter = %d, Res = %g, b = %g, a = %g\n",k,Temp,alpha,beta,rsold);
}
__global__ void multiGpuPreConjugateGradient(int *I, int *J, float *val, float *M, float *x,  float *Ax, float *p, float *r, float *Z, 
            int nnz, int N, float tol, double *d_result)
{
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    cg::multi_grid_group multi_grid = cg::this_multi_grid();

    //int TID = blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    int k = 0;
    int max_iter = 100000;
    float a = 1.0;
    float na = -1.0;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;

    rsold = 0.0; 
    cg::sync(grid);
    MultigpuSpMV(I, J, val, nnz, N, a, x, Ax, cta, multi_grid); 
    cg::sync(grid);
    MultigpuSaxpy(Ax, r, na, N, multi_grid); 
    cg::sync(grid);
    MultigpuProductVector(M, r, Z, N, cta, multi_grid);
    cg::sync(grid);
    MultigpuCopyVector(Z, p, N, multi_grid);
    cg::sync(grid);
    MultigpuDotProduct(r, Z, N, cta, multi_grid); 
    cg::sync(grid);
    if (grid.thread_rank() == 0) {
        atomicAdd_system(d_result, grid_dot_result);
        grid_dot_result = 0.0;
    }
    cg::sync(multi_grid);
    rsold = *d_result;
    //if (threadIdx.x == 0 && grid.thread_rank() == 0) printf("start : rsold1 = %g\n",rsold);
    while (rsold > tol*tol && k <= max_iter){
        k++;
        cg::sync(multi_grid);
        MultigpuSpMV(I, J, val, nnz, N, a, p, Ax, cta, multi_grid);
        if (multi_grid.thread_rank() == 0) {
            setDotResultToZero(d_result);
        }   
        cg::sync(multi_grid);
        MultigpuDotProduct(p, Ax, N, cta, multi_grid);
        cg::sync(grid);
        if (grid.thread_rank() == 0) {
            atomicAdd_system(d_result, grid_dot_result);
            grid_dot_result = 0.0;
        }
        cg::sync(multi_grid);
        Temp = *d_result;
        //if (threadIdx.x == 0 && grid.thread_rank() == 0) printf("Iter = %d, Temp = %g,",k,Temp);
        alpha = (Temp)? rsold/Temp:0.0f;
        MultigpuSaxpy(p, x, alpha, N, multi_grid);
        nalpha = -alpha;
        MultigpuSaxpy(Ax, r, nalpha, N, multi_grid);
        MultigpuProductVector(M, r, Z, N, cta, multi_grid);
        cg::sync(multi_grid);
        if (multi_grid.thread_rank() == 0) {
            setDotResultToZero(d_result);
        }   
        cg::sync(multi_grid);
        MultigpuDotProduct(r, Z, N, cta, multi_grid);
        cg::sync(grid);
        if (grid.thread_rank() == 0) {
            atomicAdd_system(d_result, grid_dot_result);
            grid_dot_result = 0.0;
        }
        cg::sync(multi_grid);
        rnew = *d_result;
        //if (threadIdx.x == 0 && grid.thread_rank() == 0) printf("rnew = %g\n",rnew);
        beta = (rsold) ? rnew/rsold: 0.0f;
        MultigpuRSaxpy(Z, p, beta, N, multi_grid);
        rsold = rnew;
        rnew = 0.0;
    }
    //if(threadIdx.x == 0 && blockIdx.x == 0 ) printf("End Iter = %d, Res = %g, b = %g, a = %g\n",k,Temp,alpha,beta,rsold);
}
__device__ void MultigpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha, float *inputVecX, 
                        float *outputVecY, cg::thread_block &cta, const cg::multi_grid_group &multi_grid) {
    for (int i = multi_grid.thread_rank(); i < num_rows; i += multi_grid.size()) {
        // i = 0 ~ A_size-1; 
        int row_elem = I[i];
        int next_row_elem = I[i+1];
        int num_elems_this_row = next_row_elem - row_elem;
        float output = 0.0;
        for (int j=row_elem-1; j < next_row_elem-1; j++){
            output +=  alpha*val[j] * inputVecX[J[j]-1];
            //if(i==num_rows-1) printf("val[%d][%d] = %g, %g, %g\n",j,J[j]-1,val[j],inputVecX[J[j]-1],output);
        }
        //printf("output[%d] = %g\n",i,output);
        outputVecY[i] = output;
    }
}
__device__ void MultigpuSaxpy(float *x, float *y, float a, int size, const cg::multi_grid_group &multi_grid) {
    for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {  
        y[i] = a*x[i] + y[i];
    }
}
__device__ void MultigpuRSaxpy(float *x, float *y, float a, int size, const cg::multi_grid_group &multi_grid) {
    for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {  
        y[i] = a*y[i] + x[i];
    }
}
__device__ void MultigpuDotProduct(float *vecA, float *vecB, int size, const cg::thread_block &cta, const cg::multi_grid_group &multi_grid) {
   __shared__ double tmp[THREADS_PER_BLOCK];
    double temp_sum = 0.0;
    for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
        temp_sum += (double) (vecA[i] * vecB[i]);
    }
    tmp[cta.thread_rank()] = temp_sum;
    cg::sync(cta);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
    double beta  = temp_sum;
    double temp;
    for (int i = tile32.size() / 2; i > 0; i >>= 1) {
        if (tile32.thread_rank() < i) {
            temp       = tmp[cta.thread_rank() + i];
            beta       += temp;
            tmp[cta.thread_rank()] = beta;
        }
        cg::sync(tile32);
    }
    cg::sync(cta);
    if (cta.thread_rank() == 0) {
        beta  = 0.0;
        for (int i = 0; i < cta.size(); i += tile32.size()) {
            beta  += tmp[i];
        }
        atomicAdd(&grid_dot_result, beta);
    }
}
__device__ void MultigpuCopyVector(float *srcA, float *destB, int size, const cg::multi_grid_group &multi_grid) {
    for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
        destB[i] = srcA[i];
    }
}
__device__ void MultigpuScaleVector(float *vec, float alpha, int size, const cg::multi_grid_group &multi_grid) {
    for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
        vec[i] = alpha*vec[i];
    }
}
__device__ void MultigpuProductVector(float *vecA, float *vecB, float *vecC, int size, const cg::thread_block &cta, const cg::multi_grid_group &multi_grid) {
    for (int i = multi_grid.thread_rank(); i < size; i += multi_grid.size()) {
        vecC[i] = (vecA[i] * vecB[i]);
    }
}
__device__ void setDotResultToZero(double *dot_result) {
  unsigned long long int *address_as_ull = (unsigned long long int *)dot_result;
  unsigned long long int old = *address_as_ull, assumed;
  do {
    assumed = old;
    old = atomicCAS_system(address_as_ull, assumed, 0);

  } while (assumed != old);
}