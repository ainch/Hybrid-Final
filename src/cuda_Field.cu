#include "cuda_Field.cuh"
// FOR Field method 4
#define THREADS_PER_BLOCK 512   
//
void PCG_SOLVER_Laplace(){
    // OUTPUT
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit

    //Make a Field DATA set  
    //Host_PCG_DATA = (DPS_Data*)malloc(A_size*sizeof(DPS_Data));
    //checkCudaErrors(hipMalloc((void**)&dev_PCG_DATA, A_size*sizeof(DPS_Data)));
    //Make_PCG_DATA_Init<<<A_size/4,4>>>(dev_PCG_DATA,A_size,dev_M);
    //checkCudaErrors(hipMemcpy(Host_PCG_DATA, dev_PCG_DATA, A_size*sizeof(DPS_Data), hipMemcpyDeviceToHost));
    // Laplace Solution
    //hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
    // hipMemset((void *) array, 0, Gsize * sizeof(int));
    

           // Find good grid and block size
        //hipOccupancyMaxPotentialBlockSize(&mingrid,&FIELD_BLOCK,(void*)hipsparseSpMV,0,Gsize); 
        //FIELD_GRID = (Gsize + FIELD_BLOCK - 1) / FIELD_BLOCK;
        //printf("blockSize = %d\n",FIELD_BLOCK);
        //printf("gridSize = %d\n",FIELD_GRID);
    // For test

    //if (matA       ) { checkCudaErrors(hipsparseDestroySpMat(matA)); }
    //if (vecx       ) { checkCudaErrors(hipsparseDestroyDnVec(vecx)); }
    //if (vecAP      ) { checkCudaErrors(hipsparseDestroyDnVec(vecAP)); }
    //if (vecp       ) { checkCudaErrors(hipsparseDestroyDnVec(vecp)); }
}
void Set_MatrixPCG_cuda(){

}
