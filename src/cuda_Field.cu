#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"

void PCG_SOLVER_Laplace(){
    // Solve Laplace Equation. (To use every time step.)
    // Goal
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    int i,j; 

    //////////////////////////////////////////////////////////////////////////////
    int grid,block,mingrid;
    float *buf;
    int IIter;
    buf = VFMalloc(A_size);
    printf("<FIELD SOVER>\n");
	printf(" Laplace eq. using PCG\n");
	printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)PCG_LAP,0,Gsize); 
    grid = (Gsize + block - 1) / block;
    printf("minGridSize = %d\n",mingrid);
    printf("blockSize = %d\n",block);
    printf("gridSize = %d\n",grid);
  
    for (i = 0; i < CondNUMR; i++) {
        hipMemcpy(dev_b, cond_b[i], A_size * sizeof(float),hipMemcpyHostToDevice);
        for(j=0;j<A_size;j++){
			//if(cond_b[i][j] !=0 )
				//printf("cond_b[%d][%d] = %g\n",i,j,cond_b[i][j]);
		} 
        IIter = 10;
        //printf("IIter = %d\n",IIter);
        //PCG_LAP<<<grid,block>>>(&IIter,Gsize,A_size,dev_A,dev_Ai,dev_Aj,dev_M,dev_AP,dev_R,dev_Z,dev_P,dev_X,dev_b);
        //printf("IIter = %d\n",IIter);
        //hipMemcpy(buf, dev_R, A_size * sizeof(float),hipMemcpyDeviceToHost);
        //printf("buf = %d\n",buf[1]);
    }
    //exit(1);
}
void Set_MatrixPCG_cuda(){
    int TID;
    int i,j,k; 
    int PCG_Laplace_SINGLECPU_Flag=0;
    float **CPUsol;
	//vec_cond_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_boundary_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_face_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_area_Garray = (float *) malloc(ngx * ngy * sizeof(float));
	//vec_eps_Carray = (float *) malloc(ncx * ncy * sizeof(float));

    vec_A_idx = (int *) malloc(ngx * ngy * sizeof(int));
    for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
			//vec_cond_Garray[j + i * ngy] = cond_Garray[i][j];
			//vec_boundary_Garray[j + i * ngy] = boundary_Garray[i][j];
			//vec_face_Garray[j + i * ngy] = face_Garray[i][j];
			//vec_area_Garray[j + i * ngy] = area_Garray[i][j];
		} // matrix save direction ^ >
	}
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  A_size * sizeof(float)));
    // Initialize
    checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
    //Copy
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, A_size * sizeof(int), hipMemcpyHostToDevice));
    // Laplace Solution
    hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
    // hipMemset((void *) array, 0, Gsize * sizeof(int));
    
    //Make a Field constant set  
    checkCudaErrors(hipMalloc((void**)&dev_PCG_const,sizeof(DPS_Const)));
    Make_PCG_Const_Init<<<1,1>>>(dev_PCG_const,A_size,PCGtol);
    Host_PCG_const = (DPS_Const*)malloc(sizeof(DPS_Const));
    checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
   
    //Make a Field DATA set  
    checkCudaErrors(hipMalloc((void**)&dev_PCG_DATA, A_size*sizeof(DPS_Data)));
    Make_PCG_DATA_Init<<<A_size/4,4>>>(dev_PCG_DATA,A_size,dev_M);

    Host_PCG_DATA = (DPS_Data*)malloc(A_size*sizeof(DPS_Data));
    checkCudaErrors(hipMemcpy(Host_PCG_DATA, dev_PCG_DATA, A_size*sizeof(DPS_Data), hipMemcpyDeviceToHost));
    for (i=0;i<A_size;i++){
        printf("[%d]%g R=%g,Z=%g,P=%g,AP=%g,M=%g\n",i,MatM[i],Host_PCG_DATA[i].vecR,Host_PCG_DATA[i].vecZ,Host_PCG_DATA[i].vecP,Host_PCG_DATA[i].vecAP,Host_PCG_DATA[i].vecM);
    }
    exit(1);
    //printf("testKernel results:\n");
    //printf("point.a: %g, point.b: %g\n",CPU_BUF[0].tol,CPU_BUF[0].tol2);
    //printf("point.a: %g, point.b: %g\n",CPU_BUF[0].tol2,CPU_BUF[0].tol2);
    //printf("point.a: %d, point.b: %g\n",CPU_BUF[0].A_size,CPU_BUF[0].tol);
    //free(CPU_BUF);
    // retrieve the results
    

    
        // deallocate memory
    
    if(PCG_Laplace_SINGLECPU_Flag==1){
		printf(" Preconditioner[Jacovi]\n"); 
        printf(" Main Library set[Single CPU PCG]\n"); 
        printf(" Laplace Equation TEST\n"); 
        X = VFMalloc(A_size);
        B = VFMalloc(A_size);
        R0 = VFMalloc(A_size);
        AX = VFMalloc(A_size);
        Z0 = VFMalloc(A_size);
        P0 = VFMalloc(A_size);
        AP = VFMalloc(A_size);
        PAP = VFMalloc(A_size);
        VFInit(X,0.0,A_size);
        VFInit(B,0.0,A_size);
        VFInit(AX,0.0,A_size);
        VFInit(R0,0.0,A_size);
        VFInit(Z0,0.0,A_size);
        VFInit(P0,0.0,A_size);
        VFInit(AP,0.0,A_size);
        VFInit(PAP,0.0,A_size);
        CPUsol = MFMalloc(CondNUMR,Gsize);
        for (k = 0; k < CondNUMR; k++) {
            VFCopy(B,cond_b[k],A_size);
            FieldIter = PCG_SINGLECPU();
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Save
            VFInit(CPUsol[k],0.0,Gsize);
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = X[vec_A_idx[TID]-1];
                    }
                }
            }
            /*
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    printf("%6.2g",CPUsol[k][TID]);
                }printf("\n");
            }printf("\n");
            */
            CPU_PCG_Laplace_Solution_Save(CPUsol);
        }
        exit(1);
	}
}
__global__ void Make_PCG_DATA_Init(DPS_Data *p, int size,float *MatrixM){
    int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if(TID>=size) return;
    printf("TID = %d, M = %g\n",TID,MatrixM[TID]);
    p[TID].vecR = 0.0;
    p[TID].vecZ = 0.0;
    p[TID].vecP = 0.0;
    p[TID].vecAP = 0.0;
    p[TID].vecM = MatrixM[TID];
}
__global__ void Make_PCG_Const_Init(DPS_Const *p,int Asize, float tol){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].A_size = Asize;
    p[i].Iter = 0;
    p[i].tol = tol;
    p[i].tol2 = tol*tol;
    p[i].rsold = 0.0;
    p[i].Temp = 0.0;
    p[i].rnew = 0.0;
    p[i].alpha = 0.0;
    p[i].beta = 0.0;    
}
int PCG_SINGLECPU(){
    int TID,i,Iter=0;
    float tol2;
    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        Z0[TID] = MatM[TID] * R0[TID];
        P0[TID] = Z0[TID];
        rsold += R0[TID]*Z0[TID]; //AtomicAdd!!
    }
    tol2 = PCGtol*PCGtol;
    while(rsold>tol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
            }
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //AtomicAdd!!
        }
        alpha = rsold/Temp;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            Z0[TID] = MatM[TID] * R0[TID];
            rnew += R0[TID]*Z0[TID];  //AtomicAdd!!
        }
        beta = rnew/rsold;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = Z0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
    }
    return Iter;
}
__global__ void PCG_LAP(int *Iter,int Gsize,int Asize,float *A,int *Ai,int *Aj,float *M,float *AP,float *R,float *Z,float *P,float *X,float *b){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=Asize) return;
    int i;
    //float tol2;
    float a;

    for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
        AP[TID] += A[i]*X[Aj[i]-Ai[0]];
    }
    R[TID] = b[TID] - AP[TID];
    Z[TID] = M[TID] * R[TID];
    P[TID] = Z[TID];
    a = R[TID]*Z[TID];
    *Iter = 100;
    
    //__syncthreads();
    //printf("TID = %d, a = %g\n",TID,a);  
    //atomicAdd(&rsold,a);
    //if(TID==0) printf("rsold %d= %g\n",TID,rsold);       
    //if(TID==10) printf("rsold %d= %g\n",TID,rsold);  

    /*
    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        Z0[TID] = MatM[TID] * R0[TID];
        P0[TID] = Z0[TID];
        rsold += R0[TID]*Z0[TID];
    }
    tol2 = PCGtol*PCGtol;
    //printf("rsold=%g\n",rsold);
    while(rsold>tol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
                //printf("[%d],[%d] %g, %g\n",TID,Aj[i],A_val[i],P0[TID]);
            }
            //printf("PAP[%d] = %g\n",TID,AP[TID]);
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //Reduction
        }
        alpha = rsold/Temp;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            Z0[TID] = MatM[TID] * R0[TID];
            rnew += R0[TID]*Z0[TID];  //Reduction
        }
        beta = rnew/rsold;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = Z0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
        //printf("alpha %d = %2g, Temp = %g, Beta %d = %2g, rsold=%2g\n",FieldIter-1,alpha,Temp,FieldIter-1,beta,rsold);
    }
    */

}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}
