#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"
//
void PCG_SOLVER_Laplace(){
    int i,j,k,TID;
    dim3 grid(ngx * ngy / 1024 + 1, 1);
	dim3 block(1024);
    hipEvent_t start, stop; // SPEED TEST
    float gputime; // SPEED TEST
    // 
    int blockSize,gridSize;
    int blockSize2,gridSize2;
    hipOccupancyMaxPotentialBlockSize(&gridSize,&blockSize,(void*)PCG_Deposit_Lap,0,Gsize); 
    gridSize = (Gsize + blockSize - 1) / blockSize;
    hipOccupancyMaxPotentialBlockSize(&gridSize2,&blockSize2,(void*)Cond_Sigma_Lap,0,Gsize); 
    gridSize2 = (Gsize + blockSize2 - 1) / blockSize2;
    hipOccupancyMaxPotentialBlockSize(&gridSize2,&blockSize2,(void*)SaveAT2D,0,Gsize); 
    gridSize2 = (Gsize + blockSize2 - 1) / blockSize2;
    printf("g = %d, b = %d\n",gridSize,blockSize);
    printf("g2 = %d, b2 = %d\n",gridSize2,blockSize2);
    printf("g3 = %d, b3 = %d\n",FIELD_GRID,FIELD_BLOCK);
    //
    // OUTPUT
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    void *kernelArgs[] = {
        (void*)&dev_Ai,
        (void*)&dev_Aj,
        (void*)&dev_A,
        (void*)&dev_X,
        (void*)&dev_M,
        (void*)&dev_AP,
        (void*)&dev_P,
        (void*)&dev_R,
        (void*)&dev_Z,
        (void*)&N,
        (void*)&nz,
        (void*)&PCGtol2,
        (void*)&FIter,
        (void*)&dot_result,
    };
    char Namebuf[256];
    float **CPUsol;
    CPUsol = MFMalloc(CondNUMR,Gsize);
    for (k = 0; k < CondNUMR; k++) {
        printf(" Laplace Solution %d",k);
        checkCudaErrors(hipMemcpy(dev_R, cond_b[k], N * sizeof(float),hipMemcpyHostToDevice));
        checkCudaErrors(hipMemset((void *) dev_X, 0, N * sizeof(float)));
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
        checkCudaErrors(hipLaunchCooperativeKernel((void *)PCG,FIELD_GRID,FIELD_BLOCK, kernelArgs, sMemSize, NULL));
        checkCudaErrors(hipDeviceSynchronize());
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
        printf(" - Iter = %d, time = %2.3f (ms), rsold^2 = %g\n",*FIter,gputime,*dot_result);
        //
        checkCudaErrors(hipMemset((void *) dev_phi, 0.0, Gsize * sizeof(float)));
        PCG_Deposit_Lap<<<gridSize,blockSize>>>(Gsize, dev_A_idx, dev_GvecSet, k, dev_X, dev_phi);
        //
        checkCudaErrors(hipMemset((void *) dev_phi_buf, 0.0, Gsize * sizeof(float)));
        Cond_Sigma_Lap<<<gridSize,blockSize>>>(ngx, ngy, dx, dy, zlength, dev_GvecSet, dev_CvecSet, dev_phi, dev_phi_buf);
        VFInit(Host_G_buf,0.0,Gsize);
        checkCudaErrors(hipMemcpy(Host_G_buf, dev_phi_buf, Gsize * sizeof(float),hipMemcpyDeviceToHost));
		for (j = 0; j < Gsize; j++) {
			if (vec_G[j].CondID){
                Lap_SIG_Sol[k][vec_G[j].CondID - 1] += Host_G_buf[j] * vec_G[j].Area;
            } 
		}
        for (j = 0; j < CondNUMR; j++)
			printf(" - Lap_SIG_Sol[%d][%d]= %g\n", k, j, Lap_SIG_Sol[k][j]);
        SaveAT2D<<<gridSize,blockSize>>>(Lap_PHI_Sol, pitch, i, dev_phi, Gsize);
    }
    printf("/***********Calculate temperature distribution**********/\n");
    checkCudaErrors(hipMemcpy(dev_R, dev_Tb, N * sizeof(float),hipMemcpyDeviceToDevice));
    checkCudaErrors(hipMemset((void *) dev_X, 0, N * sizeof(float)));
	checkCudaErrors(hipLaunchCooperativeKernel((void *)PCG,FIELD_GRID,FIELD_BLOCK, kernelArgs, sMemSize, NULL));
	printf(" - Iter = %d, rsold^2 = %g\n",*FIter,*dot_result);
	printf("/*******************************************************/\n");
    PCG_Deposit_Temp<<<gridSize,blockSize>>>(Gsize, dev_A_idx, dev_X, dev_GvecSet);

    
    checkCudaErrors(hipMemcpy(vec_G, dev_GvecSet, Gsize * sizeof(GGA), hipMemcpyDeviceToHost));
    for(i=0;i<Gsize;i++){
        CPUsol[0][i] = vec_G[i].Temp;
        CPUsol[1][i] = vec_G[i].BackDens;
    }
    sprintf(Namebuf,"Test");
    Field_Laplace_Solution_Save(Namebuf,CPUsol);

    // Laplace Solution
    //hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
    // hipMemset((void *) array, 0, Gsize * sizeof(int));

    exit(1);
}
void Set_MatrixPCG_cuda(){
    int i,j;
    N = A_size;
    nz = 5 * A_size;
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d, ngx x ngy = %d X %d = %d\n", N, ngx, ngy, Gsize);
    // Real Solution
    // Laplace Solution
    checkCudaErrors(hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR)); // for Laplace Solution
    checkCudaErrors(hipMalloc((void**) &dev_phi, Gsize * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_phi, 0.0, Gsize * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_phi_buf, Gsize * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_phi_buf, 0.0, Gsize * sizeof(float)));
    Lap_SIG_Sol = MFMalloc(CondNUMR,CondNUMR);
    MFInit(Lap_SIG_Sol,0.0,CondNUMR,CondNUMR);
    // Allocate
    checkCudaErrors(hipMalloc((void**) &dev_A, nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, nz * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (N + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_b, 0.0, N * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_X, 0.0, N * sizeof(float)));
    //
    checkCudaErrors(hipMalloc((void**) &dev_TA, nz * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Tb,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_Tb, 0.0, N * sizeof(float)));
    //
    checkCudaErrors(hipMalloc((void**) &dev_AP,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_AP, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_R, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_P, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_M, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Z,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_Z, 0.0, N * sizeof(float)));
    // Data cpu > gpu
    checkCudaErrors(hipMemcpy(dev_A, A_val, nz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_TA,TA_val,nz*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, nz * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (N + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_Tb,temp_b, N * sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, N * sizeof(float), hipMemcpyHostToDevice));
    // GGA GCA copy
    checkCudaErrors(hipMalloc((void**)&dev_GvecSet, Gsize * sizeof(GGA)));
    checkCudaErrors(hipMemcpy(dev_GvecSet, vec_G, Gsize * sizeof(GGA), hipMemcpyHostToDevice));
    for(i=0;i<Csize;i++){
      vec_C[i].eps = EPS0 * vec_C[i].eps_r;
    }
    checkCudaErrors(hipMalloc((void**)&dev_CvecSet, Csize * sizeof(GCA)));
    checkCudaErrors(hipMemcpy(dev_CvecSet, vec_C, Csize * sizeof(GCA), hipMemcpyHostToDevice));
    //Unified memory value for Field residual
    hipMallocManaged((void **)&dot_result, sizeof(double));
    *dot_result = 0.0;
    hipMallocManaged((void **)&FIter, sizeof(int));
    *FIter = 0;
    //
    vec_A_idx = (int *) malloc(Gsize * sizeof(int));
    for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
		} 
	}
    checkCudaErrors(hipMalloc((void**) &dev_A_idx, Gsize * sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_A_idx, vec_A_idx, Gsize * sizeof(int),hipMemcpyHostToDevice));
}
__device__ void Mat_x_Vec(int *I, int *J, float *val, int nnz, int num_rows, float alpha, float *inputVecX, 
                        float *outputVecY, cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < num_rows; i+= grid.size())    {
        int row_elem = I[i];
        int next_row_elem = I[i+1];
        int num_elems_this_row = next_row_elem - row_elem;
        float output = 0.0;
        for (int j=row_elem-1; j < next_row_elem-1; j++){
            //if(i==0) printf("val[%d][]\n",j);
            // I or J or val arrays - can be put in shared memory 
            // as the access is random and reused in next calls of gpuSpMV function.
            output +=  alpha*val[j] * inputVecX[J[j]-1];
            //if(i==0) printf("val[%d][%d] = %g, %g, %g\n",j,J[j]-1,val[j],inputVecX[J[j]-1],output);
        }
        outputVecY[i] = output;
    }
}
__device__ void A_x_X_p_Y(float a, float *x, float *y, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) y[i] = a*x[i] + y[i];
}
__device__ void A_x_Y_p_X(float a, float *x, float *y, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) y[i] = a*y[i] + x[i];
}
__device__ void Vec_Dot_Sum(float *vecA, float *vecB, double *result, int size, const cg::thread_block &cta, const cg::grid_group &grid)
{
   __shared__ double tmp[THREADS_PER_BLOCK];
    double temp_sum = 0.0;
    for (int i=grid.thread_rank(); i < size; i+=grid.size()){
        temp_sum += (double) (vecA[i] * vecB[i]);
    }
    tmp[cta.thread_rank()] = temp_sum;
    cg::sync(cta);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
    double beta  = temp_sum;
    double temp;
    for (int i = tile32.size() / 2; i > 0; i >>= 1) {
        if (tile32.thread_rank() < i) {
            temp       = tmp[cta.thread_rank() + i];
            beta       += temp;
            tmp[cta.thread_rank()] = beta;
        }
        cg::sync(tile32);
    }
    cg::sync(cta);
    if (cta.thread_rank() == 0) {
        beta  = 0.0;
        for (int i = 0; i < cta.size(); i += tile32.size()) {
            beta  += tmp[i];
        }
        atomicAdd(result, beta);
    }
}
__device__ void CopyVector(float *srcA, float *destB, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) destB[i] = srcA[i];
}
__device__ void Vec_x_Vec(float *vecA, float *vecB, float *vecC, int size, const cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+=grid.size()) vecC[i] = (vecA[i] * vecB[i]);
}
__global__ void PCG(int *I, int *J, float *val, float *x, float *M, float *Ax, float *p, float *r, float *Z, 
            int N, int nnz, float tol2, int *Iter, double *d_result){
    //Jacovi diagonal preconditioner version
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    //int TID = blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    int max_iter = 100000;
    float a = 1.0;
    float na = -1.0;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;
    rsold = 0.0;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *Iter = 0;
        *d_result = 0.0;  
    } 
    Mat_x_Vec(I, J, val, nnz, N, a, x, Ax, cta, grid); 
    A_x_X_p_Y(na, Ax, r, N, grid); 
    Vec_x_Vec(M, r, Z, N, cta, grid);
    CopyVector(Z, p, N, grid);
    //if(r[TID] !=0) printf("r[%d] = %g\n",TID,r[TID]);
    cg::sync(grid);
    Vec_Dot_Sum(r, Z, d_result, N, cta, grid); 
    cg::sync(grid);
    rsold = *d_result;
    //if(threadIdx.x == 0 && blockIdx.x == 0) printf("First:rsold = %g N = %d, nnz = %d\n",rsold,N,nnz);
    //return;
    while (rsold > tol2 && *Iter <= max_iter){
        Mat_x_Vec(I, J, val, nnz, N, a, p, Ax, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *Iter = *Iter + 1;
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        //if(Ax[TID] !=0) printf("Ax[%d] = %g\n",TID,Ax[TID]);
        Vec_Dot_Sum(p, Ax, d_result, N, cta, grid);
        cg::sync(grid);
        Temp = *d_result;
        //if(threadIdx.x == 0 && blockIdx.x == 0) printf("Temp = %g\n",Temp);
        //return;
        alpha = (Temp)? rsold/Temp:0.0f;
        A_x_X_p_Y(alpha, p, x, N, grid);
        nalpha = -alpha;
        A_x_X_p_Y(nalpha, Ax, r, N, grid);
        Vec_x_Vec(M, r, Z, N, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        Vec_Dot_Sum(r, Z, d_result, N, cta, grid);
        cg::sync(grid);
        rnew = *d_result;
        beta = (rsold) ? rnew/rsold: 0.0f;
        A_x_Y_p_X(beta, Z, p, N, grid);
        rsold = rnew;
        rnew = 0.0;
        //if(threadIdx.x == 0 && blockIdx.x == 0 && k<20) printf("Iter = %d, temp = %g,  AL = %g, BE = %g Res = %g\n",k,Temp,alpha,beta,rsold);
    }
    //if(threadIdx.x == 0 && blockIdx.x == 0 ) printf("End Iter = %d, Res = %g, b = %g, a = %g\n",k,Temp,alpha,beta,rsold);
}
__global__ void Cond_Sigma_Lap(int ngx, int ngy, float dx, float dy, float zlength, GGA *vecG, GCA *vecC, float *Phi, float *Sigma)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	int x, y;
	int Phi_left, Phi_right, Phi_up, Phi_down;
	int EPS_TID, EPS_left, EPS_down, EPS_cross, EPS_xover, EPS_yover;

	if(TID>=ngx*ngy) return;
	x=TID/ngy; y=TID%ngy;
	//Calculate surface charge
	EPS_TID=x*(ngy-1)+y;
	EPS_left=(x) ? EPS_TID-ngy+1:EPS_TID;
	EPS_down=(y) ? EPS_TID-1:EPS_TID;
	EPS_cross=EPS_left+EPS_down-EPS_TID;
	EPS_xover=(x==ngx-1) ? ngy-1: 0;
	EPS_yover=(y==ngy-1) ? 1: 0;
	EPS_TID-=(EPS_xover+EPS_yover);
	EPS_left-=EPS_yover;
	EPS_down-=EPS_xover;

	Phi_left =(x) ? TID-ngy:TID;
	Phi_right=(x==ngx-1) ? TID:TID+ngy;
	Phi_down =(y) ? TID-1:TID;
	Phi_up =(y==ngy-1) ? TID:TID+1;

	if((vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET) && vecG[TID].Face!=NO_FACE) {
		if(vecG[TID].Face==UP) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])/dy;
		}
		else if(vecG[TID].Face==DOWN) {
			Sigma[TID] = 0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])/dy;
		}
		else if(vecG[TID].Face==LEFT) {
			Sigma[TID] = 0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])/dx;
		}
		else if(vecG[TID].Face==RIGHT) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])/dx;
		}
		else if(vecG[TID].Face==UL_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])/dy+0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])/dx;
		}
		else if(vecG[TID].Face==UR_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])/dy+0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])/dx;
		}
		else if(vecG[TID].Face==LL_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])/dy+0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])/dx;
		}
		else if(vecG[TID].Face==LR_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])/dy+0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])/dx;
		}
	}
}
__global__ void PCG_Deposit_Lap(int Gsize, int *IDX, GGA *vecG, int k, float *X, float *PHI){
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	if(TID>=Gsize) return;
	if(IDX[TID])
		PHI[TID]=X[IDX[TID]-1];
	else if(vecG[TID].CondID == k+1)
		PHI[TID]=1.0;   
}
__global__ void PCG_Deposit_Temp(int Gsize, int *IDX, float *X, GGA *vecG)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=Gsize) return;
	else if(IDX[TID]) vecG[TID].Temp = X[IDX[TID]-1];
}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}
