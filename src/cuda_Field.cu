#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"

void PCG_SOLVER_Laplace(){
    // Solve Laplace Equation. (To use every time step.)
    // Goal
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    int i,j;
    float *dev_A, *dev_b, *dev_R, *dev_P;	// PCG device parameter
    int *dev_Aj,*dev_Ai;			
    float *dev_AP, *dev_M, *dev_Z, *dev_X, *dev_Tmp;	// PCG device parameter
    int   *vec_A_idx;
    int   *vec_cond_Garray;
    int   *vec_boundary_Garray;
    int   *vec_face_Garray;
    float *vec_area_Garray;
    float *vec_eps_Carray;
    float *dev_Sigma;
    int   *dev_face_Garray;
    float *dev_area_Garray;
    float *dev_eps_Carray;
    hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float));
	hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int));
	hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int));
	hipMalloc((void**) &dev_b, A_size * sizeof(float));
	hipMalloc((void**) &dev_R, A_size * sizeof(float));
	hipMalloc((void**) &dev_Tmp, A_size * sizeof(float));
	hipMalloc((void**) &dev_Z, A_size * sizeof(float));
	hipMalloc((void**) &dev_P, A_size * sizeof(float));
	hipMalloc((void**) &dev_AP, A_size * sizeof(float));
	hipMalloc((void**) &dev_X, A_size * sizeof(float));
	hipMalloc((void**) &dev_M, A_size * sizeof(float));
    hipMalloc((void**) &dev_b, A_size * sizeof(float));
    // Initialize
    hipMemset((void *) dev_X, 0, A_size * sizeof(float));
	hipMemset((void *) dev_AP, 0, A_size * sizeof(float));
    //Copy
    hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_M, MatM, A_size * sizeof(float), hipMemcpyHostToDevice);
	vec_A_idx = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_cond_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_boundary_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_face_Garray = (int *) malloc(ngx * ngy * sizeof(int));
	//vec_area_Garray = (float *) malloc(ngx * ngy * sizeof(float));
	//vec_eps_Carray = (float *) malloc(ncx * ncy * sizeof(float));
	for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
			//vec_cond_Garray[j + i * ngy] = cond_Garray[i][j];
			//vec_boundary_Garray[j + i * ngy] = boundary_Garray[i][j];
			//vec_face_Garray[j + i * ngy] = face_Garray[i][j];
			//vec_area_Garray[j + i * ngy] = area_Garray[i][j];
		} // matrix save direction ^ >
	}
    //////////////////////////////////////////////////////////////////////////////
	
    int grid,block,mingrid;
    // Find good grid and block size
    hipOccupancyMaxPotentialBlockSize(&mingrid,&block,(void*)PCG,0,Gsize); 
    grid = (Gsize + block - 1) / block;
    printf("minGridSize = %d\n",mingrid);
    printf("blockSize = %d\n",block);
    printf("gridSize = %d\n",grid);

    PCGtol *= 1e-3;
    for (i = 0; i < CondNUMR; i++) {
        hipMemcpy(dev_b, cond_b[i], A_size * sizeof(float),hipMemcpyHostToDevice);
        for(j=0;j<A_size;j++){
			//if(cond_b[i][j] !=0 )
				//printf("cond_b[%d][%d] = %g\n",i,j,cond_b[i][j]);
		} 
        PCG<<<grid,block>>>(FieldIter,Gsize,A_size,dev_A,dev_Ai,dev_Aj,dev_X,dev_b);
        printf("FieldIter = %d\n",FieldIter);
    }
    PCGtol *= 1e3;
    exit(1);
}
void Set_MatrixPCG_cuda(){
    

    float *dev_phi_dw;
    float *dev_phi_u;
    printf("<FIELD SOVER>\n");
	printf(" Laplace eq. using PCG\n");
	printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    if(PCG_Method==0){
		printf(" Preconditioner[Jacovi]\n"); 
        printf(" Main Library set[hipsparseScsrmv]\n"); 
	}else if(PCG_Method==1){
        printf(" Preconditioner[Jacovi]\n");
        printf(" Main Library set[hipsparseSpMV]\n"); 
	}else if(PCG_Method==2){
        printf(" Preconditioner[Jacovi]\n");
        printf(" Main Library set[hipsparseSpMV+GraphLaunch]\n"); 
	}else if(PCG_Method==3){

	}

    // Laplace Solution
    hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
   // hipMemset((void *) array, 0, Gsize * sizeof(int));
}
__global__ void PCG(int Iter,int Gsize,int Asize,float *A,int *Ai,int *Aj,float *X,float *b){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=Asize) return;
    int i;
    float AP;
    // r0 = b-AX
    // Data access
    // TID = 0~(Asize-1)
    // St_ID = Ai[0];
    // ID = Ai[TID] - 1;
    // Dn = Ai[TID]-Ai[TID-1]

    
    Iter = TID;
    //printf("Iter = %d\n",Iter);
}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}