#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"
void Field_Method0_Initial(){
    // CPU Conjugate Gradient
    printf(" Field Solver : [CPU] Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    X = VFMalloc(A_size);
    B = VFMalloc(A_size);
    R0 = VFMalloc(A_size);
    AX = VFMalloc(A_size);
    P0 = VFMalloc(A_size);
    AP = VFMalloc(A_size);
    PAP = VFMalloc(A_size);
    VFInit(B,0.0,A_size);
    VFInit(AX,0.0,A_size);
    VFInit(R0,0.0,A_size);
    VFInit(P0,0.0,A_size);
    VFInit(AP,0.0,A_size);
    VFInit(PAP,0.0,A_size);
}
void Field_Method1_Initial(){
    // CPU Preconditioned Conjugate Gradient [Jacovi]
    printf(" Field Solver : [CPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Preconditioner[Jacovi]\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    X = VFMalloc(A_size);
    B = VFMalloc(A_size);
    R0 = VFMalloc(A_size);
    AX = VFMalloc(A_size);
    Z0 = VFMalloc(A_size);
    P0 = VFMalloc(A_size);
    AP = VFMalloc(A_size);
    PAP = VFMalloc(A_size);
    VFInit(B,0.0,A_size);
    VFInit(AX,0.0,A_size);
    VFInit(R0,0.0,A_size);
    VFInit(Z0,0.0,A_size);
    VFInit(P0,0.0,A_size);
    VFInit(AP,0.0,A_size);
    VFInit(PAP,0.0,A_size);
}
void Field_Method2_Initial(){
    printf(" Field Solver : [GPU] Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Cuda Handle setting
    cublasHandle = 0;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    checkCudaErrors(hipblasStatus_t);
    cusparseHandle = 0;
    checkCudaErrors(hipsparseCreate(&cusparseHandle));
    // Data cpu > gpu
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_AP,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    /* Wrap raw data into cuSPARSE generic API objects */
    matA = NULL;
    vecx = NULL;
    vecp = NULL;
    vecAP = NULL;
    checkCudaErrors(hipsparseCreateCsr(
        &matA, A_size, A_size, 5*A_size, dev_Ai, dev_Aj, dev_A, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecx, A_size, dev_X, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecp, A_size, dev_P, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecAP, A_size, dev_AP, HIP_R_32F));
}
void Field_Method3_Initial(){
    printf(" Field Solver : [GPU] Conjugate Gradient\n"); 
    printf(" Cuda Function : Cuda Graphs launch\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    // Cuda Handle setting
    cublasHandle = 0;
    hipblasStatus_t = hipblasCreate(&cublasHandle);
    checkCudaErrors(hipblasStatus_t);
    cusparseHandle = 0;
    checkCudaErrors(hipsparseCreate(&cusparseHandle));
    cusparseStatus = hipsparseCreate(&cusparseHandle);
    checkCudaErrors(cusparseStatus);
    // Data cpu > gpu
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_AP,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  A_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    // stream
    checkCudaErrors(hipStreamCreate(&stream1));
    /* Wrap raw data into cuSPARSE generic API objects */
    matA = NULL;
    vecx = NULL;
    vecp = NULL;
    vecAP = NULL;
    checkCudaErrors(hipsparseCreateCsr(
        &matA, A_size, A_size, 5*A_size, dev_Ai, dev_Aj, dev_A, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ONE, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecx, A_size, dev_X, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecp, A_size, dev_P, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnVec(&vecAP, A_size, dev_AP, HIP_R_32F));
}
void Field_Method4_Initial(){
    printf(" Field Solver : [GPU] Conjugate Gradient\n"); 
    printf(" Cuda Function : Multi Block\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    // Initialize
    checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
    //Copy
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
}
void Field_Method5_Initial(){
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Preconditioner[Jacovi]\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  A_size * sizeof(float)));
    // Initialize
    checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
    //Copy
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, A_size * sizeof(int), hipMemcpyHostToDevice));
}
void Field_Method6_Initial(){
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Preconditioner[ILU]\n"); 
    printf(" Matrix Size = %d X %d = %d\n", A_size, A_size, A_size*A_size);
    checkCudaErrors(hipMalloc((void**) &dev_A, 5 * A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, 5 * A_size * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (A_size + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  A_size * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  A_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  A_size * sizeof(float)));
    // Initialize
    checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
    //Copy
    checkCudaErrors(hipMemcpy(dev_A, A_val, 5 * A_size * sizeof(float), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, 5 * A_size * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (A_size + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, A_size * sizeof(int), hipMemcpyHostToDevice));
}
void PCG_SOLVER_Laplace(){
    // Solve Laplace Equation. 
    // INPUT
    // Field_Solver_Flag 0 - [CPU] Conjugate Gradient 
    // Field_Solver_Flag 1 - [CPU] Preconditioned Conjugate Gradient 
    // Field_Solver_Flag 2 - [GPU] Conjugate Gradient 
    // Field_Solver_Flag 3 - [GPU] Conjugate Gradient + Cuda Graphs launch
    // Field_Solver_Flag 4 - [GPU] Conjugate Gradient + Multi Block
    // Field_Solver_Flag 5 - [GPU] [Jacovi] Preconditioned Conjugate Gradient 
    // Field_Solver_Flag 6 - [GPU] [ILU] Preconditioned Conjugate Gradient 
    // OUTPUT
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    int i,j,k,TID; 
    int mingrid;
    int IIter;
    char Namebuf[256];
    float *buf;
    float **CPUsol;
    buf = VFMalloc(A_size);
    CPUsol = MFMalloc(CondNUMR,Gsize);
    vec_A_idx = (int *) malloc(ngx * ngy * sizeof(int));
    for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
		} 
	}
    if(Field_Solver_Flag == 0){// [CPU] Conjugate Gradient 
        Field_Method0_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {
            VFCopy(B,cond_b[k],A_size);
            VFInit(X,0.0,A_size);
            FieldIter = CG_CPU();
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = X[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"CPU_CG");
        Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Field_Solver_Flag == 1){// [CPU] Preconditioned Conjugate Gradient 
        Field_Method1_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {
            VFCopy(B,cond_b[k],A_size);
            VFInit(X,0.0,A_size);
            FieldIter = PCG_CPU();
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = X[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"CPU_PCG");
        Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Field_Solver_Flag == 2){// [GPU] Conjugate Gradient 
		Field_Method2_Initial(); // Initial Setting 
        for (k = 0; k < CondNUMR; k++) {
            checkCudaErrors(hipMemcpy(dev_R, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
            FieldIter = CG_GPU();
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_CG");
        Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Field_Solver_Flag == 3){// [GPU] Conjugate Gradient + Cuda Graphs launch
		Field_Method3_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {
            checkCudaErrors(hipMemcpy(dev_R, cond_b[k], A_size * sizeof(float),hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset((void *) dev_X, 0, A_size * sizeof(float)));
            FieldIter = CG_GPU_CudaGraphs();
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpyAsync(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost, streamForGraph));
            checkCudaErrors(hipStreamSynchronize(streamForGraph));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_CG_Graph");
        Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Field_Solver_Flag == 4){// [GPU] Conjugate Gradient + Multi Block
		Field_Method4_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {

            
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_CG_MultiBlock");
        Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Field_Solver_Flag == 5){// [GPU] [Jacovi] Preconditioned Conjugate Gradient
		Field_Method5_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {

            
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_PCG_jacovi");
        Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else if(Field_Solver_Flag == 6){// [GPU] [ILU] Preconditioned Conjugate Gradient
		Field_Method6_Initial(); // Initial Setting
        for (k = 0; k < CondNUMR; k++) {

            
            printf("Solution %d",k);
            printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
            printf("FieldIter = %d\n",FieldIter);
            // Make a Solution
            VFInit(CPUsol[k],0.0,Gsize);
            checkCudaErrors(hipMemcpy(buf, dev_X, A_size * sizeof(float),hipMemcpyDeviceToHost));
            for(j=ngy-1;j>=0;j--){
                for(i=0;i<ngx;i++){
                    TID = i*ngy+j;
                    if((vec_G[TID].CondID-1)==k){
                        CPUsol[k][TID] = 1.0;
                    }
                    if(vec_A_idx[TID]){
                        CPUsol[k][TID] = buf[vec_A_idx[TID]-1];
                    }
                }
            }
        }
        sprintf(Namebuf,"GPU_PCG_ILU");
        Field_Laplace_Solution_Save(Namebuf,CPUsol);
    }else{

    }
    
    // Laplace Solution
    hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
    // hipMemset((void *) array, 0, Gsize * sizeof(int));
    
    //Make a Field constant set  
    Host_PCG_const = (DPS_Const*)malloc(sizeof(DPS_Const));
    checkCudaErrors(hipMalloc((void**)&dev_PCG_const,sizeof(DPS_Const)));
    Make_PCG_Const_Init<<<1,1>>>(dev_PCG_const,A_size,PCGtol);
    checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
    //checkCudaErrors(hipMemcpy(Host_PCG_const, dev_PCG_const, sizeof(DPS_Const), hipMemcpyDeviceToHost));
    //Make a Field DATA set  
    Host_PCG_DATA = (DPS_Data*)malloc(A_size*sizeof(DPS_Data));
    checkCudaErrors(hipMalloc((void**)&dev_PCG_DATA, A_size*sizeof(DPS_Data)));
    Make_PCG_DATA_Init<<<A_size/4,4>>>(dev_PCG_DATA,A_size,dev_M);
    //checkCudaErrors(hipMemcpy(Host_PCG_DATA, dev_PCG_DATA, A_size*sizeof(DPS_Data), hipMemcpyDeviceToHost));
           // Find good grid and block size
        //hipOccupancyMaxPotentialBlockSize(&mingrid,&FIELD_BLOCK,(void*)hipsparseSpMV,0,Gsize); 
        //FIELD_GRID = (Gsize + FIELD_BLOCK - 1) / FIELD_BLOCK;
        //printf("blockSize = %d\n",FIELD_BLOCK);
        //printf("gridSize = %d\n",FIELD_GRID);
    // For test
    hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);
    if (matA       ) { checkCudaErrors(hipsparseDestroySpMat(matA)); }
    if (vecx       ) { checkCudaErrors(hipsparseDestroyDnVec(vecx)); }
    if (vecAP      ) { checkCudaErrors(hipsparseDestroyDnVec(vecAP)); }
    if (vecp       ) { checkCudaErrors(hipsparseDestroyDnVec(vecp)); }
    exit(1);

}
void Set_MatrixPCG_cuda(){

}
int CG_CPU(){
    int TID,i,Iter=0;
    float tol2;
    float rsold,rnew,Temp;
    float alpha,beta;

    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        P0[TID] = R0[TID];
        rsold += R0[TID]*R0[TID]; //AtomicAdd!!
    }
    tol2 = PCGtol*PCGtol;
    while(rsold>tol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
            }
            //printf("AP[%d] = %g\n",TID,P0[TID]);
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //AtomicAdd!!
        }
        alpha = (Temp)? rsold/Temp:0.0f ;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            rnew += R0[TID]*R0[TID];  //AtomicAdd!!
        }
        beta = (rsold) ? rnew/rsold: 0.0f;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = R0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
        //if(Iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",Iter,Temp,alpha,beta,rsold);
    }
    return Iter;
}
int PCG_CPU(){
    int TID,i,Iter=0;
    float tol2;
    float rsold,rnew,Temp;
    float alpha,beta;

    rsold = 0;
    for(TID=0;TID<A_size;TID++){
        AX[TID] = 0;
        for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
            AX[TID] += A_val[i]*X[Aj[i]-Ai[0]];
        }
        R0[TID] = B[TID] - AX[TID];
        Z0[TID] = MatM[TID] * R0[TID];
        P0[TID] = Z0[TID];
        rsold += R0[TID]*Z0[TID]; //AtomicAdd!!
    }
    while(rsold>PCGtol2){
        Iter++;
        Temp = 0.0;
        for(TID=0;TID<A_size;TID++){
            AP[TID] = 0;
            for(i=Ai[TID]-Ai[0];i<Ai[TID+1]-Ai[0];i++){
                AP[TID] += A_val[i]*P0[Aj[i]-Ai[0]];
            }
            //printf("AP[%d] = %g\n",TID,P0[TID]);
            PAP[TID] = P0[TID] * AP[TID];
            Temp += PAP[TID]; //AtomicAdd!!
        }
        alpha = (Temp)? rsold/Temp:0.0f ;
        for(TID=0;TID<A_size;TID++){
            X[TID] = X[TID] + alpha * P0[TID];
            R0[TID] = R0[TID] - alpha * AP[TID];
            Z0[TID] = MatM[TID] * R0[TID];
            rnew += R0[TID]*Z0[TID];  //AtomicAdd!!
        }
        beta = (rsold) ? rnew/rsold: 0.0f;
        for(TID=0;TID<A_size;TID++){ 
            P0[TID] = Z0[TID] + beta*P0[TID];
        }
        rsold = rnew;
        rnew = 0.0;
        //if(Iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",Iter,Temp,alpha,beta,rsold);
    }
    return Iter;
}
int CG_GPU(){
    int iter;
    int max_iter = 10000;
    float a = 1.0;
    float b = 0.0;
    float na = -1.0;
    float rnew = 0.;
    float rsold,Temp;
    float alpha,beta;
    float nalpha;
    /* Allocate workspace for cuSPARSE */
    size_t bufferSize = 0;
    checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, &bufferSize));
    void *buffer = NULL;
    checkCudaErrors(hipMalloc(&buffer, bufferSize));
    /* Begin CG */
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, buffer));
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, &na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, &rsold));
    iter = 1;
    while (rsold > PCGtol2 && iter <= max_iter){
         if (iter > 1)
        {
            beta = rnew / rsold;
            rsold = rnew;
            hipblasSscal(cublasHandle, A_size, &beta, dev_P, 1);
            hipblasSaxpy(cublasHandle, A_size, &a, dev_R, 1, dev_P, 1);
        }
        else
        {
            hipblasScopy(cublasHandle, A_size, dev_R, 1, dev_P, 1);
        }
        checkCudaErrors(hipsparseSpMV(
            cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA,
            vecp, &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, buffer));
        hipblasSdot(cublasHandle, A_size, dev_P, 1, dev_AP, 1, &Temp);
        alpha = (Temp) ? rsold / Temp: 0.0f;
        hipblasSaxpy(cublasHandle, A_size, &alpha, dev_P, 1, dev_X, 1);
        nalpha = -alpha;
        hipblasSaxpy(cublasHandle, A_size, &nalpha, dev_AP, 1, dev_R, 1);
        hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, &rnew);
        hipDeviceSynchronize();
        if(iter<10) printf("Iter = %d, Temp = %g, alpha = %g, beta = %g, rsold = %g\n",iter,Temp,a,b,rsold);
        iter++;
    }
    return iter;

}
int CG_GPU_CudaGraphs(){
    int iter;
    int max_iter = 10000;
    float a = 1.0;
    float b = 0.0;
    float na = -1.0;
    float r1;
    float *d_r1, *d_r0, *d_dot, *d_a, *d_na, *d_b;
    /* Allocate workspace for cuSPARSE */
    size_t bufferSize = 0;
    checkCudaErrors(hipsparseSpMV_bufferSize(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_CSRMV_ALG1, &bufferSize));
    void *buffer = NULL;
    checkCudaErrors(hipMalloc(&buffer, bufferSize));
    // DEVICE variable
    checkCudaErrors(hipMalloc((void **)&d_r1, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_r0, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_dot, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_a, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_na, sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(float)));
    //
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecx,
        &b, vecAP, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, &na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, d_r1));

    iter = 1;
    // First Iteration when iter=1 starts
    checkCudaErrors(hipblasScopy(cublasHandle, A_size, dev_R, 1, dev_P, 1));
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecp,
        &b, vecAP, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_P, 1, dev_AP, 1, d_dot));
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_a, dev_P, 1, dev_X, 1));
    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),hipMemcpyDeviceToDevice, stream1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, d_r1));
    checkCudaErrors(hipMemcpyAsync(&r1, d_r1, sizeof(float),hipMemcpyDeviceToHost, stream1));
    checkCudaErrors(hipStreamSynchronize(stream1));
    printf("iteration = %3d, residual = %e\n", iter, sqrt(r1));
    // First Iteration when k=1 ends
    iter++;
    checkCudaErrors(hipStreamCreate(&streamForGraph));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
    // Capture start
    checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    checkCudaErrors(hipblasSscal(cublasHandle, A_size, d_b, dev_P, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, &a, dev_R, 1, dev_P, 1));
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    checkCudaErrors(hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST));
    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &a, matA, vecp,
        &b, vecAP, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
    checkCudaErrors(hipMemsetAsync(d_dot, 0, sizeof(float), stream1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_P, 1, dev_AP, 1, d_dot));
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_a, dev_P, 1, dev_X, 1));
    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
    checkCudaErrors(hipblasSaxpy(cublasHandle, A_size, d_na, dev_AP, 1, dev_R, 1));
    checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),hipMemcpyDeviceToDevice, stream1));
    checkCudaErrors(hipMemsetAsync(d_r1, 0, sizeof(float), stream1));
    checkCudaErrors(hipblasSdot(cublasHandle, A_size, dev_R, 1, dev_R, 1, d_r1));
    checkCudaErrors(hipMemcpyAsync((float *)&r1, d_r1, sizeof(float),hipMemcpyDeviceToHost, stream1));
    checkCudaErrors(hipStreamEndCapture(stream1, &initGraph));
    // Capture End
    checkCudaErrors(hipGraphInstantiate(&graphExec, initGraph, NULL, NULL, 0));
    checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
    checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
    while (r1 > PCGtol2 && iter <= max_iter) {
        checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
        checkCudaErrors(hipStreamSynchronize(streamForGraph));
        if(iter<10) printf("iteration = %3d, residual = %e\n", iter, sqrt(r1));
    iter++;
    }
    return iter;
}
__global__ void PCG_LAP(float *A,int *Ai,int *Aj,DPS_Const *PCG_C,DPS_Data *PCG_D,float *X,float *b){
    int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    if(TID>=PCG_C[0].A_size) return;
    int i;
    float sum;
    int MAXITER = 20;
    //Initial
    if(TID==0){
        PCG_C[0].Iter = 0;
        PCG_C[0].rsold = 0;
        PCG_C[0].Temp = 0;
        PCG_C[0].rnew = 0;
    } 
    __syncthreads();
    // cal  AP = A * P
    for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
        PCG_D[TID].AP += A[i] * X[Aj[i]-1];
    }
    PCG_D[TID].R = b[TID] - PCG_D[TID].AP;
    PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
    PCG_D[TID].P = PCG_D[TID].Z;
    sum = PCG_D[TID].R*PCG_D[TID].Z;
    atomicAdd(&PCG_C[0].rsold,sum);
    //if(TID==0) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
    //if(TID==10) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
    //if(TID==0) printf(" [%d]Initial rsold = %g, tol2 = %g\n",TID,PCG_C[0].rsold,PCG_C[0].tol2);
    while(PCG_C[0].rsold > PCG_C[0].tol2){
        if(TID==0){
            PCG_C[0].Iter++;
            PCG_C[0].Temp = 0;
            PCG_C[0].rnew = 0;
        } 
        __syncthreads();
        //if(TID==0) printf(" [%d]Iter %d start!\n",TID,PCG_C[0].Iter);
        PCG_D[TID].AP = 0;
        for(i=Ai[TID]-1;i<Ai[TID+1]-1;i++){
            PCG_D[TID].AP += A[i] * PCG_D[Aj[i]-1].P;
        }
        //printf("[%d] AP = %g\n",TID,PCG_D[TID].P);
        sum = PCG_D[TID].P * PCG_D[TID].AP;
        atomicAdd(&PCG_C[0].Temp,sum);
        //if(TID==0) printf(" [%d]Temp = %g\n",TID,PCG_C[0].Temp);       
        if(PCG_C[0].Iter>MAXITER){
            //if(TID==0) printf("Iteration[%d] = %g\n",TID,PCG_C[0].Iter);    
            break;
        } 
        if(TID==0){
            PCG_C[0].alpha = (PCG_C[0].Temp)? PCG_C[0].rsold/PCG_C[0].Temp : 0.0f; 
        } 
        __syncthreads();     
        X[TID] = X[TID] + PCG_C[0].alpha * PCG_D[TID].P;
        PCG_D[TID].R = PCG_D[TID].R - PCG_C[0].alpha * PCG_D[TID].AP;
        PCG_D[TID].Z = PCG_D[TID].M * PCG_D[TID].R;
        sum = PCG_D[TID].R * PCG_D[TID].Z;
        atomicAdd(&PCG_C[0].rnew,sum);
       //if(TID==0) printf(" [%d]rnew = %g\n",TID,PCG_C[0].rnew);     
        if(TID==0){
             PCG_C[0].beta = (PCG_C[0].rsold)? PCG_C[0].rnew/PCG_C[0].rsold : 0.0f;
        }
        __syncthreads();
        PCG_D[TID].P = PCG_D[TID].Z + PCG_C[0].beta*PCG_D[TID].P;    
        if(TID==0) PCG_C[0].rsold = PCG_C[0].rnew;
        __syncthreads();
        if(TID==0) printf(" [%d]Iter %d,Temp = %g,alpha = %g,beat = %g, rsold = %g\n",TID,PCG_C[0].Iter,PCG_C[0].Temp,PCG_C[0].alpha,PCG_C[0].beta,PCG_C[0].rsold);
    }
    if(TID==0) printf("Iter [%d]= %d, ",TID,PCG_C[0].Iter);  
    if(TID==0) printf("maxNorm [%d]= %g\n",TID,PCG_C[0].rsold);  
}

__global__ void Make_PCG_DATA_Init(DPS_Data *p, int size,float *MatrixM){
    int TID = blockIdx.x * blockDim.x + threadIdx.x;
    if(TID>=size) return;
    //printf("TID = %d, M = %g\n",TID,MatrixM[TID]);
    p[TID].R = 0.0;
    p[TID].Z = 0.0;
    p[TID].P = 0.0;
    p[TID].AP = 0.0;
    p[TID].M = MatrixM[TID];
}
__global__ void Make_PCG_Const_Init(DPS_Const *p,int Asize, float tol){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    p[i].A_size = Asize;
    p[i].Iter = 0;
    p[i].tol = tol;
    p[i].tol2 = tol*tol;
    p[i].rsold = 0.0;
    p[i].Temp = 0.0;
    p[i].rnew = 0.0;
    p[i].alpha = 0.0;
    p[i].beta = 0.0;    
}
__global__ void initVectors(float *rhs, float *x, int N) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  for (size_t i = gid; i < N; i += gridDim.x * blockDim.x) {
    rhs[i] = 1.0;
    x[i] = 0.0;
  }
}
__global__ void r1_div_x(float *r1, float *r0, float *b) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    b[0] = r1[0] / r0[0];
  }
}
__global__ void a_minus(float *a, float *na) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    na[0] = -(a[0]);
  }
}

__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}
