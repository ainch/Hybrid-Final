#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"
void Efield_cuda_Basic(){ // No circuit
    // Function
    int i,j,isp;
    float K_t;
    float q_conv;   
	hipMemcpy(CondCharge, dev_CondCharge, nsp * CondNUMR * sizeof(float),hipMemcpyDeviceToHost);
	hipMemset((void *) dev_CondCharge, 0, nsp * CondNUMR * sizeof(float));
    // Get voltage 
    for(i=0;i<CondNUMR;i++) {
		phi_cond[i] = 0.0f;
		if(Efield_Flag[i]){
			for(j=0;j<SrcNUM;j++){
                if(SrcM_ID[j] == i+1){
                    phi_cond[i] += SrcDC[j] + SrcAC[j]*sin(Src2piFREQ[j]*t+SrcRPHASE[j]);
					V_t[i] = phi_cond[i];
                }
            }
		}
    }
    // Potential summation
	hipMemset((void *) TotPotential, 0.0, Gsize * sizeof(float));
    for(i=0;i<CondNUMR;i++){
        LoadAT2D<<<FIELD_GRID2,FIELD_BLOCK2>>>(Lap_PHI_Sol, pitch, i, dev_phi_buf, Gsize);
		VectorSum<<<FIELD_GRID2, FIELD_BLOCK2>>>(Gsize, TotPotential, phi_cond[i], dev_phi_buf);
		//hipDeviceSynchronize();
    }
	// TotPotential = laplace * votage
    hipMemcpy(LapPotential, TotPotential, Gsize * sizeof(float),hipMemcpyDeviceToDevice);
	// TotPotential = laplace * votage 
	// LapPotential = laplace * votage 
	VectorSum<<<FIELD_GRID2, FIELD_BLOCK2>>>(Gsize, TotPotential, 1, dev_phi);
	// TotPotential = laplace * votage + Poisson's
	// LapPotential = laplace * votage 
    GGACopy_Potential<<<FIELD_GRID2, FIELD_BLOCK2>>>(Gsize, dev_GvecSet, LapPotential, dev_phi);
	Cond_Sigma<<<FIELD_GRID2,FIELD_BLOCK2>>>(ngx, ngy, hdx, hdy, idxy, zlength, dev_GvecSet, dev_CvecSet, dev_info_sp, dev_G_sp, dev_phi, dev_Sigma);
	//Cond_Sigma_v2<<<FIELD_GRID2,FIELD_BLOCK2>>>(ngx, ngy, hdx, hdy, idxy, zlength, dev_GvecSet, dev_CvecSet, dev_info_sp, dev_G_sp, dev_phi, dev_Sigma);
	VtoEfield<<<FIELD_GRID2, FIELD_BLOCK2>>>(ngx,ngy,dx,dy,hdx,hdy,idx,idy, dev_Sigma, TotPotential, dev_G_sp, dev_CvecSet, dev_GvecSet);
    //hipMemcpy(vec_G, dev_GvecSet, Gsize * sizeof(GGA), hipMemcpyDeviceToHost); // for TEST
    //Main_Variable_printorSave(); // for TEST
	hipDeviceSynchronize();
}
void Efield_cuda(){ // External circuit
    // Function
    int i,j,isp;
    float K_t;
    float q_conv;   
	hipMemcpy(CondCharge, dev_CondCharge, nsp * CondNUMR * sizeof(float),hipMemcpyDeviceToHost);
	hipMemset((void *) dev_CondCharge, 0, nsp * CondNUMR * sizeof(float));
    // Get voltage 
    for(i=0;i<CondNUMR;i++) {
		V_t[i] = 0.0;
		if(Efield_Flag[i]){
			for(j=0;j<SrcNUM;j++){
                if(SrcM_ID[j] == i+1){
                    V_t[i] += SrcDC[j] + SrcAC[j]*sin(Src2piFREQ[j]*t+SrcRPHASE[j]);
                }
            }
		}
		K_t = (CC_a[i][1]*extq[i] + CC_a[i][2]*extq_1[i] + CC_a[i][3]*extq_2[i] + CC_a[i][4]*extq_3[i])/CC_a[i][0];
        //convective charge
		q_conv=0;
		for(isp=0;isp<nsp;isp++) {
			q_conv-=SP[isp].q_density*CondCharge[isp*CondNUMR+i];
		}
        b_t[i]=(V_t[i]-K_t)/CC_a[i][0] - extq[i] - q_conv + Surf_charge[i] - Pois_SIG_Sol[i];
		for(j=0;j<CondNUMR;j++){
			if(i==j) AM[i][j] = Lap_SIG_Sol[j][i]+1/CC_a[i][0];
			else AM[i][j] = Lap_SIG_Sol[j][i];
		}
    }
    cofactor(AM, CondNUMR);
    for (i = 0; i < CondNUMR; i++) {
		phi_cond[i] = 0.0;
		for (j = 0; j < CondNUMR; j++) {
			phi_cond[i] += AM[i][j] * b_t[j];
		}
        extq_3[i] = extq_2[i];
		extq_2[i] = extq_1[i];
		extq_1[i] = extq[i];
		extq[i] = (V_t[i] - phi_cond[i]) / CC_a[i][0] - K_t;
    }
    // Potential summation
	hipMemset((void *) TotPotential, 0.0, Gsize * sizeof(float));
    for(i=0;i<CondNUMR;i++){
        LoadAT2D<<<FIELD_GRID2,FIELD_BLOCK2>>>(Lap_PHI_Sol, pitch, i, dev_phi_buf, Gsize);
		VectorSum<<<FIELD_GRID2, FIELD_BLOCK2>>>(Gsize, TotPotential, phi_cond[i], dev_phi_buf);
		hipDeviceSynchronize();
    }
	// TotPotential = laplace * votage
    hipMemcpy(LapPotential, TotPotential, Gsize * sizeof(float),hipMemcpyDeviceToDevice);
	// TotPotential = laplace * votage 
	// LapPotential = laplace * votage 
	VectorSum<<<FIELD_GRID2, FIELD_BLOCK2>>>(Gsize, TotPotential, 1, dev_phi);
	// TotPotential = laplace * votage + Poisson's
	// LapPotential = laplace * votage 
    GGACopy_Potential<<<FIELD_GRID2, FIELD_BLOCK2>>>(Gsize, dev_GvecSet, LapPotential, dev_phi);
	Cond_Sigma<<<FIELD_GRID2,FIELD_BLOCK2>>>(ngx, ngy,  hdx, hdy, idxy, zlength, dev_GvecSet, dev_CvecSet, dev_info_sp, dev_G_sp, dev_phi, dev_Sigma);
    VtoEfield<<<FIELD_GRID2, FIELD_BLOCK2>>>(ngx,ngy,dx,dy,hdx,hdy,idx,idy, dev_Sigma, TotPotential, dev_G_sp, dev_CvecSet, dev_GvecSet);
    //hipMemcpy(vec_G, dev_GvecSet, Gsize * sizeof(GGA), hipMemcpyDeviceToHost); // for TEST
    //Main_Variable_printorSave(); // for TEST
	hipDeviceSynchronize();
}

static float * Pois_SIG_Sol_tmp;

static void __global__ update
(
	float * Pois_SIG_Sol_tmp,
	GGA * const Field, 
	float * const dev_phi_buf, 
	int Gsize,
	int CondNUMR
)
{
	cg::thread_block group_block  = cg::this_thread_block();
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float a_shd[128];
	
	int id_cond = 0;
	float area;
	float potential;
	if(tid < Gsize)
	{
		id_cond = Field[tid].CondID;
		area = Field[tid].Area;
		potential = dev_phi_buf[tid];
		if(id_cond != 0)
			atomicAdd(Pois_SIG_Sol_tmp + id_cond - 1, area * potential);
	}
}

void PCG_SOLVER(){
    int i;
	void *kernelArgs[] = {
        (void*)&dev_Ai,(void*)&dev_Aj,(void*)&dev_A,(void*)&dev_X,
        (void*)&dev_M, (void*)&dev_AP,(void*)&dev_P,(void*)&dev_R,
        (void*)&dev_Z, (void*)&N,     (void*)&nz,   (void*)&PCGtol2,
        (void*)&FIter, (void*)&dot_result2,
    };
	checkCudaErrors(hipMemset((void *) dot_result2,  0, 1e3 * sizeof(float)));
	hipLaunchCooperativeKernel((void *)PCG_float,FIELD_GRID,FIELD_BLOCK, kernelArgs, sMemSize, NULL);
    hipDeviceSynchronize();
    PCG_Deposit<<<FIELD_GRID2,FIELD_BLOCK2>>>(Gsize, dev_A_idx, dev_GvecSet, dev_X, dev_phi);
    hipMemset((void *) dev_phi_buf, 0.0, Gsize * sizeof(float));
    Cond_Sigma<<<FIELD_GRID2,FIELD_BLOCK2>>>(ngx, ngy, hdx, hdy, idxy, zlength, dev_GvecSet, dev_CvecSet, dev_info_sp, dev_G_sp, dev_phi, dev_phi_buf);
	hipMemcpy(Host_G_buf, dev_phi_buf, Gsize * sizeof(float),hipMemcpyDeviceToHost);
	VFInit(Pois_SIG_Sol,0.0,CondNUMR); 
	hipDeviceSynchronize();
	
	if(false)
	{
		for (i = 0; i < Gsize; i++) 
		if (vec_G[i].CondID) 
		Pois_SIG_Sol[vec_G[i].CondID - 1] += Host_G_buf[i] * vec_G[i].Area;
		for(int j=0;j<CondNUMR;j++)
			printf("%.12f ", Pois_SIG_Sol[j]);
		puts("");
	}

	if(true){
		dim3 dim_num_block = dim3(Gsize / 128 + 1);
		dim3 dim_size_block = dim3(128);
		checkCudaErrors(hipMemset((void *) Pois_SIG_Sol_tmp, 0.0f, CondNUMR * sizeof(float)));
		
		update<<<dim_num_block, dim_size_block>>>
		(
			Pois_SIG_Sol_tmp, dev_GvecSet,
			dev_phi_buf, Gsize, CondNUMR
		);
	
		hipDeviceSynchronize();
		checkCudaErrors
		(
			hipMemcpy
			(
				Pois_SIG_Sol, Pois_SIG_Sol_tmp, CondNUMR * sizeof(float), hipMemcpyDeviceToHost
			)
		);
		
		//printf("%.12f %.12f %.12f\n", Pois_SIG_Sol[0],Pois_SIG_Sol[1],Pois_SIG_Sol[2]);
	}

}
void PCG_SOLVER_Laplace(){
    int j,k;
    hipEvent_t start, stop; // SPEED TEST
    float gputime = 0.0f; // SPEED TEST
    void *kernelArgs[] = {
        (void*)&dev_Ai,(void*)&dev_Aj,(void*)&dev_A,(void*)&dev_X,
        (void*)&dev_M, (void*)&dev_AP,(void*)&dev_P,(void*)&dev_R,
        (void*)&dev_Z, (void*)&N,     (void*)&nz,   (void*)&PCGtol2,
        (void*)&FIter, (void*)&dot_result2,
    };
    for (k = 0; k < CondNUMR; k++) {
        printf(" Laplace Solution %d",k);
        checkCudaErrors(hipMemcpy(dev_R, cond_b[k], N * sizeof(float),hipMemcpyHostToDevice));
        checkCudaErrors(hipMemset((void *) dev_X, 0, N * sizeof(float)));
        hipEventCreate(&start); hipEventCreate(&stop);
	    hipEventRecord( start, 0 );
		checkCudaErrors(hipMemset((void *) dot_result2,  0, 1e4 * sizeof(float)));
        checkCudaErrors(hipLaunchCooperativeKernel((void *)PCG_float,FIELD_GRID,FIELD_BLOCK, kernelArgs, sMemSize, NULL));
        checkCudaErrors(hipDeviceSynchronize());
        hipEventRecord( stop, 0 ); hipEventSynchronize( stop );
	    hipEventElapsedTime( &gputime, start, stop );
	    hipEventDestroy( start );hipEventDestroy( stop );
        printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k+1);
        printf(" - Iter = %d, time = %2.3g (ms), rsold^2 = %g\n",*FIter,gputime,*dot_result2);
        //
        checkCudaErrors(hipMemset((void *) dev_phi, 0.0, Gsize * sizeof(float)));
        PCG_Deposit_Lap<<<FIELD_GRID2,FIELD_BLOCK2>>>(Gsize, dev_A_idx, dev_GvecSet, k, dev_X, dev_phi);
        //
        checkCudaErrors(hipMemset((void *) dev_phi_buf, 0.0, Gsize * sizeof(float)));
        Cond_Sigma_Lap<<<FIELD_GRID2,FIELD_BLOCK2>>>(ngx, ngy, dx, dy, zlength, dev_GvecSet, dev_CvecSet, dev_phi, dev_phi_buf);
        VFInit(Host_G_buf,0.0,Gsize);
        checkCudaErrors(hipMemcpy(Host_G_buf, dev_phi_buf, Gsize * sizeof(float),hipMemcpyDeviceToHost));
		for (j = 0; j < Gsize; j++) {
			if (vec_G[j].CondID) Lap_SIG_Sol[k][vec_G[j].CondID - 1] += Host_G_buf[j] * vec_G[j].Area;
		}
        for (j = 0; j < CondNUMR; j++) printf(" - Lap_SIG_Sol[%d][%d]= %g\n", k, j, Lap_SIG_Sol[k][j]);
        SaveAT2D<<<FIELD_GRID2,FIELD_BLOCK2>>>(Lap_PHI_Sol, pitch, k, dev_phi, Gsize);
    }
	checkCudaErrors(hipMemset((void *) dev_phi_buf, 0.0, Gsize * sizeof(float)));
	if(DumpFlag == 0){
    	printf("/***********Calculate temperature distribution**********/\n");
    	checkCudaErrors(hipMemcpy(dev_R, dev_Tb, N * sizeof(float),hipMemcpyDeviceToDevice));
    	checkCudaErrors(hipMemset((void *) dev_X, 0, N * sizeof(float)));
		
		checkCudaErrors(hipMemset((void *) dot_result2,  0, 1e4 * sizeof(float)));
		checkCudaErrors(hipLaunchCooperativeKernel((void *)PCG_float,FIELD_GRID,FIELD_BLOCK, kernelArgs, sMemSize, NULL));
		printf(" - Iter = %d, rsold^2 = %g\n",*FIter,*dot_result2);
		printf("/*******************************************************/\n");
    	PCG_Deposit_Temp<<<FIELD_GRID2,FIELD_BLOCK2>>>(Gsize, dev_A_idx, dev_X, dev_GvecSet);
    	if(MainGas == ARGON || MainGas == OXYGEN) Calculate_1GasPara<<<FIELD_GRID2,FIELD_BLOCK2>>>(Gsize, BG[0].mass, BG[0].Pres, dev_GvecSet); 
    	else if(MainGas == ARO2) Calculate_2GasPara<<<FIELD_GRID2,FIELD_BLOCK2>>>(Gsize, BG[0].mass, BG[0].Pres, BG[1].mass, BG[1].Pres, dev_GvecSet);
    	checkCudaErrors(hipMemcpy(vec_G, dev_GvecSet, Gsize * sizeof(GGA), hipMemcpyDeviceToHost));
	}
	checkCudaErrors(hipMemset((void *) dev_X, 0, N * sizeof(float)));
}
void Set_MatrixPCG_cuda(){
    int i,j;
    N = A_size;
    nz = 5 * A_size;
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d, ngx x ngy = %d X %d = %d\n", N, ngx, ngy, Gsize);
    // Real Solution
    // Laplace Solution
    checkCudaErrors(hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR)); // for Laplace Solution
    checkCudaErrors(hipMalloc((void**) &dev_phi, Gsize * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_phi, 0.0, Gsize * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_phi_buf, Gsize * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_phi_buf, 0.0, Gsize * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &Pois_SIG_Sol_tmp, CondNUMR * sizeof(float)));
    Lap_SIG_Sol = MFMalloc(CondNUMR,CondNUMR);
    MFInit(Lap_SIG_Sol,0.0,CondNUMR,CondNUMR);
    Pois_SIG_Sol = VFMalloc(CondNUMR);
    VFInit(Pois_SIG_Sol,0.0,CondNUMR);
    // Allocate
    checkCudaErrors(hipMalloc((void**) &dev_A, nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, nz * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (N + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_b, 0.0, N * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_X, 0.0, N * sizeof(float)));
    //
    checkCudaErrors(hipMalloc((void**) &dev_TA, nz * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Tb,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_Tb, 0.0, N * sizeof(float)));
    //
    checkCudaErrors(hipMalloc((void**) &dev_AP,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_AP, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_R, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_P, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_M, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Z,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_Z, 0.0, N * sizeof(float)));
    // Data cpu > gpu
    checkCudaErrors(hipMemcpy(dev_A, A_val, nz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_TA,TA_val,nz*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, nz * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (N + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_Tb,temp_b, N * sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, N * sizeof(float), hipMemcpyHostToDevice));
    // GGA GCA copy
    checkCudaErrors(hipMalloc((void**)&dev_GvecSet, Gsize * sizeof(GGA)));
    checkCudaErrors(hipMemcpy(dev_GvecSet, vec_G, Gsize * sizeof(GGA), hipMemcpyHostToDevice));
    for(i=0;i<Csize;i++){
      vec_C[i].eps = EPS0 * vec_C[i].eps_r;
    }
    checkCudaErrors(hipMalloc((void**)&dev_CvecSet, Csize * sizeof(GCA)));
    checkCudaErrors(hipMemcpy(dev_CvecSet, vec_C, Csize * sizeof(GCA), hipMemcpyHostToDevice));
    //Unified memory value for Field residual
    hipMallocManaged((void **)&dot_result, sizeof(double));
    *dot_result = 0.0;
	hipMallocManaged((void **)&dot_result2, sizeof(float) * 10000);
    *dot_result2 = 0.0;
    hipMallocManaged((void **)&FIter, sizeof(int));
    *FIter = 0;
    //
    vec_A_idx = (int *) malloc(Gsize * sizeof(int));
    for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
		} 
	}
    checkCudaErrors(hipMalloc((void**) &dev_A_idx, Gsize * sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_A_idx, vec_A_idx, Gsize * sizeof(int),hipMemcpyHostToDevice));
    // dev_Sigma, dev_Source, TotPotential
    checkCudaErrors(hipMalloc((void**) &dev_Sigma, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *) dev_Sigma, 0, Gsize * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Source, Gsize * sizeof(float))); // Total charge --> dev_b or dev_R
	checkCudaErrors(hipMemset((void *) dev_Source, 0, Gsize * sizeof(float))); // Total charge --> dev_b or dev_R
    checkCudaErrors(hipMalloc((void**) &TotPotential, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *) TotPotential, 0, Gsize * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &LapPotential, Gsize * sizeof(float)));
	checkCudaErrors(hipMemset((void *) LapPotential, 0, Gsize * sizeof(float)));
	// Condunctor 
	checkCudaErrors(hipMalloc((void**)&dev_CondCharge, nsp * CondNUMR * sizeof(float)));
	checkCudaErrors(hipMemset((void *)dev_CondCharge, 0, nsp * CondNUMR * sizeof(float)));
}
__global__ void GGACopy_Potential(int Gsize, GGA *vecG, float *V1, float *V2){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	if(TID>=Gsize) return;
	vecG[TID].Lap_Pot=V1[TID];
    vecG[TID].Pois_Pot=V2[TID];
}
__global__ void VectorSum(int Gsize,float *TotPhi,float V,float *Phi){
	int TID = threadIdx.x + blockIdx.x * blockDim.x;
	if(TID>=Gsize) return;
	//if(TID<5) printf(" VectorSum:TID[%d]\n",TID);
	TotPhi[TID]+=V*Phi[TID];
}
__device__ void Mat_x_Vec(int *I, int *J, float *val, int nnz, int num_rows, float alpha, float *inputVecX, 
                        float *outputVecY, cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < num_rows; i+= grid.size())    {
        int row_elem = I[i];
        int next_row_elem = I[i+1];
        float output = 0.0;
        for (int j=row_elem-1; j < next_row_elem-1; j++){
            //if(i==0) printf("val[%d][]\n",j);
            // I or J or val arrays - can be put in shared memory 
            // as the access is random and reused in next calls of gpuSpMV function.
            output +=  alpha*val[j] * inputVecX[J[j]-1];
            //if(i==0) printf("val[%d][%d] = %g, %g, %g\n",j,J[j]-1,val[j],inputVecX[J[j]-1],output);
        }
        outputVecY[i] = output;
    }
}
__device__ void A_x_X_p_Y(float a, float *x, float *y, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) y[i] = a*x[i] + y[i];
}
__device__ void A_x_Y_p_X(float a, float *x, float *y, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) y[i] = a*y[i] + x[i];
}
__device__ void Vec_Dot_Sum(float *vecA, float *vecB, double *result, int size, const cg::thread_block &cta, const cg::grid_group &grid)
{
   __shared__ double tmp[THREADS_PER_BLOCK];
    double temp_sum = 0.0;
    for (int i=grid.thread_rank(); i < size; i+=grid.size()){
        temp_sum += (double) (vecA[i] * vecB[i]);
    }
    tmp[cta.thread_rank()] = temp_sum;
    cg::sync(cta);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
    double beta  = temp_sum;
    double temp;
    for (int i = tile32.size() / 2; i > 0; i >>= 1) {
        if (tile32.thread_rank() < i) {
            temp       = tmp[cta.thread_rank() + i];
            beta       += temp;
            tmp[cta.thread_rank()] = beta;
        }
        cg::sync(tile32);
    }
    cg::sync(cta);
    if (cta.thread_rank() == 0) {
        beta  = 0.0;
        for (int i = 0; i < cta.size(); i += tile32.size()) {
            beta  += tmp[i];
        }
        atomicAdd(result, beta);
    }
}
__device__ void CopyVector(float *srcA, float *destB, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) destB[i] = srcA[i];
}
__device__ void Vec_x_Vec(float *vecA, float *vecB, float *vecC, int size, const cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+=grid.size()) 
		vecC[i] = (vecA[i] * vecB[i]);
}
__global__ void PCG(int *I, int *J, float *val, float *x, float *M, float *Ax, float *p, float *r, float *Z, 
            int N, int nnz, float tol2, int *Iter, double *d_result){
    //Jacovi diagonal preconditioner version
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    //int TID = blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    int max_iter = 100000;
    float a = 1.0;
    float na = -1.0;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;
    rsold = 0.0;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *Iter = 0;
        *d_result = 0.0;  
    } 
    Mat_x_Vec(I, J, val, nnz, N, a, x, Ax, cta, grid); 
    A_x_X_p_Y(na, Ax, r, N, grid); 
    Vec_x_Vec(M, r, Z, N, cta, grid);
    CopyVector(Z, p, N, grid);
    cg::sync(grid);
    Vec_Dot_Sum(r, Z, d_result, N, cta, grid); 
    cg::sync(grid);
    rsold = *d_result;
    while (rsold > tol2 && *Iter <= max_iter){
        Mat_x_Vec(I, J, val, nnz, N, a, p, Ax, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *Iter = *Iter + 1;
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        Vec_Dot_Sum(p, Ax, d_result, N, cta, grid);
        cg::sync(grid);
        Temp = *d_result;
        alpha = (Temp)? rsold/Temp:0.0f;
		//if(threadIdx.x == 0 && blockIdx.x == 0) printf("Iter = %d, temp = %g,  Res = %g AL = %g\n",*Iter,Temp,rsold,alpha);
        A_x_X_p_Y(alpha, p, x, N, grid);
        nalpha = -alpha;
        A_x_X_p_Y(nalpha, Ax, r, N, grid);
        Vec_x_Vec(M, r, Z, N, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        Vec_Dot_Sum(r, Z, d_result, N, cta, grid);
        cg::sync(grid);
        rnew = *d_result;
        beta = (rsold) ? rnew/rsold: 0.0f;
        A_x_Y_p_X(beta, Z, p, N, grid);
        rsold = rnew;
        //rnew = 0.0;
    }
    //if(threadIdx.x == 0 && blockIdx.x == 0 ) printf("End Iter = %d, Res = %g, b = %g, a = %g\n",*Iter,Temp,alpha,beta,rsold);
}
__global__ void Cond_Sigma_Lap(int ngx, int ngy, float dx, float dy, float zlength, GGA *vecG, GCA *vecC, float *Phi, float *Sigma)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	int x, y;
	int Phi_left, Phi_right, Phi_up, Phi_down;
	int EPS_TID, EPS_left, EPS_down, EPS_cross, EPS_xover, EPS_yover;

	if(TID>=ngx*ngy) return;
	x=TID/ngy; y=TID%ngy;
	//Calculate surface charge
	EPS_TID=x*(ngy-1)+y;
	EPS_left=(x) ? EPS_TID-ngy+1:EPS_TID;
	EPS_down=(y) ? EPS_TID-1:EPS_TID;
	EPS_cross=EPS_left+EPS_down-EPS_TID;
	EPS_xover=(x==ngx-1) ? ngy-1: 0;
	EPS_yover=(y==ngy-1) ? 1: 0;
	EPS_TID-=(EPS_xover+EPS_yover);
	EPS_left-=EPS_yover;
	EPS_down-=EPS_xover;

	Phi_left =(x) ? TID-ngy:TID;
	Phi_right=(x==ngx-1) ? TID:TID+ngy;
	Phi_down =(y) ? TID-1:TID;
	Phi_up =(y==ngy-1) ? TID:TID+1;

	if((vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET) && vecG[TID].Face!=NO_FACE) {
		if(vecG[TID].Face==UP) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])/dy;
		}
		else if(vecG[TID].Face==DOWN) {
			Sigma[TID] = 0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])/dy;
		}
		else if(vecG[TID].Face==LEFT) {
			Sigma[TID] = 0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])/dx;
		}
		else if(vecG[TID].Face==RIGHT) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])/dx;
		}
		else if(vecG[TID].Face==UL_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])/dy+0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])/dx;
		}
		else if(vecG[TID].Face==UR_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])/dy+0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])/dx;
		}
		else if(vecG[TID].Face==LL_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])/dy+0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])/dx;
		}
		else if(vecG[TID].Face==LR_CORN) {
			Sigma[TID] = 0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])/dy+0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])/dx;
		}
	}
}
__global__ void Cond_Sigma(int ngx, int ngy, float hdx, float hdy, float idxy, float zlength, 
					GGA *vecG, GCA *vecC, Species *info, GPG *data, float *Phi, float *Sigma)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	int x, y;
    int i;
    float SumSig;
	int Phi_left, Phi_right, Phi_up, Phi_down;
	int EPS_TID, EPS_left, EPS_down, EPS_cross, EPS_xover, EPS_yover;

	if(TID>=ngx*ngy) return;
	//if(TID<5) printf("Cond_Sigma:TID[%d]\n",TID);
	x=TID/ngy; y=TID%ngy;
	//Calculate surface charge
	EPS_TID=x*(ngy-1)+y;
	EPS_left=(x) ? EPS_TID-ngy+1:EPS_TID;
	EPS_down=(y) ? EPS_TID-1:EPS_TID;
	EPS_cross=EPS_left+EPS_down-EPS_TID;
	EPS_xover=(x==ngx-1) ? ngy-1: 0;
	EPS_yover=(y==ngy-1) ? 1: 0;
	EPS_TID-=(EPS_xover+EPS_yover);
	EPS_left-=EPS_yover;
	EPS_down-=EPS_xover;

	Phi_left =(x) ? TID-ngy:TID;
	Phi_right=(x==ngx-1) ? TID:TID+ngy;
	Phi_down =(y) ? TID-1:TID;
	Phi_up =(y==ngy-1) ? TID:TID+1;

    SumSig = 0;
    for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].den * info[i].q_density;

	if((vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET) && vecG[TID].Face!=NO_FACE) {
		if(vecG[TID].Face==UP) {
			Sigma[TID] = hdy*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])-hdx*SumSig;
		}
		else if(vecG[TID].Face==DOWN) {
			Sigma[TID] = hdy*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])-hdx*SumSig;
		}
		else if(vecG[TID].Face==LEFT) {
			Sigma[TID] = hdx*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])-hdy*SumSig;
		}
		else if(vecG[TID].Face==RIGHT) {
			Sigma[TID] = hdx*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])-hdy*SumSig;
		}
		else if(vecG[TID].Face==UL_CORN) {
			Sigma[TID] = hdy*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])+hdx*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])-idxy*SumSig;
		}
		else if(vecG[TID].Face==UR_CORN) {
			Sigma[TID] = hdy*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up])+hdx*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])-idxy*SumSig;
		}
		else if(vecG[TID].Face==LL_CORN) {
			Sigma[TID] = hdy*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])+hdx*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left])-idxy*SumSig;
		}
		else if(vecG[TID].Face==LR_CORN) {
			Sigma[TID] = hdy*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down])+hdx*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right])-idxy*SumSig;
		}
	}
}
__global__ void Cond_Sigma_v2(int ngx, int ngy, float hdx, float hdy, float idxy, float zlength, 
					GGA *vecG, GCA *vecC, Species *info, GPG *data, float *Phi, float *Sigma)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	int x, y;
    int i;
    float SumSig;
	int Phi_left, Phi_right, Phi_up, Phi_down;
	int EPS_TID, EPS_left, EPS_down, EPS_cross, EPS_xover, EPS_yover;
	float a,b,c;
	if(TID>=ngx*ngy) return;
	//if(TID<5) printf("Cond_Sigma:TID[%d]\n",TID);
	x=TID/ngy; y=TID%ngy;
	//Calculate surface charge
	EPS_TID=x*(ngy-1)+y;
	EPS_left=(x) ? EPS_TID-ngy+1:EPS_TID;
	EPS_down=(y) ? EPS_TID-1:EPS_TID;
	EPS_cross=EPS_left+EPS_down-EPS_TID;
	EPS_xover=(x==ngx-1) ? ngy-1: 0;
	EPS_yover=(y==ngy-1) ? 1: 0;
	EPS_TID-=(EPS_xover+EPS_yover);
	EPS_left-=EPS_yover;
	EPS_down-=EPS_xover;

	Phi_left =(x) ? TID-ngy:TID;
	Phi_right=(x==ngx-1) ? TID:TID+ngy;
	Phi_down =(y) ? TID-1:TID;
	Phi_up =(y==ngy-1) ? TID:TID+1;

    SumSig = 0;
    for(i=0;i<info[0].spnum;i++) SumSig += data[TID + i*ngx*ngy].den * info[i].q_density;

	if((vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET) && vecG[TID].Face!=NO_FACE) {
		if(vecG[TID].Face==UP) {
			a = hdy*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up]);
			b = 0.0f;
			c = hdx*SumSig;
		}
		else if(vecG[TID].Face==DOWN) {
			a = hdy*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down]);
			b = 0.0f;
			c = hdx*SumSig;
		}
		else if(vecG[TID].Face==LEFT) {
			a = hdx*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left]);
			b = 0.0f;
			c = hdy*SumSig;
		}
		else if(vecG[TID].Face==RIGHT) {
			a = hdx*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right]);
			b = 0.0f;
			c = hdy*SumSig;
		}
		else if(vecG[TID].Face==UL_CORN) {
			a = hdy*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up]);
			b = hdx*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left]);
			c = idxy*SumSig;
		}
		else if(vecG[TID].Face==UR_CORN) {
			a = hdy*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[TID]-Phi[Phi_up]);
			b = hdx*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right]);
			c = idxy*SumSig;
		}
		else if(vecG[TID].Face==LL_CORN) {
			a = hdy*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down]);
			b = hdx*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_left]);
			c = idxy*SumSig;
		}
		else if(vecG[TID].Face==LR_CORN) {
			a = hdy*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[TID]-Phi[Phi_down]);
			b = hdx*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[TID]-Phi[Phi_right]);
			c = idxy*SumSig;
		}else{
			return;
		}
	}else{
		return;
	}
	Sigma[TID] = a + b - c;
}
__global__ void PCG_Deposit(int Gsize, int *IDX, GGA *vecG, float *X, float *PHI){
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	if(TID>=Gsize) return;
	//if(TID<5) printf("PCG_Deposit:TID[%d]\n",TID);
	if(IDX[TID]) PHI[TID]=X[IDX[TID]-1];
    else PHI[TID] = 0;
}
__global__ void PCG_Deposit_Lap(int Gsize, int *IDX, GGA *vecG, int k, float *X, float *PHI){
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	if(TID>=Gsize) return;
	if(IDX[TID])
		PHI[TID]=X[IDX[TID]-1];
	else if(vecG[TID].CondID == k+1)
		PHI[TID]=1.0;   
}
__global__ void PCG_Deposit_Temp(int Gsize, int *IDX, float *X, GGA *vecG)
{
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=Gsize) return;
	else if(IDX[TID]) vecG[TID].Temp = X[IDX[TID]-1];
}
__global__ void Calculate_1GasPara(int Gsize, float mass, float press, GGA *vecG){
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	if(TID>=Gsize) return;
	if(vecG[TID].DensRegion){
    	vecG[TID].BackDen1 = NperTORR*press/(vecG[TID].Temp*8.6142e-5+DBL_MIN);
		vecG[TID].BackVel1 = sqrt(vecG[TID].Temp*1.38e-23/mass);
	}
}
__global__ void Calculate_2GasPara(int Gsize, float mass1, float press1, float mass2, float press2, GGA *vecG){
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
	if(TID>=Gsize) return;
	if(vecG[TID].DensRegion){
    	vecG[TID].BackDen1 = NperTORR*press1/(vecG[TID].Temp*8.6142e-5+DBL_MIN);
		vecG[TID].BackVel1 = sqrt(vecG[TID].Temp*1.38e-23/mass1);
    	vecG[TID].BackDen2 = NperTORR*press2/(vecG[TID].Temp*8.6142e-5+DBL_MIN);
		vecG[TID].BackVel2 = sqrt(vecG[TID].Temp*1.38e-23/mass2);
	}
}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;
	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;
	//if(TID<5) printf("LoadAT2D:TID[%d]\n",TID);
	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}
__global__ void VtoEfield(int ngx,int ngy,float dx,float dy,float hdx,float hdy,float idx,
                            float idy, float *Sigma, float *Phi, GPG *data, GCA *vecC, GGA *vecG){
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x ;
	int x, y;
	int Phi_left, Phi_right, Phi_up, Phi_down;
	int EPS_TID, EPS_left, EPS_down, EPS_cross, EPS_xover, EPS_yover;

	if(TID>=ngx*ngy) return;
	//if(TID<5) printf("VtoEfield:TID[%d]\n",TID);

	x=TID/ngy; y=TID%ngy;
	Phi_left =(x) ? TID-ngy:TID;
	Phi_right=(x==ngx-1) ? TID:TID+ngy;
	Phi_down =(y) ? TID-1:TID;
	Phi_up =(y==ngy-1) ? TID:TID+1;
	EPS_TID=x*(ngy-1)+y;
	EPS_left=(x) ? EPS_TID-ngy+1:EPS_TID;
	EPS_down=(y) ? EPS_TID-1:EPS_TID;
	EPS_cross=EPS_left+EPS_down-EPS_TID;
	EPS_xover=(x==ngx-1) ? ngy-1: 0;
	EPS_yover=(y==ngy-1) ? 1: 0;
	EPS_TID-=(EPS_xover+EPS_yover);
	EPS_left-=EPS_yover;
	EPS_down-=EPS_xover;

	if(vecG[TID].Face == LEFT) {
		if(vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==NEUMANN || vecG[TID].Boundary==DIRICHLET) {
			vecG[TID].Ex = -2.0*Sigma[TID]/(vecC[EPS_left].eps+vecC[EPS_cross].eps);
			vecG[TID].Ey = 0.0;
		}
		else if(vecG[TID].Boundary==DIELECTRIC || vecG[TID].Boundary>=100) {
			vecG[TID].Ex = idx*(Phi[Phi_left]-Phi[TID]);
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
		else {
			vecG[TID].Ex = 0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps)*(Phi[Phi_right]-Phi[TID])*idx;
			vecG[TID].Ex-= (0.5*dx/dy)*vecC[EPS_TID].eps*(Phi[TID]-Phi[Phi_up])*idy;
			vecG[TID].Ex-= (0.5*dx/dy)*vecC[EPS_down].eps*(Phi[TID]-Phi[Phi_down])*idy;
			vecG[TID].Ex/= -0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps);
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
	}
	else if(vecG[TID].Face == RIGHT) {
		if(vecG[TID].Boundary==CONDUCTOR ||  vecG[TID].Boundary==NEUMANN || vecG[TID].Boundary==DIRICHLET) {
			vecG[TID].Ex = 2.0*Sigma[TID]/(vecC[EPS_TID].eps+vecC[EPS_down].eps);
			vecG[TID].Ey = 0.0;
		}
		else if(vecG[TID].Boundary==DIELECTRIC || vecG[TID].Boundary>=100) {
			vecG[TID].Ex = idx*(Phi[TID]-Phi[Phi_right]);
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
		else {
			vecG[TID].Ex = 0.5*(vecC[EPS_left].eps+vecC[EPS_cross].eps)*(Phi[Phi_left]-Phi[TID])*idx;
			vecG[TID].Ex-= (0.5*dx/dy)*vecC[EPS_left].eps*(Phi[TID]-Phi[Phi_up])*idy;
			vecG[TID].Ex-= (0.5*dx/dy)*vecC[EPS_cross].eps*(Phi[TID]-Phi[Phi_down])*idy;
			vecG[TID].Ex/= 0.5*(vecC[EPS_TID].eps+vecC[EPS_down].eps);

			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
	}
	else if(vecG[TID].Face == UP) {
		if(vecG[TID].Boundary==CONDUCTOR ||  vecG[TID].Boundary==NEUMANN || vecG[TID].Boundary==DIRICHLET) {
			vecG[TID].Ex = 0.0;
			vecG[TID].Ey = 2.0*Sigma[TID]/(vecC[EPS_TID].eps+vecC[EPS_left].eps);
		}
		else if(vecG[TID].Boundary==DIELECTRIC || vecG[TID].Boundary>=100) {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = idy*(Phi[TID]-Phi[Phi_up]);
		}
		else {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = 0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps)*(Phi[Phi_down]-Phi[TID])*idy;
			vecG[TID].Ey-= (0.5*dy/dx)*vecC[EPS_down].eps*(Phi[TID]-Phi[Phi_right])*idx;
			vecG[TID].Ey-= (0.5*dy/dx)*vecC[EPS_cross].eps*(Phi[TID]-Phi[Phi_left])*idx;
			vecG[TID].Ey/= 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps);
		}
	}
	else if(vecG[TID].Face == DOWN) {
		if(vecG[TID].Boundary==CONDUCTOR ||  vecG[TID].Boundary==NEUMANN || vecG[TID].Boundary==DIRICHLET) {
			vecG[TID].Ex = 0.0;
			vecG[TID].Ey = -2.0*Sigma[TID]/(vecC[EPS_down].eps+vecC[EPS_cross].eps);
		}
		else if(vecG[TID].Boundary==DIELECTRIC || vecG[TID].Boundary>=100) {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = idy*(Phi[Phi_down]-Phi[TID]);
		}
		else {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = 0.5*(vecC[EPS_TID].eps+vecC[EPS_left].eps)*(Phi[Phi_up]-Phi[TID])*idy;
			vecG[TID].Ey-= (0.5*dy/dx)*vecC[EPS_TID].eps*(Phi[TID]-Phi[Phi_right])*idx;
			vecG[TID].Ey-= (0.5*dy/dx)*vecC[EPS_left].eps*(Phi[TID]-Phi[Phi_left])*idx;
			vecG[TID].Ey/= -0.5*(vecC[EPS_down].eps+vecC[EPS_cross].eps);
		}
	}
	else if(vecG[TID].Face == UL_CORN) {
		if(vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET || vecG[TID].Boundary==DIELECTRIC) {
			vecG[TID].Ex = idx*(Phi[Phi_left]-Phi[TID]);
			vecG[TID].Ey = idy*(Phi[TID]-Phi[Phi_up]);
		}
		else {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
	}
	else if(vecG[TID].Face == UR_CORN) {
		if(vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET || vecG[TID].Boundary==DIELECTRIC) {
			vecG[TID].Ex = idx*(Phi[TID]-Phi[Phi_right]);
			vecG[TID].Ey = idy*(Phi[TID]-Phi[Phi_up]);
		}
		else {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
	}
	else if(vecG[TID].Face == LL_CORN) {
		if(vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET || vecG[TID].Boundary==DIELECTRIC) {
			vecG[TID].Ex = idx*(Phi[Phi_left]-Phi[TID]);
			vecG[TID].Ey = idy*(Phi[Phi_down]-Phi[TID]);
		}
		else {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
	}
	else if(vecG[TID].Face == LR_CORN) {
		if(vecG[TID].Boundary==CONDUCTOR || vecG[TID].Boundary==DIRICHLET || vecG[TID].Boundary==DIELECTRIC) {
			vecG[TID].Ex = idx*(Phi[TID]-Phi[Phi_right]);
			vecG[TID].Ey = idy*(Phi[Phi_down]-Phi[TID]);
		}
		else {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
	}
	else if(vecG[TID].Boundary==NEUMANN) {
		if(x==0 || x==ngx-1) {
			vecG[TID].Ex = 0.0;
			vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
		}
		else if(y==0 || y==ngy-1) {
			vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
			vecG[TID].Ey = 0.0;
		}
	}
	else {
		vecG[TID].Ex = hdx*(Phi[Phi_left]-Phi[Phi_right]);
		vecG[TID].Ey = hdy*(Phi[Phi_down]-Phi[Phi_up]);
	}
}
void transpose(float **matrix, float matrix_cofactor[N_MAX][N_MAX],
		float size) {
	int i, j;
	float m_transpose[N_MAX][N_MAX], m_inverse[N_MAX][N_MAX], d;

	for (i = 0; i < size; i++) {
		for (j = 0; j < size; j++) {
			m_transpose[i][j] = matrix_cofactor[j][i];
		}
	}
	d = determinant(matrix, size);
	for (i = 0; i < size; i++) {
		for (j = 0; j < size; j++) {
			m_inverse[i][j] = m_transpose[i][j] / d;
		}
	}

	for (i = 0; i < size; i++) {
		for (j = 0; j < size; j++) {
			matrix[i][j] = m_inverse[i][j];
		}
	}
}
float determinant(float **matrix, float size) {
	float s = 1, det = 0;
	float **m_minor;
	int i, j, m, n, c;

	if (size == 1) {
		return (matrix[0][0]);
	} else {
		m_minor = (float **) malloc(size * sizeof(float *));
		for (i = 0; i < size; i++)
			m_minor[i] = (float *) malloc(size * sizeof(float));
		det = 0;
		for (c = 0; c < size; c++) {
			m = 0;
			n = 0;
			for (i = 0; i < size; i++) {
				for (j = 0; j < size; j++) {
					m_minor[i][j] = 0;
					if (i != 0 && j != c) {
						m_minor[m][n] = matrix[i][j];
						if (n < (size - 2))
							n++;
						else {
							n = 0;
							m++;
						}
					}
				}
			}
			det = det + s * (matrix[0][c] * determinant(m_minor, size - 1));
			s = -1 * s;
		}
	}
	for (i = 0; i < size; i++)
		free(m_minor[i]);
	free(m_minor);
	return (det);
}
void cofactor(float **matrix, float size) {
	float matrix_cofactor[N_MAX][N_MAX];
	float **m_cofactor;
	int p, q, m, n, i, j;

	m_cofactor = (float **) malloc(size * sizeof(float *));
	for (i = 0; i < size; i++)
		m_cofactor[i] = (float *) malloc(size * sizeof(float));

	for (q = 0; q < size; q++) {
		for (p = 0; p < size; p++) {
			m = 0;
			n = 0;
			for (i = 0; i < size; i++) {
				for (j = 0; j < size; j++) {
					if (i != q && j != p) {
						m_cofactor[m][n] = matrix[i][j];
						if (n < (size - 2))
							n++;
						else {
							n = 0;
							m++;
						}
					}
				}
			}
			matrix_cofactor[q][p] = pow(-1, q + p) * determinant(m_cofactor, size - 1);
		}
	}
	transpose(matrix, matrix_cofactor, size);
	for (i = 0; i < size; i++)
		free(m_cofactor[i]);
	free(m_cofactor);
}
__global__ void PCG_float2(int *I, int *J, float *val, float *x, float *M, float *Ax, float *p, float *r, float *Z, 
            int N, int nnz, float tol2, int *Iter, float *d_result){
    //Jacovi diagonal preconditioner version
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    //int TID = blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    int max_iter = 100000;
    float a = 1.0f;
    float na = -1.0f;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;
    rsold = 0.0f;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *Iter = 0;
        *d_result = 0.0f;  
    } 
    Mat_x_Vec(I, J, val, nnz, N, a, x, Ax, cta, grid); 
    A_x_X_p_Y(na, Ax, r, N, grid); 
    Vec_x_Vec(M, r, Z, N, cta, grid);
    CopyVector(Z, p, N, grid);
    cg::sync(grid);
    Vec_Dot_Sum_F(r, Z, d_result, N, cta, grid); 
    cg::sync(grid);
    rsold = *d_result;
    while (rsold > tol2 && *Iter <= max_iter){
        Mat_x_Vec(I, J, val, nnz, N, a, p, Ax, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *Iter = *Iter + 1;
            *d_result = 0.0f;  
        }
        cg::sync(grid);
        Vec_Dot_Sum_F(p, Ax, d_result, N, cta, grid);
        cg::sync(grid);
        Temp = *d_result;
        alpha = (Temp)? rsold/Temp:0.0f;
		A_x_X_p_Y(alpha, p, x, N, grid);
        nalpha = -alpha;
        A_x_X_p_Y(nalpha, Ax, r, N, grid);
        Vec_x_Vec(M, r, Z, N, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0f;  
        } 
        cg::sync(grid);
        Vec_Dot_Sum_F(r, Z, d_result, N, cta, grid);
        cg::sync(grid);
        rnew = *d_result;
        beta = (rsold) ? rnew/rsold: 0.0f;
        A_x_Y_p_X(beta, Z, p, N, grid);
        rsold = rnew;
        //rnew = 0.0;
    }
    //if(threadIdx.x == 0 && blockIdx.x == 0 ) printf("End Iter = %d, Res = %g, b = %g, a = %g\n",*Iter,Temp,alpha,beta,rsold);
}
__global__ void PCG_float(int *I, int *J, float *val, float *x, float *M, float *Ax, float *p, float *r, float *Z, 
            int N, int nnz, float tol2, int *Iter, float *d_result){
    //Jacovi diagonal preconditioner version
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    int max_iter = 100000;
    float a = 1.0f;
    float na = -1.0f;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;
    rsold = 0.0f;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *Iter = 0;
    } 
    Mat_x_Vec(I, J, val, nnz, N, a, x, Ax, cta, grid); 
    A_x_X_p_Y(na, Ax, r, N, grid); 
    Vec_x_Vec(M, r, Z, N, cta, grid);
    CopyVector(Z, p, N, grid);
    cg::sync(grid);
    Vec_Dot_Sum_F(r, Z, d_result, N, cta, grid); 
    cg::sync(grid);
    rsold = *d_result;

	float p_local, r_local, z_local;
	int iter_local = 0;
	
	int cnt;

	float val_reg_0, val_reg_1, val_reg_2, val_reg_3, val_reg_4;
	int J_reg_0, J_reg_1, J_reg_2, J_reg_3, J_reg_4;
	float M_reg_0;

	cnt = 0;
	for (int i=grid.thread_rank(); i < N; i+= grid.size())
	{
		if(cnt == 0)
		{
			M_reg_0 = M[i];
		}
		++cnt;
	}

	cnt = 0;
	for (int i=grid.thread_rank(); i < N; i+= grid.size())    
	{
		int row_elem = I[i];
		int next_row_elem = I[i+1];
		float output = 0.0;
		for (int j=row_elem-1; j < next_row_elem-1; j++){
			if(cnt == 0)
			{
				val_reg_0 = val[j];
				J_reg_0 = J[j];
			}
			else if(cnt == 1)
			{
				val_reg_1 = val[j];
				J_reg_1 = J[j];
			}
			else if(cnt == 2)
			{
				val_reg_2 = val[j];
				J_reg_2 = J[j];
			}
			else if(cnt == 3)
			{
				val_reg_3 = val[j];
				J_reg_3 = J[j];
			}
			else if(cnt == 4)
			{
				val_reg_4 = val[j];
				J_reg_4 = J[j];
			}
			++cnt;
		}
	}

    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
	
	Mat_x_Vec(I, J, val, nnz, N, a, p, Ax, cta, grid);
	iter_local++;
	Vec_Dot_Sum_F(p, Ax, d_result + iter_local * 2 - 1, N, cta, grid);
	cg::sync(grid);

    while(rsold > tol2 && iter_local <= max_iter)
	{
        Temp = d_result[iter_local * 2 - 1];
        alpha = (Temp)? rsold/Temp:0.0f;
		nalpha = -alpha;

        //A_x_X_p_Y(nalpha, Ax, r, N, grid);
        //Vec_x_Vec(M, r, Z, N, cta, grid);
		//Vec_Dot_Sum_F(r, Z, d_result + 2 * iter_local, N, cta, grid);
		{
			float temp_sum = 0.0f;
			cnt = 0;
			for (int i=grid.thread_rank(); i < N; i+= grid.size())
			{
				r_local = nalpha*Ax[i] + r[i];
				if(cnt == 0)
				{
					z_local = M_reg_0 * r_local;
				}
				else
				{
					z_local = M[i] * r_local;
				}
				temp_sum += z_local * r_local;
				r[i] = r_local;
				Z[i] = z_local;
			}
			for (int offset = 16; offset > 0; offset /= 2)
				temp_sum += __shfl_down_sync(0xffffffff, temp_sum, offset);
			if (tile32.thread_rank() == 0) 
			{
				atomicAdd(d_result + 2 * iter_local, temp_sum);
			}
		}

		cg::sync(grid);
        rnew = d_result[2 * iter_local];
        beta = (rsold) ? rnew/rsold: 0.0f;
		//A_x_X_p_Y(alpha, p, x, N, grid);
		//A_x_Y_p_X(beta, Z, p, N, grid);
        //Mat_x_Vec(I, J, val, nnz, N, a, p, Ax, cta, grid);
		{
			for (int i=grid.thread_rank(); i < N; i+= grid.size()) 
			{
				p_local = p[i];
				x[i] = alpha*p_local + x[i];
				p[i] = beta*p_local + Z[i];
			}
		}
		rsold = rnew;
		{
			cnt = 0;
			for (int i=grid.thread_rank(); i < N; i+= grid.size())
			{
				int row_elem = I[i];
				int next_row_elem = I[i+1];
				float output = 0.0;
				for (int j=row_elem-1; j < next_row_elem-1; j++){
					if(cnt == 0)
					{
						output +=  val_reg_0 * p[J_reg_0-1];
					}
					else if(cnt == 1)
					{
						output +=  val_reg_1 * p[J_reg_1-1];
					}
					else if(cnt == 2)
					{
						output +=  val_reg_2 * p[J_reg_2-1];
					}
					else if(cnt == 3)
					{
						output +=  val_reg_3 * p[J_reg_3-1];
					}
					else if(cnt == 4)
					{
						output +=  val_reg_4 * p[J_reg_4-1];
					}
					else
					{
						output += val[j] * p[J[j]-1];
					}
					cnt++;
				}
				Ax[i] = a * output;
			}
		}
		iter_local++;
        //Vec_Dot_Sum_F(p, Ax, d_result + iter_local * 2 - 1, N, cta, grid);
		{
			float temp_sum = 0.0f;
			cnt = 0;
			for (int i=grid.thread_rank(); i < N; i+= grid.size())
			{
				temp_sum += Ax[i] * p[i];
			}
			for (int offset = 16; offset > 0; offset /= 2)
				temp_sum += __shfl_down_sync(0xffffffff, temp_sum, offset);
			if (tile32.thread_rank() == 0) 
			{
				atomicAdd(d_result + 2 * iter_local - 1, temp_sum);
			}
		}
        cg::sync(grid);
    }
	if (threadIdx.x == 0 && blockIdx.x == 0){
		d_result[0] = d_result[2 * iter_local - 2];
		*Iter = iter_local;
	}
}
__device__ void Vec_Dot_Sum_F(float *vecA, float *vecB, float *result, int size, const cg::thread_block &cta, const cg::grid_group &grid)
{
   __shared__ float tmp[THREADS_PER_BLOCK];
    float temp_sum = 0.0;
    for (int i=grid.thread_rank(); i < size; i+=grid.size()){
        temp_sum += (float) (vecA[i] * vecB[i]);
    }
    tmp[cta.thread_rank()] = temp_sum;
    cg::sync(cta);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
    float beta  = temp_sum;
    float temp;
    for (int i = tile32.size() / 2; i > 0; i >>= 1) {
        if (tile32.thread_rank() < i) {
            temp       = tmp[cta.thread_rank() + i];
            beta       += temp;
            tmp[cta.thread_rank()] = beta;
        }
        cg::sync(tile32);
    }
    cg::sync(cta);
    if (cta.thread_rank() == 0) {
        beta  = 0.0;
        for (int i = 0; i < cta.size(); i += tile32.size()) {
            beta  += tmp[i];
        }
        atomicAdd(result, beta);
    }
}