#include "hip/hip_runtime.h"
#include "cuda_Field.cuh"
//
void PCG_SOLVER_Laplace(){
    int i,j,k,TID;
    // OUTPUT
    // Lap_TEMP_Sol[Gsize] : Temperature Profile
    // Lap_PHI_Sol[CondNUMR][Gsize] : Each of conductor Phi Profile, This is Device value
    // Lap_SIG_Sol[CondNUMR][CondNUMR] : Each of conductor Sigma Profile for external circuit
    void *kernelArgs[] = {
        (void*)&dev_Ai,
        (void*)&dev_Aj,
        (void*)&dev_A,
        (void*)&dev_X,
        (void*)&dev_M,
        (void*)&dev_AP,
        (void*)&dev_P,
        (void*)&dev_R,
        (void*)&dev_Z,
        (void*)&N,
        (void*)&nz,
        (void*)&PCGtol2,
        (void*)&FIter,
        (void*)&dot_result,
    };
    for (k = 0; k < CondNUMR; k++) {
        checkCudaErrors(hipMemcpy(dev_R, cond_b[k], N * sizeof(float),hipMemcpyHostToDevice));
        checkCudaErrors(hipMemset((void *) dev_X, 0, N * sizeof(float)));
        checkCudaErrors(hipLaunchCooperativeKernel((void *)PCG,FIELD_GRID,FIELD_BLOCK, kernelArgs, sMemSize, NULL));
        checkCudaErrors(hipDeviceSynchronize());
        printf(" Laplace Solution %d",k);
        printf(" : Conductor %d = 1 V, Other CondUCTOR = 0 V\n",k);
        printf(" - Iter = %d (ms), rsold^2 = %g\n",*FIter,*dot_result);

    }

    // Laplace Solution
    //hipMallocPitch(&Lap_PHI_Sol, &pitch, Gsize * sizeof(float), CondNUMR); // for Laplace Solution
    //hipMalloc((void**) &Lap_TEMP_Sol, Gsize * sizeof(int));
    // hipMemset((void *) array, 0, Gsize * sizeof(int));

    exit(1);
}
void Set_MatrixPCG_cuda(){
    int i,j;
    N = A_size;
    nz = 5 * A_size;
    printf(" Field Solver : [GPU] Preconditioned Conjugate Gradient\n"); 
    printf(" Laplace Equation\n"); 
    printf(" Matrix Size = %d, ngx x ngy = %d X %d = %d\n", N, ngx, ngy, Gsize);
    // Allocate
    checkCudaErrors(hipMalloc((void**) &dev_A, nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_Aj, nz * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_Ai, (N + 1) * sizeof(int)));
	checkCudaErrors(hipMalloc((void**) &dev_b,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_b, 0.0, N * sizeof(float)));
	checkCudaErrors(hipMalloc((void**) &dev_X,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_X, 0.0, N * sizeof(float)));
    //
    checkCudaErrors(hipMalloc((void**) &dev_TA, nz * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Tb,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_Tb, 0.0, N * sizeof(float)));
    //
    checkCudaErrors(hipMalloc((void**) &dev_AP,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_AP, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_R,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_R, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_P,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_P, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_M,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_M, 0.0, N * sizeof(float)));
    checkCudaErrors(hipMalloc((void**) &dev_Z,  N * sizeof(float)));
    checkCudaErrors(hipMemset((void *) dev_Z, 0.0, N * sizeof(float)));
    // Data cpu > gpu
    checkCudaErrors(hipMemcpy(dev_A, A_val, nz * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_TA,TA_val,nz*sizeof(float),hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Aj, Aj, nz * sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_Ai, Ai, (N + 1) * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_Tb,temp_b, N * sizeof(float),hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_M, MatM, N * sizeof(float), hipMemcpyHostToDevice));
    //Unified memory value for Field residual
    hipMallocManaged((void **)&dot_result, sizeof(double));
    *dot_result = 0.0;
    hipMallocManaged((void **)&FIter, sizeof(int));
    *FIter = 0;
    //
    vec_A_idx = (int *) malloc(Gsize * sizeof(int));
    for (i = 0; i < ngx; i++) {
		for (j = 0; j < ngy; j++) {
			vec_A_idx[j + i * ngy] = A_idx[i][j];
		} 
	}
    checkCudaErrors(hipMalloc((void**) &dev_A_idx, Gsize * sizeof(int)));
    checkCudaErrors(hipMemcpy(dev_A_idx, A_idx, Gsize * sizeof(int),hipMemcpyHostToDevice));
}
__device__ void Mat_x_Vec(int *I, int *J, float *val, int nnz, int num_rows, float alpha, float *inputVecX, 
                        float *outputVecY, cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < num_rows; i+= grid.size())    {
        int row_elem = I[i];
        int next_row_elem = I[i+1];
        int num_elems_this_row = next_row_elem - row_elem;
        float output = 0.0;
        for (int j=row_elem-1; j < next_row_elem-1; j++){
            //if(i==0) printf("val[%d][]\n",j);
            // I or J or val arrays - can be put in shared memory 
            // as the access is random and reused in next calls of gpuSpMV function.
            output +=  alpha*val[j] * inputVecX[J[j]-1];
            //if(i==0) printf("val[%d][%d] = %g, %g, %g\n",j,J[j]-1,val[j],inputVecX[J[j]-1],output);
        }
        outputVecY[i] = output;
    }
}
__device__ void A_x_X_p_Y(float a, float *x, float *y, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) y[i] = a*x[i] + y[i];
}
__device__ void A_x_Y_p_X(float a, float *x, float *y, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) y[i] = a*y[i] + x[i];
}
__device__ void Vec_Dot_Sum(float *vecA, float *vecB, double *result, int size, const cg::thread_block &cta, const cg::grid_group &grid)
{
   __shared__ double tmp[THREADS_PER_BLOCK];
    double temp_sum = 0.0;
    for (int i=grid.thread_rank(); i < size; i+=grid.size()){
        temp_sum += (double) (vecA[i] * vecB[i]);
    }
    tmp[cta.thread_rank()] = temp_sum;
    cg::sync(cta);
    cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
    double beta  = temp_sum;
    double temp;
    for (int i = tile32.size() / 2; i > 0; i >>= 1) {
        if (tile32.thread_rank() < i) {
            temp       = tmp[cta.thread_rank() + i];
            beta       += temp;
            tmp[cta.thread_rank()] = beta;
        }
        cg::sync(tile32);
    }
    cg::sync(cta);
    if (cta.thread_rank() == 0) {
        beta  = 0.0;
        for (int i = 0; i < cta.size(); i += tile32.size()) {
            beta  += tmp[i];
        }
        atomicAdd(result, beta);
    }
}
__device__ void CopyVector(float *srcA, float *destB, int size, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+= grid.size()) destB[i] = srcA[i];
}
__device__ void Vec_x_Vec(float *vecA, float *vecB, float *vecC, int size, const cg::thread_block &cta, const cg::grid_group &grid){
    for (int i=grid.thread_rank(); i < size; i+=grid.size()) vecC[i] = (vecA[i] * vecB[i]);
}
__global__ void PCG(int *I, int *J, float *val, float *x, float *M, float *Ax, float *p, float *r, float *Z, 
            int N, int nnz, float tol2, int *Iter, double *d_result){
    //Jacovi diagonal preconditioner version
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    //int TID = blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;
    int max_iter = 100000;
    float a = 1.0;
    float na = -1.0;
    float rsold,rnew,Temp;
    float nalpha,alpha,beta;
    rsold = 0.0;
    if (threadIdx.x == 0 && blockIdx.x == 0){
        *Iter = 0;
        *d_result = 0.0;  
    } 
    Mat_x_Vec(I, J, val, nnz, N, a, x, Ax, cta, grid); 
    A_x_X_p_Y(na, Ax, r, N, grid); 
    Vec_x_Vec(M, r, Z, N, cta, grid);
    CopyVector(Z, p, N, grid);
    //if(r[TID] !=0) printf("r[%d] = %g\n",TID,r[TID]);
    cg::sync(grid);
    Vec_Dot_Sum(r, Z, d_result, N, cta, grid); 
    cg::sync(grid);
    rsold = *d_result;
    //if(threadIdx.x == 0 && blockIdx.x == 0) printf("First:rsold = %g N = %d, nnz = %d\n",rsold,N,nnz);
    //return;
    while (rsold > tol2 && *Iter <= max_iter){
        Mat_x_Vec(I, J, val, nnz, N, a, p, Ax, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *Iter = *Iter + 1;
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        //if(Ax[TID] !=0) printf("Ax[%d] = %g\n",TID,Ax[TID]);
        Vec_Dot_Sum(p, Ax, d_result, N, cta, grid);
        cg::sync(grid);
        Temp = *d_result;
        //if(threadIdx.x == 0 && blockIdx.x == 0) printf("Temp = %g\n",Temp);
        //return;
        alpha = (Temp)? rsold/Temp:0.0f;
        A_x_X_p_Y(alpha, p, x, N, grid);
        nalpha = -alpha;
        A_x_X_p_Y(nalpha, Ax, r, N, grid);
        Vec_x_Vec(M, r, Z, N, cta, grid);
        if (threadIdx.x == 0 && blockIdx.x == 0){
            *d_result = 0.0;  
        } 
        cg::sync(grid);
        Vec_Dot_Sum(r, Z, d_result, N, cta, grid);
        cg::sync(grid);
        rnew = *d_result;
        beta = (rsold) ? rnew/rsold: 0.0f;
        A_x_Y_p_X(beta, Z, p, N, grid);
        rsold = rnew;
        rnew = 0.0;
        //if(threadIdx.x == 0 && blockIdx.x == 0 && k<20) printf("Iter = %d, temp = %g,  AL = %g, BE = %g Res = %g\n",k,Temp,alpha,beta,rsold);
    }
    //if(threadIdx.x == 0 && blockIdx.x == 0 ) printf("End Iter = %d, Res = %g, b = %g, a = %g\n",k,Temp,alpha,beta,rsold);
}
__global__ void SaveAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable 
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	row[TID]=PHI[TID];
}
__global__ void LoadAT2D(float *A, size_t pitch, int height, float *PHI, int n){
    // High save and load for Matrix type variable
	int TID=blockDim.x*(gridDim.x*blockIdx.y+blockIdx.x)+threadIdx.x;

	if(TID>=n) return;

	float *row=(float *)((char *)A+height*pitch);

	PHI[TID]=row[TID];
}