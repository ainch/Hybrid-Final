#include "hip/hip_runtime.h"
#include "cuda_Move.cuh"

void Move_cuda() {
    MoveE_Basic<<<MOVE_GRID, MOVE_BLOCK>>>(Gsize, ngy, dt_dx, dt_dy, dev_info_sp, dev_sp, dev_G_sp, dev_GvecSet);
	hipDeviceSynchronize();
}
__global__ void MoveE_Basic(int Gsize,int ngy, float dt_dx,float dt_dy, Species *info, GCP *sp, GPG *data, GGA *Field){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
	int PNC,isp,ID;
	isp = (int)TID/Gsize; //species number [< nsp]
    ID = (int)TID%Gsize; // Grid ID [< Gsize]
    if(TID>Gsize*info[isp].spnum) return;
	
    PNC = data[TID].PtNumInCell;
    if(PNC==0){
        data[TID].PtNumMoveInterCell=0;
        return;
    } 
	float ex_ws,ex_wn,ex_es,ex_en;
	float ey_ws,ey_wn,ey_es,ey_en;
	float lx,ly;
	int MPNC;
	int PNMC,index;
	int i,k,xp;
	float mvX,mvY,mvZ;
	float del_vx,del_vy;
    float id_cell;
	
	PNMC=0;
	
	ex_ws=Field[ID].Ex; 
	ex_wn=Field[ID+1].Ex; 
	ex_es=Field[ID+ngy].Ex; 
	ex_en=Field[ID+ngy+1].Ex;
	ey_ws=Field[ID].Ey; 
	ey_wn=Field[ID+1].Ey; 
	ey_es=Field[ID+ngy].Ey; 
	ey_en=Field[ID+ngy+1].Ey;

	MPNC = data[TID].MaxPtNumInCell;
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
		lx=sp[i].x; 
		ly=sp[i].y;
		del_vx=ex_ws*(1-lx)*(1-ly)+ex_wn*(1-lx)*ly+ex_es*lx*(1-ly)+ex_en*lx*ly;
		del_vy=ey_ws*(1-lx)*(1-ly)+ey_wn*(1-lx)*ly+ey_es*lx*(1-ly)+ey_en*lx*ly;

		mvX=sp[i].vx+del_vx*info[isp].Ascale;
		mvY=sp[i].vy+del_vy*info[isp].Ascale;
		mvZ=sp[i].vz;

		lx+=mvX*dt_dx;
		ly+=mvY*dt_dy;

        if(ly>=1 || ly<0 || lx>=1 || lx<0){ // out of cell
            PNMC++;
            index = info[isp].St_num + ID + (MPNC-PNMC)*Gsize;
			id_cell = 0.0f;
            if(ly>=1){								//top
				id_cell+=1;
				ly-=1.0;
			}
			else if(ly<0){							//bottom
				id_cell-=1;
				ly+=1.0;
			}
			if(lx>=1){								//right
				id_cell+=ngy;
				lx-=1.0;
			}
			else if(lx<0){							//left
				id_cell-=ngy;
				lx+=1.0;
			}
            sp[index].CellID = id_cell;
        }else{
            index = i-PNMC*Gsize;
        }
        sp[index].vx=mvX;
		sp[index].vy=mvY;
		sp[index].vz=mvZ;
        sp[index].x=lx;
		sp[index].y=ly;
		if(sp[index].vx==0) printf("B[%g]->A[%g]\n",sp[i].x,sp[index].x);
        i+=Gsize;
    }
	data[TID].PtNumMoveInterCell=PNMC;
	data[TID].PtNumInCell-=PNMC;

}