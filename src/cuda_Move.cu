#include "hip/hip_runtime.h"
#include "cuda_Move.cuh"

void Move_cuda() {
    MoveE<<<MOVE_GRID, MOVE_BLOCK>>>(Gsize, ngy, dt_dx, dt_dy, SP, dev_sp, dev_G_sp, dev_GvecSet);
					
}
__global__ void MoveE(int Gsize,int ngy,float dt_dx,float dt_dy, Species *info, GCP *sp, GPG *data, GGA *Field){
    int TID = threadIdx.x + blockIdx.x * blockDim.x;
	int PNC,isp,ID;
    if(TID>Gsize*info[0].spnum) return;
    PNC = data[TID].PtNumInCell;
    if(PNC==0){
        data[TID].PtNumMoveInterCell=0;
        return;
    } 
    isp = (int)TID/Gsize; //species number [< nsp]
    ID = (int)TID%Gsize; // Grid ID [< Gsize]

	float ex_ws,ex_wn,ex_es,ex_en;
	float ey_ws,ey_wn,ey_es,ey_en;
	float lx,ly;
	int MPNC;
	int PNMC,index;
	int i,k,xp;
	float mvX,mvY,mvZ;
	float del_vx,del_vy;
    float id_cell;

	PNMC=0;

	ex_ws= Field[ID].Ex; ex_wn=Field[ID+1].Ex; ex_es=Field[ID+ngy].Ex; ex_en=Field[ID+ngy+1].Ex;
	ey_ws=Field[ID].Ey; ey_wn=Field[ID+1].Ey; ey_es=Field[ID+ngy].Ey; ey_en=Field[ID+ngy+1].Ey;

	MPNC = data[TID].MaxPtNumInCell;
	i = info[isp].St_num + ID;

	for(k=0;k<PNC;k++){
		lx=sp[i].x; ly=sp[i].y;
		//weighting
		del_vx=ex_ws*(1-lx)*(1-ly)+ex_wn*(1-lx)*ly+ex_es*lx*(1-ly)+ex_en*lx*ly;
		del_vy=ey_ws*(1-lx)*(1-ly)+ey_wn*(1-lx)*ly+ey_es*lx*(1-ly)+ey_en*lx*ly;

		mvX=sp[i].vx+del_vx*info[isp].Ascale;
		mvY=sp[i].vy+del_vy*info[isp].Ascale;
		mvZ=sp[i].vz;

		lx+=mvX*dt_dx;
		ly+=mvY*dt_dy;
        if(ly>=1 || ly<0 || lx>=1 || lx<0){ // out of cell
            PNMC++;
            index = TID+(MPNC-PNMC)*Gsize;
			id_cell = 0.0f;
            if(ly>=1){								//top
				id_cell+=1;
				ly-=1.0;
			}
			else if(ly<0){							//bottom
				id_cell-=1;
				ly+=1.0;
			}
			if(lx>=1){								//right
				id_cell+=ngy;
				lx-=1.0;
			}
			else if(lx<0){							//left
				id_cell-=ngy;
				lx+=1.0;
			}
            sp[index].CellID = id_cell;
        }else{
            index = i-PNMC*Gsize;
        }
        sp[index].vx=mvX;
		sp[index].vy=mvY;
		sp[index].vz=mvZ;
        sp[index].x=lx;
		sp[index].y=ly;
        i+=Gsize;
    }
	data[TID].PtNumMoveInterCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}