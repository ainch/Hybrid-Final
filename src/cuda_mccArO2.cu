#include "hip/hip_runtime.h"
#include "cuda_mccArO2.cuh"
__device__ void ArO2_Electron(int Gsize, int ngy, int ID, int MCCn, float dtm, float dx, float dy, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, GGA *BG, GFC *Fluid){
/*
	int i,j,k,n,index;
	int CID,PNC,PNC2;
	int nx,ny,ngx;
	int Target,oldPNC;
	int Colltype;
	float Tprob,Prob1,Prob2;
    int Randn,AddPt1;
	float R1,R2;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,vel2,engy,rengy;
	float SumSigma,SumEngyLoss;
	PNC = data[ID].PtNumInCell;
    PNC2 = data[ID+Gsize].PtNumInCell;
	hiprandState LocalStates = states[ID];
    nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);
	Prob2 = 1.0f - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);
	Tprob = Prob1 + Prob2;
    // Calculate total Collision probability.
	Randn = MCCn;
    AddPt1 = 0;
	i = info[0].St_num + ID;
	for(k=0;k<PNC;k++){
        Colltype = 0;
		for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob){
                Colltype = 1;
                break;
            }
		}
        if(Colltype == 0){
            i+=Gsize;
			continue;
        }
		R1 = Tprob * R1;
        if(R1 <= Prob1)	Target = (int)0;
		else			Target = (int)1;
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 2;
        switch(Target){
			case 0:{
                R2 = hiprand_uniform(&LocalStates) * sigv[0].val / vel;
				// 0. e + Ar > e + Ar 			Elastic Scattering
				SumSigma = Argon_CrossSection(0, engy, N_LOGX, idLOGX, CX);
				if(R2<=SumSigma){
				// 1. e + Ar > e + Ar* 			Excitation to Total Excited state
				}else if(engy > info_CX[1].Th_e && R2<=(SumSigma += Argon_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
				// 2. e + Ar > e + Ar* 			Excitation to AR4SM
				}else if(engy > info_CX[2].Th_e && R2<=(SumSigma += Argon_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
				// 3. e + Ar > e + e + Ar^		Direct ionization
				}else if(engy > info_CX[3].Th_e && R2<=(SumSigma += Argon_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					AddPt1++;
				}else{
					Colltype = 1;
				}
				break;
			}
			case 1:{
                R2 = hiprand_uniform(&LocalStates)*sigv[1].val / vel;
				// 4. e + Ar* > e + e + Ar^		step ionization
				SumSigma = Argon_CrossSection(4, engy, N_LOGX, idLOGX, CX);
				if(engy > info_CX[4].Th_e && R2<=SumSigma){
					Colltype = 3;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					AddPt1++;
				}else{
					Colltype = 1;
				}
				break;
			}
		}
        if(Colltype == 2){ // Just energy loss
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			sp[i].vx = VX;
			sp[i].vy = VY;
			sp[i].vz = VZ;
		}else if(Colltype == 3){ //ionization 
			//printf("Ionization 1 ! \n");
            ///// scatter the created electron
			index = info[0].St_num + ID + (PNC + AddPt1 - 1) * Gsize; 
			sp[index].CellID = sp[i].CellID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            ///// assign velocities to the created ion
            index = info[1].St_num + ID + (PNC2 + AddPt1 - 1) * Gsize; 
            sp[index].CellID = sp[i].CellID + Gsize;
            sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
            sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			///// scatter the incident electron
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,info_CX[4].mofM,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            sp[i].vx = VX;
			sp[i].vy = VY;
			sp[i].vz = VZ;
			//AddPt1--;
		}
		i+=Gsize;
	}
    data[ID].PtNumInCell = PNC + AddPt1;
    data[ID+Gsize].PtNumInCell = PNC2 + AddPt1;
	states[ID]=LocalStates;
    */
}
__device__ void ArO2_ArIon(int Gsize, int ngy, int ID, int MCCn, float dt, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, GGA *BG, GFC *Fluid){
	/*
    int i,j,k,n,index;
	int PNC;
	float Prob;
	float R1;
	float VX,VY,VZ,VX_buf,VY_buf,VZ_buf;
	float dum,vel,engy;
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

    PNC = data[ID+Gsize].PtNumInCell;
	hiprandState LocalStates = states[ID];
	Prob = 1.0f - exp(-1*dt*sigv[2].val*BG[ID].BackDen1);

    // Calculate total Collision probability.
	i = info[1].St_num + ID;
	for(k=0;k<PNC;k++){
		R1 = hiprand_uniform(&LocalStates);
		if(R1>Prob){
			i+=Gsize;
			continue;
		} 
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX=sp[i].vx-vneutx;
	  	VY=sp[i].vy-vneutz;
	  	VZ=sp[i].vz-vneuty;
		dum=VX*VX+VY*VY+VZ*VZ;
		engy=info[1].Escale*dum;
	  	vel=sqrt(dum);
		R1 = hiprand_uniform(&LocalStates) * sigv[2].val / vel;
		SumSigma = Argon_CrossSection(5, engy, N_LOGX, idLOGX, CX);
		if(R1<=SumSigma){
			// 5. Ar + Ar^ > Ar + Ar^		Charge Exchange
			VX_buf = vneutx;
			VY_buf = vneuty;
			VZ_buf = vneutz;
		}else if(R1<=(SumSigma += Argon_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
			// 6. AR + AR^ > AR + AR^		ELASTIC SCATTERING
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			VX_buf = VX+vneutx;
			VY_buf = VY+vneuty;
			VZ_buf = VZ+vneutz;
		}else{
			VX_buf = sp[i].vx;
			VY_buf = sp[i].vy;
			VZ_buf = sp[i].vz;
		}
		sp[i].vx = VX_buf;
		sp[i].vy = VY_buf;
		sp[i].vz = VZ_buf;	
		i+=Gsize;
	}
	states[ID]=LocalStates;
    */
}
__device__ float ArO2_CrossSection(int R, float engy, int N_LOGX, float idLOGX, ArO2CollD *data){
	if(engy == 0) return 0.0;
	float lengy = log10(engy);
	float ee1, a1, a2;
	int ee2;
    lengy = lengy - data[0].xe;
	ee1 = idLOGX * lengy;
	ee2 = (int)ee1;
	a1 = ee1 - ee2;
	a2 = 1 - a1;
	switch (R) {
        case 0 : 
            if(lengy < data[0].xe){
			    return data[0].cx_0;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_0 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_0+a1*data[ee2+1].cx_0;
            break;
        case 1 :
            if(lengy < data[0].xe){
			    return data[0].cx_1;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_1 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_1+a1*data[ee2+1].cx_1;
            break;
        case 2 :
            if(lengy < data[0].xe){
			    return data[0].cx_2;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_2 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_2+a1*data[ee2+1].cx_2;
            break;
        case 3 :
            if(lengy < data[0].xe){
			    return data[0].cx_3;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_3 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_3+a1*data[ee2+1].cx_3;
            break;
        case 4 :
            if(lengy < data[0].xe){
			    return data[0].cx_4;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_4 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_4+a1*data[ee2+1].cx_4;
            break;
        case 5 :
            if(lengy < data[0].xe){
			    return data[0].cx_5;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_5 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_5+a1*data[ee2+1].cx_5;
            break;
        case 6 :
            if(lengy < data[0].xe){
			    return data[0].cx_6;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_6 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_6+a1*data[ee2+1].cx_6;
            break;
        case 7 :
            if(lengy < data[0].xe){
			    return data[0].cx_7;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_7 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_7+a1*data[ee2+1].cx_7;
            break;
        case 8 :
            if(lengy < data[0].xe){
			    return data[0].cx_8;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_8 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_8+a1*data[ee2+1].cx_8;
            break;
        case 9 :
            if(lengy < data[0].xe){
			    return data[0].cx_9;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_9 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_9+a1*data[ee2+1].cx_9;
            break;
        case 10 :
            if(lengy < data[0].xe){
			    return data[0].cx_10;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_10 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_10+a1*data[ee2+1].cx_10;
            break;
        case 11 :
            if(lengy < data[0].xe){
			    return data[0].cx_11;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_11 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_11+a1*data[ee2+1].cx_11;
            break;
        case 12 :
            if(lengy < data[0].xe){
			    return data[0].cx_12;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_12 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_12+a1*data[ee2+1].cx_12;
            break;
        case 13 :
            if(lengy < data[0].xe){
			    return data[0].cx_13;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_13 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_13+a1*data[ee2+1].cx_13;
            break;
        case 14 :
            if(lengy < data[0].xe){
			    return data[0].cx_14;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_14 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_14+a1*data[ee2+1].cx_14;
            break;
        case 15 :
            if(lengy < data[0].xe){
			    return data[0].cx_15;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_15 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_15+a1*data[ee2+1].cx_15;
            break;
        case 16 :
            if(lengy < data[0].xe){
			    return data[0].cx_16;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_16 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_16+a1*data[ee2+1].cx_16;
            break;
        case 17 :
            if(lengy < data[0].xe){
			    return data[0].cx_17;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_17 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_17+a1*data[ee2+1].cx_17;
            break;
        case 18 :
            if(lengy < data[0].xe){
			    return data[0].cx_18;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_18 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_18+a1*data[ee2+1].cx_18;
            break;
        case 19 :
            if(lengy < data[0].xe){
			    return data[0].cx_19;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_19 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_19+a1*data[ee2+1].cx_19;
            break;
        case 20 :
            if(lengy < data[0].xe){
			    return data[0].cx_20;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_20 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_20+a1*data[ee2+1].cx_20;
            break;
        case 21 :
            if(lengy < data[0].xe){
			    return data[0].cx_21;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_21 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_21+a1*data[ee2+1].cx_21;
            break;
        case 22 :
            if(lengy < data[0].xe){
			    return data[0].cx_22;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_22 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_22+a1*data[ee2+1].cx_22;
            break;
        case 23 :
            if(lengy < data[0].xe){
			    return data[0].cx_23;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_23 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_23+a1*data[ee2+1].cx_23;
            break;
        case 24 :
            if(lengy < data[0].xe){
			    return data[0].cx_24;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_24 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_24+a1*data[ee2+1].cx_24;
            break;
        case 25 :
            if(lengy < data[0].xe){
			    return data[0].cx_25;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_25 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_25+a1*data[ee2+1].cx_25;
            break;
        case 26 :
            if(lengy < data[0].xe){
			    return data[0].cx_26;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_26 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_26+a1*data[ee2+1].cx_26;
            break;
        case 27 :
            if(lengy < data[0].xe){
			    return data[0].cx_27;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_27 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_27+a1*data[ee2+1].cx_27;
            break;
        case 28 :
            if(lengy < data[0].xe){
			    return data[0].cx_28;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_28 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_28+a1*data[ee2+1].cx_28;
            break;
        case 29 :
            if(lengy < data[0].xe){
			    return data[0].cx_29;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_29 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_29+a1*data[ee2+1].cx_29;
            break;
        case 30 :
            if(lengy < data[0].xe){
			    return data[0].cx_30;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_30 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_30+a1*data[ee2+1].cx_30;
            break;
        case 31 :
            if(lengy < data[0].xe){
			    return data[0].cx_31;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_31 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_31+a1*data[ee2+1].cx_31;
            break;
        case 32 :
            if(lengy < data[0].xe){
			    return data[0].cx_32;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_32 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_32+a1*data[ee2+1].cx_32;
            break;
        case 33 :
            if(lengy < data[0].xe){
			    return data[0].cx_33;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_33 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_33+a1*data[ee2+1].cx_33;
            break;
        case 34 :
            if(lengy < data[0].xe){
			    return data[0].cx_34;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_34 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_34+a1*data[ee2+1].cx_34;
            break;
        case 35 :
            if(lengy < data[0].xe){
			    return data[0].cx_35;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_35 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_35+a1*data[ee2+1].cx_35;
            break;
        case 36 :
            if(lengy < data[0].xe){
			    return data[0].cx_36;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_36 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_36+a1*data[ee2+1].cx_36;
            break;
        case 37 :
            if(lengy < data[0].xe){
			    return data[0].cx_37;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_37 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_37+a1*data[ee2+1].cx_37;
            break;
        case 38 :
            if(lengy < data[0].xe){
			    return data[0].cx_38;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_38 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_38+a1*data[ee2+1].cx_38;
            break;
        case 39 :
            if(lengy < data[0].xe){
			    return data[0].cx_39;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_39 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_39+a1*data[ee2+1].cx_39;
            break;
        case 40 :
            if(lengy < data[0].xe){
			    return data[0].cx_40;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_40 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_40+a1*data[ee2+1].cx_40;
            break;
        case 41 :
            if(lengy < data[0].xe){
			    return data[0].cx_41;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_41 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_41+a1*data[ee2+1].cx_41;
            break;
        case 42 :
            if(lengy < data[0].xe){
			    return data[0].cx_42;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_42 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_42+a1*data[ee2+1].cx_42;
            break;
        case 43 :
            if(lengy < data[0].xe){
			    return data[0].cx_43;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_43 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_43+a1*data[ee2+1].cx_43;
            break;
        case 44 :
            if(lengy < data[0].xe){
			    return data[0].cx_44;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_44 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_44+a1*data[ee2+1].cx_44;
            break;
        case 45 :
            if(lengy < data[0].xe){
			    return data[0].cx_45;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_45 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_45+a1*data[ee2+1].cx_45;
            break;
        case 46 :
            if(lengy < data[0].xe){
			    return data[0].cx_46;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_46 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_46+a1*data[ee2+1].cx_46;
            break;
        case 47 :
            if(lengy < data[0].xe){
			    return data[0].cx_47;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_47 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_47+a1*data[ee2+1].cx_47;
            break;
        case 48 :
            if(lengy < data[0].xe){
			    return data[0].cx_48;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_48 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_48+a1*data[ee2+1].cx_48;
            break;
        case 49 :
            if(lengy < data[0].xe){
			    return data[0].cx_49;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_49 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_49+a1*data[ee2+1].cx_49;
            break;
        case 50 :
            if(lengy < data[0].xe){
			    return data[0].cx_50;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_50 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_50+a1*data[ee2+1].cx_50;
            break;
        case 51 :
            if(lengy < data[0].xe){
			    return data[0].cx_51;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_51 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_51+a1*data[ee2+1].cx_51;
            break;
        case 52 :
            if(lengy < data[0].xe){
			    return data[0].cx_52;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_52 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_52+a1*data[ee2+1].cx_52;
            break;
        case 53 :
            if(lengy < data[0].xe){
			    return data[0].cx_53;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_53 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_53+a1*data[ee2+1].cx_53;
            break;
        case 54 :
            if(lengy < data[0].xe){
			    return data[0].cx_54;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_54 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_54+a1*data[ee2+1].cx_54;
            break;
        case 55 :
            if(lengy < data[0].xe){
			    return data[0].cx_55;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_55 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_55+a1*data[ee2+1].cx_55;
            break;
        case 56 :
            if(lengy < data[0].xe){
			    return data[0].cx_56;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_56 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_56+a1*data[ee2+1].cx_56;
            break;
        case 57 :
            if(lengy < data[0].xe){
			    return data[0].cx_57;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_57 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_57+a1*data[ee2+1].cx_57;
            break;
        default :
            printf("\nError : Call about cross section data in ArO2MCC.\n\n");
            return 0.0;
    }
}