#include "hip/hip_runtime.h"
#include "cuda_mccArO2.cuh"
__device__ void ArO2_Electron(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float rengy,engy,dum,vel,vel2;
	int Iz_isp1,Iz_isp2;		
	float SumSigma,SumEngyLoss;

    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
	// Calculate total Collision probability
	i = info[0].St_num + TID + (MPNC-1)*Gsize;
	//printf("PNMC = %d\n",PNMC);
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
		//printf("[%d]:[%3.3e, %3.3e, %3.3e] = [%g], case[%d] engy = %g \n",k,sp[i].vx,sp[i].vy,sp[i].vz,dum,Flag,engy);
        Colltype = 1;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Energy loss
        // 2 : Attachment using maxwellv
		// 3 : ionization 1 -> 3 charged
		// 4 : dissociative  recombination just delete
		// 5 : Detachment 
		R1 = hiprand_uniform(&LocalStates);
        switch(Flag){
			case 0:{ // E + Ar
				mofm = info_CX[0].mofM;
				R1 *= sigv[0].val / vel;
				//printf("k[%d]:c[%d]:engy[%g]:R1[%g] \n",k,Flag,engy,R1);
				//printf("k[%d]:CX0[%1.3e]\n",k,ArO2_CrossSection(0, engy, N_LOGX, idLOGX, CX));
				if(engy > info_CX[0].Th_e && R1<=(SumSigma=ArO2_CrossSection(0, engy, N_LOGX, idLOGX, CX))){
					MCCR[TID*TnRct]++;
				}else if(engy > info_CX[1].Th_e && R1<=(SumSigma += ArO2_CrossSection(1, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[1].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+1]++;
				}else if(engy > info_CX[2].Th_e && R1<=(SumSigma += ArO2_CrossSection(2, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[2].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+2]++;
				}else if(engy > info_CX[3].Th_e && R1<=(SumSigma += ArO2_CrossSection(3, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3;
					engy-=info_CX[3].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+3]++;
				}else{
					Colltype = 0;
					Null++;
				}
				
				break;
			}
			case 1:{ // E + Ar*
				mofm = info_CX[4].mofM;
				R1 *= sigv[1].val / vel;
				if(engy > info_CX[4].Th_e && R1<=(SumSigma=ArO2_CrossSection(4, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3;
					engy-=info_CX[4].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+4]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 2:{ // E + O2
				mofm = info_CX[5].mofM;
                R1 *= sigv[2].val / vel;
				if(engy > info_CX[5].Th_e &&R1<=(SumSigma=ArO2_CrossSection(5, engy, N_LOGX, idLOGX, CX))){
                    // R0 Elastic
					MCCR[TID*TnRct+5]++;
				}else if(engy > info_CX[6].Th_e && R1<=(SumSigma += ArO2_CrossSection(6, engy, N_LOGX, idLOGX, CX))){
                    //"6.e+O2>e+O2*");
					engy-=info_CX[6].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+6]++;
				}else if(engy > info_CX[7].Th_e && R1<=(SumSigma += ArO2_CrossSection(7, engy, N_LOGX, idLOGX, CX))){
                    //"7.e+O2>e+O2*");
					engy-=info_CX[7].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+7]++;
				}else if(engy > info_CX[8].Th_e && R1<=(SumSigma += ArO2_CrossSection(8, engy, N_LOGX, idLOGX, CX))){
                    //"8.e+O2>e+O2A");
					engy-=info_CX[8].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+8]++;
				}else if(engy > info_CX[9].Th_e && R1<=(SumSigma += ArO2_CrossSection(9, engy, N_LOGX, idLOGX, CX))){
					//"9.e+O2>e+O2B");
                    engy-=info_CX[9].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+9]++;
				}else if(engy > info_CX[10].Th_e && R1<=(SumSigma += ArO2_CrossSection(10, engy, N_LOGX, idLOGX, CX))){
                    //"10.e+O2>e+O2*");
					engy-=info_CX[10].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+10]++;
				}else if(engy > info_CX[11].Th_e && R1<=(SumSigma += ArO2_CrossSection(11, engy, N_LOGX, idLOGX, CX))){
                    //"11.e+O2>OP+O-"
                    Colltype = 2;
					MCCR[TID*TnRct+11]++;
                }else if(engy > info_CX[12].Th_e && R1<=(SumSigma += ArO2_CrossSection(12, engy, N_LOGX, idLOGX, CX))){
                    //"12.e+O2>e+2OP");
					engy-=info_CX[12].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+12]++;
                }else if(engy > info_CX[13].Th_e && R1<=(SumSigma += ArO2_CrossSection(13, engy, N_LOGX, idLOGX, CX))){
                    //"13.e+O2>e+OP+OD");
					engy-=info_CX[13].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+13]++;
                }else if(engy > info_CX[14].Th_e && R1<=(SumSigma += ArO2_CrossSection(14, engy, N_LOGX, idLOGX, CX))){
                    //"14.e+O2>e+2OD");
					engy-=info_CX[14].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+14]++;
                }else if(engy > info_CX[15].Th_e && R1<=(SumSigma += ArO2_CrossSection(15, engy, N_LOGX, idLOGX, CX))){
                    //"15.e+O2>2e+O2^");
                    Colltype = 3;
					engy-=info_CX[15].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+15]++;
                }else if(engy > info_CX[16].Th_e && R1<=(SumSigma += ArO2_CrossSection(16, engy, N_LOGX, idLOGX, CX))){
                    //"16.e+O2>e+OP+O*");
					engy-=info_CX[11].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+16]++;
                }else if(engy > info_CX[17].Th_e && R1<=(SumSigma += ArO2_CrossSection(17, engy, N_LOGX, idLOGX, CX))){
                    //"17.e+O2>e+O^+O-");
                    Colltype = 3;
					engy-=info_CX[17].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 3;
					Iz_isp2 = 4;
					MCCR[TID*TnRct+17]++;
                }else if(engy > info_CX[18].Th_e && R1<=(SumSigma += ArO2_CrossSection(18, engy, N_LOGX, idLOGX, CX))){
                    //"18.e+O2>2e+O^+OP");  
                    Colltype = 3;
					engy-=info_CX[18].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+18]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 3:{ // E + O2A
				mofm = info_CX[19].mofM;
                R1 *= sigv[3].val / vel;
				if(engy > info_CX[19].Th_e &&R1<=(SumSigma=ArO2_CrossSection(19, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"19.e+O2A>2e+O2+");
					engy-=info_CX[19].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+19]++;
				}else if(engy > info_CX[20].Th_e && R1<=(SumSigma += ArO2_CrossSection(20, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; //"20.e+O2A>OP+O-");
					MCCR[TID*TnRct+20]++;
				}else if(engy > info_CX[21].Th_e && R1<=(SumSigma += ArO2_CrossSection(21, engy, N_LOGX, idLOGX, CX))){
					engy+=0.977f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+21]++;
				}else if(engy > info_CX[22].Th_e && R1<=(SumSigma += ArO2_CrossSection(22, engy, N_LOGX, idLOGX, CX))){
					engy+=0.977f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+22]++;
				}else if(engy > info_CX[23].Th_e && R1<=(SumSigma += ArO2_CrossSection(23, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[23].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+23]++;
				}else if(engy > info_CX[24].Th_e && R1<=(SumSigma += ArO2_CrossSection(24, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[24].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+24]++;
				}else if(engy > info_CX[25].Th_e && R1<=(SumSigma += ArO2_CrossSection(25, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[25].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+25]++;
				}else if(engy > info_CX[26].Th_e && R1<=(SumSigma += ArO2_CrossSection(26, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"26.e+O2A>2e+O^+OP");
					engy-=info_CX[26].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+26]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 4:{ // E + O2B
				mofm = info_CX[27].mofM;
                R1 *= sigv[4].val / vel;
                if(engy > info_CX[27].Th_e && R1<=(SumSigma=ArO2_CrossSection(27, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"22.e+O2B>2e+O2^");
					engy-=info_CX[27].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+27]++;
				}else if(engy > info_CX[28].Th_e && R1<=(SumSigma += ArO2_CrossSection(28, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; //"23.e+O2B>OP+O-");
					MCCR[TID*TnRct+28]++;
				}else if(engy > info_CX[29].Th_e && R1<=(SumSigma += ArO2_CrossSection(29, engy, N_LOGX, idLOGX, CX))){
					engy+=1.627f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+29]++;
				}else if(engy > info_CX[30].Th_e && R1<=(SumSigma += ArO2_CrossSection(30, engy, N_LOGX, idLOGX, CX))){
					engy+=1.627f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+30]++;
				}else if(engy > info_CX[31].Th_e && R1<=(SumSigma += ArO2_CrossSection(31, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[31].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+31]++;
				}else if(engy > info_CX[32].Th_e && R1<=(SumSigma += ArO2_CrossSection(32, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[32].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+32]++;
				}else if(engy > info_CX[33].Th_e && R1<=(SumSigma += ArO2_CrossSection(33, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[33].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+33]++;
				}else if(engy > info_CX[34].Th_e && R1<=(SumSigma += ArO2_CrossSection(34, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"34.e+O2B>2e+O^+OP");
					engy-=info_CX[34].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+34]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 5:{ // E + O-
				mofm = info_CX[35].mofM;
                R1 *= sigv[5].val / vel;
                if(engy > info_CX[35].Th_e && R1<=(SumSigma=ArO2_CrossSection(35, engy, N_LOGX, idLOGX, CX))){
					Colltype = 5; //"30.e+O->2e+OP");
					engy-=info_CX[35].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+35]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 6:{ // E + O2+
				mofm = info_CX[36].mofM;
                R1 *= sigv[6].val / vel;
                if(engy > info_CX[36].Th_e &&R1<=(SumSigma=ArO2_CrossSection(36, engy, N_LOGX, idLOGX, CX))){
					Colltype = 4; //"36.e+O2^>OP+OD");
					MCCR[TID*TnRct+36]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 7:{ // E + OP
				mofm = info_CX[37].mofM;
                R1 *= sigv[7].val / vel;
                if(engy > info_CX[37].Th_e &&R1<=(SumSigma=ArO2_CrossSection(37, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[37].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+37]++;
				}else if(engy > info_CX[38].Th_e && R1<=(SumSigma += ArO2_CrossSection(38, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[38].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+38]++;
				}else if(engy > info_CX[39].Th_e && R1<=(SumSigma += ArO2_CrossSection(39, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[39].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+39]++;
				}else if(engy > info_CX[40].Th_e && R1<=(SumSigma += ArO2_CrossSection(40, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[40].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+40]++;
				}else if(engy > info_CX[41].Th_e && R1<=(SumSigma += ArO2_CrossSection(41, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[41].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+41]++;
				}else if(engy > info_CX[42].Th_e && R1<=(SumSigma += ArO2_CrossSection(42, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"42.e+OP>2e+O^");
					engy-=info_CX[42].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+42]++;
				}else if(engy > info_CX[43].Th_e && R1<=(SumSigma += ArO2_CrossSection(43, engy, N_LOGX, idLOGX, CX))){
					engy-=info_CX[43].Th_e;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+43]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 8:{ // E + OD
				mofm = info_CX[44].mofM;
                R1 *= sigv[8].val / vel;
                if(engy > info_CX[44].Th_e &&R1<=(SumSigma=ArO2_CrossSection(44, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; //"44.e+OD>2e+O^");
					engy-=info_CX[44].Th_e;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+44]++;
				}else if(engy > info_CX[45].Th_e && R1<=(SumSigma += ArO2_CrossSection(45, engy, N_LOGX, idLOGX, CX))){
					engy+=1.96f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+45]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		} 
        switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Energy loss
            dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : Attachment using maxwellv
			oldPNC = atomicAdd(&data[TID+4*Gsize].PtNumInCell,1);
			index = info[4].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+4*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            break;
        case 3: // 3 : ionization 1 -> 3 charged
			// second charged create
			oldPNC = atomicAdd(&data[TID+Iz_isp1*Gsize].PtNumInCell,1);
			index = info[Iz_isp1].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp1*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp1 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
				dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}
			// Third charged create
			oldPNC = atomicAdd(&data[TID+Iz_isp2*Gsize].PtNumInCell,1);
			index = info[Iz_isp2].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp2*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp2 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
				dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}
			// energy loss electron 
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 4: // 4 : dissociative  recombination just delete
            break;
		case 5: // 5 : // Detachment 
			//"30.e+O->2e+OP");
			// new electron
			//printf("Ecollision case 5\n");
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			// energy loss
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			// delete O-
			oldPNC = atomicAdd(&data[TID+4*Gsize].PtNumInCell,0);
			if(oldPNC>1){
				R2 = hiprand_uniform(&LocalStates);
				index = info[4].St_num + TID + oldPNC*Gsize;
				index2 = (int)((float)oldPNC * R2);
				index3 = info[4].St_num + TID + index2*Gsize;
				sp[index3].CellID = sp[index].CellID;
				sp[index3].x = sp[index].x;
				sp[index3].y = sp[index].y;
				sp[index3].vx = sp[index].vx;
				sp[index3].vy = sp[index].vy;
				sp[index3].vz = sp[index].vz;
				atomicAdd(&data[TID+4*Gsize].PtNumInCell,-1);
				//printf("2[%d][%d]: %g,%g,%g,%g,%g,\n",TID,sp[index].x,sp[index].y,sp[index].vx,sp[index].vy,sp[i].vz);
			}else if(oldPNC == 1){
				atomicAdd(&data[TID+4*Gsize].PtNumInCell,-1);
			}
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_Ar_ion(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneut,vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[1].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		Flag = sp[i].CellID;
		if(Flag == 0) vneut = BG[ID].BackVel1;
		else  vneut = BG[ID].BackVel2;
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],vneut,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[1].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange 
		R1 = hiprand_uniform(&LocalStates);
        switch(Flag){
			case 0:{
				mofm = info_CX[65].mofM;
        		R1 = hiprand_uniform(&LocalStates)*sigv[23].val / vel;
				if(engy > info_CX[65].Th_e &&R1<=(SumSigma=ArO2_CrossSection(65, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+65]++;
				}else if(engy > info_CX[66].Th_e &&R1<=(SumSigma=ArO2_CrossSection(66, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+66]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[67].mofM;
        		R1 = hiprand_uniform(&LocalStates)*sigv[24].val / vel;
				if(engy > info_CX[67].Th_e &&R1<=(SumSigma=ArO2_CrossSection(67, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+67]++;
				}else{
					Null++;
				}
				break;
			}
			default:{

				break;
			}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange 
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_O2_ion(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[2].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[2].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange O2+
		// 3 : Charge exchange O+
		// 4 : Charge exchange AR+
        switch(Flag){
			case 0:{
				mofm = info_CX[52].mofM;
                R1 = hiprand_uniform(&LocalStates) * sigv[14].val / vel;
				if(engy > info_CX[52].Th_e &&R1<=(SumSigma=ArO2_CrossSection(52, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+52]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[53].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[15].val / vel;
				if(engy > info_CX[53].Th_e &&R1<=(SumSigma=ArO2_CrossSection(53, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+53]++;
				}else if(engy > info_CX[54].Th_e &&R1<=(SumSigma=ArO2_CrossSection(54, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+54]++;
				}else if(engy > info_CX[55].Th_e &&R1<=(SumSigma=ArO2_CrossSection(55, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+55]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[56].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[16].val / vel;
                if(engy > info_CX[56].Th_e &&R1<=(SumSigma=ArO2_CrossSection(56, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+56]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[57].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[17].val / vel;
                if(engy > info_CX[57].Th_e && R1<=(SumSigma=ArO2_CrossSection(57, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+57]++;
				}else{
					Null++;
				}
                break;
			}
			case 4:{
				mofm = info_CX[58].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[18].val / vel;
                if(engy > info_CX[58].Th_e && R1<=(SumSigma=ArO2_CrossSection(58, engy, N_LOGX, idLOGX, CX))){
					Colltype = 4; 
					MCCR[ID*TnRct+58]++;
				}else if(engy > info_CX[59].Th_e && R1<=(SumSigma=ArO2_CrossSection(59, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+59]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        case 3: // 3 : Charge exchange o+
			oldPNC = atomicAdd(&data[TID+Gsize].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID+Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
		case 4: // 4 : Charge exchange AR+
			oldPNC = atomicAdd(&data[TID-Gsize].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID-Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_O_ion(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[3].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel2,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[3].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange O2+
		// 3 : Charge exchange O+
        switch(Flag){
			case 0:{
				mofm = info_CX[60].mofM;
                R1 = hiprand_uniform(&LocalStates) * sigv[19].val / vel;
				if(engy > info_CX[60].Th_e &&R1<=(SumSigma=ArO2_CrossSection(60, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+60]++;
				}else if(engy > info_CX[61].Th_e &&R1<=(SumSigma=ArO2_CrossSection(61, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+61]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[62].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[20].val / vel;
				if(engy > info_CX[62].Th_e &&R1<=(SumSigma=ArO2_CrossSection(62, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+62]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[63].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[21].val / vel;
                if(engy > info_CX[63].Th_e &&R1<=(SumSigma=ArO2_CrossSection(63, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+63]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[64].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[22].val / vel;
                if(engy > info_CX[64].Th_e && R1<=(SumSigma=ArO2_CrossSection(64, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+64]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID-Gsize].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID-Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        case 3: // 3 : Charge exchange o+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_O_negative(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;
	int Iz_isp1,Iz_isp2;		
	float SumSigma,SumEngyLoss;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[4].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
		//if(Flag !=0) printf("\n[%d] : Flag = %d \n\n",TID,Flag);
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[4].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Detachment using maxwellv
		// 3 : dissociative  recombination just delete
        switch(Flag){
			case 0:{
				mofm = info_CX[46].mofM;
                R1 = hiprand_uniform(&LocalStates) * sigv[9].val / vel;
				if(engy > info_CX[46].Th_e &&R1<=(SumSigma=ArO2_CrossSection(46, engy, N_LOGX, idLOGX, CX))){
					Colltype = 1; 
					MCCR[ID*TnRct+46]++;
				}else if(engy > info_CX[47].Th_e && R1<=(SumSigma += ArO2_CrossSection(47, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+47]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[48].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[10].val / vel;
				if(engy > info_CX[48].Th_e &&R1<=(SumSigma=ArO2_CrossSection(48, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+48]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[49].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[11].val / vel;
                if(engy > info_CX[49].Th_e &&R1<=(SumSigma=ArO2_CrossSection(49, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+49]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[50].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[12].val / vel;
                if(engy > info_CX[50].Th_e && R1<=(SumSigma=ArO2_CrossSection(50, engy, N_LOGX, idLOGX, CX))){
					Colltype = 3; 
					MCCR[ID*TnRct+50]++;
				}else{
					Null++;
				}
                break;
			}
			case 4:{
				mofm = info_CX[51].mofM;
                R1 = hiprand_uniform(&LocalStates)*sigv[13].val / vel;
                if(engy > info_CX[51].Th_e &&R1<=(SumSigma=ArO2_CrossSection(51, engy, N_LOGX, idLOGX, CX))){
					Colltype = 2; 
					MCCR[ID*TnRct+51]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[4].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[4].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : Detachment using maxwellv
			oldPNC = atomicAdd(&data[ID].PtNumInCell,1);
			index = info[0].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = ID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			break;
        case 3: // 3 : dissociative  recombination just delete
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void  ArO2_Collision_Check(int Gsize, int Csize, int ngy, int TID, float dt, int MCCn, float dtm, float dx, float dy,
                                        hiprandState *states, Species *info, GPG *data, GCP *sp, MCC_sigmav *sigv, GGA *BG, GFC *Fluid){
	int i,j,k,index,Randn;
	int ID,isp,CID,PNMC,MPNC;
    int PNC,Flag;
	int nx,ny,ngx;
	float Tprob,Prob1,Prob2,Prob3,Prob4,Prob5,Prob6,Prob7,Prob8,Prob9;
	float R1;
	ID = TID%Gsize;
    isp = TID/Gsize;
	nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	hiprandState LocalStates = states[TID];
	PNC = data[TID].PtNumInCell;
	MPNC = data[TID].MaxPtNumInCell;
	PNMC = 0;
	// Calculate total Collision probability.
    switch (isp){
    case 0: // Electron
		Prob1 = 1.0f - exp(-1*dtm*sigv[0].val*BG[ID].BackDen1);  // E + Ar
		Prob2 = Prob1 + 1.0f - exp(-1*dtm*sigv[1].val*Fluid[CID].ave_den);  // E + Ar*
        Prob3 = Prob2 + 1.0f - exp(-1*dtm*sigv[2].val*BG[ID].BackDen2);  // E + O2
	    Prob4 = Prob3 + 1.0f - exp(-1*dtm*sigv[3].val*Fluid[CID+Csize].ave_den);  // E + O2A
        Prob5 = Prob4 + 1.0f - exp(-1*dtm*sigv[4].val*Fluid[CID+2*Csize].ave_den);  // E + O2B
        Prob6 = Prob5 + 1.0f - exp(-1*dtm*sigv[5].val*data[ID+4*Gsize].den*info[4].np2c*dx*dy);  // E + O-
        Prob7 = Prob6 + 1.0f - exp(-1*dtm*sigv[6].val*data[ID+2*Gsize].den*info[2].np2c*dx*dy);  // E + O2+
        Prob8 = Prob7 + 1.0f - exp(-1*dtm*sigv[7].val*Fluid[CID+3*Csize].ave_den);  // E + OP
        Prob9 = Prob8 + 1.0f - exp(-1*dtm*sigv[8].val*Fluid[CID+4*Csize].ave_den);  // E + OD
	    Tprob = Prob9; 
		//printf("Case[%d] : 1[%1.2e] 2[%1.2e] 3[%1.2e] 4[%1.2e] 5[%1.2e] 6[%1.2e] 7[%1.2e] 8[%1.2e] 9[%1.2e]\n",isp,Prob1, Prob2, Prob3, Prob4, Prob5, Prob6, Prob7, Prob8, Prob9);
		//printf("Case[%d] : Ar[%1.2e] O2[%1.2e]\n",isp,BG[ID].BackDen1,BG[ID].BackDen2);
		//printf("Case[%d] : Ar*[%1.2e] O2A[%1.2e] O2B[%1.2e] OP[%1.2e] OD[%1.2e]\n",isp,Fluid[CID].ave_den,Fluid[CID+1*Csize].ave_den,Fluid[CID+2*Csize].ave_den,Fluid[CID+3*Csize].ave_den,Fluid[CID+4*Csize].ave_den);
		Randn = MCCn;
        break;
	case 1: // Ar+
		Prob1 = 1.0 - exp(-1*dt*sigv[23].val*BG[ID].BackDen1); // Ar+ + Ar
	    Prob2 = Prob1 + 1.0 - exp(-1*dt*sigv[24].val*BG[ID].BackDen2); // Ar+ + O2
		Tprob = Prob2;
		//printf("Case[%d] : 1[%1.2e] 2[%1.2e]\n",isp,Prob1, Prob2);
		Randn = 1;
		break;
    case 2: // O2+
        Prob1 = 1.0 - exp(-1*dt*sigv[14].val*Fluid[CID+3*Csize].ave_den); // O2+ + OP
	    Prob2 = Prob1 + 1.0 - exp(-1*dt*sigv[15].val*BG[ID].BackDen2); // O2+ + O2
	    Prob3 = Prob2 + 1.0 - exp(-1*dt*sigv[16].val*Fluid[CID+Csize].ave_den); // O2+ + O2A
	    Prob4 = Prob3 + 1.0 - exp(-1*dt*sigv[17].val*Fluid[CID+2*Csize].ave_den); // O2+ + O2B
		Prob5 = Prob4 + 1.0 - exp(-1*dt*sigv[18].val*BG[ID].BackDen1); // O2+ + AR
        Tprob = Prob5;
		//printf("Case[%d] : 1[%1.2e] 2[%1.2e] 3[%1.2e] 4[%1.2e] 5[%1.2e]\n",isp,Prob1, Prob2, Prob3, Prob4, Prob5);
		Randn = 1;
        break;
    case 3: // O+
        Prob1 = 1.0 - exp(-1*dt*sigv[19].val*BG[ID].BackDen2); // O+ + O2
	    Prob2 = Prob1 + 1.0 - exp(-1*dt*sigv[20].val*Fluid[CID+3*Csize].ave_den); // O+ + OP
	    Prob3 = Prob2 + 1.0 - exp(-1*dt*sigv[21].val*Fluid[CID+Csize].ave_den); // O+ + O2A
	    Prob4 = Prob3 + 1.0 - exp(-1*dt*sigv[22].val*Fluid[CID+2*Csize].ave_den); // O+ + O2B
	    Tprob = Prob4;
		//printf("Case[%d] : 1[%1.2e] 2[%1.2e] 3[%1.2e] 4[%1.2e]\n",isp,Prob1, Prob2, Prob3, Prob4);
		Randn = 1;
        break;
    case 4: // O-
        Prob1 = 1.0 - exp(-1*dt*sigv[9].val*BG[ID].BackDen2); // O- + O2
	    Prob2 = Prob1 + 1.0 - exp(-1*dt*sigv[10].val*Fluid[CID+3*Csize].ave_den); // O- + OP
	    Prob3 = Prob2 + 1.0 - exp(-1*dt*sigv[11].val*data[ID+2*Gsize].den*info[2].np2c*dx*dy); // O- + O2+
	    Prob4 = Prob3 + 1.0 - exp(-1*dt*sigv[12].val*data[ID+3*Gsize].den*info[3].np2c*dx*dy); // O- + O+
	    Prob5 = Prob4 + 1.0 - exp(-1*dt*sigv[13].val*Fluid[CID+Csize].ave_den); // O- + O2A
	    Tprob = Prob5;
		//printf("Case[%d] : 1[%1.2e] 2[%1.2e] 3[%1.2e] 4[%1.2e] 5[%1.2e]\n",isp,Prob1, Prob2, Prob3, Prob4, Prob5);
		Randn = 1;
        break;
    default:
        break;
    }
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
        for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob) break;
		}
		if(R1 >= Tprob){ // no collision
			index = i - PNMC*Gsize;
            Flag = sp[i].CellID;
		}else{ // collision
			PNMC++;
			index = info[isp].St_num + ID + (MPNC-PNMC)*Gsize;
			//printf("k[%d], PNMC[%d], R1[%g], Tprob[%g]\n",k,PNMC,R1,Tprob);
            switch (isp){
            case 0:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
                else if(R1 <= Prob4)	Flag = (int)3;
                else if(R1 <= Prob5)	Flag = (int)4;
                else if(R1 <= Prob6)	Flag = (int)5;
				else if(R1 <= Prob7)	Flag = (int)6;
				else if(R1 <= Prob8)	Flag = (int)7;
		        else			        Flag = (int)8;
				//printf("k[%d], PNMC[%d], R1[%g], Tprob[%g], Flag[%d]\n",k,PNMC,R1,Tprob,Flag);
                break;
            case 1:
                if(R1 <= Prob1)	        Flag = (int)0;
		        else			        Flag = (int)1;
                break;
            case 2:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
				else if(R1 <= Prob4)	Flag = (int)3;
		        else			        Flag = (int)4;
                break;
            case 3:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
		        else			        Flag = (int)3;
                break;
            case 4:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
                else if(R1 <= Prob4)	Flag = (int)3;
		        else			        Flag = (int)4;
                break;
            default:
                break;
            }
		}
		sp[index].CellID = Flag;
		sp[index].vx=sp[i].vx;
		sp[index].vy=sp[i].vy;
		sp[index].vz=sp[i].vz;
        sp[index].x=sp[i].x;
		sp[index].y=sp[i].y;
		i+=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNumMCCInCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}
__device__ void ArO2_Electron_TEST(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float rengy,engy,dum,vel,vel2;
	int Iz_isp1,Iz_isp2;		
	float SumSigma,SumEngyLoss;

    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
	// Calculate total Collision probability
	i = info[0].St_num + TID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[0].Escale * dum;
        Colltype = 1;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Energy loss
        // 2 : Attachment using maxwellv
		// 3 : ionization 1 -> 3 charged
		// 4 : dissociative  recombination just delete
		// 5 : Detachment 
		R1 = hiprand_uniform(&LocalStates);
        switch(Flag){
			case 0:{ // E + Ar
				mofm = info_CX[0].mofM;
				if(R1<=0.2){
					MCCR[TID*TnRct]++;
				}else if(R1<=0.4){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+1]++;
				}else if(R1<=0.6){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+2]++;
				}else if(R1<=0.8){
					Colltype = 3;
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+3]++;
				}else{
					Colltype = 0;
					Null++;
				}
				
				break;
			}
			case 1:{ // E + Ar*
				mofm = info_CX[4].mofM;
				if(R1<=0.5){
					Colltype = 3;
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 1;
					MCCR[TID*TnRct+4]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 2:{ // E + O2
				mofm = info_CX[5].mofM;
				if(R1<=0.0625){
                    // R0 Elastic
					MCCR[TID*TnRct+5]++;
				}else if(R1<=0.125){
                    //"6.e+O2>e+O2*");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+6]++;
				}else if(R1<=0.1875){
                    //"7.e+O2>e+O2*");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+7]++;
				}else if(R1<=0.25){
                    //"8.e+O2>e+O2A");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+8]++;
				}else if(R1<=0.3125){
					//"9.e+O2>e+O2B");
                    engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+9]++;
				}else if(R1<=0.375){
                    //"10.e+O2>e+O2*");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+10]++;
				}else if(R1<=0.4375){
                    //"11.e+O2>OP+O-"
                    Colltype = 2;
					MCCR[TID*TnRct+11]++;
                }else if(R1<=0.5){
                    //"12.e+O2>e+2OP");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+12]++;
                }else if(R1<=0.5625){
                    //"13.e+O2>e+OP+OD");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+13]++;
                }else if(R1<=0.625){
                    //"14.e+O2>e+2OD");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+14]++;
                }else if(R1<=0.6875){
                    //"15.e+O2>2e+O2^");
                    Colltype = 3;
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+15]++;
                }else if(R1<=0.75){
                    //"16.e+O2>e+OP+O*");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+16]++;
                }else if(R1<=0.8125){
                    //"17.e+O2>e+O^+O-");
                    Colltype = 3;
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 3;
					Iz_isp2 = 4;
					MCCR[TID*TnRct+17]++;
                }else if(R1<=0.875){
                    //"18.e+O2>2e+O^+OP");  
                    Colltype = 3;
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+18]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 3:{ // E + O2A
				mofm = info_CX[19].mofM;
				if(R1<=0.1){
					Colltype = 3; //"19.e+O2A>2e+O2+");
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+19]++;
				}else if(R1<=0.2){
					Colltype = 2; //"20.e+O2A>OP+O-");
					MCCR[TID*TnRct+20]++;
				}else if(R1<=0.3){
					engy+=0.977f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+21]++;
				}else if(R1<=0.4){
					engy+=0.977f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+22]++;
				}else if(R1<=0.5){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+23]++;
				}else if(R1<=0.6){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+24]++;
				}else if(R1<=0.7){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+25]++;
				}else if(R1<=0.8){
					Colltype = 3; //"26.e+O2A>2e+O^+OP");
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+26]++;
				}else{
					Colltype = 0;
					Null++;
				}
				break;
			}
			case 4:{ // E + O2B
				mofm = info_CX[27].mofM;
                if(R1<=0.1){
					Colltype = 3; //"22.e+O2B>2e+O2^");
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 2;
					MCCR[TID*TnRct+27]++;
				}else if(R1<=0.2){
					Colltype = 2; //"23.e+O2B>OP+O-");
					MCCR[TID*TnRct+28]++;
				}else if(R1<=0.3){
					engy+=1.627f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+29]++;
				}else if(R1<=0.4){
					engy+=1.627f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+30]++;
				}else if(R1<=0.5){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+31]++;
				}else if(R1<=0.6){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+32]++;
				}else if(R1<=0.7){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+33]++;
				}else if(R1<=0.8){
					Colltype = 3; //"34.e+O2B>2e+O^+OP");
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+34]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 5:{ // E + O-
				mofm = info_CX[35].mofM;
                if(R1<=0.5){
					Colltype = 5; //"30.e+O->2e+OP");
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+35]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 6:{ // E + O2+
				mofm = info_CX[36].mofM;
                if(R1<=0.5){
					Colltype = 4; //"36.e+O2^>OP+OD");
					MCCR[TID*TnRct+36]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 7:{ // E + OP
				mofm = info_CX[37].mofM;
                if(R1<=0.125){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+37]++;
				}else if(R1<=0.25){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+38]++;
				}else if(R1<=0.375){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+39]++;
				}else if(R1<=0.5){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+40]++;
				}else if(R1<=0.625){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+41]++;
				}else if(R1<=0.75){
					Colltype = 3; //"42.e+OP>2e+O^");
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+42]++;
				}else if(R1<=0.875){
					engy/=2.0f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+43]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			case 8:{ // E + OD
				mofm = info_CX[44].mofM;
                if(R1<=0.3333){
					Colltype = 3; //"44.e+OD>2e+O^");
					engy/=2.0f;
					rengy=10.0*__tanf(hiprand_uniform(&LocalStates)*atan(engy/20.0));
					engy-=rengy;
					vel = sqrt(fabs(rengy)/info[0].Escale);
					vel2 = sqrt(fabs(engy)/info[0].Escale);
					Iz_isp1 = 0;
					Iz_isp2 = 3;
					MCCR[TID*TnRct+44]++;
				}else if(R1<=0.6666){
					engy+=1.96f;
					vel=sqrt(fabs(engy)/info[0].Escale);
					MCCR[TID*TnRct+45]++;
				}else{
					Colltype = 0;
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		} 
        switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Energy loss
            dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : Attachment using maxwellv
			oldPNC = atomicAdd(&data[TID+4*Gsize].PtNumInCell,1);
			index = info[4].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+4*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			n = (nvel-1)*hiprand_uniform(&LocalStates);
			dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
            break;
        case 3: // 3 : ionization 1 -> 3 charged
			// second charged create
			/*
			oldPNC = atomicAdd(&data[TID+Iz_isp1*Gsize].PtNumInCell,1);
			index = info[Iz_isp1].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp1*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp1 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
				dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}*/
			// Third charged create
			oldPNC = atomicAdd(&data[TID+Iz_isp2*Gsize].PtNumInCell,1);
			index = info[Iz_isp2].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID+Iz_isp2*Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			if(Iz_isp2 == 0){
				sp[index].vx = VX;
				sp[index].vy = VY;
				sp[index].vz = VZ;
				dev_anewvel(rengy,vel,&sp[index].vx,&sp[index].vy,&sp[index].vz,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}else{
				n = (nvel-1)*hiprand_uniform(&LocalStates);
				dev_maxwellv(&sp[index].vx,&sp[index].vy,&sp[index].vz,vsave[n],BG[TID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			}
			// energy loss electron 
			dev_anewvel(engy,vel2,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 4: // 4 : dissociative  recombination just delete
            break;
		case 5: // 5 : // Detachment 
			//"30.e+O->2e+OP");
			// new electron
			//printf("Ecollision case 5\n");
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			// energy loss
			dev_anewvel(engy,vel,&VX,&VY,&VZ,0,mofm,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[0].St_num + TID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			// delete O-
			oldPNC = atomicAdd(&data[TID+4*Gsize].PtNumInCell,0);
			if(oldPNC>1){
				R2 = hiprand_uniform(&LocalStates);
				index = info[4].St_num + TID + oldPNC*Gsize;
				index2 = (int)((float)oldPNC * R2);
				index3 = info[4].St_num + TID + index2*Gsize;
				sp[index3].CellID = sp[index].CellID;
				sp[index3].x = sp[index].x;
				sp[index3].y = sp[index].y;
				sp[index3].vx = sp[index].vx;
				sp[index3].vy = sp[index].vy;
				sp[index3].vz = sp[index].vz;
				atomicAdd(&data[TID+4*Gsize].PtNumInCell,-1);
				//printf("2[%d][%d]: %g,%g,%g,%g,%g,\n",TID,sp[index].x,sp[index].y,sp[index].vx,sp[index].vy,sp[i].vz);
			}else if(oldPNC == 1){
				atomicAdd(&data[TID+4*Gsize].PtNumInCell,-1);
			}
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_Ar_ion_TEST(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneut,vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[1].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		Flag = sp[i].CellID;
		if(Flag == 0) vneut = BG[ID].BackVel1;
		else  vneut = BG[ID].BackVel2;
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],vneut,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[1].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange 
		R1 = hiprand_uniform(&LocalStates);
        switch(Flag){
			case 0:{
				mofm = info_CX[65].mofM;
				if(engy > info_CX[65].Th_e&& R1<=0.33333){
					Colltype = 2; 
					MCCR[ID*TnRct+65]++;
				}else if(engy > info_CX[66].Th_e && R1<=0.66666){
					Colltype = 1; 
					MCCR[ID*TnRct+66]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[67].mofM;
				if(engy > info_CX[67].Th_e && R1<=0.5){
					Colltype = 1; 
					MCCR[ID*TnRct+67]++;
				}else{
					Null++;
				}
				break;
			}
			default:{

				break;
			}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange 
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_O2_ion_TEST(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[2].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel1,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[2].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange O2+
		// 3 : Charge exchange O+
		// 4 : Charge exchange AR+
        switch(Flag){
			case 0:{
				mofm = info_CX[52].mofM;
                R1 = hiprand_uniform(&LocalStates);
				if(R1<=0.5){
					Colltype = 3; 
					MCCR[ID*TnRct+52]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[53].mofM;
                R1 = hiprand_uniform(&LocalStates);
				if(R1<=0.25){
					Colltype = 2; 
					MCCR[ID*TnRct+53]++;
				}else if(R1<=0.5){
					Colltype = 1; 
					MCCR[ID*TnRct+54]++;
				}else if(R1<=0.75){
					Colltype = 3; 
					MCCR[ID*TnRct+55]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[56].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(R1<=0.5){
					Colltype = 2; 
					MCCR[ID*TnRct+56]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[57].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(R1<=0.5){
					Colltype = 2; 
					MCCR[ID*TnRct+57]++;
				}else{
					Null++;
				}
                break;
			}
			case 4:{
				mofm = info_CX[58].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(R1<=0.33333){
					Colltype = 4; 
					MCCR[ID*TnRct+58]++;
				}else if(R1<=0.66666){
					Colltype = 2; 
					MCCR[ID*TnRct+59]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        case 3: // 3 : Charge exchange o+
			oldPNC = atomicAdd(&data[TID+Gsize].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID+Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
		case 4: // 4 : Charge exchange AR+
			oldPNC = atomicAdd(&data[TID-Gsize].PtNumInCell,1);
			index = info[1].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID-Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_O_ion_TEST(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;	
	float SumSigma,SumEngyLoss;
	float vneutx,vneuty,vneutz;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[3].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		n = (nvel-1)*hiprand_uniform(&LocalStates);
		dev_maxwellv(&vneutx,&vneuty,&vneutz,vsave[n],BG[ID].BackVel2,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
        VX = sp[i].vx - vneutx;
		VY = sp[i].vy - vneuty;
		VZ = sp[i].vz - vneutz;
		dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		engy = info[3].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Charge exchange O2+
		// 3 : Charge exchange O+
        switch(Flag){
			case 0:{
				mofm = info_CX[60].mofM;
                R1 = hiprand_uniform(&LocalStates);
				if(engy > info_CX[60].Th_e && R1<=0.33333){
					Colltype = 2; 
					MCCR[ID*TnRct+60]++;
				}else if(engy > info_CX[61].Th_e && R1<=0.66666){
					Colltype = 1; 
					MCCR[ID*TnRct+61]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[62].mofM;
                R1 = hiprand_uniform(&LocalStates);
				if(engy > info_CX[62].Th_e && R1<=0.5){
					Colltype = 3; 
					MCCR[ID*TnRct+62]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[63].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(engy > info_CX[63].Th_e && R1<=0.5){
					Colltype = 2; 
					MCCR[ID*TnRct+63]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[64].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(engy > info_CX[64].Th_e && R1<=0.5){
					Colltype = 2; 
					MCCR[ID*TnRct+64]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX+vneutx;
			sp[index].vy = VY+vneuty;
			sp[index].vz = VZ+vneutz;
            break;
        case 2: // 2 : Charge exchange o2+
			oldPNC = atomicAdd(&data[TID-Gsize].PtNumInCell,1);
			index = info[2].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID-Gsize;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
			break;
        case 3: // 3 : Charge exchange o+
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[3].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = vneutx;
			sp[index].vy = vneuty;
			sp[index].vz = vneutz;
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void ArO2_O_negative_TEST(int Gsize, int ngy, int TID, int nvel, float *vsave, hiprandState *states, 
											Species *info, GPG *data, GCP *sp, int N_LOGX, float idLOGX, 
											MCC_sigmav *sigv, CollF *info_CX, ArO2CollD *CX, int TnRct,float *MCCR, GGA *BG){
	int i,j,k,n,index,index2,index3;
	int ID,PNMC,MPNC,Null,Flag;
	int Target,oldPNC;
	int Colltype;
	float mofm,R1,R2;
	float VX,VY,VZ;
	float engy,dum,vel;
	int Iz_isp1,Iz_isp2;		
	float SumSigma,SumEngyLoss;

	ID = TID%Gsize;
    hiprandState LocalStates = states[TID];
	PNMC = data[TID].PtNumMCCInCell;
	MPNC = data[TID].MaxPtNumInCell;
	Null = 0;
    // Calculate total Collision probability
	i = info[4].St_num + ID + (MPNC-1)*Gsize;
	for(k=0;k<PNMC;k++){
        // Calculate energy
		VX = sp[i].vx;
		VY = sp[i].vy;
		VZ = sp[i].vz;
        Flag = sp[i].CellID;
		//if(Flag !=0) printf("\n[%d] : Flag = %d \n\n",TID,Flag);
        dum = VX*VX+VY*VY+VZ*VZ;
		vel = sqrt(dum);
		VX/=vel; VY/=vel; VZ/=vel;
		engy = info[4].Escale * dum;
        Colltype = 0;
        //Start
        // Colltype
        // 0 : Null collision
        // 1 : Scattering
        // 2 : Detachment using maxwellv
		// 3 : dissociative  recombination just delete
        switch(Flag){
			case 0:{
				mofm = info_CX[46].mofM;
                R1 = hiprand_uniform(&LocalStates);
				if(R1<=0.3333){
					Colltype = 1; 
					MCCR[ID*TnRct+46]++;
				}else if(R1<=0.6666){
					Colltype = 2; 
					MCCR[ID*TnRct+47]++;
				}else{
					Null++;
				}
				break;
			}
			case 1:{
				mofm = info_CX[48].mofM;
                R1 = hiprand_uniform(&LocalStates);
				if(R1<=0.5){
					Colltype = 2; 
					MCCR[ID*TnRct+48]++;
				}else{
					Null++;
				}
				break;
			}
			case 2:{
				mofm = info_CX[49].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(R1<=0.5){
					Colltype = 3; 
					MCCR[ID*TnRct+49]++;
				}else{
					Null++;
				}
                break;
			}
			case 3:{
				mofm = info_CX[50].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(R1<=0.5){
					Colltype = 3; 
					MCCR[ID*TnRct+50]++;
				}else{
					Null++;
				}
                break;
			}
			case 4:{
				mofm = info_CX[51].mofM;
                R1 = hiprand_uniform(&LocalStates);
                if(R1<=0.5){
					Colltype = 2; 
					MCCR[ID*TnRct+51]++;
				}else{
					Null++;
				}
                break;
			}
			default:{
            	break;
        	}
		}
		
		switch (Colltype){
        case 0: // 0 : Null collision
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[4].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = sp[i].vx;
			sp[index].vy = sp[i].vy;
			sp[index].vz = sp[i].vz;
            break;
        case 1: // 1 : Scattering
			dev_newvel_IONSC(&VX,&VY,&VZ,vel,hiprand_uniform(&LocalStates),hiprand_uniform(&LocalStates));
			oldPNC = atomicAdd(&data[TID].PtNumInCell,1);
			index = info[4].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = TID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
            break;
        case 2: // 2 : Detachment using maxwellv
			oldPNC = atomicAdd(&data[ID].PtNumInCell,1);
			index = info[0].St_num + ID + oldPNC*Gsize;
			sp[index].CellID = ID;
			sp[index].x = sp[i].x;
			sp[index].y = sp[i].y;
			sp[index].vx = VX;
			sp[index].vy = VY;
			sp[index].vz = VZ;
			break;
        case 3: // 3 : dissociative  recombination just delete
            break;
        default:
            break;
        }
		i-=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNullMCCInCell = Null;
}
__device__ void  ArO2_Collision_Check_TEST(int Gsize, int Csize, int ngy, int TID, float dt, int MCCn, float dtm, float dx, float dy,
                                        hiprandState *states, Species *info, GPG *data, GCP *sp, MCC_sigmav *sigv, GGA *BG, GFC *Fluid){
	int i,j,k,index,Randn;
	int ID,isp,CID,PNMC,MPNC;
    int PNC,Flag;
	int nx,ny,ngx;
	float Tprob,Prob1,Prob2,Prob3,Prob4,Prob5,Prob6,Prob7,Prob8,Prob9;
	float R1;
	ID = TID%Gsize;
    isp = TID/Gsize;
	nx = ID/ngy;
	ny = ID%ngy;
	ngx = Gsize/ngy;
	if(nx == ngx-1) nx--;
	if(ny == ngy-1) ny--;
	CID = ny + (ngy-1)*nx;
	hiprandState LocalStates = states[TID];
	PNC = data[TID].PtNumInCell;
	MPNC = data[TID].MaxPtNumInCell;
	PNMC = 0;
	// Calculate total Collision probability.
    switch (isp){
    case 0: // Electron
		Prob1 = 0.01;
		Prob2 = Prob1 + 0.01;
        Prob3 = Prob2 + 0.01;
	    Prob4 = Prob3 + 0.01;
        Prob5 = Prob4 + 0.01;
        Prob6 = Prob5 + 0.01;
        Prob7 = Prob6 + 0.01;
        Prob8 = Prob7 + 0.01;
        Prob9 = Prob8 + 0.01;
	    Tprob = Prob9; 
		Randn = MCCn;
        break;
	case 1: // Ar+
		Prob1 = 0.1;
	    Prob2 = Prob1 + 0.1;
		Tprob = Prob2;
		Randn = MCCn;
		break;
    case 2: // O2+
        Prob1 = 0.01;
	    Prob2 = Prob1 + 0.01;
	    Prob3 = Prob2 + 0.01;
	    Prob4 = Prob3 + 0.01;
		Prob5 = Prob4 + 0.01;
        Tprob = Prob5;
		Randn = MCCn;
        break;
    case 3: // O+
        Prob1 = 0.01;
	    Prob2 = Prob1 + 0.01;
	    Prob3 = Prob2 + 0.01;
	    Prob4 = Prob3 + 0.01;
	    Tprob = Prob4;
		Randn = MCCn;
        break;
    case 4: // O-
        Prob1 = 0.01;
	    Prob2 = Prob1 + 0.01;
	    Prob3 = Prob2 + 0.01;
	    Prob4 = Prob3 + 0.01;
	    Prob5 = Prob4 + 0.01;
	    Tprob = Prob5;
		Randn = MCCn;
        break;
    default:
        break;
    }
	i = info[isp].St_num + ID;
	for(k=0;k<PNC;k++){
        for(j=0;j<Randn;j++){
			R1 = hiprand_uniform(&LocalStates);
			if(R1<Tprob) break;
		}
		if(R1 >= Tprob){ // no collision
			index = i - PNMC*Gsize;
            Flag = sp[i].CellID;
		}else{ // collision
			PNMC++;
			index = info[isp].St_num + ID + (MPNC-PNMC)*Gsize;
			//printf("k[%d], PNMC[%d], R1[%g], Tprob[%g]\n",k,PNMC,R1,Tprob);
            switch (isp){
            case 0:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
                else if(R1 <= Prob4)	Flag = (int)3;
                else if(R1 <= Prob5)	Flag = (int)4;
                else if(R1 <= Prob6)	Flag = (int)5;
				else if(R1 <= Prob7)	Flag = (int)6;
				else if(R1 <= Prob8)	Flag = (int)7;
		        else			        Flag = (int)8;
				//printf("k[%d], PNMC[%d], R1[%g], Tprob[%g], Flag[%d]\n",k,PNMC,R1,Tprob,Flag);
                break;
            case 1:
                if(R1 <= Prob1)	        Flag = (int)0;
		        else			        Flag = (int)1;
                break;
            case 2:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
				else if(R1 <= Prob4)	Flag = (int)3;
		        else			        Flag = (int)4;
                break;
            case 3:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
		        else			        Flag = (int)3;
                break;
            case 4:
                if(R1 <= Prob1)	        Flag = (int)0;
                else if(R1 <= Prob2)	Flag = (int)1;
                else if(R1 <= Prob3)	Flag = (int)2;
                else if(R1 <= Prob4)	Flag = (int)3;
		        else			        Flag = (int)4;
                break;
            default:
                break;
            }
		}
		sp[index].CellID = Flag;
		sp[index].vx=sp[i].vx;
		sp[index].vy=sp[i].vy;
		sp[index].vz=sp[i].vz;
        sp[index].x=sp[i].x;
		sp[index].y=sp[i].y;
		i+=Gsize;
	}
	states[TID]=LocalStates;
	data[TID].PtNumMCCInCell=PNMC;
	data[TID].PtNumInCell-=PNMC;
}
__device__ float ArO2_CrossSection(int R, float engy, int N_LOGX, float idLOGX, ArO2CollD *data){
	if(engy == 0) return 0.0;
	float lengy = log10(engy);
	float ee1, a1, a2;
	int ee2;
	ee1 = idLOGX * (lengy - data[0].xe);
	ee2 = (int)ee1;
	a1 = ee1 - ee2;
	a2 = 1 - a1;
	switch (R) {
        case 0 : 
            if(lengy < data[0].xe){
			    return data[0].cx_0;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_0 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_0+a1*data[ee2+1].cx_0;
            break;
        case 1 :
            if(lengy < data[0].xe){
			    return data[0].cx_1;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_1 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_1+a1*data[ee2+1].cx_1;
            break;
        case 2 :
            if(lengy < data[0].xe){
			    return data[0].cx_2;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_2 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_2+a1*data[ee2+1].cx_2;
            break;
        case 3 :
            if(lengy < data[0].xe){
			    return data[0].cx_3;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_3 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_3+a1*data[ee2+1].cx_3;
            break;
        case 4 :
            if(lengy < data[0].xe){
			    return data[0].cx_4;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_4 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_4+a1*data[ee2+1].cx_4;
            break;
        case 5 :
            if(lengy < data[0].xe){
			    return data[0].cx_5;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_5 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_5+a1*data[ee2+1].cx_5;
            break;
        case 6 :
            if(lengy < data[0].xe){
			    return data[0].cx_6;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_6 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_6+a1*data[ee2+1].cx_6;
            break;
        case 7 :
            if(lengy < data[0].xe){
			    return data[0].cx_7;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_7 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_7+a1*data[ee2+1].cx_7;
            break;
        case 8 :
            if(lengy < data[0].xe){
			    return data[0].cx_8;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_8 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_8+a1*data[ee2+1].cx_8;
            break;
        case 9 :
            if(lengy < data[0].xe){
			    return data[0].cx_9;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_9 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_9+a1*data[ee2+1].cx_9;
            break;
        case 10 :
            if(lengy < data[0].xe){
			    return data[0].cx_10;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_10 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_10+a1*data[ee2+1].cx_10;
            break;
        case 11 :
            if(lengy < data[0].xe){
			    return data[0].cx_11;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_11 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_11+a1*data[ee2+1].cx_11;
            break;
        case 12 :
            if(lengy < data[0].xe){
			    return data[0].cx_12;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_12 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_12+a1*data[ee2+1].cx_12;
            break;
        case 13 :
            if(lengy < data[0].xe){
			    return data[0].cx_13;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_13 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_13+a1*data[ee2+1].cx_13;
            break;
        case 14 :
            if(lengy < data[0].xe){
			    return data[0].cx_14;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_14 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_14+a1*data[ee2+1].cx_14;
            break;
        case 15 :
            if(lengy < data[0].xe){
			    return data[0].cx_15;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_15 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_15+a1*data[ee2+1].cx_15;
            break;
        case 16 :
            if(lengy < data[0].xe){
			    return data[0].cx_16;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_16 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_16+a1*data[ee2+1].cx_16;
            break;
        case 17 :
            if(lengy < data[0].xe){
			    return data[0].cx_17;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_17 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_17+a1*data[ee2+1].cx_17;
            break;
        case 18 :
            if(lengy < data[0].xe){
			    return data[0].cx_18;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_18 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_18+a1*data[ee2+1].cx_18;
            break;
        case 19 :
            if(lengy < data[0].xe){
			    return data[0].cx_19;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_19 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_19+a1*data[ee2+1].cx_19;
            break;
        case 20 :
            if(lengy < data[0].xe){
			    return data[0].cx_20;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_20 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_20+a1*data[ee2+1].cx_20;
            break;
        case 21 :
            if(lengy < data[0].xe){
			    return data[0].cx_21;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_21 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_21+a1*data[ee2+1].cx_21;
            break;
        case 22 :
            if(lengy < data[0].xe){
			    return data[0].cx_22;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_22 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_22+a1*data[ee2+1].cx_22;
            break;
        case 23 :
            if(lengy < data[0].xe){
			    return data[0].cx_23;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_23 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_23+a1*data[ee2+1].cx_23;
            break;
        case 24 :
            if(lengy < data[0].xe){
			    return data[0].cx_24;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_24 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_24+a1*data[ee2+1].cx_24;
            break;
        case 25 :
            if(lengy < data[0].xe){
			    return data[0].cx_25;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_25 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_25+a1*data[ee2+1].cx_25;
            break;
        case 26 :
            if(lengy < data[0].xe){
			    return data[0].cx_26;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_26 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_26+a1*data[ee2+1].cx_26;
            break;
        case 27 :
            if(lengy < data[0].xe){
			    return data[0].cx_27;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_27 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_27+a1*data[ee2+1].cx_27;
            break;
        case 28 :
            if(lengy < data[0].xe){
			    return data[0].cx_28;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_28 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_28+a1*data[ee2+1].cx_28;
            break;
        case 29 :
            if(lengy < data[0].xe){
			    return data[0].cx_29;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_29 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_29+a1*data[ee2+1].cx_29;
            break;
        case 30 :
            if(lengy < data[0].xe){
			    return data[0].cx_30;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_30 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_30+a1*data[ee2+1].cx_30;
            break;
        case 31 :
            if(lengy < data[0].xe){
			    return data[0].cx_31;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_31 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_31+a1*data[ee2+1].cx_31;
            break;
        case 32 :
            if(lengy < data[0].xe){
			    return data[0].cx_32;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_32 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_32+a1*data[ee2+1].cx_32;
            break;
        case 33 :
            if(lengy < data[0].xe){
			    return data[0].cx_33;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_33 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_33+a1*data[ee2+1].cx_33;
            break;
        case 34 :
            if(lengy < data[0].xe){
			    return data[0].cx_34;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_34 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_34+a1*data[ee2+1].cx_34;
            break;
        case 35 :
            if(lengy < data[0].xe){
			    return data[0].cx_35;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_35 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_35+a1*data[ee2+1].cx_35;
            break;
        case 36 :
            if(lengy < data[0].xe){
			    return data[0].cx_36;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_36 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_36+a1*data[ee2+1].cx_36;
            break;
        case 37 :
            if(lengy < data[0].xe){
			    return data[0].cx_37;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_37 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_37+a1*data[ee2+1].cx_37;
            break;
        case 38 :
            if(lengy < data[0].xe){
			    return data[0].cx_38;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_38 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_38+a1*data[ee2+1].cx_38;
            break;
        case 39 :
            if(lengy < data[0].xe){
			    return data[0].cx_39;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_39 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_39+a1*data[ee2+1].cx_39;
            break;
        case 40 :
            if(lengy < data[0].xe){
			    return data[0].cx_40;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_40 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_40+a1*data[ee2+1].cx_40;
            break;
        case 41 :
            if(lengy < data[0].xe){
			    return data[0].cx_41;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_41 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_41+a1*data[ee2+1].cx_41;
            break;
        case 42 :
            if(lengy < data[0].xe){
			    return data[0].cx_42;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_42 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_42+a1*data[ee2+1].cx_42;
            break;
        case 43 :
            if(lengy < data[0].xe){
			    return data[0].cx_43;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_43 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_43+a1*data[ee2+1].cx_43;
            break;
        case 44 :
            if(lengy < data[0].xe){
			    return data[0].cx_44;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_44 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_44+a1*data[ee2+1].cx_44;
            break;
        case 45 :
            if(lengy < data[0].xe){
			    return data[0].cx_45;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_45 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_45+a1*data[ee2+1].cx_45;
            break;
        case 46 :
            if(lengy < data[0].xe){
			    return data[0].cx_46;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_46 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_46+a1*data[ee2+1].cx_46;
            break;
        case 47 :
            if(lengy < data[0].xe){
			    return data[0].cx_47;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_47 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_47+a1*data[ee2+1].cx_47;
            break;
        case 48 :
            if(lengy < data[0].xe){
			    return data[0].cx_48;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_48 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_48+a1*data[ee2+1].cx_48;
            break;
        case 49 :
            if(lengy < data[0].xe){
			    return data[0].cx_49;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_49 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_49+a1*data[ee2+1].cx_49;
            break;
        case 50 :
            if(lengy < data[0].xe){
			    return data[0].cx_50;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_50 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_50+a1*data[ee2+1].cx_50;
            break;
        case 51 :
            if(lengy < data[0].xe){
			    return data[0].cx_51;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_51 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_51+a1*data[ee2+1].cx_51;
            break;
        case 52 :
            if(lengy < data[0].xe){
			    return data[0].cx_52;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_52 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_52+a1*data[ee2+1].cx_52;
            break;
        case 53 :
            if(lengy < data[0].xe){
			    return data[0].cx_53;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_53 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_53+a1*data[ee2+1].cx_53;
            break;
        case 54 :
            if(lengy < data[0].xe){
			    return data[0].cx_54;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_54 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_54+a1*data[ee2+1].cx_54;
            break;
        case 55 :
            if(lengy < data[0].xe){
			    return data[0].cx_55;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_55 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_55+a1*data[ee2+1].cx_55;
            break;
        case 56 :
            if(lengy < data[0].xe){
			    return data[0].cx_56;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_56 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_56+a1*data[ee2+1].cx_56;
            break;
        case 57 :
            if(lengy < data[0].xe){
			    return data[0].cx_57;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_57 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_57+a1*data[ee2+1].cx_57;
            break;
        case 58 :
            if(lengy < data[0].xe){
			    return data[0].cx_58;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_58 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_58+a1*data[ee2+1].cx_58;
            break;
        case 59 :
            if(lengy < data[0].xe){
			    return data[0].cx_59;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_59 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_59+a1*data[ee2+1].cx_59;
            break;
        case 60 :
            if(lengy < data[0].xe){
			    return data[0].cx_60;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_60 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_60+a1*data[ee2+1].cx_60;
            break;
        case 61 :
            if(lengy < data[0].xe){
			    return data[0].cx_61;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_61 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_61+a1*data[ee2+1].cx_61;
            break;
        case 62 :
            if(lengy < data[0].xe){
			    return data[0].cx_62;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_62 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_62+a1*data[ee2+1].cx_62;
            break;
        case 63 :
            if(lengy < data[0].xe){
			    return data[0].cx_63;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_63 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_63+a1*data[ee2+1].cx_63;
            break;
        case 64 :
            if(lengy < data[0].xe){
			    return data[0].cx_64;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_64 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_64+a1*data[ee2+1].cx_64;
            break;
        case 65 :
            if(lengy < data[0].xe){
			    return data[0].cx_65;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_65 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_65+a1*data[ee2+1].cx_65;
            break;
        case 66 :
            if(lengy < data[0].xe){
			    return data[0].cx_66;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_66 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_66+a1*data[ee2+1].cx_66;
            break;
        case 67 :
            if(lengy < data[0].xe){
			    return data[0].cx_67;
		    }else if(lengy > data[N_LOGX-1].xe){
			    return data[N_LOGX-1].cx_67 * 0.1 * exp(-1 * (lengy - data[N_LOGX-1].xe));
		    }
		    return a2*data[ee2].cx_67+a1*data[ee2+1].cx_67;
            break;
        default :
            printf("\nError : Call about cross section data in ArO2MCC.\n\n");
            return 0.0;
    }
}