#include "cuda_SaveDump.cuh"
void V001_DUMP(FILE *SF){

}
void V000_DUMP(FILE *SF){
    int isp,i;
    // time
    fwrite(&t, 8, 1, SF);
    fwrite(&tstep, 4, 1, SF);
    fwrite(&cstep, 4, 1, SF);
    // Particle Data gpu >> cpu
    hipMemcpy(SP, dev_info_sp,  nsp * sizeof(Species), hipMemcpyDeviceToHost);
    hipMemcpy(Host_sp, dev_sp, Total_maxnp * sizeof(GCP), hipMemcpyDeviceToHost);
    hipMemcpy(Host_G_sp, dev_G_sp, nsp * Gsize * sizeof(GPG), hipMemcpyDeviceToHost);
    Copy_GCPtoHCP(SP, Host_sp, PtD, Host_G_sp);
    // Gas info
    fwrite(&MainGas, 4, 1, SF);
    for(isp=0;isp<nsp;isp++){
        // NP2C
        fwrite(&SP[isp].np2c, 4, 1, SF);
        fwrite(&SP[isp].np, 4, 1, SF);
        for (i = 0; i < SP[isp].np; i++) {
            fwrite(&PtD[isp].CellID[i], 4, 1, SF);
            fwrite(&PtD[isp].x[i], 4, 1, SF);
            fwrite(&PtD[isp].y[i], 4, 1, SF);
            fwrite(&PtD[isp].vx[i], 4, 1, SF);
            fwrite(&PtD[isp].vy[i], 4, 1, SF);
            fwrite(&PtD[isp].vz[i], 4, 1, SF);
        }
    }
    // History data
    fwrite(&hist_count, 4, 1, SF);
    fwrite(&dHIST, 4, 1, SF);
	fwrite(t_array, 4, hist_count, SF);
	for (isp = 0; isp < nsp; isp++) {
		fwrite(HistPt[isp].np, 4, hist_count, SF);
	}
	fwrite(iter_array, 4, hist_count, SF);
    for (i = 0; i < CondNUMR; i++) {
		fwrite(Current_hist[i], 4, hist_count, SF);
		for (isp = 0; isp < nsp; isp++) {
			fwrite(SP_current_hist[isp][i], 4, hist_count, SF);
		}
		fwrite(Volt_hist[i], 4, hist_count, SF);
		fwrite(Volt_cond_hist[i], 4, hist_count, SF);
		fwrite(Surf_charge_hist[i], 4, hist_count, SF);
	}
    // 2D data
    for (i = 0; i < nsp*Gsize; i++) {
        fwrite(&Host_G_sp[i].den, 4, 1, SF);    // pt density : GPG
        fwrite(&Host_G_sp[i].ave_den, 4, 1, SF); // pt Ave_density : GPG
        fwrite(&Host_G_sp[i].sigma, 4, 1, SF); // pt sigma
    }
    hipMemcpy(vec_G, dev_GvecSet, Gsize * sizeof(GGA), hipMemcpyDeviceToHost);
    for (i = 0; i < Gsize; i++) {
        fwrite(&vec_G[i].Temp, 4, 1, SF); // BG density,Tem,vel : GGA
        fwrite(&vec_G[i].BackVel1, 4, 1, SF); // BG density,Tem,vel : GGA
        fwrite(&vec_G[i].BackDen1, 4, 1, SF); // BG density,Tem,vel : GGA
        fwrite(&vec_G[i].BackVel2, 4, 1, SF); // BG density,Tem,vel : GGA
        fwrite(&vec_G[i].BackDen2, 4, 1, SF); // BG density,Tem,vel : GGA
        fwrite(&vec_G[i].Lap_Pot, 4, 1, SF); // BG Lap_Pot
        fwrite(&vec_G[i].Pois_Pot, 4, 1, SF); // BG Pois_Pot
        fwrite(&vec_G[i].Ex, 4, 1, SF); // BG Ex
        fwrite(&vec_G[i].Ey, 4, 1, SF); // BG Ey
    } 
    // Field
    hipMemcpy(vec_Potential, TotPotential, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    fwrite(vec_Potential, 4, Gsize, SF); 
    hipMemcpy(ave_Potential, dev_ave_Potential, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    fwrite(ave_Potential, 4, Gsize, SF); 
    hipMemcpy(ave_Source, dev_ave_Source, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    fwrite(ave_Source, 4, Gsize, SF); 
    hipMemcpy(ave_Sigma, dev_ave_Sigma, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    fwrite(ave_Sigma, 4, Gsize, SF); 
    hipMemcpy(ave_Ex, dev_ave_Ex, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    fwrite(ave_Ex, 4, Gsize, SF); 
    hipMemcpy(ave_Ey, dev_ave_Ey, Gsize * sizeof(float), hipMemcpyDeviceToHost);
    fwrite(ave_Ey, 4, Gsize, SF); 
    // MCC
    hipMemcpy(ave_MCC_rate, dev_ave_MCC_rate, TnRct * Gsize * sizeof(float), hipMemcpyDeviceToHost);
    fwrite(ave_MCC_rate, 4, TnRct * Gsize, SF); // MCC AVE RATE : mccrate
}
void SaveDumpFile(int KEY2,int KEY1,int KEY0){
    FILE *SaveFile;
	char FileName[512];
    int isp;
    float time_sum;
    /////// Function access management ///////
    if (init_dump_num){
		while (1){ // find dump_order
			if (tstep <= dump_cycle[dump_order])
				break;
			else
				dump_order++;
		}
		init_dump_num--;
	}
	if (tstep < dump_cycle[dump_order]){
		return;
    }
    if(dump_order >= dump_num){
        OVER_dump_order++;
        if(OVER_dump_order == 100 * DT_PIC){
            OVER_dump_order = 0;
        }else{
            return;
        }
    }
	dump_order++;
    //////////////////////////////////////////
    // Dump load version SAVE.  Ver.[KEY2].[KEY1].[KEY0]
    //  KEY0 : 0~9 If you just add and remove the storage variable
    //  KEY1 : 0~9 If you just add and remove the storage variable
    //  KEY2 : When there is a change in a significant calculation module
    // Version History
    //  Ver.0.0.0 : Time, Particle information, np2c, Number of particle,
    //
    /// Open Dump File
	fprintf(stderr,"\n-------------------------Dumping File Ver.[%d][%d][%d]---------------------------\n",KEY2,KEY1,KEY0);
    sprintf(FileName, "%s.dmp%d", InputFile, dump_order);
	if ((SaveFile = fopen(FileName, "w")) == NULL) {
		puts("Dump: open failed");
		exit(-1);
	}
    //Save Start
    fwrite(&KEY2, 4, 1, SaveFile);
    fwrite(&KEY1, 4, 1, SaveFile);
    fwrite(&KEY0, 4, 1, SaveFile);
    if(KEY2>=0 && KEY1>=0 && KEY0>=0)  V000_DUMP(SaveFile);
    if(KEY2>=0 && KEY1>=0 && KEY0>=1)  V001_DUMP(SaveFile);
    //Save End
    fclose(SaveFile);

    // time calculate
	while(totaltime > 1000){
		TotalT_S++;
		totaltime = totaltime - 1000;
	}
	while(TotalT_S >= 60){
			TotalT_M++;
			TotalT_S -= 60;
	}
	while(TotalT_M >= 60){
			TotalT_H++;
			TotalT_M -= 60;
	}
	while(TotalT_H >= 24){
			TotalT_D++;
			TotalT_H -= 24;
	}
    fprintf(stderr, "Dump at t=%1.5e(s),Step[%d]Cycle[%d] %s\n", t,tstep,cstep,FileName);
	for (isp = 0; isp < nsp; isp++){
		fprintf(stderr, "%s : %d,  ", SP[isp].name, SP[isp].np);
	}fprintf(stderr, "\n");
	fprintf(stderr, "Domain size : %d X %d =%d,  ", ngx, ngy, ngx * ngy);
	fprintf(stderr, "Time: %d(d), %d(h), %d(m), %d(s)\n",TotalT_D,TotalT_H,TotalT_M,TotalT_S);
	time_sum = gputime_field+gputime_efield+gputime_move+gputime_sort+gputime_mcc+gputime_continue+gputime_deposit+gputime_diag+gputime_Tec+gputime_dump;
	fprintf(stderr, "Total : time = %2.8f	(s)\n", time_sum * 0.001);
	fprintf(stderr, "Field	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_field * 0.001, gputime_field * 100 / time_sum);
	fprintf(stderr, "Efield	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_efield * 0.001, gputime_efield * 100 / time_sum);
	fprintf(stderr, "Move	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_move * 0.001, gputime_move * 100 / time_sum);
	fprintf(stderr, "Sort	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_sort * 0.001, gputime_sort * 100 / time_sum);
	fprintf(stderr, "Mcc	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_mcc * 0.001, gputime_mcc * 100 / time_sum);
	fprintf(stderr, "CONTI	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_continue * 0.001, gputime_continue * 100 / time_sum);
	fprintf(stderr, "Depo	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_deposit * 0.001, gputime_deposit * 100 / time_sum);
	fprintf(stderr, "Diag	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_diag * 0.001, gputime_diag * 100 / time_sum);
	fprintf(stderr, "Tecplot: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_Tec * 0.001, gputime_Tec * 100 / time_sum);
	fprintf(stderr, "Dump	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_dump * 0.001, gputime_dump * 100 / time_sum);
	fprintf(stderr, "------------------------------------------------------------------------------\n");
}