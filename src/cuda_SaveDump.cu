#include "cuda_SaveDump.cuh"
void V001_DUMP(FILE *SF){

}
void V000_DUMP(FILE *SF){
    int isp,i;
    // time
    fwrite(&t, 8, 1, SF);
    fwrite(&tstep, 4, 1, SF);
    fwrite(&cstep, 4, 1, SF);
    // Particle Data gpu >> cpu
    checkCudaErrors(hipMemcpy(SP, dev_info_sp,  nsp * sizeof(Species), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(Host_sp, dev_sp, Total_maxnp * sizeof(GCP), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(Host_G_sp, dev_G_sp, Gsize * nsp * sizeof(GPG), hipMemcpyDeviceToHost));
    Copy_GCPtoHCP(SP, Host_sp, PtD, Host_G_sp);
    // Gas info
    fwrite(&MainGas, 4, 1, SF);
    for(isp=0;isp<nsp;isp++){
        // NP2C
        fwrite(&SP[isp].np2c, 4, 1, SF);
        fwrite(&SP[isp].np, 4, 1, SF);
        for (i = 0; i < SP[isp].np; i++) {
            fwrite(&PtD[isp].CellID[i], 4, 1, SF);
            fwrite(&PtD[isp].x[i], 4, 1, SF);
            fwrite(&PtD[isp].y[i], 4, 1, SF);
            fwrite(&PtD[isp].vx[i], 4, 1, SF);
            fwrite(&PtD[isp].vy[i], 4, 1, SF);
            fwrite(&PtD[isp].vz[i], 4, 1, SF);
        }
    }
}
void SaveDumpFile(int KEY2,int KEY1,int KEY0){
    FILE *SaveFile;
	char filename[512];
    int isp;
    float time_sum;
    /////// Function access management ///////
    if (init_dump_num){
		while (1){ // find dump_order
			if (tstep <= dump_cycle[dump_order])
				break;
			else
				dump_order++;
		}
		init_dump_num--;
	}
	if (tstep < dump_cycle[dump_order]){
		return;
    }
    if(dump_order >= dump_num){
        OVER_dump_order++;
        if(OVER_dump_order == 100 * DT_PIC){
            OVER_dump_order = 0;
        }else{
            return;
        }
    }
	dump_order++;
    //////////////////////////////////////////
    // Dump load version SAVE.  Ver.[KEY2].[KEY1].[KEY0]
    //  KEY0 : 0~9 If you just add and remove the storage variable
    //  KEY1 : 0~9 If you just add and remove the storage variable
    //  KEY2 : When there is a change in a significant calculation module
    // Version History
    //  Ver.0.0.0 : Time, Particle information, np2c, Number of particle,
    //
    /// Open Dump File
	fprintf(stderr,"\n-------------------------Dumping File Ver.[%d][%d][%d]---------------------------\n",KEY2,KEY1,KEY0);
	sprintf(filename, "%s.dmp%d", InputFile, dump_order);
	if ((SaveFile = fopen(filename, "w")) == NULL) {
		puts("Dump: open failed");
		exit(-1);
	}
    //Save Start
    fwrite(&KEY2, 4, 1, SaveFile);
    fwrite(&KEY1, 4, 1, SaveFile);
    fwrite(&KEY0, 4, 1, SaveFile);
    if(KEY2==0 && KEY2==0 && KEY2==0)       V000_DUMP(SaveFile);
    else if(KEY2==0 && KEY2==0 && KEY2==1)  V001_DUMP(SaveFile);
    //Save End
    fclose(SaveFile);

    // time calculate
	while(totaltime > 1000){
		TotalT_S++;
		totaltime = totaltime - 1000;
	}
	while(TotalT_S >= 60){
			TotalT_M++;
			TotalT_S -= 60;
	}
	while(TotalT_M >= 60){
			TotalT_H++;
			TotalT_M -= 60;
	}
	while(TotalT_H >= 24){
			TotalT_D++;
			TotalT_H -= 24;
	}
    fprintf(stderr, "Dump at t=%1.5e(s),Step[%d]Cycle[%d] %s\n", t,tstep,cstep,filename);
	for (isp = 0; isp < nsp; isp++){
		fprintf(stderr, "%s : %d,  ", SP[isp].name, SP[isp].np);
	}fprintf(stderr, "\n");
	fprintf(stderr, "Domain size : %d X %d =%d,  ", ngx, ngy, ngx * ngy);
	fprintf(stderr, "Time: %d(d), %d(h), %d(m), %d(s)\n",TotalT_D,TotalT_H,TotalT_M,TotalT_S);
	time_sum = gputime_field+gputime_efield+gputime_move+gputime_sort+gputime_mcc+gputime_continue+gputime_deposit+gputime_diag+gputime_Tec+gputime_dump;
	fprintf(stderr, "Total : time = %2.8f	(s)\n", time_sum * 0.001);
	fprintf(stderr, "Field	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_field * 0.001, gputime_field * 100 / time_sum);
	fprintf(stderr, "Efield	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_efield * 0.001, gputime_efield * 100 / time_sum);
	fprintf(stderr, "Move	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_move * 0.001, gputime_move * 100 / time_sum);
	fprintf(stderr, "Sort	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_sort * 0.001, gputime_sort * 100 / time_sum);
	fprintf(stderr, "Mcc	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_mcc * 0.001, gputime_mcc * 100 / time_sum);
	fprintf(stderr, "CONTI	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_continue * 0.001, gputime_continue * 100 / time_sum);
	fprintf(stderr, "Depo	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_deposit * 0.001, gputime_deposit * 100 / time_sum);
	fprintf(stderr, "Diag	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_diag * 0.001, gputime_diag * 100 / time_sum);
	fprintf(stderr, "Tecplot: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_Tec * 0.001, gputime_Tec * 100 / time_sum);
	fprintf(stderr, "Dump	: time = %2.8f	(s)		rate = %g	(%)\n",	gputime_dump * 0.001, gputime_dump * 100 / time_sum);
	fprintf(stderr, "------------------------------------------------------------------------------\n");
}